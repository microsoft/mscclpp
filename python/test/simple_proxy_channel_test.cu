#include "hip/hip_runtime.h"
#include <mscclpp/packet.hpp>
#include <mscclpp/proxy_channel_device.hpp>

#include "mscclpp_common.h"

// BEGIN_DEFINES //

#ifndef PARAMETRIZE
#define KERNEL simple_proxy_channel
#define N_SHARDS 8
#define TD int
#define USE_PACKET false
#endif

// END_DEFINES //

// be careful about using channels[my_rank] as it is inavlie and it is there just for simplicity of indexing
extern "C" __global__ void __launch_bounds__(1024, 1)
    KERNEL(Plist<mscclpp::SimpleProxyChannelDeviceHandle, N_SHARDS> channels, int my_rank, int nranks, TD* data,
           TD* scratch, int num_elements) {
  int tid = threadIdx.x;
  int nthreads = blockDim.x;
  uint64_t size_per_rank = (num_elements * sizeof(TD)) / nranks;
  uint64_t my_offset = size_per_rank * my_rank;
  int nthreads_per_rank = nthreads / nranks;
  int my_nghr = tid / nthreads_per_rank;
  uint64_t my_nghr_offset = size_per_rank * my_nghr;
  __syncthreads();
  int flag = 123;
  if (USE_PACKET) {
    mscclpp::putPackets(scratch, 2 * my_offset, data, my_offset, size_per_rank, tid, nthreads, flag);
    __syncthreads();
    if (tid < nranks && tid != my_rank) {
      channels[tid].put(2 * my_offset, 2 * my_offset, 2 * size_per_rank);
    }
    if (my_nghr != my_rank && my_nghr < nranks)
      mscclpp::getPackets(data, my_nghr_offset, scratch, 2 * my_nghr_offset, size_per_rank, tid % nthreads_per_rank,
                          nthreads_per_rank, flag);
  } else {
    if (tid < nranks && tid != my_rank) {
      channels[tid].putWithSignal(my_offset, my_offset, size_per_rank);
      channels[tid].wait();
    }
  }
}
