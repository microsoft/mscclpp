#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/semaphore_device.hpp>

// be careful about using semaphore[my_rank] as it is an invalid semaphore and it is there just for simplicity of
// indexing
extern "C" __global__ void __launch_bounds__(1024, 1)
    d2d_semaphore(mscclpp::SmDevice2DeviceSemaphoreDeviceHandle* semaphores, int my_rank, int nranks) {
  int tid = threadIdx.x;
  if (tid < nranks && tid != my_rank) {
    semaphores[tid].signal();
    semaphores[tid].wait();
  }
}
