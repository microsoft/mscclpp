#include "hip/hip_runtime.h"
#include "mscclpp.h"

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <unistd.h>

#include "common.h"

#define MSCCLPP_USE_MPI_FOR_TESTS
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
#include <mpi.h>
#endif // MSCCLPP_USE_MPI_FOR_TESTS

#define RANKS_PER_NODE 8
#define USE_DMA_FOR_P2P 1
#define TEST_CONN_TYPE 0 // 0: P2P(for local)+IB(for remote), 1: IB-Only
#define BLOCK_THREADS_NUM 256

#define MSCCLPPCHECK(call)                                                                                             \
  do {                                                                                                                 \
    mscclppResult_t res = call;                                                                                        \
    if (res != mscclppSuccess && res != mscclppInProgress) {                                                           \
      /* Print the back trace*/                                                                                        \
      printf("Failure at %s:%d -> %d\n", __FILE__, __LINE__, res);                                                     \
      return res;                                                                                                      \
    }                                                                                                                  \
  } while (0);

// Check CUDA RT calls
#define CUDACHECK(cmd)                                                                                                 \
  do {                                                                                                                 \
    hipError_t err = cmd;                                                                                             \
    if (err != hipSuccess) {                                                                                          \
      printf("%s:%d Cuda failure '%s'\n", __FILE__, __LINE__, hipGetErrorString(err));                                \
      exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (false)

// Measure current time in second.
static double getTime(void)
{
  struct timespec tspec;
  if (clock_gettime(CLOCK_MONOTONIC, &tspec) == -1) {
    printf("clock_gettime failed\n");
    exit(EXIT_FAILURE);
  }
  return (tspec.tv_nsec / 1.0e9) + tspec.tv_sec;
}


void parse_arguments(int argc, const char* argv[], const char** ip_port, int* rank, int* world_size)
{
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  if (argc != 2 && argc != 4) {
    print_usage(argv[0]);
    exit(-1);
  }
  *ip_port = argv[1];
  if (argc == 4) {
    *rank = atoi(argv[2]);
    *world_size = atoi(argv[3]);
  } else {
    MPI_Comm_rank(MPI_COMM_WORLD, rank);
    MPI_Comm_size(MPI_COMM_WORLD, world_size);
  }
#else
  if (argc != 4) {
    print_usage(argv[0]);
    exit(-1);
  }
  *ip_port = argv[1];
  *rank = atoi(argv[2]);
  *world_size = atoi(argv[3]);
#endif
}

__global__ void initKernel(char* data_d, int dataSize)
{
  for (size_t i = threadIdx.x; i < dataSize; i += blockDim.x) {
    data_d[i] = i % 256;
  }
}

__constant__ mscclppDevConn_t sendConnConst;
__constant__ mscclppDevConn_t recvConnConst;

__global__ void smKernel(bool root, size_t dataSize)
{
  mscclppDevConn_t sendConn = sendConnConst;
  mscclppDevConn_t recvConn = recvConnConst;

  if (root)
  {
    sendConn.putDirect(0, dataSize, threadIdx.x, blockDim.x);
    // make sure all the threads have put their data
    __syncthreads();
    if (threadIdx.x == 0){
      sendConn.signalDirect();
      recvConn.waitDirectSingal();
    }
  }
  else
  {
    if (threadIdx.x == 0) {
      recvConn.waitDirectSingal();
    }
    // make sure we get the latest data
    __syncthreads();
    sendConn.putDirect(0, dataSize, threadIdx.x, blockDim.x);
    __syncthreads();
    if (threadIdx.x == 0) {
      sendConn.signalDirect();
    }
  }
}

void resetData(char* data_d, size_t data_size, bool isRoot)
{
  if (isRoot) {
    initKernel<<<1, BLOCK_THREADS_NUM>>>(data_d, data_size);
  } else {
    CUDACHECK(hipMemset(data_d, 0, data_size));
  }
}

int main(int argc, const char* argv[])
{
  const char* ip_port;
  int rank, world_size;
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  MPI_Init(NULL, NULL);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &world_size);
#endif
  parse_arguments(argc, argv, &ip_port, &rank, &world_size);

  bool isRoot = rank == 0;

  CUDACHECK(hipSetDevice(rank));

  if (rank == 0)
    printf("Initializing MSCCL++\n");
  mscclppComm_t comm;
  MSCCLPPCHECK(mscclppCommInitRank(&comm, world_size, ip_port, rank));

  char* data_d;
  // size_t data_size = 1 << 10; // Kilobyte
  // size_t data_size = 1 << 20; // Megabyte
  size_t data_size = 1 << 30; // Gigabyte
  CUDACHECK(hipMalloc(&data_d, data_size));
  resetData(data_d, data_size, isRoot);

  if (rank == 0) {
    MSCCLPPCHECK(mscclppConnect(comm, 1, 0, data_d, data_size, mscclppTransportP2P));
  } else {
    MSCCLPPCHECK(mscclppConnect(comm, 0, 0, data_d, data_size, mscclppTransportP2P));
  }
  if (rank == 0)
    printf("Finished connection\n");

  MSCCLPPCHECK(mscclppConnectionSetup(comm));
  if (rank == 0)
    printf("Finished Setup\n");

  MSCCLPPCHECK(mscclppProxyLaunch(comm));
  if (rank == 0)
    printf("Finished proxy launch\n");

  mscclppDevConn_t *sendDevConn;
  mscclppDevConn_t *recvDevConn;
  if (rank == 0) {
    MSCCLPPCHECK(mscclppGetDeviceConnection(comm, 1, 0, &sendDevConn));
    MSCCLPPCHECK(mscclppGetDeviceConnection(comm, 1, 0, &recvDevConn));
  } else {
    MSCCLPPCHECK(mscclppGetDeviceConnection(comm, 0, 0, &sendDevConn));
    MSCCLPPCHECK(mscclppGetDeviceConnection(comm, 0, 0, &recvDevConn));
  }
  if (rank == 0)
    printf("Finished device connection\n");

  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(sendConnConst), sendDevConn, sizeof(mscclppDevConn_t)));
  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(recvConnConst), recvDevConn, sizeof(mscclppDevConn_t)));

  hipStream_t stream;
  CUDACHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  if (rank == 0)
    printf("Start running kernel\n");
  smKernel<<<1, BLOCK_THREADS_NUM, 0, stream>>>(isRoot, data_size);
  CUDACHECK(hipDeviceSynchronize());

  // Read results from GPU
  char* buf = (char*)calloc(data_size, 1);
  if (buf == nullptr) {
    printf("calloc failed\n");
    return -1;
  }
  CUDACHECK(hipMemcpy(buf, data_d, data_size, hipMemcpyDeviceToHost));

  bool failed = false;
  for (size_t i = 0; i < data_size; ++i) {
    char expected = (char)(i % 256);
    if (buf[i] != expected) {
      printf("rank: %d, wrong data: %d, expected %d\n", rank, buf[i], expected);
      failed = true;
    }
  }
  if (failed) {
    return -1;
  }

  MSCCLPPCHECK(mscclppProxyStop(comm));

  MSCCLPPCHECK(mscclppCommDestroy(comm));

#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  if (argc == 2) {
    MPI_Finalize();
  }
#endif
  printf("Succeeded! %d\n", rank);
  return 0;
}
