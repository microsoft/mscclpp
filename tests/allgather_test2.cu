#include "hip/hip_runtime.h"
#include "mscclpp.h"
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
#include "mpi.h"
#endif // MSCCLPP_USE_MPI_FOR_TESTS
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <unistd.h>

#define RANKS_PER_NODE 8

// Check CUDA RT calls
#define CUDACHECK(cmd)                                                                                                 \
  do {                                                                                                                 \
    hipError_t err = cmd;                                                                                             \
    if (err != hipSuccess) {                                                                                          \
      printf("%s:%d Cuda failure '%s'\n", __FILE__, __LINE__, hipGetErrorString(err));                                \
      exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (false)

// Measure current time in second.
static double getTime(void)
{
  struct timespec tspec;
  if (clock_gettime(CLOCK_MONOTONIC, &tspec) == -1) {
    printf("clock_gettime failed\n");
    exit(EXIT_FAILURE);
  }
  return (tspec.tv_nsec / 1.0e9) + tspec.tv_sec;
}

__constant__ mscclppDevConn_t constDevConns[16];

__global__ void kernel(int rank, int world_size, size_t nelemsPerGPU)
{
  if (threadIdx.x % 32 != 0)
    return;

  int warpId = threadIdx.x / 32;
  bool isIB = false;
  if (warpId >= world_size - 1)
    isIB = true;
  if (isIB)
    warpId = warpId - (world_size - 1);
  int remoteRank = (warpId < rank) ? warpId : warpId + 1;
  mscclppDevConn_t devConn = constDevConns[remoteRank];
  if (isIB)
    devConn = constDevConns[remoteRank + world_size];

    // Each warp receives data from different ranks
#if 1

  // Trigger sending data, flag and synchronize after
  devConn.putWithSignal(rank * nelemsPerGPU * sizeof(int), nelemsPerGPU * sizeof(int));

  devConn.wait();

#else
  for (int i = 1; i < world_size; i++) {
    __syncthreads();
    if (remoteRank != ((rank + i) % world_size))
      continue;

    // Trigger sending data, flag and synchronize after
    size_t ibPortion = nelemsPerGPU / 12; // nelemsPerGPU/12;
    if (isIB)
      devConn.fifo.setTrigger(trig, mscclppFlag | mscclppData | mscclppSync,
                              rank * nelemsPerGPU * sizeof(int) + (nelemsPerGPU - ibPortion) * sizeof(int),
                              rank * nelemsPerGPU * sizeof(int) + (nelemsPerGPU - ibPortion) * sizeof(int),
                              ibPortion * sizeof(int));
    else
      devConn.fifo.setTrigger(trig, mscclppFlag | mscclppData | mscclppSync, rank * nelemsPerGPU * sizeof(int),
                              rank * nelemsPerGPU * sizeof(int), (nelemsPerGPU - ibPortion) * sizeof(int));
    // Wait on the request to make sure it is safe to reuse buffer and flag
    auto req = devConn.fifo.putWithSignal(dataOffset, dataSize);
    devConn.fifo.sync(req);
  }
  // Wait for receiving data from remote rank
  while (*proxyFlag == baseFlag)
    ;
#endif
}

int rankToLocalRank(int rank)
{
  return rank % RANKS_PER_NODE;
}

int rankToNode(int rank)
{
  return rank / RANKS_PER_NODE;
}

int cudaNumToIbNum(int cudaNum)
{
  int ibNum;
  if (cudaNum == 0) {
    ibNum = 0;
  } else if (cudaNum == 1) {
    ibNum = 4;
  } else if (cudaNum == 2) {
    ibNum = 1;
  } else if (cudaNum == 3) {
    ibNum = 5;
  } else if (cudaNum == 4) {
    ibNum = 2;
  } else if (cudaNum == 5) {
    ibNum = 6;
  } else if (cudaNum == 6) {
    ibNum = 3;
  } else if (cudaNum == 7) {
    ibNum = 7;
  } else {
    printf("Invalid cudaNum: %d\n", cudaNum);
    exit(EXIT_FAILURE);
  }
  return ibNum;
}

void print_usage(const char* prog)
{
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  printf("usage: %s IP:PORT [rank nranks]\n", prog);
#else
  printf("usage: %s IP:PORT rank nranks\n", prog);
#endif
}

int main(int argc, const char* argv[])
{
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  if (argc != 2 && argc != 4) {
    print_usage(argv[0]);
    return -1;
  }
  const char* ip_port = argv[1];
  int rank;
  int world_size;
  if (argc == 4) {
    rank = atoi(argv[2]);
    world_size = atoi(argv[3]);
  } else {
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
  }
#else
  if (argc != 4) {
    print_usage(argv[0]);
    return -1;
  }
  const char* ip_port = argv[1];
  int rank = atoi(argv[2]);
  int world_size = atoi(argv[3]);
#endif
  int localRank = rankToLocalRank(rank);
  int thisNode = rankToNode(rank);
  int cudaNum = localRank;
  int ibNum = cudaNumToIbNum(cudaNum);

  CUDACHECK(hipSetDevice(cudaNum));
  std::string ibDevStr = "mlx5_ib" + std::to_string(localRank);

  mscclppComm_t comm;
  MSCCLPPCHECK(mscclppCommInitRank(&comm, world_size, rank, ip_port));

  int* data_d;
  uint64_t* flag_d;
  size_t data_size = 1536 * 1024 * 1024;
  size_t nelemsPerGPU = data_size / sizeof(int) / world_size;
  CUDACHECK(hipMalloc(&data_d, data_size));
  CUDACHECK(hipMalloc(&flag_d, sizeof(uint64_t)));
  CUDACHECK(hipMemset(data_d, 0, data_size));
  CUDACHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));

  int* data_h = new int[nelemsPerGPU * world_size];
  for (int i = 0; i < nelemsPerGPU * world_size; i++) {
    size_t val = i + 1;
    if (i / nelemsPerGPU == rank) {
      data_h[i] = val;
    } else {
      data_h[i] = 0;
    }
  }
  CUDACHECK(hipMemcpy(data_d, data_h, data_size, hipMemcpyHostToDevice));

  mscclppDevConn_t devConns[16];
  for (int r = 0; r < world_size; ++r) {
    if (r == rank)
      continue;
    mscclppTransport_t transportType;
    const char* ibDev = NULL;
    transportType = mscclppTransportP2P;
    // Connect with all other ranks
    MSCCLPPCHECK(mscclppConnect(comm, &devConns[r], r, 0, data_d, data_size, flag_d, transportType, ibDev));
  }
  for (int r = 0; r < world_size; ++r) {
    if (r == rank)
      continue;
    mscclppTransport_t transportType;
    const char* ibDev = ibDevStr.c_str();
    transportType = mscclppTransportIB;
    // Connect with all other ranks
    MSCCLPPCHECK(
      mscclppConnect(comm, &devConns[r + world_size], r, 0, data_d, data_size, flag_d, transportType, ibDev));
  }

  MSCCLPPCHECK(mscclppConnectionSetup(comm));

  MSCCLPPCHECK(mscclppProxyLaunch(comm));

  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(constDevConns), devConns, sizeof(mscclppDevConn_t) * 2 * world_size));

  hipStream_t stream;
  CUDACHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  CUDACHECK(hipDeviceSynchronize());
  kernel<<<1, 32 * 2 * (world_size - 1), 0, stream>>>(rank, world_size, nelemsPerGPU);
  CUDACHECK(hipDeviceSynchronize());
  CUDACHECK(hipMemcpy(data_h, data_d, data_size, hipMemcpyDeviceToHost));
  CUDACHECK(hipDeviceSynchronize());

  for (size_t i = 0; i < nelemsPerGPU * world_size; i++) {
    int val = i + 1;
    if (data_h[i] != val) {
      printf("oh uh things went wrong! data_h[%d] (%d) != val (%d)\n", i, data_h[i], val);
      break;
    }
  }
  int tmp[16];
  MSCCLPPCHECK(mscclppBootstrapAllGather(comm, tmp, sizeof(int)));

  //   // Perf test
  //   hipEvent_t ev_start;
  //   hipEvent_t ev_end;
  //   CUDACHECK(hipEventCreate(&ev_start));
  //   CUDACHECK(hipEventCreate(&ev_end));

  // warm up
  // int warmupiter = 1000;
  // for (int i = 0; i < warmupiter; ++i) {
  //   kernel<<<1, 32 * (world_size - 1), 0, stream>>>(rank, world_size, nelemsPerGPU);
  // }
  // CUDACHECK(hipDeviceSynchronize());
  // MSCCLPPCHECK(mscclppBootstrapAllGather(comm, tmp, sizeof(int)));

  // cudaGraph Capture
  hipGraph_t graph;
  hipGraphExec_t instance;
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  int cudagraphiter = 10;
  for (int i = 0; i < cudagraphiter; ++i) {
    kernel<<<1, 32 * 2 * (world_size - 1), 0, stream>>>(rank, world_size, nelemsPerGPU);
  }
  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

  int cudagraphwarmup = 10;
  for (int i = 0; i < cudagraphwarmup; ++i) {
    hipGraphLaunch(instance, stream);
  }
  CUDACHECK(hipStreamSynchronize(stream));

  // measure runtime
  //  CUDACHECK(hipEventRecord(ev_start, stream));
  double t0 = getTime();
  int cudagraphlaunch = 10;
  for (int i = 0; i < cudagraphlaunch; ++i) {
    // kernel<<<1, 32 * (world_size - 1), 0, stream>>>(rank, world_size);
    hipGraphLaunch(instance, stream);
  }
  //  CUDACHECK(hipEventRecord(ev_end, stream));
  CUDACHECK(hipStreamSynchronize(stream));

  double t1 = getTime();
  float ms = (t1 - t0) * 1000.0;
  //  CUDACHECK(hipEventElapsedTime(&ms, ev_start, ev_end));
  double time_in_us = ms * 1000. / (float)cudagraphlaunch / (float)cudagraphiter;
  printf("rank: %d, time: %f us/iter algBW %f\n", rank, time_in_us,
         (double)(data_size) / 1024. / 1024. / 1024. / (time_in_us / 1e6));

  MSCCLPPCHECK(mscclppBootstrapAllGather(comm, tmp, sizeof(int)));
  MSCCLPPCHECK(mscclppProxyStop(comm));

  MSCCLPPCHECK(mscclppCommDestroy(comm));

#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  if (argc == 2) {
    MPI_Finalize();
  }
#endif
  printf("Succeeded! %d\n", rank);
  return 0;
}
