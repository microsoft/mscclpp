#include "hip/hip_runtime.h"
#include "mpi.h"
#include "mscclpp.h"
#include "prims_ll.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
// Check CUDA RT calls
#define CUDACHECK(cmd)                                                         \
    do {                                                                       \
        hipError_t err = cmd;                                                 \
        if (err != hipSuccess) {                                              \
            printf("Cuda failure '%s'", hipGetErrorString(err));              \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (false)

__device__ void test_send_ll(void *data_src, void *recvbuff, void *sendConnHead,
                             int size)
{
    // using Proto = ProtoLL;
    int tid = threadIdx.x;
    int nthreads = blockDim.x;
    // Primitives<float, FuncSum<float>, FanSymmetric<1>, 1, Proto, 0> prims(
    //     tid, nthreads, ncclDevSum, 0);
    // prims.send(0, size);
    return;
}

__device__ void test_recv_ll(void *data_dst, void *recvbuff, void *sendConnHead,
                             int size)
{
    // using Proto = ProtoLL;
    int tid = threadIdx.x;
    int nthreads = blockDim.x;
    // Primitives<float, FuncSum<float>, FanSymmetric<1>, 1, Proto, 0> prims(
    //     tid, nthreads, ncclDevSum, 0);
    // prims.recv(0, size);
    return;
}

__global__ void kernel(mscclppDevConn_t devConns, int rank, int world_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if (tid == 0) {
    //   // Set my data
    //   volatile int *data = (volatile int *)devConns[rank].localBuff;
    //   volatile int *flag = (volatile int *)devConns[rank].localFlag;
    //   data[rank] = rank;

    //   // Inform that the data is set
    //   *flag = 1;

    //   for (int i = 0; i < (world_size - 1) * 2; ++i) {
    //     mscclppDevConn_t devConn = &devConns[i];
    //     int tag = devConn->tag;
    //     int rankRecv = tag / world_size;
    //     int rankSend = tag % world_size;

    //     if (rankRecv != rank) continue;

    //     volatile int *remoteData = (volatile int *)devConn->remoteBuff;
    //     volatile int *remoteFlag = (volatile int *)devConn->remoteFlag;

    //     // Wait until the remote data is set
    //     while (*remoteFlag != 1) {}

    //     // Read remote data
    //     data[rankSend] = remoteData[rankSend];
    //   }
    // }
    if (rank == 0) {
        test_send_ll(devConns[0].localBuff, devConns[0].remoteBuff,
                     devConns[0].remoteFlag, 1);
    } else {
        test_recv_ll(devConns[0].localBuff, devConns[0].remoteBuff,
                     devConns[0].remoteFlag, 1);
    }
}

void print_usage(const char *prog) { printf("usage: %s IP:PORT\n", prog); }

int main(int argc, const char *argv[])
{
    if (argc != 2) {
        print_usage(argv[0]);
        return -1;
    }

    MPI_Init(NULL, NULL);

    int rank;
    int world_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    mscclppComm_t comm;
    const char *ip_port = argv[1];
    mscclppCommInitRank(&comm, world_size, rank, ip_port);

    int *data_src;
    int *data_dst;
    int elem_num = 1024;
    int data_size = sizeof(float) * elem_num;
    int *flag_d;
    CUDACHECK(hipMalloc(&data_src, data_size));
    CUDACHECK(hipMalloc(&flag_d, sizeof(int)));

    mscclppResult_t res;

    // Read from all other ranks
    for (int r = 0; r < world_size; ++r) {
        if (r == rank)
            continue;
        int tag = rank * world_size + r;
        res = mscclppConnect(comm, rank, r, data_src, data_size,
                             flag_d, tag, mscclppTransportP2P);
        if (res != mscclppSuccess) {
            printf("mscclppConnect failed\n");
            return -1;
        }
    }
    // Let others read from me
    for (int r = 0; r < world_size; ++r) {
        if (r == rank)
            continue;
        int tag = r * world_size + rank;
        res = mscclppConnect(comm, r, rank, data_src, data_size,
                             flag_d, tag, mscclppTransportP2P);
        if (res != mscclppSuccess) {
            printf("mscclppConnect failed\n");
            return -1;
        }
    }
    res = mscclppConnectionSetup(comm);
    if (res != mscclppSuccess) {
        printf("mscclppConnectionSetup failed\n");
        return -1;
    }

    mscclppDevConn_t devConns;
    mscclppGetDevConns(comm, &devConns);

    kernel<<<1, 1>>>(devConns, rank, world_size);
    CUDACHECK(hipDeviceSynchronize());

    int *buf = (int *)calloc(world_size, sizeof(int));
    if (buf == nullptr) {
        printf("calloc failed\n");
        return -1;
    }
    CUDACHECK(hipMemcpy(buf, data_dst, data_size,
                         hipMemcpyDeviceToHost));


    res = mscclppCommDestroy(comm);
    if (res != mscclppSuccess) {
        printf("mscclppDestroy failed\n");
        return -1;
    }

    MPI_Finalize();

    printf("Succeeded! %d\n", rank);
    return 0;
}
