#include "hip/hip_runtime.h"
#include "mpi.h"
#include "mscclpp.h"
#include "prims_ll.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define MSCCLPPCHECK(call)                                                     \
    do {                                                                       \
        mscclppResult_t res = call;                                            \
        if (res != mscclppSuccess && res != mscclppInProgress) {               \
            /* Print the back trace*/                                          \
            printf("Failure at %s:%d -> %d", __FILE__, __LINE__, res);         \
            return res;                                                        \
        }                                                                      \
    } while (0);

// Check CUDA RT calls
#define CUDACHECK(cmd)                                                         \
    do {                                                                       \
        hipError_t err = cmd;                                                 \
        if (err != hipSuccess) {                                              \
            printf("Cuda failure '%s'", hipGetErrorString(err));              \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (false)

#define STEPLINES 4096

__global__ void ring_all_reduce(mscclppDevConn_t devConns, int rank, int nranks,
                                void *data_src, void *data_dst, void *recvBuff,
                                int elem_num)
{
    int tid = threadIdx.x;
    int nthreads = blockDim.x;
    // in the ring allreduce algorithm, a primitive need to connect to the prev
    // and next GPU.
    Primitives_LL<float> prims(tid, nthreads, 0, 0, STEPLINES);
    // devConns[0] is the connection to the next GPU and devConns[1] is the
    // connection to the previous GPU
    prims.data_src = (float *)data_src;
    prims.data_dst = (float *)data_dst;
    prims.sendBuff = (mscclppLLFifoLine *)devConns[0].remoteBuff;
    prims.recvBuff = (mscclppLLFifoLine *)recvBuff;
    // the control flag, might be a little buggy, still need to be developed
    prims.sendConnHeadPtr = (volatile uint64_t *)devConns[0].localFlag;
    prims.recvConnHeadPtr = (volatile uint64_t *)devConns[1].remoteFlag;
    int ChunkSize = elem_num / nranks;

    ssize_t offset;
    int nelem = ChunkSize;
    int chunk;

    // step 0: push data to next GPU
    chunk = (rank + nranks - 1) % nranks;
    offset = chunk * ChunkSize;
    // nelem = min(ChunkSize, size - offset);
    prims.send(offset, nelem);
    // k-2 steps: reduce and copy to next GPU
    for (int j = 2; j < nranks; ++j) {
        chunk = (rank + nranks - j) % nranks;
        offset = chunk * ChunkSize;
        // nelem = min(ChunkSize, size - offset);
        prims.recvReduceSend(offset, nelem);
    }

    // step k-1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    chunk = rank + 0;
    offset = chunk * ChunkSize;
    // nelem = min(ChunkSize, size - offset);
    prims.recvReduceCopySend(offset, offset, nelem,
                             /*postOp=*/true);
    // k-2 steps: copy to next GPU
    for (int j = 1; j < nranks - 1; ++j) {
        chunk = (rank + nranks - j) % nranks;
        offset = chunk * ChunkSize;
        // nelem = min(ChunkSize, size - offset);
        prims.recvCopySend(offset, nelem);
    }

    // Make final copy from buffer to dest.
    chunk = (rank + 1) % nranks;
    offset = chunk * ChunkSize;
    // nelem = min(ChunkSize, size - offset);
    prims.recv(offset, nelem);
}

void print_usage(const char *prog) { printf("usage: %s IP:PORT\n", prog); }

int main(int argc, const char *argv[])
{
    if (argc != 2) {
        print_usage(argv[0]);
        return -1;
    }

    MPI_Init(NULL, NULL);

    int rank;
    int world_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    mscclppComm_t comm;
    const char *ip_port = argv[1];
    MSCCLPPCHECK(mscclppCommInitRank(&comm, world_size, rank, ip_port));
    CUDACHECK(hipSetDevice(rank));

    int elem_num = 1024;
    int data_size = sizeof(float) * elem_num;

    float *data_src;
    float *data_dst;
    char *recvbuff;
    int *sendConnhead;

    CUDACHECK(hipMalloc(&data_src, data_size));
    float *h_data_src = (float *)malloc(data_size);
    for (int i = 0; i < elem_num; ++i) {
        h_data_src[i] = i % 23;
    }
    CUDACHECK(
        hipMemcpy(data_src, h_data_src, data_size, hipMemcpyHostToDevice));
    CUDACHECK(hipMalloc(&data_dst, data_size));
    // the recvbuff's size is MSCCLPP_STEPS * STEPLINES, MSCCLPP has a concept of
    // MSCCLPP_STEPS, MSCCLPP will allocate multiple recvbuff , So we can have
    // multiple send recv operations happen at the same time.
    CUDACHECK(hipMalloc(&recvbuff, MSCCLPP_STEPS * STEPLINES));
    CUDACHECK(hipMalloc(&sendConnhead, sizeof(int)));
    mscclppResult_t res;
    int tag = 0;
    int rank_next = (rank + 1) % world_size;
    int rank_prev = (rank + world_size - 1) % world_size;
    // in the ring all reduce, we need to connect to the next and previous GPU
    MSCCLPPCHECK(mscclppConnect(comm, rank_next, rank, recvbuff, data_size,
                                sendConnhead, tag, mscclppTransportP2P));
    MSCCLPPCHECK(mscclppConnect(comm, rank, rank_prev, recvbuff, data_size,
                                sendConnhead, tag, mscclppTransportP2P));

    MSCCLPPCHECK(mscclppConnectionSetup(comm));

    mscclppDevConn_t devConns;
    MSCCLPPCHECK(mscclppGetDevConns(comm, &devConns));

    ring_all_reduce<<<1, 32>>>(devConns, rank, world_size, data_src, data_dst,
                               recvbuff, elem_num);
    CUDACHECK(hipDeviceSynchronize());
    float *h_data_dst = (float *)malloc(data_size);
    CUDACHECK(
        hipMemcpy(h_data_dst, data_dst, data_size, hipMemcpyDeviceToHost));
    for (int i = 0; i < elem_num; ++i) {
        if (h_data_dst[i] != world_size * (i % 23)) {
            printf("data_dst[%d] = %f, expected %f", i, h_data_dst[i],
                   1.0 * world_size * (i % 23));
            return -1;
        }
    }
    MSCCLPPCHECK(mscclppCommDestroy(comm));
    MPI_Finalize();

    printf("Succeeded! %d\n", rank);
    return 0;
}
