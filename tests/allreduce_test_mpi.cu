#include "hip/hip_runtime.h"
#include "mpi.h"
#include "mscclpp.h"
#include "prims_ll.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define MSCCLPPCHECK(call)                                                     \
    do {                                                                       \
        mscclppResult_t res = call;                                            \
        if (res != mscclppSuccess && res != mscclppInProgress) {               \
            /* Print the back trace*/                                          \
            printf("Failure at %s:%d -> %d", __FILE__, __LINE__, res);         \
            return res;                                                        \
        }                                                                      \
    } while (0);

// Check CUDA RT calls
#define CUDACHECK(cmd)                                                         \
    do {                                                                       \
        hipError_t err = cmd;                                                 \
        if (err != hipSuccess) {                                              \
            printf("Cuda failure '%s'", hipGetErrorString(err));              \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (false)

__global__ void ring_all_reduce(mscclppDevConn_t devConns, int rank, int nranks,
                                void *data_src, void *data_dst, void *recvBuff,
                                int elem_num)
{
    int tid = threadIdx.x;
    int nthreads = blockDim.x;
    Primitives_LL<float> prims(tid, nthreads, 0, 0, 4096);
    // devConns[0] is the connection to the previous GPU and devConns[1] is the
    // connection to the next GPU
    prims.data_src = (float *)data_src;
    prims.data_dst = (float *)data_dst;
    prims.sendBuff = (ncclLLFifoLine *)devConns[1].remoteBuff;
    prims.recvBuff = (ncclLLFifoLine *)recvBuff;
    prims.sendConnHeadPtr = (volatile uint64_t *)devConns[1].localFlag;
    prims.recvConnHeadPtr = (volatile uint64_t *)devConns[0].remoteFlag;
    if (tid == 0)
        printf("data_src: %p, data_dst: %p, sendBuff: %p, recvBuff: %p "
               "sendConnHeadPtr: %p, recvConnHeadPtr: %p\n",
               prims.data_src, prims.data_dst, prims.sendBuff, prims.recvBuff,
               prims.sendConnHeadPtr, prims.recvConnHeadPtr);
    int ChunkSize = elem_num / nranks;

    ssize_t offset;
    int nelem = ChunkSize;
    int chunk;

    // step 0: push data to next GPU
    chunk = (rank + nranks - 1) % nranks;
    offset = chunk * ChunkSize;
    // nelem = min(ChunkSize, size - offset);
    prims.send(offset, nelem);
    // return;
    // k-2 steps: reduce and copy to next GPU
    for (int j = 2; j < nranks; ++j) {
        chunk = (rank + nranks - j) % nranks;
        offset = chunk * ChunkSize;
        // nelem = min(ChunkSize, size - offset);
        printf("recvReduceCopySend1");
        prims.recvReduceSend(offset, nelem);
    }

    // step k-1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    chunk = rank + 0;
    offset = chunk * ChunkSize;
    // nelem = min(ChunkSize, size - offset);
    printf("recvReduceCopySend2\n");
    printf("offset: %ld, nelem: %d", offset, nelem);
    prims.recvReduceCopySend(offset, offset, nelem,
                         /*postOp=*/true);
    // prims.recv(offset, nelem,
    //                       /*postOp=*/true);
    // return;
    // k-2 steps: copy to next GPU
    for (int j = 1; j < nranks - 1; ++j) {
        chunk = (rank + nranks - j) % nranks;
        offset = chunk * ChunkSize;
        // nelem = min(ChunkSize, size - offset);
        printf("recvCopySend");
        prims.recvCopySend(offset, nelem);
    }

    // Make final copy from buffer to dest.
    chunk = (rank + 1) % nranks;
    offset = chunk * ChunkSize;
    // nelem = min(ChunkSize, size - offset);
    printf("recv\n");
    prims.recv(offset, nelem);
}

void print_usage(const char *prog) { printf("usage: %s IP:PORT\n", prog); }

int main(int argc, const char *argv[])
{
    if (argc != 2) {
        print_usage(argv[0]);
        return -1;
    }

    MPI_Init(NULL, NULL);

    int rank;
    int world_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    mscclppComm_t comm;
    const char *ip_port = argv[1];
    MSCCLPPCHECK(mscclppCommInitRank(&comm, world_size, rank, ip_port));

    float *data_src;
    float *data_dst;
    char *recvbuff;
    int elem_num = 1024;

    int data_size = sizeof(float) * elem_num;

    int *flag_d;
    CUDACHECK(hipMalloc(&data_src, data_size));
    float *h_data_src = (float *)malloc(data_size);
    for (int i = 0; i < elem_num; ++i) {
        h_data_src[i] = i % 23;
    }
    CUDACHECK(
        hipMemcpy(data_src, h_data_src, data_size, hipMemcpyHostToDevice));
    // mscclppBootStrapAllGather(comm, data_src, data_size);
    CUDACHECK(hipMalloc(&data_dst, data_size));
    CUDACHECK(hipMalloc(&recvbuff, 2 * data_size));
    CUDACHECK(hipMalloc(&flag_d, sizeof(int)));

    mscclppResult_t res;
    int tag = 0;
    int rank_next = (rank + 1) % world_size;
    int rank_prev = (rank + world_size - 1) % world_size;
    // in the ring all reduce, we need to connect to the next and previous GPU
    MSCCLPPCHECK(mscclppConnect(comm, rank_next, rank, data_src, data_size,
                                flag_d, tag, mscclppTransportP2P));
    MSCCLPPCHECK(mscclppConnect(comm, rank, rank_prev, recvbuff, data_size,
                                flag_d, tag, mscclppTransportP2P));
    MSCCLPPCHECK(mscclppConnectionSetup(comm));

    mscclppDevConn_t devConns;
    MSCCLPPCHECK(mscclppGetDevConns(comm, &devConns));
    printf("data_src: %p, data_dst: %p, recvbuff %p\n", data_src, data_dst,
           recvbuff);
    ring_all_reduce<<<1, 32>>>(devConns, rank, world_size, data_src, data_dst,
                               recvbuff, elem_num);
    CUDACHECK(hipDeviceSynchronize());
    float *h_data_dst = (float *)malloc(data_size);
    CUDACHECK(
        hipMemcpy(h_data_dst, data_dst, data_size, hipMemcpyDeviceToHost));
    for (int i = 0; i < elem_num; ++i) {
        if (h_data_dst[i] != world_size * (i % 23)) {
            printf("data_dst[%d] = %f, expected %f", i, h_data_dst[i],
                   (i % 23));
            return -1;
        }
    }
    MSCCLPPCHECK(mscclppCommDestroy(comm));
    MPI_Finalize();

    printf("Succeeded! %d\n", rank);
    return 0;
}
