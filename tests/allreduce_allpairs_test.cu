#include "hip/hip_runtime.h"
#include "mscclpp.h"
#include <cuda/barrier>
#include <tuple>
#include <vector>

#include "common.h"

#define ALIGN 4

__constant__ mscclppDevConn_t constDevConns[16];

struct Volume
{
  size_t offset;
  size_t size;
};

__host__ __device__ Volume chunkVolume(size_t totalSize, size_t totalChunks, size_t chunkIdx, size_t chunkCount)
{
  size_t remainder = totalSize % totalChunks;
  size_t smallChunk = totalSize / totalChunks;
  size_t largeChunk = smallChunk + 1;
  size_t numLargeChunks = chunkIdx < remainder ? remainder - chunkIdx : 0;
  size_t numSmallChunks = chunkCount - numLargeChunks;
  size_t offset =
    (remainder - numLargeChunks) * largeChunk + (chunkIdx > remainder ? chunkIdx - remainder : 0) * smallChunk;
  return Volume{offset, numLargeChunks * largeChunk + numSmallChunks * smallChunk};
}

template <typename T, void (*reduce)(T*, T*, size_t)> struct AllreduceAllpairs
{
  int rank;
  int nRanks;
  T* userData;
  size_t userSize;
  T* scratch;
  size_t scratchSize;
  mscclppDevConn_t* conns;
  uint64_t* connFlags;
  cuda::barrier<cuda::thread_scope_device>* barrier;
  typedef T valueType;

  __device__ void run(int idx)
  {
    int myPeer = peerRank(idx, rank);
    mscclppDevConn_t phase1SendConn = conns[phase1SendConnIdx(myPeer)];
    mscclppDevConn_t phase1RecvConn = conns[phase1RecvConnIdx(myPeer)];
    mscclppDevConn_t phase2Conn = conns[phase2ConnIdx(myPeer)];

    // 1st communication phase: send data to the scratch buffer of the peer associated with this block
    Volume toPeer = chunkVolume(userSize, nRanks, myPeer, 1);
    // Now we need to figure out the offset of this chunk in the scratch buffer of the destination.
    // The destination will have allocated a scratch buffer of size numPeers() * toPeer.size and
    // inside that each of the destination's peers send to the nth chunk, where n is the index of the
    // source peer from the destination's perspective.
    size_t dstOffset = peerIdx(rank, myPeer) * toPeer.size;
    send(phase1SendConn, toPeer.offset, dstOffset, toPeer.size);
    recv(phase1RecvConn);

    if (threadIdx.x == 0)
      barrier->arrive_and_wait();
    __syncthreads();

    // Local reduction: every block reduces a slice of each chunk in the scratch buffer into the user buffer
    Volume rankUserChunk = chunkVolume(userSize, nRanks, rank, 1);
    T* userChunk = userData + rankUserChunk.offset;
    Volume blockUserChunk = chunkVolume(rankUserChunk.size, numBlocks(), idx, 1);
    for (int peerIdx = 0; peerIdx < numPeers(); ++peerIdx) {
      assert(scratchSize % numPeers() == 0);
      assert(scratchSize / numPeers() == rankUserChunk.size);
      size_t scratchChunkSize = scratchSize / numPeers();
      T* scratchChunk = scratch + peerIdx * scratchChunkSize;
      Volume blockScratchChunk = chunkVolume(scratchChunkSize, numBlocks(), idx, 1);
      assert(blockScratchChunk.size == blockUserChunk.size);
      reduce(userChunk + blockUserChunk.offset, scratchChunk + blockScratchChunk.offset, blockScratchChunk.size);
    }

    if (threadIdx.x == 0)
      barrier->arrive_and_wait();
    __syncthreads();

    // 2nd communication phase: send the now reduced data between the user buffers
    Volume srcVolume2 = chunkVolume(userSize, nRanks, rank, 1);
    send(phase2Conn, srcVolume2.offset, srcVolume2.offset, srcVolume2.size);
    recv(phase2Conn);
  }

  __device__ void send(mscclppDevConn_t& conn, size_t srcOffset, size_t dstOffset, size_t size)
  {
    if (threadIdx.x == 0) {
      conn.putWithSignalAndFlush(dstOffset * sizeof(T), srcOffset * sizeof(T), size * sizeof(T));
    }
    __syncthreads();
  }

  __device__ void recv(mscclppDevConn_t& conn)
  {
    if (threadIdx.x == 0) {
      conn.wait();
    }
    __syncthreads();
  }

  __host__ __device__ int numPeers()
  {
    return nRanks - 1;
  }

  __host__ __device__ int numBlocks()
  {
    return numPeers();
  }

  __host__ __device__ int peerIdx(int peerRank, int myRank)
  {
    return peerRank < myRank ? peerRank : peerRank - 1;
  }

  __host__ __device__ int peerRank(int peerIdx, int myRank)
  {
    return peerIdx < myRank ? peerIdx : peerIdx + 1;
  }

  __host__ __device__ int phase1SendConnIdx(int peerRank)
  {
    return peerIdx(peerRank, rank) * 3;
  }

  __host__ __device__ int phase1RecvConnIdx(int peerRank)
  {
    return peerIdx(peerRank, rank) * 3 + 1;
  }

  __host__ __device__ int phase2ConnIdx(int peerRank)
  {
    return peerIdx(peerRank, rank) * 3 + 2;
  }

  void freeGPUResources()
  {
    if (scratch)
      CUDACHECK(hipFree(scratch));
    scratch = nullptr;
    if (connFlags)
      CUDACHECK(hipFree(connFlags));
    connFlags = nullptr;
    if (conns)
      CUDACHECK(hipFree(conns));
    conns = nullptr;
    if (barrier)
      CUDACHECK(hipFree(barrier));
    barrier = nullptr;
  }
};

// The builder class encapsulates the
template <class T, void (*reduce)(T*, T*, size_t)> class AllreduceAllpairsBuilder
{
  AllreduceAllpairs<T, reduce> d;
  std::vector<mscclppDevConn_t> hostConns;

public:
  // The constructor is called after the user has allocated the buffer to be allreduced
  AllreduceAllpairsBuilder(T* data, size_t size)
  {
    d.userData = data;
    d.userSize = size;
    d.scratch = nullptr;
    d.connFlags = nullptr;
    d.conns = nullptr;
    d.barrier = nullptr;
  }

  // connect is called after rank initialization but before connection setup
  mscclppResult_t connect(mscclppComm_t comm)
  {
    MSCCLPPCHECK(mscclppCommRank(comm, &d.rank));
    MSCCLPPCHECK(mscclppCommSize(comm, &d.nRanks));

    Volume myChunks = chunkVolume(d.userSize, d.nRanks, d.rank, 1);
    d.scratchSize = myChunks.size * d.numPeers();

    CUDACHECK(hipMalloc(&d.scratch, d.scratchSize * sizeof(T)));
    CUDACHECK(hipMalloc(&d.connFlags, 3 * sizeof(uint64_t)));
    CUDACHECK(hipMemset(d.connFlags, 0, 3 * sizeof(uint64_t)));

    hostConns.resize(d.numPeers() * 3);
    for (int peer = 0; peer < d.nRanks; ++peer) {
      if (peer != d.rank) {
        int sendTag = d.rank < peer ? 0 : 1;
        int recvTag = d.rank < peer ? 1 : 0;
        MSCCLPPCHECK(mscclppConnect(comm, hostConns.data() + d.phase1SendConnIdx(peer), peer, d.userData,
                                    d.userSize * sizeof(T), d.connFlags + 0, sendTag, mscclppTransportP2P, nullptr));
        MSCCLPPCHECK(mscclppConnect(comm, hostConns.data() + d.phase1RecvConnIdx(peer), peer, d.scratch,
                                    d.scratchSize * sizeof(T), d.connFlags + 1, recvTag, mscclppTransportP2P, nullptr));
        MSCCLPPCHECK(mscclppConnect(comm, hostConns.data() + d.phase2ConnIdx(peer), peer, d.userData,
                                    d.userSize * sizeof(T), d.connFlags + 2, 2, mscclppTransportP2P, nullptr));
      }
    }

    return mscclppSuccess;
  }

  // finishSetup is called after connection setup and returns an algorithm object that is ready to be passed to a GPU
  // kernel
  AllreduceAllpairs<T, reduce> finishSetup()
  {
    CUDACHECK(hipMalloc(&d.conns, hostConns.size() * sizeof(mscclppDevConn_t)));
    CUDACHECK(
      hipMemcpy(d.conns, hostConns.data(), hostConns.size() * sizeof(mscclppDevConn_t), hipMemcpyHostToDevice));
    CUDACHECK(hipMalloc(&d.barrier, sizeof(cuda::barrier<cuda::thread_scope_device>)));
    cuda::barrier<cuda::thread_scope_device> initBarrier(d.numBlocks());
    CUDACHECK(
      hipMemcpy(d.barrier, &initBarrier, sizeof(cuda::barrier<cuda::thread_scope_device>), hipMemcpyHostToDevice));
    return d;
  }
};

template <class T> __device__ void reduceSum(T* dst, T* src, size_t size)
{
  for (int i = threadIdx.x; i < size; i += blockDim.x) {
    dst[i] += src[i];
  }
}

template <class T> __global__ void init(T* data, size_t size, int rank)
{
  for (int i = threadIdx.x; i < size; i += blockDim.x) {
    data[i] = rank;
  }
}

// The main test kernel
template <class T> __global__ void kernel0(AllreduceAllpairs<T, reduceSum> d)
{
  d.run(blockIdx.x);
}

void AllReduceGetCollByteCount(size_t* sendcount, size_t* recvcount, size_t* paramcount, size_t* sendInplaceOffset,
                               size_t* recvInplaceOffset, size_t count, int nranks)
{
  size_t base = (count / (ALIGN * nranks)) * ALIGN;
  *sendcount = base;
  *recvcount = base * nranks;
  *sendInplaceOffset = base;
  *recvInplaceOffset = 0;
  *paramcount = base;
}

void AllReduceGetBuffSize(size_t* sendcount, size_t* recvcount, size_t count, int nranks)
{
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  AllReduceGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

struct testColl allReduceTest = {"AllReduce", AllGatherGetCollByteCount, AllGatherInitData, AllGatherGetBw,
                                 AllGatherRunColl};

testResult_t AllReduceSetupMscclppConnections(struct testArgs* args)
{
  Volume myChunks = chunkVolume(args->nbytes, args->totalProcs, args->proc, 1);
  d.scratchSize = myChunks.size * d.numPeers();

  CUDACHECK(hipMalloc(&d.scratch, d.scratchSize * sizeof(T)));
  CUDACHECK(hipMalloc(&d.connFlags, 3 * sizeof(uint64_t)));
  CUDACHECK(hipMemset(d.connFlags, 0, 3 * sizeof(uint64_t)));

  hostConns.resize(d.numPeers() * 3);
  for (int peer = 0; peer < d.nRanks; ++peer) {
    if (peer != d.rank) {
      int sendTag = d.rank < peer ? 0 : 1;
      int recvTag = d.rank < peer ? 1 : 0;
      MSCCLPPCHECK(mscclppConnect(args->comm, hostConns.data() + d.phase1SendConnIdx(peer), peer, d.userData,
                                  d.userSize * sizeof(T), d.connFlags + 0, sendTag, mscclppTransportP2P, nullptr));
      MSCCLPPCHECK(mscclppConnect(args->comm, hostConns.data() + d.phase1RecvConnIdx(peer), peer, d.scratch,
                                  d.scratchSize * sizeof(T), d.connFlags + 1, recvTag, mscclppTransportP2P, nullptr));
      MSCCLPPCHECK(mscclppConnect(args->comm, hostConns.data() + d.phase2ConnIdx(peer), peer, d.userData,
                                  d.userSize * sizeof(T), d.connFlags + 2, 2, mscclppTransportP2P, nullptr));
    }
  }

  return testSuccess;
}

testResult_t AllReduceRunTest(struct testArgs* args)
{
  args->collTest = &allReduceTest;
  mscclppDevConn_t* devConns;
  int nCons;
  MSCCLPPCHECK(mscclppGetAllDeviceConnections(args->comm, &devConns, &nCons));
  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(constDevConns), devConns, sizeof(mscclppDevConn_t) * nCons));
  TESTCHECK(TimeTest(args));
  return testSuccess;
}

struct testEngine allReduceEngine = {AllReduceGetBuffSize, AllReduceRunTest, AllReduceSetupMscclppConnections};

#pragma weak mscclppTestEngine = allReduceEngine