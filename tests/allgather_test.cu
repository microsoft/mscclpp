#include "hip/hip_runtime.h"
#include "mscclpp.h"
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
#include "mpi.h"
#endif // MSCCLPP_USE_MPI_FOR_TESTS
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string>

#define RANKS_PER_NODE 8
#define USE_DMA_FOR_P2P 1

#define MSCCLPPCHECK(call) do { \
  mscclppResult_t res = call; \
  if (res != mscclppSuccess && res != mscclppInProgress) { \
    /* Print the back trace*/ \
    printf("Failure at %s:%d -> %d\n", __FILE__, __LINE__, res);    \
    return res; \
  } \
} while (0);

// Check CUDA RT calls
#define CUDACHECK(cmd) do {                                   \
    hipError_t err = cmd;                                    \
    if( err != hipSuccess ) {                                \
        printf("%s:%d Cuda failure '%s'\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
} while(false)

// Measure current time in second.
static double getTime(void)
{
  struct timespec tspec;
  if (clock_gettime(CLOCK_MONOTONIC, &tspec) == -1) {
    printf("clock_gettime failed\n");
    exit(EXIT_FAILURE);
  }
  return (tspec.tv_nsec / 1.0e9) + tspec.tv_sec;
}

__constant__ mscclppDevConn_t constDevConns[16];

__global__ void kernel(int rank, int world_size, int nelemsPerGPU)
{
  if (threadIdx.x % 32 != 0) return;

  int warpId = threadIdx.x / 32;
  int remoteRank = (warpId < rank) ? warpId : warpId + 1;
  mscclppDevConn_t devConn = constDevConns[remoteRank];
  // volatile int *data = (volatile int *)devConn.localBuff;
  volatile uint64_t *localFlag = devConn.localFlag;
#if (USE_DMA_FOR_P2P == 0)
  volatile uint64_t *remoteFlag = devConn.remoteFlag;
#endif
  volatile uint64_t *proxyFlag = devConn.proxyFlag;
  mscclppTrigger *trig = devConn.getTrigger();

  uint64_t baseFlag = *localFlag;

  __syncthreads();
  if (threadIdx.x == 0) {
    // Do we need a sys fence?
    // __threadfence_system();
    *localFlag = baseFlag + 1;
  }

  // Each warp receives data from different ranks
#if (USE_DMA_FOR_P2P == 1)

  // Prevent overwriting trigger
  devConn.waitTrigger(trig);

  // Trigger sending data and flag
  devConn.setTrigger(trig, mscclppFlag | mscclppData | mscclppSync, rank * nelemsPerGPU * sizeof(int), nelemsPerGPU*sizeof(int));

  // Wait until the proxy have sent my data and flag
  devConn.waitTrigger(trig);

  // Wait for receiving data from remote rank
  while (*proxyFlag == baseFlag) {}

#else // USE_DMA_FOR_P2P == 0

  if (devConn.remoteBuff == NULL) { // IB
    // Wait until the proxy have sent my data and flag
    devConn.waitTrigger(trig);

    // Trigger sending data and flag
    devConn.setTrigger(trig, mscclppFlag | mscclppData, rank * nelemsPerGPU * sizeof(int), nelemsPerGPU*sizeof(int));

    // Wait for receiving data from remote rank
    while (*proxyFlag == baseFlag) {}
  } else { // P2P
    // Directly read data
    volatile int *remoteData = (volatile int *)devConn.remoteBuff;

    // Wait until the remote data is set
    while (*remoteFlag == baseFlag) {}

    // Read remote data
    data[remoteRank] = remoteData[remoteRank];
  }

#endif
}

int rankToLocalRank(int rank)
{
  return rank % RANKS_PER_NODE;
}

int rankToNode(int rank)
{
  return rank / RANKS_PER_NODE;
}

int cudaNumToIbNum(int cudaNum)
{
  int ibNum;
  if (cudaNum == 0) {
    ibNum = 0;
  } else if (cudaNum == 1) {
    ibNum = 4;
  } else if (cudaNum == 2) {
    ibNum = 1;
  } else if (cudaNum == 3) {
    ibNum = 5;
  } else if (cudaNum == 4) {
    ibNum = 2;
  } else if (cudaNum == 5) {
    ibNum = 6;
  } else if (cudaNum == 6) {
    ibNum = 3;
  } else if (cudaNum == 7) {
    ibNum = 7;
  } else {
    printf("Invalid cudaNum: %d\n", cudaNum);
    exit(EXIT_FAILURE);
  }
  return ibNum;
}

void print_usage(const char *prog)
{
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  printf("usage: %s IP:PORT [rank nranks]\n", prog);
#else
  printf("usage: %s IP:PORT rank nranks\n", prog);
#endif
}

int main(int argc, const char *argv[])
{
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  if (argc != 2 && argc != 4) {
    print_usage(argv[0]);
    return -1;
  }
  const char *ip_port = argv[1];
  int rank;
  int world_size;
  if (argc == 4) {
    rank = atoi(argv[2]);
    world_size = atoi(argv[3]);
  } else {
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
  }
#else
  if (argc != 4) {
    print_usage(argv[0]);
    return -1;
  }
  const char *ip_port = argv[1];
  int rank = atoi(argv[2]);
  int world_size = atoi(argv[3]);
#endif
  int localRank = rankToLocalRank(rank);
  int thisNode = rankToNode(rank);
  int cudaNum = localRank;
  int ibNum = cudaNumToIbNum(cudaNum);

  CUDACHECK(hipSetDevice(cudaNum));
  std::string ibDevStr = "mlx5_ib" + std::to_string(ibNum);

  mscclppComm_t comm;
  MSCCLPPCHECK(mscclppCommInitRank(&comm, world_size, rank, ip_port));

  int *data_d;
  uint64_t *flag_d;
  size_t data_size = 1024*1024*16;
  int nelemsPerGPU = data_size / sizeof(int) / world_size;
  CUDACHECK(hipMalloc(&data_d, data_size));
  CUDACHECK(hipMalloc(&flag_d, sizeof(uint64_t)));
  CUDACHECK(hipMemset(data_d, 0, data_size));
  CUDACHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));

  int* data_h = new int[nelemsPerGPU*world_size];
  for (int i = 0; i < nelemsPerGPU*world_size; i++){
    int val = i + 1;
    if (i / nelemsPerGPU == rank){
      data_h[i] = val;
    } else {
      data_h[i] = 0;
    }
  }
  CUDACHECK(hipMemcpy(data_d, data_h, data_size, hipMemcpyHostToDevice));

  mscclppDevConn_t devConns[16];
  for (int r = 0; r < world_size; ++r) {
    if (r == rank) continue;
    mscclppTransport_t transportType;
    const char* ibDev = ibDevStr.c_str();
    if (rankToNode(r) == thisNode){
      ibDev = NULL;
      transportType = mscclppTransportP2P;
    } else {
      transportType = mscclppTransportIB;
    }
    // Connect with all other ranks
    MSCCLPPCHECK(mscclppConnect(comm, &devConns[r], r, data_d, data_size, flag_d, 0, transportType, ibDev));
  }

  MSCCLPPCHECK(mscclppConnectionSetup(comm));

  MSCCLPPCHECK(mscclppProxyLaunch(comm));

  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(constDevConns), devConns, sizeof(mscclppDevConn_t) * world_size));

  hipStream_t stream;
  CUDACHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));


  CUDACHECK(hipDeviceSynchronize());
  kernel<<<1, 32 * (world_size - 1), 0, stream>>>(rank, world_size, nelemsPerGPU);
  CUDACHECK(hipDeviceSynchronize());
  CUDACHECK(hipMemcpy(data_h, data_d, data_size, hipMemcpyDeviceToHost));
  CUDACHECK(hipDeviceSynchronize());

  for (int i = 0; i < nelemsPerGPU*world_size; i++){
    int val = i + 1;
    if (data_h[i] != val){
      printf("oh uh things went wrong! data_h[%d] (%d) != val (%d)\n", i, data_h[i], val);
      break;
    }
  }
  int tmp[16];
  MSCCLPPCHECK(mscclppBootStrapAllGather(comm, tmp, sizeof(int)));

//   // Perf test
//   hipEvent_t ev_start;
//   hipEvent_t ev_end;
//   CUDACHECK(hipEventCreate(&ev_start));
//   CUDACHECK(hipEventCreate(&ev_end));

  // warm up
  // int warmupiter = 1000;
  // for (int i = 0; i < warmupiter; ++i) {
  //   kernel<<<1, 32 * (world_size - 1), 0, stream>>>(rank, world_size, nelemsPerGPU);
  // }
  // CUDACHECK(hipDeviceSynchronize());
  // MSCCLPPCHECK(mscclppBootStrapAllGather(comm, tmp, sizeof(int)));

  // cudaGraph Capture
  hipGraph_t graph;
  hipGraphExec_t instance;
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  int cudagraphiter = 10;
  for (int i = 0; i < cudagraphiter; ++i) {
  	kernel<<<1, 32 * (world_size - 1), 0, stream>>>(rank, world_size, nelemsPerGPU);
  }
  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

  int cudagraphwarmup = 10;
  for (int i = 0; i < cudagraphwarmup; ++i) {
	  hipGraphLaunch(instance, stream);
  }
  CUDACHECK(hipStreamSynchronize(stream));

  // measure runtime 
//  CUDACHECK(hipEventRecord(ev_start, stream));
  double t0 = getTime();
  int cudagraphlaunch = 10;
  for (int i = 0; i < cudagraphlaunch; ++i) {
  // kernel<<<1, 32 * (world_size - 1), 0, stream>>>(rank, world_size);
     hipGraphLaunch(instance, stream);
  }
//  CUDACHECK(hipEventRecord(ev_end, stream));
  CUDACHECK(hipStreamSynchronize(stream));

  double t1 = getTime();
  float ms = (t1-t0)*1000.0;
//  CUDACHECK(hipEventElapsedTime(&ms, ev_start, ev_end));
  double time_in_us = ms * 1000. / (float) cudagraphlaunch / (float) cudagraphiter;
  printf("rank: %d, time: %f us/iter algBW %f\n", rank, time_in_us, (double) (data_size) / 1024./1024./1024./(time_in_us/1e6));

  MSCCLPPCHECK(mscclppProxyStop(comm));

  MSCCLPPCHECK(mscclppCommDestroy(comm));

#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  if (argc == 2) {
    MPI_Finalize();
  }
#endif
  printf("Succeeded! %d\n", rank);
  return 0;
}
