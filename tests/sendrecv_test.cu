#include "hip/hip_runtime.h"
#include "comm.h"
#include "common.h"

#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <unistd.h>

constexpr size_t BLOCK_THREADS_NUM = 1024;
// Try to use more blocks if per-block data size exceeds this threshold
constexpr size_t THRES_BYTES_PER_BLOCK = 8192;
// Let it no more than the number of SMs on a GPU
constexpr size_t MAX_BLOCKS_NUM = 32;

#define ALIGN 4

__constant__ mscclppDevConn_t sendConnConst;
__constant__ mscclppDevConn_t recvConnConst;

struct SyncGpuState
{
  volatile int flag;
  int cnt;
  int is_add;
};

// Synchronize multiple thread blocks inside a kernel. Guarantee that all
// previous work of all threads in cooperating blocks is finished and
// visible to all threads in the device.
__forceinline__ __device__ void sync_gpu(SyncGpuState& state, int blockNum)
{
  int maxOldCnt = blockNum - 1;
  __syncthreads();
  if (threadIdx.x == 0) {
    int is_add_ = state.is_add ^ 1;
    if (is_add_) {
      if (atomicAdd(&state.cnt, 1) == maxOldCnt) {
        state.flag = 1;
      }
      while (!state.flag) {
      }
    } else {
      if (atomicSub(&state.cnt, 1) == 1) {
        state.flag = 0;
      }
      while (state.flag) {
      }
    }
    state.is_add = is_add_;
  }
  // We need sync here because only a single thread is checking whether
  // the flag is flipped.
  __syncthreads();
}

inline int getSendTag(int rank, int peer)
{
  return rank < peer ? 0 : 1;
}

inline int getRecvTag(int rank, int peer)
{
  return rank < peer ? 1 : 0;
}

inline int getBlockNum(size_t count)
{
  return std::min((count + THRES_BYTES_PER_BLOCK - 1) / THRES_BYTES_PER_BLOCK, MAX_BLOCKS_NUM);
}

__device__ SyncGpuState GLOBAL_SYNC_STATE;

__global__ void kernel(int rank, size_t dataSize, size_t dataPerBlock)
{
  mscclppDevConn_t sendConn = sendConnConst;
  mscclppDevConn_t recvConn = recvConnConst;
  size_t startIndex = blockIdx.x * dataPerBlock;
  size_t blockDataSize = min(dataSize - startIndex, dataPerBlock);
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  sendConn.putDirect(startIndex, blockDataSize, threadIdx.x, blockDim.x);
  sync_gpu(GLOBAL_SYNC_STATE, gridDim.x);
  if (tid == 0) {
    sendConn.signalDirect();
    recvConn.waitDirect();
  }
}

void SendRecvGetCollByteCount(size_t* sendcount, size_t* recvcount, size_t* paramcount, size_t* sendInplaceOffset,
                              size_t* recvInplaceOffset, size_t count, int nranks)
{
  size_t base = (count / ALIGN) * ALIGN;
  *sendcount = base;
  *recvcount = base;
  *sendInplaceOffset = base;
  *recvInplaceOffset = 0;
  *paramcount = base;
}

testResult_t SendRecvInitColl()
{
  SyncGpuState state = {};
  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(GLOBAL_SYNC_STATE), &state, sizeof(SyncGpuState)));
  return testSuccess;
}

testResult_t SendRecvInitData(struct testArgs* args, int in_place)
{
  size_t sendCount = args->sendBytes / sizeof(int);
  size_t recvCount = args->expectedBytes / sizeof(int);
  size_t maxCount = std::max(sendCount, recvCount);

  int rank = args->proc;
  CUDACHECK(hipMemset(args->sendbuff, 0, args->sendBytes));
  std::vector<int> dataHost(maxCount, rank);
  CUDACHECK(hipMemcpy(args->sendbuff, dataHost.data(), sendCount * sizeof(int), hipMemcpyHostToDevice));

  int recvPeerRank = (rank - 1 + args->totalProcs) % args->totalProcs;
  for (size_t i = 0; i < recvCount; i++) {
    dataHost[i] = recvPeerRank;
  }
  CUDACHECK(hipMemcpy(args->expected, dataHost.data(), recvCount * sizeof(int), hipMemcpyHostToDevice));
  MSCCLPPCHECK(mscclppBootstrapBarrier(args->comm));

  return testSuccess;
}

void SendRecvGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks)
{
  double baseBw = (double)(count * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = 1;
  *busBw = baseBw * factor;
}

testResult_t SendRecvRunColl(void* sendbuff, void* recvbuff, int nranksPerNode, size_t count, mscclppComm_t comm,
                             hipStream_t stream, int kernel_num)
{
  int blockNum = getBlockNum(count);
  size_t bytesPerBlock = (count + blockNum - 1) / blockNum;
  kernel<<<blockNum, BLOCK_THREADS_NUM, 0, stream>>>(comm->rank, count, bytesPerBlock);
  return testSuccess;
}

struct testColl sendRecvTest = {"SendRecvTest", SendRecvGetCollByteCount, SendRecvInitColl, SendRecvInitData,
                                SendRecvGetBw,  SendRecvRunColl};

void SendRecvGetBuffSize(size_t* sendcount, size_t* recvcount, size_t count, int nranks)
{
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  SendRecvGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t SendRecvSetupConnections(struct testArgs* args)
{
  int rank = args->proc;
  int worldSize = args->totalProcs;
  int ranksPerNode = args->nranksPerNode;
  int thisNode = rank / ranksPerNode;
  int localRank = rank % ranksPerNode;
  std::string ibDevStr = "mlx5_ib" + std::to_string(localRank);
  int sendToRank = (rank + 1) % worldSize;
  int recvFromRank = (rank - 1 + worldSize) % worldSize;
  std::array<int, 2> ranks = {sendToRank, recvFromRank};

  for (int i = 0; i < 2; i++) {
    int r = ranks[i];
    const char* ibDev = r / ranksPerNode == thisNode ? nullptr : ibDevStr.c_str();
    mscclppTransport_t transportType = ibDev == nullptr ? mscclppTransportP2P : mscclppTransportIB;
    void* buff = (i == 0) ? args->sendbuff : args->recvbuff;
    int tag = (i == 0) ? getSendTag(rank, r) : getRecvTag(rank, r);
    MSCCLPPCHECK(mscclppConnect(args->comm, r, tag, buff, args->maxbytes, transportType, ibDev));
  }
  MSCCLPPCHECK(mscclppConnectionSetup(args->comm));

  return testSuccess;
}

testResult_t SendRecvRunTest(struct testArgs* args)
{
  args->collTest = &sendRecvTest;
  int rank = args->proc, worldSize = args->totalProcs;

  // only support out-of-place for sendrecv test
  args->in_place = 0;

  mscclppDevConn_t* sendDevConn;
  mscclppDevConn_t* recvDevConn;
  MSCCLPPCHECK(mscclppGetDeviceConnection(args->comm, (rank + 1) % worldSize, getSendTag(rank, (rank + 1) % worldSize),
                                          &sendDevConn));
  MSCCLPPCHECK(mscclppGetDeviceConnection(args->comm, (rank - 1 + worldSize) % worldSize,
                                          getRecvTag(rank, (rank - 1 + worldSize) % worldSize), &recvDevConn));
  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(sendConnConst), sendDevConn, sizeof(mscclppDevConn_t)));
  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(recvConnConst), recvDevConn, sizeof(mscclppDevConn_t)));
  TESTCHECK(TimeTest(args));
  return testSuccess;
}

struct testEngine sendRecvTestEngine = {SendRecvGetBuffSize, SendRecvRunTest, SendRecvSetupConnections, nullptr};

#pragma weak mscclppTestEngine = sendRecvTestEngine
