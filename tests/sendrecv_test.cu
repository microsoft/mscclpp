#include "hip/hip_runtime.h"
#include "comm.h"
#include "common.h"

#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <unistd.h>

#include <cuda/barrier>

constexpr int BLOCK_THREADS_NUM = 128;
constexpr int MAX_BLOCKS_NUM = 1024;
constexpr int BYTES_SEND_PER_THREAD = 8;
constexpr int DEFAULT_BYTES_PER_BLOCK = BLOCK_THREADS_NUM * BYTES_SEND_PER_THREAD * 2; // loop twice

#define ALIGN 4

__constant__ mscclppDevConn_t sendConnConst;
__constant__ mscclppDevConn_t recvConnConst;

cuda::barrier<cuda::thread_scope_device>* barrier;

inline int getSendTag(int rank, int peer)
{
  return rank < peer ? 0 : 1;
}

inline int getRecvTag(int rank, int peer)
{
  return rank < peer ? 1 : 0;
}

inline int getBlockNum(size_t count)
{
  return std::min((count + DEFAULT_BYTES_PER_BLOCK - 1) / DEFAULT_BYTES_PER_BLOCK,
                  static_cast<size_t>(MAX_BLOCKS_NUM));
}

__global__ void kernel(int rank, size_t dataSize, size_t dataPerBlock, cuda::barrier<cuda::thread_scope_device>* barrier)
{
  mscclppDevConn_t sendConn = sendConnConst;
  mscclppDevConn_t recvConn = recvConnConst;
  size_t startIndex = blockIdx.x * dataPerBlock;
  size_t blockDataSize = min(dataSize - startIndex, dataPerBlock);
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  sendConn.putDirect(startIndex, blockDataSize, threadIdx.x, blockDim.x);
  if (threadIdx.x == 0)
    barrier->arrive_and_wait();
  __syncthreads();
  if (tid == 0) {
    sendConn.signalDirect();
    recvConn.waitDirect();
  }
}

void SendRecvGetCollByteCount(size_t* sendcount, size_t* recvcount, size_t* paramcount, size_t* sendInplaceOffset,
                              size_t* recvInplaceOffset, size_t count, int nranks)
{
  size_t base = (count / ALIGN) * ALIGN;
  *sendcount = base;
  *recvcount = base;
  *sendInplaceOffset = base;
  *recvInplaceOffset = 0;
  *paramcount = base;
  int blockNum = getBlockNum(count * sizeof(int));
  cuda::barrier<cuda::thread_scope_device> initBarrier(blockNum);
  hipMemcpy(barrier, &initBarrier, sizeof(cuda::barrier<cuda::thread_scope_device>), hipMemcpyHostToDevice);
}

testResult_t SendRecvInitData(struct testArgs* args, int in_place)
{
  size_t sendCount = args->sendBytes / sizeof(int);
  size_t recvCount = args->expectedBytes / sizeof(int);
  size_t maxCount = std::max(sendCount, recvCount);

  int rank = args->proc;
  CUDACHECK(hipMemset(args->sendbuff, 0, args->sendBytes));
  std::vector<int> dataHost(maxCount, rank);
  CUDACHECK(hipMemcpy(args->sendbuff, dataHost.data(), sendCount * sizeof(int), hipMemcpyHostToDevice));

  int recvPeerRank = (rank - 1 + args->totalProcs) % args->totalProcs;
  for (size_t i = 0; i < recvCount; i++) {
    dataHost[i] = recvPeerRank;
  }
  CUDACHECK(hipMemcpy(args->expected, dataHost.data(), recvCount * sizeof(int), hipMemcpyHostToDevice));
  MSCCLPPCHECK(mscclppBootstrapBarrier(args->comm));

  return testSuccess;
}

void SendRecvGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks)
{
  double baseBw = (double)(count * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = 1;
  *busBw = baseBw * factor;
}

testResult_t SendRecvRunColl(void* sendbuff, void* recvbuff, int nranksPerNode, size_t count, mscclppComm_t comm,
                             hipStream_t stream, int kernel_num)
{
  int blockNum =
    std::min((count + DEFAULT_BYTES_PER_BLOCK - 1) / DEFAULT_BYTES_PER_BLOCK, static_cast<size_t>(MAX_BLOCKS_NUM));
  size_t bytesPerBlock = (count + blockNum - 1) / blockNum;
  kernel<<<blockNum, BLOCK_THREADS_NUM, 0, stream>>>(comm->rank, count, bytesPerBlock, barrier);
  return testSuccess;
}

struct testColl sendRecvTest = {"SendRecvTest", SendRecvGetCollByteCount, SendRecvInitData, SendRecvGetBw,
                                SendRecvRunColl};

void SendRecvGetBuffSize(size_t* sendcount, size_t* recvcount, size_t count, int nranks)
{
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  SendRecvGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t SendRecvSetupConnections(struct testArgs* args)
{
  int rank = args->proc;
  int worldSize = args->totalProcs;
  int ranksPerNode = args->nranksPerNode;
  int thisNode = rank / ranksPerNode;
  int localRank = rank % ranksPerNode;
  std::string ibDevStr = "mlx5_ib" + std::to_string(localRank);
  int sendToRank = (rank + 1) % worldSize;
  int recvFromRank = (rank - 1 + worldSize) % worldSize;
  std::array<int, 2> ranks = {sendToRank, recvFromRank};

  for (int i = 0; i < 2; i++) {
    int r = ranks[i];
    const char* ibDev = r / ranksPerNode == thisNode ? nullptr : ibDevStr.c_str();
    mscclppTransport_t transportType = ibDev == nullptr ? mscclppTransportP2P : mscclppTransportIB;
    void* buff = (i == 0) ? args->sendbuff : args->recvbuff;
    int tag = (i == 0) ? getSendTag(rank, r) : getRecvTag(rank, r);
    MSCCLPPCHECK(mscclppConnect(args->comm, r, tag, buff, args->maxbytes, transportType, ibDev));
  }
  MSCCLPPCHECK(mscclppConnectionSetup(args->comm));

  return testSuccess;
}

testResult_t SendRecvRunTest(struct testArgs* args)
{
  args->collTest = &sendRecvTest;
  int rank = args->proc, worldSize = args->totalProcs;

  // only support out-of-place for sendrecv test
  args->in_place = 0;

  mscclppDevConn_t* sendDevConn;
  mscclppDevConn_t* recvDevConn;
  MSCCLPPCHECK(mscclppGetDeviceConnection(args->comm, (rank + 1) % worldSize, getSendTag(rank, (rank + 1) % worldSize),
                                          &sendDevConn));
  MSCCLPPCHECK(mscclppGetDeviceConnection(args->comm, (rank - 1 + worldSize) % worldSize,
                                          getRecvTag(rank, (rank - 1 + worldSize) % worldSize), &recvDevConn));
  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(sendConnConst), sendDevConn, sizeof(mscclppDevConn_t)));
  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(recvConnConst), recvDevConn, sizeof(mscclppDevConn_t)));
  CUDACHECK(hipMalloc(&barrier, sizeof(cuda::barrier<cuda::thread_scope_device>)));
  TESTCHECK(TimeTest(args));
  CUDACHECK(hipFree(barrier));
  return testSuccess;
}

struct testEngine sendRecvTestEngine = {SendRecvGetBuffSize, SendRecvRunTest, SendRecvSetupConnections, nullptr};

#pragma weak mscclppTestEngine = sendRecvTestEngine
