#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <iostream>
#include <mscclpp/core.hpp>
#include <mscclpp/gpu_utils.hpp>
#include <mscclpp/memory_channel.hpp>
#include <mscclpp/memory_channel_device.hpp>
#include <sstream>

template <typename... Args>
void log(Args &&...args) {
  std::stringstream ss;
  (ss << ... << args);
  ss << std::endl;
  std::cout << ss.str();
}

__device__ void spin_cycles(unsigned long long cycles) {
  unsigned long long start = clock64();
  while (clock64() - start < cycles) {
    // spin
  }
}

__global__ void gpuKernel0(mscclpp::BaseMemoryChannelDeviceHandle *devHandle, int iter) {
  if (threadIdx.x + blockIdx.x * blockDim.x == 0) {
    for (int i = 0; i < iter; ++i) {
      devHandle->relaxedWait();
      // spin for a few ms
      spin_cycles(1e7);
      devHandle->relaxedSignal();
    }
  }
}

__global__ void gpuKernel1(mscclpp::BaseMemoryChannelDeviceHandle *devHandle, int iter) {
  if (threadIdx.x + blockIdx.x * blockDim.x == 0) {
    for (int i = 0; i < iter; ++i) {
      devHandle->relaxedSignal();
      devHandle->relaxedWait();
    }
  }
}

int main() {
  // Optional: check if we have at least two GPUs
  int deviceCount;
  MSCCLPP_CUDATHROW(hipGetDeviceCount(&deviceCount));
  if (deviceCount < 2) {
    log("Error: At least two GPUs are required.");
    return 1;
  }

  // Optional: check if the two GPUs can peer-to-peer access each other
  int canAccessPeer;
  MSCCLPP_CUDATHROW(hipDeviceCanAccessPeer(&canAccessPeer, 0, 1));
  if (!canAccessPeer) {
    log("Error: GPU 0 cannot access GPU 1. Make sure that the GPUs are connected peer-to-peer. You can check this "
        "by running `nvidia-smi topo -m` (the connection between GPU 0 and 1 should be either NV# or PIX).");
    return 1;
  }

  const int iter = 100;
  const mscclpp::Transport transport = mscclpp::Transport::CudaIpc;

  log("Creating endpoints ...");

  auto ctx = mscclpp::Context::create();
  mscclpp::Endpoint ep0 = ctx->createEndpoint({transport, {mscclpp::DeviceType::GPU, 0}});
  mscclpp::Endpoint ep1 = ctx->createEndpoint({transport, {mscclpp::DeviceType::GPU, 1}});

  log("GPU 0: Creating a connection and a semaphore stub ...");

  MSCCLPP_CUDATHROW(hipSetDevice(0));
  std::shared_ptr<mscclpp::Connection> conn0 = ctx->connect(/*localEndpoint*/ ep0, /*remoteEndpoint*/ ep1);
  mscclpp::SemaphoreStub semaStub0(conn0);

  log("GPU 1: Creating a connection and a semaphore stub ...");

  MSCCLPP_CUDATHROW(hipSetDevice(1));
  std::shared_ptr<mscclpp::Connection> conn1 = ctx->connect(/*localEndpoint*/ ep1, /*remoteEndpoint*/ ep0);
  mscclpp::SemaphoreStub semaStub1(conn1);

  log("GPU 0: Creating a semaphore and a memory channel ...");

  MSCCLPP_CUDATHROW(hipSetDevice(0));
  mscclpp::Semaphore sema0(/*localSemaphoreStub*/ semaStub0, /*remoteSemaphoreStub*/ semaStub1);
  mscclpp::BaseMemoryChannel memChan0(sema0);
  mscclpp::BaseMemoryChannelDeviceHandle memChanHandle0 = memChan0.deviceHandle();
  void *devHandle0;
  MSCCLPP_CUDATHROW(hipMalloc(&devHandle0, sizeof(mscclpp::BaseMemoryChannelDeviceHandle)));
  MSCCLPP_CUDATHROW(hipMemcpy(devHandle0, &memChanHandle0, sizeof(memChanHandle0), hipMemcpyHostToDevice));

  log("GPU 1: Creating a semaphore and a memory channel ...");

  MSCCLPP_CUDATHROW(hipSetDevice(1));
  mscclpp::Semaphore sema1(/*localSemaphoreStub*/ semaStub1, /*remoteSemaphoreStub*/ semaStub0);
  mscclpp::BaseMemoryChannel memChan1(sema1);
  mscclpp::BaseMemoryChannelDeviceHandle memChanHandle1 = memChan1.deviceHandle();
  void *devHandle1;
  MSCCLPP_CUDATHROW(hipMalloc(&devHandle1, sizeof(mscclpp::BaseMemoryChannelDeviceHandle)));
  MSCCLPP_CUDATHROW(hipMemcpy(devHandle1, &memChanHandle1, sizeof(memChanHandle1), hipMemcpyHostToDevice));

  log("GPU 0: Launching gpuKernel0 ...");

  MSCCLPP_CUDATHROW(hipSetDevice(0));
  gpuKernel0<<<1, 1>>>(reinterpret_cast<mscclpp::BaseMemoryChannelDeviceHandle *>(devHandle0), iter);
  MSCCLPP_CUDATHROW(hipGetLastError());

  log("GPU 1: Launching gpuKernel1 ...");

  MSCCLPP_CUDATHROW(hipSetDevice(1));
  hipEvent_t start, end;
  MSCCLPP_CUDATHROW(hipEventCreate(&start));
  MSCCLPP_CUDATHROW(hipEventCreate(&end));
  MSCCLPP_CUDATHROW(hipEventRecord(start));
  gpuKernel1<<<1, 1>>>(reinterpret_cast<mscclpp::BaseMemoryChannelDeviceHandle *>(devHandle1), iter);
  MSCCLPP_CUDATHROW(hipGetLastError());
  MSCCLPP_CUDATHROW(hipEventRecord(end));
  MSCCLPP_CUDATHROW(hipEventSynchronize(end));

  float elapsedMs;
  MSCCLPP_CUDATHROW(hipEventElapsedTime(&elapsedMs, start, end));

  MSCCLPP_CUDATHROW(hipSetDevice(0));
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  float msPerIter = elapsedMs / iter;
  log("Elapsed ", msPerIter, " ms per iteration (", iter, ")");
  if (msPerIter < 1.0f) {
    log("Failed: the elapsed time per iteration is less than 1 ms, which may indicate that the relaxedSignal "
        "and relaxedWait are not working as expected.");
    return 1;
  }
  log("Succeed!");
  return 0;
}
