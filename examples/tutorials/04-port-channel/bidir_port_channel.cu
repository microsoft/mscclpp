#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <sys/wait.h>
#include <unistd.h>

#include <functional>
#include <iostream>
#include <mscclpp/concurrency_device.hpp>
#include <mscclpp/core.hpp>
#include <mscclpp/gpu_utils.hpp>
#include <mscclpp/port_channel.hpp>
#include <mscclpp/port_channel_device.hpp>
#include <sstream>

#define PORT_NUMBER "50505"

template <typename... Args>
void log(Args &&...args) {
  std::stringstream ss;
  (ss << ... << args);
  ss << std::endl;
  std::cout << ss.str();
}

int spawn_process(std::function<void()> func) {
  pid_t pid = fork();
  if (pid < 0) return -1;
  if (pid == 0) {
    // Child process
    func();
    exit(0);
  }
  return pid;
}

int wait_process(int pid) {
  int status;
  if (waitpid(pid, &status, 0) < 0) {
    return -1;
  }
  if (WIFEXITED(status)) {
    return WEXITSTATUS(status);
  }
  return -1;
}

__global__ void bidirPutKernel(mscclpp::PortChannelDeviceHandle *devHandle, size_t copyBytes, int myRank) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid == 0) {
    devHandle->signal();
    devHandle->wait();

    const uint64_t srcOffset = myRank * copyBytes;
    const uint64_t dstOffset = srcOffset;
    devHandle->putWithSignal(dstOffset, srcOffset, copyBytes);
    devHandle->wait();
  }
}

void worker(int gpuId) {
  MSCCLPP_CUDATHROW(hipSetDevice(gpuId));
  const int myRank = gpuId;
  const int remoteRank = myRank == 0 ? 1 : 0;
  const int nRanks = 2;
  const int iter = 1000;
  const mscclpp::Transport transport = mscclpp::Transport::CudaIpc;
  const size_t bufferBytes = 256 * 1024 * 1024;

  log("GPU ", gpuId, ": Preparing for tests ...");

  // Build a connection and a semaphore
  auto bootstrap = std::make_shared<mscclpp::TcpBootstrap>(myRank, nRanks);
  bootstrap->initialize("lo:127.0.0.1:" PORT_NUMBER);
  mscclpp::Communicator comm(bootstrap);
  auto conn = comm.connect({transport, {mscclpp::DeviceType::GPU, gpuId}}, remoteRank).get();
  auto sema = comm.buildSemaphore(conn, remoteRank).get();

  mscclpp::GpuBuffer buffer(bufferBytes);
  mscclpp::RegisteredMemory localRegMem = comm.registerMemory(buffer.data(), buffer.bytes(), transport);

  comm.sendMemory(localRegMem, remoteRank);
  auto remoteRegMemFuture = comm.recvMemory(remoteRank);
  mscclpp::RegisteredMemory remoteRegMem = remoteRegMemFuture.get();

  mscclpp::ProxyService proxyService;
  mscclpp::SemaphoreId semaId = proxyService.addSemaphore(sema);
  mscclpp::MemoryId localMemId = proxyService.addMemory(localRegMem);
  mscclpp::MemoryId remoteMemId = proxyService.addMemory(remoteRegMem);
  mscclpp::PortChannel portChan = proxyService.portChannel(semaId, remoteMemId, localMemId);

  auto portChanHandle = portChan.deviceHandle();

  void *devHandle;
  MSCCLPP_CUDATHROW(hipMalloc(&devHandle, sizeof(portChanHandle)));
  MSCCLPP_CUDATHROW(hipMemcpy(devHandle, &portChanHandle, sizeof(portChanHandle), hipMemcpyHostToDevice));

  hipStream_t stream;
  MSCCLPP_CUDATHROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  std::function<void(size_t)> kernels[1];

  kernels[0] = [&](size_t copyBytes) {
    bidirPutKernel<<<1, 1, 0, stream>>>(reinterpret_cast<mscclpp::PortChannelDeviceHandle *>(devHandle), copyBytes,
                                        myRank);
  };

  hipEvent_t start, end;
  if (gpuId == 0) {
    MSCCLPP_CUDATHROW(hipEventCreate(&start));
    MSCCLPP_CUDATHROW(hipEventCreate(&end));
  }
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());
  bootstrap->barrier();

  for (int kernelId = 0; kernelId < 1; ++kernelId) {
    const std::string testName = "Bidir PutWithSignal";
    for (size_t copyBytes : {1024, 1024 * 1024, 128 * 1024 * 1024}) {
      hipGraph_t graph;
      hipGraphExec_t graphExec;

      proxyService.startProxy();

      MSCCLPP_CUDATHROW(hipGraphCreate(&graph, 0));
      MSCCLPP_CUDATHROW(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

      for (int i = 0; i < iter; ++i) {
        kernels[kernelId](copyBytes);
      }

      MSCCLPP_CUDATHROW(hipStreamEndCapture(stream, &graph));
      MSCCLPP_CUDATHROW(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

      proxyService.stopProxy();

      // Synchronize before timing
      MSCCLPP_CUDATHROW(hipDeviceSynchronize());
      proxyService.startProxy();
      bootstrap->barrier();

      if (gpuId == 0) {
        MSCCLPP_CUDATHROW(hipEventRecord(start, stream));
      }

      MSCCLPP_CUDATHROW(hipGraphLaunch(graphExec, stream));

      if (gpuId == 0) {
        MSCCLPP_CUDATHROW(hipEventRecord(end, stream));
        MSCCLPP_CUDATHROW(hipEventSynchronize(end));
        float elapsedTime;
        float elapsedTimePerIter;
        float gbps;
        MSCCLPP_CUDATHROW(hipEventElapsedTime(&elapsedTime, start, end));
        elapsedTimePerIter = elapsedTime / iter;
        gbps = float(copyBytes) / elapsedTimePerIter * 1e-6f;
        log("GPU ", gpuId, ": [", testName, "] bytes ", copyBytes, ", elapsed ", elapsedTimePerIter, " ms/iter, BW ",
            gbps, " GB/s");
      }
      MSCCLPP_CUDATHROW(hipStreamSynchronize(stream));
      proxyService.stopProxy();

      MSCCLPP_CUDATHROW(hipGraphExecDestroy(graphExec));
      MSCCLPP_CUDATHROW(hipGraphDestroy(graph));
    }
  }

  bootstrap->barrier();
}

int main() {
  int pid0 = spawn_process([]() { worker(0); });
  int pid1 = spawn_process([]() { worker(1); });
  if (pid0 < 0 || pid1 < 0) {
    log("Failed to spawn processes.");
    return -1;
  }
  int status0 = wait_process(pid0);
  int status1 = wait_process(pid1);
  if (status0 < 0 || status1 < 0) {
    log("Failed to wait for processes.");
    return -1;
  }
  if (status0 != 0 || status1 != 0) {
    log("One of the processes failed.");
    return -1;
  }
  log("Succeed!");
  return 0;
}
