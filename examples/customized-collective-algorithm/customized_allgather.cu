#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/nccl.h>
#include <sys/wait.h>

#include <filesystem>
#include <fstream>
#include <iostream>
#include <memory>
#include <mscclpp/algorithm.hpp>
#include <mscclpp/core.hpp>
#include <mscclpp/gpu_utils.hpp>
#include <unordered_map>

#if defined(__HIP_PLATFORM_AMD__)
#define WARP_SIZE 64
#else
#define WARP_SIZE 32
#endif

template <typename... Args>
void log(Args&&... args) {
  std::stringstream ss;
  (ss << ... << args);
  ss << std::endl;
  std::cout << ss.str();
}

int spawn_process(std::function<void()> func) {
  pid_t pid = fork();
  if (pid < 0) return -1;
  if (pid == 0) {
    // Child process
    func();
    exit(0);
  }
  return pid;
}

int wait_process(int pid) {
  int status;
  if (waitpid(pid, &status, 0) < 0) {
    return -1;
  }
  if (WIFEXITED(status)) {
    return WEXITSTATUS(status);
  }
  return -1;
}

__global__ void __launch_bounds__(1024)
    allgather(mscclpp::DeviceHandle<mscclpp::PortChannel>* portChannels, int rank, size_t nbytesPerGPU) {
  int warpId = threadIdx.x / WARP_SIZE;

  // Each warp is responsible for one of the remote ranks
  mscclpp::DeviceHandle<mscclpp::PortChannel> portChan = portChannels[warpId];

  // this allgather is really simple and implemented as an alltoall

  // this thread's role is a sender role
  // put your data asynchronously
  if (threadIdx.x % WARP_SIZE == 0) {
    portChan.putWithSignal(rank * nbytesPerGPU, 0, nbytesPerGPU);
  }
  // make sure everyone is put their data before some thread randomly blocks everyone else in signal
  __syncthreads();
  // push with flag and sync to make sure the data is received
  if (threadIdx.x % WARP_SIZE == 0) {
    portChan.flush();
  }

  // this thread's role is a receiver role. wait on the semaphore to make sure the data is ready
  if (threadIdx.x % WARP_SIZE == 0) {
    portChan.wait();
  }
}

class AllgatherAlgoBuilder : public mscclpp::AlgorithmBuilder {
 public:
  AllgatherAlgoBuilder() = default;
  ~AllgatherAlgoBuilder() {
    if (proxyService_) {
      proxyService_->stopProxy();
    }
  }

  mscclpp::Algorithm build() {
    auto self = std::make_shared<AllgatherAlgoBuilder>();
    mscclpp::Algorithm allgatherAlgo(
        "allgather", "allgather",
        [self](std::shared_ptr<mscclpp::Communicator> comm, std::unordered_map<std::string, std::shared_ptr<void>>&) {
          self->initialize(comm);
        },
        [self](const std::shared_ptr<mscclpp::AlgorithmCtx> ctx, const void* input, void* output, size_t count,
               int dtype, hipStream_t stream, std::unordered_map<std::string, std::shared_ptr<void>>& extras) {
          return self->allgatherKernelFunc(ctx, input, output, count, static_cast<ncclDataType_t>(dtype), stream,
                                           extras);
        },
        [self](std::shared_ptr<mscclpp::Communicator> comm, const void* input, void* output, size_t count, int dtype) {
          return self->initAllgatherContext(comm, input, output, count, static_cast<ncclDataType_t>(dtype));
        },
        [self](const void* input, void* output, size_t count, int dtype) {
          return self->generateAllgatherContextKey(input, output, count, static_cast<ncclDataType_t>(dtype));
        });
    return allgatherAlgo;
  }

 private:
  std::vector<std::shared_ptr<mscclpp::Connection>> conns_;
  std::shared_ptr<mscclpp::ProxyService> proxyService_;
  int worldSize_;

  void initialize(std::shared_ptr<mscclpp::Communicator> comm) {
    std::vector<std::shared_future<std::shared_ptr<mscclpp::Connection>>> connectionFutures;
    worldSize_ = comm->bootstrap()->getNranks();
    for (int i = 0; i < worldSize_; i++) {
      if (i == comm->bootstrap()->getRank()) continue;
      connectionFutures.push_back(comm->connect(mscclpp::Transport::CudaIpc, i));
    }
    std::vector<std::shared_ptr<mscclpp::Connection>> connections;
    std::transform(connectionFutures.begin(), connectionFutures.end(), std::back_inserter(connections),
                   [](const auto& future) { return future.get(); });
    this->conns_ = std::move(connections);
    proxyService_ = std::make_shared<mscclpp::ProxyService>();
    proxyService_->startProxy();
  }

  ncclResult_t allgatherKernelFunc(const std::shared_ptr<mscclpp::AlgorithmCtx> ctx, const void* input, void* output,
                                   size_t count, [[maybe_unused]] ncclDataType_t dtype, hipStream_t stream,
                                   std::unordered_map<std::string, std::shared_ptr<void>>& extras) {
    int rank = ctx->rank;
    int worldSize = ctx->workSize;

    int nThreadsPerBlock = (worldSize - 1) * WARP_SIZE;
    allgather<<<1, nThreadsPerBlock, 0, stream>>>(ctx->portChannelDeviceHandles.get(), rank,
                                                  count * ncclTypeSize(dtype));
    if (hipGetLastError() == hipSuccess) {
      return ncclSuccess;
    }
    return ncclInternalError;
  }

  std::shared_ptr<mscclpp::AlgorithmCtx> initAllgatherContext(std::shared_ptr<mscclpp::Communicator> comm,
                                                              const void* input, void* output, size_t count,
                                                              ncclDataType_t dtype) {
    auto ctx = std::make_shared<mscclpp::AlgorithmCtx>();
    ctx->rank = comm->bootstrap()->getRank();
    ctx->workSize = comm->bootstrap()->getNranks();
    ctx->nRanksPerNode = comm->bootstrap()->getNranksPerNode();

    // register memories
    mscclpp::RegisteredMemory inputBufRegMem =
        comm->registerMemory((void*)input, count * ncclTypeSize(dtype), mscclpp::Transport::CudaIpc);
    mscclpp::RegisteredMemory outputBufRegMem =
        comm->registerMemory(output, count * ncclTypeSize(dtype) * ctx->workSize, mscclpp::Transport::CudaIpc);
    std::vector<std::shared_future<mscclpp::RegisteredMemory>> remoteRegMemories;
    for (int i = 0; i < ctx->workSize; i++) {
      if (i == ctx->rank) continue;
      comm->sendMemory(outputBufRegMem, i, 0);
      remoteRegMemories.push_back(comm->recvMemory(i, 0));
    }

    // setup channels
    std::vector<mscclpp::DeviceHandle<mscclpp::PortChannel>> portChannels;
    mscclpp::MemoryId inputMemoryId = this->proxyService_->addMemory(inputBufRegMem);
    for (int i = 0; i < this->conns_.size(); i++) {
      auto remoteMemory = remoteRegMemories[i].get();
      mscclpp::MemoryId remoteMemoryId = this->proxyService_->addMemory(remoteMemory);
      portChannels.push_back(mscclpp::deviceHandle(this->proxyService_->portChannel(
          this->proxyService_->buildAndAddSemaphore(*comm, this->conns_[i]), remoteMemoryId, inputMemoryId)));
    }
    ctx->portChannelDeviceHandles =
        mscclpp::detail::gpuCallocShared<mscclpp::DeviceHandle<mscclpp::PortChannel>>(portChannels.size());
    mscclpp::gpuMemcpy(ctx->portChannelDeviceHandles.get(), portChannels.data(), portChannels.size(),
                       hipMemcpyHostToDevice);

    // keep registered memory references
    std::transform(remoteRegMemories.begin(), remoteRegMemories.end(), std::back_inserter(ctx->registeredMemories),
                   [](const auto& fut) { return fut.get(); });
    ctx->registeredMemories.push_back(inputBufRegMem);
    ctx->registeredMemories.push_back(outputBufRegMem);

    return ctx;
  }

  mscclpp::AlgorithmCtxKey generateAllgatherContextKey(const void* input, void* output, size_t count,
                                                       ncclDataType_t dtype) {
    return {(void*)input, output, count * ncclTypeSize(dtype), count * ncclTypeSize(dtype) * worldSize_, 0};
  }
};

void worker(int rank, int worldSize, ncclUniqueId id) {
  constexpr int size = 1024 * 1024 * 64;
  const int iter = 100;
  MSCCLPP_CUDATHROW(hipSetDevice(rank));

  // register algorithm
  auto allgatherAlgoBuilder = std::make_shared<AllgatherAlgoBuilder>();
  mscclpp::AlgorithmCollectionBuilder::getInstance()->addAlgorithmBuilder(allgatherAlgoBuilder);
  mscclpp::AlgorithmCollectionBuilder::getInstance()->setAlgorithmSelector(
      [](const std::unordered_map<std::string, std::unordered_map<std::string, mscclpp::Algorithm>>&
             algoMapByCollective,
         std::string collective, const void* input, void* output, size_t messageSize, int dtype, int nRanksPerNode,
         int worldSize) {
        if (collective != "allgather") {
          return mscclpp::Algorithm();
        }
        return algoMapByCollective.at(collective).at("allgather");
      });

  float *sendbuff, *recvbuff;
  hipStream_t stream;
  MSCCLPP_CUDATHROW(hipMalloc(&sendbuff, size * sizeof(float)));
  MSCCLPP_CUDATHROW(hipMalloc(&recvbuff, size * sizeof(float) * worldSize));
  MSCCLPP_CUDATHROW(hipMemcpy(recvbuff + rank * size, sendbuff, size * sizeof(float), hipMemcpyDeviceToDevice));
  MSCCLPP_CUDATHROW(hipStreamCreate(&stream));

  ncclComm_t comm;
  hipGraphExec_t graphExec;
  hipGraph_t graph;
  MSCCLPP_CUDATHROW(hipGraphCreate(&graph, 0));

  ncclCommInitRank(&comm, worldSize, id, rank);
  MSCCLPP_CUDATHROW(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  for (int i = 0; i < iter; ++i) {
    ncclAllGather(sendbuff, recvbuff, size, ncclFloat, comm, stream);
  }
  MSCCLPP_CUDATHROW(hipStreamEndCapture(stream, &graph));
  MSCCLPP_CUDATHROW(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

  hipEvent_t start, end;
  if (rank == 0) {
    MSCCLPP_CUDATHROW(hipEventCreate(&start));
    MSCCLPP_CUDATHROW(hipEventCreate(&end));
  }

  MSCCLPP_CUDATHROW(hipDeviceSynchronize());
  if (rank == 0) {
    MSCCLPP_CUDATHROW(hipEventRecord(start, stream));
  }
  MSCCLPP_CUDATHROW(hipGraphLaunch(graphExec, stream));
  if (rank == 0) {
    MSCCLPP_CUDATHROW(hipEventRecord(end, stream));
    MSCCLPP_CUDATHROW(hipEventSynchronize(end));
    float elapsedTime;
    float elapsedTimePerIter;
    float gbps;
    MSCCLPP_CUDATHROW(hipEventElapsedTime(&elapsedTime, start, end));
    elapsedTimePerIter = elapsedTime / iter;
    gbps = float(size) * (worldSize - 1) * ncclTypeSize(ncclFloat) / elapsedTimePerIter * 1e-6f;
    log("GPU ", rank, ": bytes ", size * ncclTypeSize(ncclFloat), ", elapsed ", elapsedTimePerIter, " ms/iter, BW ",
        gbps, " GB/s");
  }

  MSCCLPP_CUDATHROW(hipStreamSynchronize(stream));

  MSCCLPP_CUDATHROW(hipFree(sendbuff));
  MSCCLPP_CUDATHROW(hipFree(recvbuff));

  ncclCommDestroy(comm);
}

int main() {
  ncclUniqueId id;
  ncclGetUniqueId(&id);

  int pid0 = spawn_process([&]() { worker(0, 4, id); });
  int pid1 = spawn_process([&]() { worker(1, 4, id); });
  int pid2 = spawn_process([&]() { worker(2, 4, id); });
  int pid3 = spawn_process([&]() { worker(3, 4, id); });

  if (pid0 < 0 || pid1 < 0 || pid2 < 0 || pid3 < 0) {
    log("Fork failed!");
    return -1;
  }

  int status0 = wait_process(pid0);
  int status1 = wait_process(pid1);
  int status2 = wait_process(pid2);
  int status3 = wait_process(pid3);
  if (status0 != 0 || status1 != 0 || status2 != 0 || status3 != 0) {
    log("Worker failed!");
    return -1;
  }

  log("Succeed!");
  return 0;
}