// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/nccl.h>

#include <mscclpp/algorithm.hpp>
#include <mscclpp/env.hpp>
#include <mscclpp/gpu.hpp>
#include <mscclpp/gpu_utils.hpp>

#include "allreduce.hpp"
#include "debug.h"

using AllreduceFunc = std::function<hipError_t(
    const void*, void*, void*, mscclpp::DeviceHandle<mscclpp::MemoryChannel>*,
    mscclpp::DeviceHandle<mscclpp::MemoryChannel>*, mscclpp::DeviceHandle<mscclpp::SwitchChannel>*,
    mscclpp::DeviceHandle<mscclpp::SwitchChannel>*, size_t, size_t, size_t, int, int, int, size_t, hipStream_t,
    uint32_t*, uint32_t*, uint32_t*, uint32_t)>;

namespace {

template <Op OpType, typename T>
struct AllpairAdapter {
  static hipError_t call(const void* buff, void* scratch, void* resultBuff,
                          mscclpp::DeviceHandle<mscclpp::MemoryChannel>* memoryChannels,
                          mscclpp::DeviceHandle<mscclpp::MemoryChannel>*,
                          mscclpp::DeviceHandle<mscclpp::SwitchChannel>*,
                          mscclpp::DeviceHandle<mscclpp::SwitchChannel>*, size_t channelInOffset, size_t,
                          size_t channelScratchOffset, int rank, int nRanksPerNode, int worldSize, size_t nelems,
                          hipStream_t stream, uint32_t* deviceFlag7, uint32_t* deviceFlag28, uint32_t*,
                          uint32_t numScratchBuff) {
    if (sizeof(T) * nelems < worldSize * sizeof(int)) {
      int nBlocks = worldSize - 1;
      int nThreadsPerBlock = 32;
      allreduceAllPairs<OpType><<<nBlocks, nThreadsPerBlock, 0, stream>>>(
          (T*)buff, (T*)scratch, (T*)resultBuff, memoryChannels, channelInOffset, channelScratchOffset, rank,
          nRanksPerNode, worldSize, nelems, deviceFlag7, numScratchBuff);
    } else if (sizeof(T) * nelems <= (1 << 14)) {
      int nBlocks = (worldSize - 1) * 4;
      int nThreadsPerBlock = 512;
      allreduceAllPairs<OpType><<<nBlocks, nThreadsPerBlock, 0, stream>>>(
          (T*)buff, (T*)scratch, (T*)resultBuff, memoryChannels, channelInOffset, channelScratchOffset, rank,
          nRanksPerNode, worldSize, nelems, deviceFlag28, numScratchBuff);
    }
    return hipGetLastError();
  }
};

template <template <Op, typename> class Adapter>
AllreduceFunc dispatch(ncclRedOp_t op, ncclDataType_t dtype) {
  Op reduceOp = getReduceOp(op);
  AllreduceFunc allreduceFunc;
  if (reduceOp == SUM) {
    if (dtype == ncclFloat16) {
      allreduceFunc = Adapter<SUM, half>::call;
    } else if (dtype == ncclFloat32) {
      allreduceFunc = Adapter<SUM, float>::call;
#if defined(__CUDA_BF16_TYPES_EXIST__)
    } else if (dtype == ncclBfloat16) {
      allreduceFunc = Adapter<SUM, __bfloat16>::call;
#endif
    } else if (dtype == ncclInt32 || dtype == ncclUint32) {
      allreduceFunc = Adapter<SUM, int>::call;
    } else {
      return nullptr;
    }
  } else if (reduceOp == MIN) {
    if (dtype == ncclFloat16) {
      allreduceFunc = Adapter<MIN, half>::call;
    } else if (dtype == ncclFloat32) {
      allreduceFunc = Adapter<MIN, float>::call;
#if defined(__CUDA_BF16_TYPES_EXIST__)
    } else if (dtype == ncclBfloat16) {
      allreduceFunc = Adapter<MIN, __bfloat16>::call;
#endif
    } else if (dtype == ncclInt32 || dtype == ncclUint32) {
      allreduceFunc = Adapter<MIN, int>::call;
    } else {
      return nullptr;
    }
  }
  return nullptr;
}
}  // namespace

enum Op getReduceOp(ncclRedOp_t op) {
  switch (op) {
    case ncclSum:
      return SUM;
    case ncclMin:
      return MIN;
    default:
      WARN("op is invalid, op: %d", op);
      throw mscclpp::Error("Invalid operation", mscclpp::ErrorCode::InternalError);
  }
}

ncclResult_t AllreduceAllpair::allreduceKernelFunc(const std::shared_ptr<mscclpp::AlgorithmCtx> ctx, const void* input,
                                                   void* output, size_t count, [[maybe_unused]] ncclDataType_t dtype,
                                                   hipStream_t stream,
                                                   std::unordered_map<std::string, std::shared_ptr<void>>& extras) {
  const size_t bytes = count * ncclTypeSize(dtype);
  const int worldSize = ctx->workSize;
  ncclRedOp_t op = *static_cast<ncclRedOp_t*>(extras.at("op").get());
  AllreduceFunc allreduce = dispatch<AllpairAdapter>(op, dtype);
  if (!allreduce) {
    WARN("Unsupported operation or data type for allreduce: op=%d, dtype=%d", op, dtype);
    return ncclInvalidArgument;
  }
  hipError_t error = allreduce(input, ctx->scratchBuffer.get(), output, ctx->memoryChannelDeviceHandles.get(), nullptr,
                                nullptr, nullptr, 0, 0, 0, ctx->rank, ctx->nRanksPerNode, ctx->workSize, count, stream,
                                nullptr, nullptr, nullptr, 0U);
  if (error != hipSuccess) {
    WARN("AllreduceAllpair failed with error: %s", hipGetErrorString(error));
    return ncclUnhandledCudaError;
  }
  return ncclSuccess;
}

std::shared_ptr<mscclpp::AlgorithmCtx> AllreduceAllpair::initAllreduceContext(
    std::shared_ptr<mscclpp::Communicator> comm, const void*, void* output, size_t, ncclDataType_t) {
      return nullptr;
    }
mscclpp::AlgorithmCtxKey AllreduceAllpair::generateAllreduceContextKey(const void*, void*, size_t, ncclDataType_t) {
  return mscclpp::AlgorithmCtxKey{nullptr, nullptr, 0, 0, 0};
}

void AllreduceAllpair::registerAlgorithm(std::shared_ptr<mscclpp::Communicator> comm) {
  auto self = shared_from_this();
  mscclpp::Algorithm allgatherAlgo(
      comm, "allreduce",
      [self](const std::shared_ptr<mscclpp::AlgorithmCtx> ctx, const void* input, void* output, size_t count,
             ncclDataType_t dtype, hipStream_t stream,
             std::unordered_map<std::string, std::shared_ptr<void>>& extras) {
        return self->allreduceKernelFunc(ctx, input, output, count, dtype, stream, extras);
      },
      [self](std::shared_ptr<mscclpp::Communicator> comm, const void* input, void* output, size_t count,
             ncclDataType_t dtype) { return self->initAllreduceContext(comm, input, output, count, dtype); },
      [self](const void* input, void* output, size_t count, ncclDataType_t dtype) {
        return self->generateAllreduceContextKey(input, output, count, dtype);
      });
  mscclpp::AlgorithmFactory::getInstance()->registerAlgorithm("allreduce", "default_allreduce_allpair", allgatherAlgo);
}