// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <algorithm>
#include <filesystem>
#include <functional>
#include <mscclpp/concurrency_device.hpp>
#include <mscclpp/core.hpp>
#include <mscclpp/env.hpp>
#include <mscclpp/executor.hpp>
#include <mscclpp/memory_channel.hpp>
#include <mscclpp/memory_channel_device.hpp>
#include <mscclpp/nvls.hpp>
#include <mscclpp/utils.hpp>
#include <queue>
#include <sstream>
#include <unordered_map>
#include <vector>
#if defined(ENABLE_NPKIT)
#include <mscclpp/npkit/npkit.hpp>
#endif
#include <dlfcn.h>
#include <mscclpp/nccl.h>

#include <mscclpp/algorithm.hpp>

#include "allgather.hpp"
#include "allreduce.hpp"
#include "broadcast.hpp"
#include "debug.h"

#define NCCL_API extern "C" __attribute__((visibility("default")))

#define CUDACHECK(cmd)                                                                      \
  do {                                                                                      \
    hipError_t e = cmd;                                                                    \
    if (e != hipSuccess) {                                                                 \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
      exit(EXIT_FAILURE);                                                                   \
    }                                                                                       \
  } while (0)

typedef enum mscclppNcclDlopenErr {
  dlopenSuccess = 0,
  dlopenError = 1,
} mscclppNcclDlopenErr_t;

typedef struct _mscclppNcclOps_t {
  ncclResult_t (*CommInitRank)(ncclComm_t* comm, int nranks, ncclUniqueId commId, int rank);
  ncclResult_t (*GetUniqueId)(ncclUniqueId* uniqueId);
  ncclResult_t (*CommDestroy)(ncclComm_t comm);
  ncclResult_t (*CommUserRank)(const ncclComm_t, int* rank);
  ncclResult_t (*AllReduce)(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype, ncclRedOp_t op,
                            ncclComm_t comm, hipStream_t stream);
  ncclResult_t (*AllGather)(const void* sendbuff, void* recvbuff, size_t sendcount, ncclDataType_t datatype,
                            ncclComm_t comm, hipStream_t stream);
  ncclResult_t (*Broadcast)(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype, int root,
                            ncclComm_t comm, hipStream_t stream);
  ncclResult_t (*ReduceScatter)(const void* sendbuff, void* recvbuff, size_t recvcount, ncclDataType_t datatype,
                                ncclRedOp_t op, ncclComm_t comm, hipStream_t stream);
  ncclResult_t (*Reduce)(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype, ncclRedOp_t op,
                         int root, ncclComm_t comm, hipStream_t stream);
  ncclResult_t (*Send)(const void* sendbuff, size_t count, ncclDataType_t datatype, int peer, ncclComm_t comm,
                       hipStream_t stream);
  ncclResult_t (*Recv)(void* recvbuff, size_t count, ncclDataType_t datatype, int peer, ncclComm_t comm,
                       hipStream_t stream);
  ncclResult_t (*GroupStart)();
  ncclResult_t (*GroupEnd)();
} mscclppNcclOps_t;

mscclppNcclOps_t mscclppNcclOps;
void* mscclppNcclDlHandle = NULL;
bool mscclppNcclDlopenSharedLib = false;

#define QUOTE(symbol) #symbol

#define NCCL_DLSYM(_struct_, _handle_, _prefix_, _function_, _type_)                               \
  do {                                                                                             \
    _struct_._function_ = (_type_)dlsym((_handle_), QUOTE(_prefix_##_function_));                  \
    if (_struct_._function_ == NULL) {                                                             \
      printf("Failed: dlsym error: Cannot open %s: %s\n", QUOTE(_prefix_##_function_), dlerror()); \
      exit(dlopenError);                                                                           \
    }                                                                                              \
  } while (0)

static inline int mscclppNcclDlopenInit() {
  const char* ncclLibPath = mscclpp::env()->ncclSharedLibPath.c_str();
  if (ncclLibPath != nullptr && ncclLibPath[0] != '\0') {
    if (std::filesystem::is_directory(ncclLibPath)) {
      WARN("The value of the environment variable %s is a directory", ncclLibPath);
      return dlopenError;
    }

    mscclppNcclDlHandle = dlopen(ncclLibPath, RTLD_LAZY | RTLD_NODELETE);
    if (!mscclppNcclDlHandle) {
      WARN("Cannot open the shared library specified by MSCCLPP_NCCL_LIB_PATH: %s\n", dlerror());
      return dlopenError;
    }
  } else {
    WARN("The value of MSCCLPP_NCCL_LIB_PATH is empty!\n");
    return dlopenError;
  }

  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, CommInitRank,
             ncclResult_t (*)(ncclComm_t*, int, ncclUniqueId, int));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, GetUniqueId, ncclResult_t (*)(ncclUniqueId*));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, CommDestroy, ncclResult_t (*)(ncclComm_t));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, CommUserRank, ncclResult_t (*)(ncclComm_t, int*));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, AllReduce,
             ncclResult_t (*)(const void*, void*, size_t, ncclDataType_t, ncclRedOp_t, ncclComm_t, hipStream_t));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, AllGather,
             ncclResult_t (*)(const void*, void*, size_t, ncclDataType_t, ncclComm_t, hipStream_t));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, Broadcast,
             ncclResult_t (*)(const void*, void*, size_t, ncclDataType_t, int, ncclComm_t, hipStream_t));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, ReduceScatter,
             ncclResult_t (*)(const void*, void*, size_t, ncclDataType_t, ncclRedOp_t, ncclComm_t, hipStream_t));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, Reduce,
             ncclResult_t (*)(const void*, void*, size_t, ncclDataType_t, ncclRedOp_t, int, ncclComm_t, hipStream_t));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, Send,
             ncclResult_t (*)(const void*, size_t, ncclDataType_t, int, ncclComm_t, hipStream_t));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, Recv,
             ncclResult_t (*)(void*, size_t, ncclDataType_t, int, ncclComm_t, hipStream_t));
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, GroupStart, ncclResult_t (*)());
  NCCL_DLSYM(mscclppNcclOps, mscclppNcclDlHandle, nccl, GroupEnd, ncclResult_t (*)());

  return dlopenSuccess;
}

static inline void mscclppNcclDlopenFinalize() {
  if (mscclppNcclDlHandle) {
    dlclose(mscclppNcclDlHandle);
  }
}

static inline int mscclppNcclInFallbackList(const char* collOps, const char* fallbackList) {
  if (strcmp(fallbackList, "all") == 0) {
    return 1;
  }

  char* fallbackListCopy = strdup(fallbackList);
  char* token = strtok(fallbackListCopy, ",");
  while (token != NULL) {
    if (strcmp(collOps, token) == 0) {
      free(fallbackListCopy);
      return 1;
    }
    token = strtok(NULL, ",");
  }

  free(fallbackListCopy);
  return 0;
}

static bool tryLoadNcclSharedLib() {
  if (mscclppNcclDlopenSharedLib) return true;
  if (!mscclpp::env()->ncclSharedLibPath.empty()) {
    if (mscclppNcclDlopenInit() == dlopenSuccess) {
      mscclppNcclDlopenSharedLib = true;
      return true;
    }
  }
  return false;
}

// Declare the global map to store associations between raw pointer and shared pointer
static std::unordered_map<void*, std::shared_ptr<char>> ptrMap;

struct planKey {
  size_t minMessageSize;
  size_t maxMessageSize;
  bool isInPlace;
};

struct executionPlanInstance {
  planKey key;
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
};

struct splitCommInfo {
  int color;
  int key;
  int originalRank;
};

struct ncclComm {
  std::shared_ptr<mscclpp::Communicator> comm;
  std::shared_ptr<mscclpp::Executor> executor;
  std::unordered_map<std::string, std::vector<executionPlanInstance>> executionPlans;
  std::shared_ptr<mscclpp::AlgorithmCollection> algorithmCollection;
  std::shared_ptr<char> scratchBuffer_;
  const size_t scratchBufferSize_ = (1 << 27);  // 128MB
  int nRanksPerNode;
  int worldSize;

  void* mscclppNcclComm;
};

static std::pair<std::string, executionPlanInstance> loadExecutionPlan(const std::string& filename, int rank) {
  std::shared_ptr<mscclpp::ExecutionPlan> plan = std::make_shared<mscclpp::ExecutionPlan>(filename, rank);
  std::string collective = plan->collective();
  planKey key{plan->minMessageSize(), plan->maxMessageSize(), plan->isInPlace()};
  return std::make_pair(collective, executionPlanInstance{key, plan});
}

static ncclResult_t executeWithPlan(std::shared_ptr<mscclpp::Executor> executor, int rank, ncclDataType_t datatype,
                                    const void* sendbuff, void* recvbuff, size_t sendBytes, size_t recvBytes,
                                    std::shared_ptr<mscclpp::ExecutionPlan> plan, hipStream_t stream) {
  switch (datatype) {
    case ncclFloat16:
      executor->execute(rank, (half*)sendbuff, (half*)recvbuff, sendBytes, recvBytes, mscclpp::DataType::FLOAT16, *plan,
                        stream);
      break;
    case ncclFloat32:
      executor->execute(rank, (float*)sendbuff, (float*)recvbuff, sendBytes, recvBytes, mscclpp::DataType::FLOAT32,
                        *plan, stream);
      break;
    case ncclBfloat16:
      executor->execute(rank, (__bfloat16*)sendbuff, (__bfloat16*)recvbuff, sendBytes, recvBytes,
                        mscclpp::DataType::BFLOAT16, *plan, stream);
      break;
    case ncclInt32:
    case ncclUint32:
      executor->execute(rank, (int*)sendbuff, (int*)recvbuff, sendBytes, recvBytes, mscclpp::DataType::UINT32, *plan,
                        stream);
      break;
    default:
      WARN("datatype is invalid");
      return ncclInvalidArgument;
  }
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGetVersion(int* version) {
  if (version == nullptr) {
    WARN("version is nullptr");
    return ncclInvalidArgument;
  }
  *version = MSCCLPP_VERSION;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGetUniqueId(ncclUniqueId* uniqueId) {
  if (uniqueId == nullptr) {
    WARN("uniqueId is nullptr");
    return ncclInvalidArgument;
  }
  if (mscclpp::UniqueIdBytes != NCCL_UNIQUE_ID_BYTES) return ncclInternalError;
  mscclpp::UniqueId id = mscclpp::TcpBootstrap::createUniqueId();
  memcpy(uniqueId, &id, sizeof(ncclUniqueId));
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommInitRankConfig(ncclComm_t* comm, int nranks, ncclUniqueId commId, int rank,
                                             ncclConfig_t*) {
  // TODO: implement config
  return ncclCommInitRank(comm, nranks, commId, rank);
}

static void registerCustomizedAlgo() {
  auto collectionBuilder = mscclpp::AlgorithmCollectionBuilder::getInstance();
  std::shared_ptr<BroadcastAlgo6> broadcastAlgo6 = std::make_shared<BroadcastAlgo6>();
  collectionBuilder->addAlgorithmBuilder(broadcastAlgo6);

  std::shared_ptr<AllgatherAlgo6> allgatherAlgo6 = std::make_shared<AllgatherAlgo6>();
  std::shared_ptr<AllgatherAlgo8> allgatherAlgo8 = std::make_shared<AllgatherAlgo8>();
  collectionBuilder->addAlgorithmBuilder(allgatherAlgo6);
  // TODO(binyli): remove allgather8 algo, use nccl by default
  collectionBuilder->addAlgorithmBuilder(allgatherAlgo8);

  std::shared_ptr<AllreducePacket> allreduceAllpairAlgo = std::make_shared<AllreducePacket>();
  std::shared_ptr<AllreduceNvls> allreduceNvlsAlgo = std::make_shared<AllreduceNvls>();
  std::shared_ptr<AllreduceNvlsWithCopy> allreduceNvlsWithCopyAlgo = std::make_shared<AllreduceNvlsWithCopy>();
  std::shared_ptr<Allreduce8> allreduceAllreduce8Algo = std::make_shared<Allreduce8>();
  collectionBuilder->addAlgorithmBuilder(allreduceAllpairAlgo);
  collectionBuilder->addAlgorithmBuilder(allreduceNvlsAlgo);
  collectionBuilder->addAlgorithmBuilder(allreduceNvlsWithCopyAlgo);
  collectionBuilder->addAlgorithmBuilder(allreduceAllreduce8Algo);
}

static mscclpp::Algorithm algoSelector(
    const std::unordered_map<std::string, std::unordered_map<std::string, mscclpp::Algorithm>>& algoMapByCollective,
    std::string collective, const void* input, void* output, size_t messageSize, int nRanksPerNode, int worldSize) {
  if (nRanksPerNode != worldSize) {
    // Fallback to nccl/rccl when multi-node
    return mscclpp::Algorithm();
  }
  bool isCuMemMapAllocated =
      mscclpp::isCuMemMapAllocated(const_cast<void*>(input)) && mscclpp::isCuMemMapAllocated(output);
  bool mscclppDisableChannelCache = mscclpp::env()->disableChannelCache;
  bool useNvlsWithZeroCopy = mscclpp::isNvlsSupported() && !mscclppDisableChannelCache && isCuMemMapAllocated;
  if (collective == "allgather") {
    if (messageSize <= 32 * (1 << 20)) {
      return algoMapByCollective.at(collective).at("default_allgather6");
    } else {
#if defined(__HIP_PLATFORM_AMD__)
      return algoMapByCollective.at(collective).at("default_allgather6");
#else
      if (!mscclppNcclDlopenSharedLib) {
        return algoMapByCollective.at(collective).at("default_allgather8");
      }
#endif
    }
  }
  if (collective == "allreduce") {
    if (messageSize <= (1 << 16) || (messageSize <= (1 << 20) && !useNvlsWithZeroCopy)) {
      return algoMapByCollective.at(collective).at("default_allreduce_packet");
    } else if (useNvlsWithZeroCopy) {
      return algoMapByCollective.at(collective).at("default_allreduce_nvls");
    } else if (mscclpp::isNvlsSupported()) {
      return algoMapByCollective.at(collective).at("default_allreduce_nvls_with_copy");
    } else {
#if defined(__HIP_PLATFORM_AMD__)
      return algoMapByCollective.at(collective).at("default_allreduce_allreduce8");
#else
      if (!mscclppNcclDlopenSharedLib) {
        return algoMapByCollective.at(collective).at("default_allreduce_allreduce8");
      }
#endif
    }
  }
  INFO(MSCCLPP_NCCL, "Failed to get algo from customized kernel, fallback to nccl/rccl");
  return mscclpp::Algorithm();
}

NCCL_API ncclResult_t ncclCommInitRank(ncclComm_t* comm, int nranks, ncclUniqueId commId, int rank) {
  INFO(MSCCLPP_NCCL, "Initializing NCCL communicator for rank %d, world_size=%d", rank, nranks);
  if (comm == nullptr) {
    WARN("comm is nullptr");
    return ncclInvalidArgument;
  }
  if (nranks < 0 || rank < 0 || rank >= nranks) {
    WARN("nranks is %d, rank is %d", nranks, rank);
    return ncclInvalidArgument;
  }
  std::shared_ptr<mscclpp::TcpBootstrap> bootstrap = std::make_shared<mscclpp::TcpBootstrap>(rank, nranks);
  mscclpp::UniqueId id;
  memcpy(id.data(), &commId, sizeof(ncclUniqueId));
  bootstrap->initialize(id);
  std::shared_ptr<mscclpp::Communicator> mscclppComm = std::make_shared<mscclpp::Communicator>(bootstrap);
  ncclComm* commPtr = new ncclComm();

  commPtr->comm = mscclppComm;
  commPtr->scratchBuffer_ = mscclpp::GpuBuffer<char>(commPtr->scratchBufferSize_).memory();
  commPtr->executor = std::make_shared<mscclpp::Executor>(mscclppComm);
  commPtr->nRanksPerNode = mscclppComm->bootstrap()->getNranksPerNode();
  commPtr->worldSize = mscclppComm->bootstrap()->getNranks();

  if (commPtr->worldSize == 1) {
    *comm = commPtr;
    return ncclSuccess;
  }

  const std::string& collectiveDir = mscclpp::env()->executionPlanDir;
  if (collectiveDir != "") {
    if (!std::filesystem::is_directory(collectiveDir)) {
      WARN("The value of the environment variable %s is not a directory", collectiveDir.c_str());
      return ncclInvalidArgument;
    }
    for (const auto& entry : std::filesystem::directory_iterator(collectiveDir)) {
      if (entry.is_regular_file()) {
        auto plan = loadExecutionPlan(entry.path(), rank);
        commPtr->executionPlans[plan.first].push_back(plan.second);
      }
    }
  }

  mscclpp::AlgorithmCollectionBuilder::getInstance()->setFallbackAlgorithmSelector(algoSelector);
  registerCustomizedAlgo();
  commPtr->algorithmCollection = mscclpp::AlgorithmCollectionBuilder::getInstance()->build();

  *comm = commPtr;
#if defined(ENABLE_NPKIT)
  if (mscclpp::env()->npkitDumpDir != "") {
    NpKit::Init(rank);
  }
#endif

  const std::string ncclLibPath = mscclpp::env()->ncclSharedLibPath;
  if (!ncclLibPath.empty() && !mscclppNcclDlopenSharedLib) {
    if (!tryLoadNcclSharedLib()) {
      WARN("Failed to load the shared library for nccl/rccl");
      return ncclInternalError;
    }
  }

  if (mscclppNcclDlopenSharedLib == true) {
    ncclUniqueId mscclppNcclUniqueId;
    if (rank == 0) {
      mscclppNcclOps.GetUniqueId(&mscclppNcclUniqueId);
    }
    // After broadcast, mscclppNcclUniqueId on each rank has the same ncclUniqueId
    bootstrap->broadcast(&mscclppNcclUniqueId, sizeof(ncclUniqueId), 0);

    commPtr->mscclppNcclComm = new ncclComm_t();
    if (commPtr->mscclppNcclComm == nullptr) {
      WARN("Failed to allocate memory for mscclppNcclComm");
      return ncclInternalError;
    }
    mscclppNcclOps.CommInitRank(reinterpret_cast<ncclComm_t*>(commPtr->mscclppNcclComm), nranks, mscclppNcclUniqueId,
                                rank);
  }

  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommInitAll(ncclComm_t* comm, int ndev, const int*) {
  if (ndev == 1) {
    ncclUniqueId Id;
    ncclGetUniqueId(&Id);
    return ncclCommInitRank(comm, ndev, Id, 0);
  }
  // TODO: implement this function
  WARN("ncclCommInitAll is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclCommFinalize(ncclComm_t comm) {
  comm->comm->bootstrap()->barrier();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommDestroy(ncclComm_t comm) {
  if (comm == nullptr) {
    WARN("comm is nullptr");
    return ncclInvalidArgument;
  }
#if defined(ENABLE_NPKIT)
  const std::string& npkitDumpDir = mscclpp::env()->npkitDumpDir;
  if (npkitDumpDir != "") {
    NpKit::Dump(npkitDumpDir);
    NpKit::Shutdown();
  }
#endif

  if (mscclppNcclDlopenSharedLib == true) {
    mscclppNcclOps.CommDestroy(*reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm));
    mscclppNcclDlopenFinalize();
    delete static_cast<ncclComm_t*>(comm->mscclppNcclComm);
  }
  delete comm;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommAbort(ncclComm_t) {
  // TODO: implement this function
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommSplit(ncclComm_t comm, int color, int key, ncclComm_t* newcomm, ncclConfig_t*) {
  *newcomm = NCCL_COMM_NULL;
  int nRanks = comm->comm->bootstrap()->getNranks();
  int rank = comm->comm->bootstrap()->getRank();
  splitCommInfo info{color, key, comm->comm->bootstrap()->getRank()};
  std::vector<splitCommInfo> infos(nRanks);
  infos[rank] = info;
  comm->comm->bootstrap()->allGather(infos.data(), sizeof(splitCommInfo));
  comm->comm->bootstrap()->barrier();
  std::vector<splitCommInfo> group;
  std::copy_if(infos.begin(), infos.end(), std::back_inserter(group),
               [color](const splitCommInfo& info) { return info.color == color; });
  std::sort(group.begin(), group.end(), [](const splitCommInfo& a, const splitCommInfo& b) { return a.key < b.key; });
  int newRank = std::distance(group.begin(),
                              std::find_if(group.begin(), group.end(),
                                           [rank](const splitCommInfo& info) { return info.originalRank == rank; }));
  int groupSize = group.size();
  ncclUniqueId uniqueId;
  if (newRank == 0) {
    ncclGetUniqueId(&uniqueId);
  }
  std::vector<ncclUniqueId> uniqueIds(nRanks);
  uniqueIds[rank] = uniqueId;
  comm->comm->bootstrap()->allGather(uniqueIds.data(), sizeof(ncclUniqueId));
  comm->comm->bootstrap()->barrier();
  uniqueId = uniqueIds[group.front().originalRank];
  if (color == NCCL_SPLIT_NOCOLOR) {
    return ncclSuccess;
  }
  return ncclCommInitRankConfig(newcomm, groupSize, uniqueId, newRank, nullptr);
}

ncclResult_t ncclCommInitRankScalable(ncclComm_t*, int, int, int, ncclUniqueId*, ncclConfig_t*) {
  WARN("ncclCommInitRankScalable is currently unavailable");
  return ncclInternalError;
}

NCCL_API const char* ncclGetErrorString(ncclResult_t result) {
  switch (result) {
    case ncclSuccess:
      return "no error";
    case ncclUnhandledCudaError:
      return "unhandled cuda error (run with MSCCLPP_DEBUG=INFO for details)";
    case ncclSystemError:
      return "unhandled system error (run with MSCCLPP_DEBUG=INFO for details)";
    case ncclInternalError:
      return "internal error (run with MSCCLPP_DEBUG=WARN for details)";
    case ncclInvalidArgument:
      return "invalid argument (run with MSCCLPP_DEBUG=WARN for details)";
    case ncclInvalidUsage:
      return "invalid usage (run with MSCCLPP_DEBUG=WARN for details)";
    case ncclRemoteError:
      return "remote process exited or there was a network error";
    case ncclInProgress:
      return "NCCL operation in progress";
    default:
      return "unknown result code";
  }
}

NCCL_API const char* ncclGetLastError(ncclComm_t) {
  // TODO: implement this function
  return "";
}

NCCL_API ncclResult_t ncclCommGetAsyncError(ncclComm_t, ncclResult_t* asyncError) {
  if (asyncError == nullptr) {
    WARN("asyncError is nullptr");
    return ncclInvalidArgument;
  }
  *asyncError = ncclSuccess;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommCount(const ncclComm_t comm, int* count) {
  if (comm == nullptr || count == nullptr) {
    WARN("comm is nullptr or count is nullptr");
    return ncclInvalidArgument;
  }
  *count = comm->comm->bootstrap()->getNranks();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommCuDevice(const ncclComm_t comm, int* device) {
  if (comm == nullptr || device == nullptr) {
    WARN("comm is nullptr or device is nullptr");
    return ncclInvalidArgument;
  }
  *device = comm->comm->bootstrap()->getRank();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommUserRank(const ncclComm_t comm, int* rank) {
  if (comm == nullptr || rank == nullptr) {
    WARN("comm is nullptr or rank is nullptr");
    return ncclInvalidArgument;
  }

  if (mscclppNcclDlopenSharedLib == true) {
    return mscclppNcclOps.CommUserRank(*reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm), rank);
  }

  *rank = comm->comm->bootstrap()->getRank();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommWindowRegister(ncclComm_t, void*, size_t, ncclWindow_t*, int) {
  WARN("ncclCommWindowRegister is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclCommWindowDeregister(ncclComm_t, ncclWindow_t) {
  WARN("ncclCommWindowDeregister is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclRedOpCreatePreMulSum(ncclRedOp_t*, void*, ncclDataType_t, ncclScalarResidence_t, ncclComm_t) {
  // TODO: implement this function
  WARN("ncclRedOpCreatePreMulSum is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclRedOpDestroy(ncclRedOp_t, ncclComm_t) {
  // TODO: implement this function
  WARN("ncclRedOpDestroy is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclReduce(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                 ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  // TODO: implement this function
  if (mscclppNcclDlopenSharedLib == true) {
    return mscclppNcclOps.Reduce(sendbuff, recvbuff, count, datatype, op, root,
                                 *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm), stream);
  }
  WARN("ncclReduce is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclBcast(void* buff, size_t count, ncclDataType_t datatype, int root, ncclComm_t comm,
                                hipStream_t stream) {
  return ncclBroadcast(buff, buff, count, datatype, root, comm, stream);
}

NCCL_API ncclResult_t ncclBroadcast(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                    int root, ncclComm_t comm, hipStream_t stream) {
  size_t bytes = count * ncclTypeSize(datatype);
  if (comm->worldSize == 1) {
    if (sendbuff != recvbuff) {
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, bytes, hipMemcpyDeviceToDevice, stream));
    }
    return ncclSuccess;
  }
  int rank = comm->comm->bootstrap()->getRank();
  if ((sendbuff == nullptr && root == rank) || recvbuff == nullptr || bytes == 0 || comm == nullptr) {
    WARN(
        "One or more of the following conditions is met: sendbuff or recvbuff pointer is nullptr, bytes is 0, "
        "or comm is nullptr.");
    return ncclInvalidArgument;
  }

  INFO(MSCCLPP_NCCL, "rank %d broadcast sendbuff %p recvbuff %p count %ld, dtype %d, comm: %p", rank, sendbuff,
       recvbuff, count, datatype, comm);

  const char* fallbackList = mscclpp::env()->forceNcclFallbackOperation.c_str();
  if (mscclppNcclDlopenSharedLib == true && mscclppNcclInFallbackList("broadcast", fallbackList)) {
    return mscclppNcclOps.Broadcast(sendbuff, recvbuff, count, datatype, root,
                                    *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm), stream);
  }

  std::vector<executionPlanInstance>& plans = comm->executionPlans["broadcast"];
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
  bool inPlace = sendbuff == recvbuff;
  for (const auto& p : plans) {
    if (bytes >= p.key.minMessageSize && bytes < p.key.maxMessageSize && inPlace == p.key.isInPlace) {
      plan = p.plan;
      break;
    }
  }

  if (plan != nullptr) {
    return executeWithPlan(comm->executor, rank, datatype, sendbuff, recvbuff, bytes, bytes, plan, stream);
  }
  auto algo = comm->algorithmCollection->selectAlgorithm(
      "broadcast", sendbuff, recvbuff, count * ncclTypeSize(datatype), comm->comm->bootstrap()->getNranksPerNode(),
      comm->comm->bootstrap()->getNranks());
  if (!algo.isEmpty()) {
    std::unordered_map<std::string, std::shared_ptr<void>> extras{
        {"root", std::make_shared<int>(root)},
        {"scratch", comm->scratchBuffer_},
        {"scratch_size", std::make_shared<size_t>(comm->scratchBufferSize_)}};
    return static_cast<ncclResult_t>(algo.launch(comm->comm, sendbuff, recvbuff, count, datatype, stream, extras));
  }

  if (mscclppNcclDlopenSharedLib == true) {
    return mscclppNcclOps.Broadcast(sendbuff, recvbuff, count, datatype, root,
                                    *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm), stream);
  }

  WARN("No FallBack implementation for broadcast");
  return ncclInvalidUsage;
}

NCCL_API ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                    ncclRedOp_t reductionOperation, ncclComm_t comm, hipStream_t stream) {
  size_t bytes = count * ncclTypeSize(datatype);
  if (comm->worldSize == 1) {
    if (sendbuff != recvbuff) {
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, bytes, hipMemcpyDeviceToDevice, stream));
    }
    return ncclSuccess;
  }
  // Checking if the parameters are valids
  if (sendbuff == nullptr || recvbuff == nullptr || count == 0 || ncclTypeSize(datatype) == 0 || comm == nullptr) {
    WARN(
        "One or more of the following conditions is met: sendbuff or recvbuff pointer is nullptr, count is 0, "
        "datatype is invalid, or comm is nullptr.");
    return ncclInvalidArgument;
  }
  // Declarating variables
  int rank = comm->comm->bootstrap()->getRank();
  INFO(MSCCLPP_NCCL, "rank %d allreduce sendbuff %p recvbuff %p count %ld, dtype %d comm is %p", rank, sendbuff,
       recvbuff, count, datatype, comm);

  const char* fallbackList = mscclpp::env()->forceNcclFallbackOperation.c_str();
  if (mscclppNcclDlopenSharedLib && mscclppNcclInFallbackList("allreduce", fallbackList)) {
    return mscclppNcclOps.AllReduce(sendbuff, recvbuff, count, datatype, reductionOperation,
                                    *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm), stream);
  }

  std::vector<executionPlanInstance>& plans = comm->executionPlans["allreduce"];
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
  bool inPlace = sendbuff == recvbuff;
  for (const auto& p : plans) {
    if (bytes >= p.key.minMessageSize && bytes < p.key.maxMessageSize && inPlace == p.key.isInPlace) {
      plan = p.plan;
      break;
    }
  }

  if (plan != nullptr) {
    return executeWithPlan(comm->executor, rank, datatype, sendbuff, recvbuff, bytes, bytes, plan, stream);
  }

  auto algo = comm->algorithmCollection->selectAlgorithm(
      "allreduce", sendbuff, recvbuff, count * ncclTypeSize(datatype), comm->comm->bootstrap()->getNranksPerNode(),
      comm->comm->bootstrap()->getNranks());
  if (!algo.isEmpty()) {
    std::unordered_map<std::string, std::shared_ptr<void>> extras{
        {"op", std::make_shared<int>(reductionOperation)},
        {"scratch", comm->scratchBuffer_},
        {"scratch_size", std::make_shared<size_t>(comm->scratchBufferSize_)}};
    return static_cast<ncclResult_t>(algo.launch(comm->comm, sendbuff, recvbuff, count, datatype, stream, extras));
  }

  if (mscclppNcclDlopenSharedLib == true) {
    return mscclppNcclOps.AllReduce(sendbuff, recvbuff, count, datatype, reductionOperation,
                                    *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm), stream);
  }

  WARN("No FallBack implementation for AllReduce");
  return ncclInvalidUsage;
}

NCCL_API ncclResult_t ncclReduceScatter(const void* sendbuff, void* recvbuff, size_t recvcount, ncclDataType_t datatype,
                                        ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
  size_t bytes = recvcount * ncclTypeSize(datatype);
  if (comm->worldSize == 1) {
    if (sendbuff != recvbuff) {
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, bytes, hipMemcpyDeviceToDevice, stream));
    }
    return ncclSuccess;
  }

  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr) {
    WARN(
        "One or more of the following conditions is met: sendbuff or recvbuff pointer is nullptr, bytes is 0, "
        "or comm is nullptr.");
    return ncclInvalidArgument;
  }

  INFO(MSCCLPP_NCCL, "ReduceScatter recvcount: %ld, datatype: %d, op: %d, messageSize: %ld", recvcount, datatype, op,
       bytes * comm->comm->bootstrap()->getNranks());

  const char* fallbackList = mscclpp::env()->forceNcclFallbackOperation.c_str();
  if (mscclppNcclDlopenSharedLib == true && mscclppNcclInFallbackList("reducescatter", fallbackList)) {
    return mscclppNcclOps.ReduceScatter(sendbuff, recvbuff, recvcount, datatype, op,
                                        *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm), stream);
  }

  int rank = comm->comm->bootstrap()->getRank();
  int nRank = comm->comm->bootstrap()->getNranks();

  std::vector<executionPlanInstance>& plans = comm->executionPlans["reducescatter"];
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
  void* basePtr = (char*)sendbuff + rank * bytes;
  bool inPlace = basePtr == recvbuff;
  const size_t totalBytes = bytes * nRank;
  for (const auto& p : plans) {
    if (totalBytes >= p.key.minMessageSize && totalBytes < p.key.maxMessageSize && inPlace == p.key.isInPlace) {
      plan = p.plan;
      break;
    }
  }

  if (plan != nullptr) {
    return executeWithPlan(comm->executor, rank, datatype, sendbuff, recvbuff, totalBytes, bytes, plan, stream);
  }

  if (mscclppNcclDlopenSharedLib == true) {
    return mscclppNcclOps.ReduceScatter(sendbuff, recvbuff, recvcount, datatype, op,
                                        *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm), stream);
  }

  WARN("No FallBack implementation for ReduceScatter");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllGather(const void* sendbuff, void* recvbuff, size_t sendcount, ncclDataType_t datatype,
                                    ncclComm_t comm, hipStream_t stream) {
  size_t bytes = sendcount * ncclTypeSize(datatype);
  if (comm->worldSize == 1) {
    if (sendbuff != recvbuff) {
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, bytes, hipMemcpyDeviceToDevice, stream));
    }
    return ncclSuccess;
  }
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr) {
    WARN(
        "One or more of the following conditions is met: sendbuff or recvbuff pointer is nullptr, bytes is 0, "
        "or comm is nullptr.");
    return ncclInvalidArgument;
  }

  int rank = comm->comm->bootstrap()->getRank();
  int nRank = comm->comm->bootstrap()->getNranks();
  INFO(MSCCLPP_NCCL, "rank %d allgather sendbuff %p recvbuff %p count %ld, dtype %d, comm %p", rank, sendbuff, recvbuff,
       sendcount, datatype, comm);

  const char* fallbackList = mscclpp::env()->forceNcclFallbackOperation.c_str();
  if (mscclppNcclDlopenSharedLib == true && mscclppNcclInFallbackList("allgather", fallbackList)) {
    return mscclppNcclOps.AllGather(sendbuff, recvbuff, sendcount, datatype,
                                    *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm), stream);
  }

  std::vector<executionPlanInstance>& plans = comm->executionPlans["allgather"];
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
  void* basePtr = (char*)sendbuff - rank * bytes;
  bool inPlace = basePtr == recvbuff;
  const size_t totalBytes = bytes * nRank;
  for (const auto& p : plans) {
    if (totalBytes >= p.key.minMessageSize && totalBytes < p.key.maxMessageSize && inPlace == p.key.isInPlace) {
      plan = p.plan;
      break;
    }
  }

  if (plan != nullptr) {
    return executeWithPlan(comm->executor, rank, datatype, sendbuff, recvbuff, bytes, totalBytes, plan, stream);
  }

  auto algo = comm->algorithmCollection->selectAlgorithm(
      "allgather", sendbuff, recvbuff, nRank * sendcount * ncclTypeSize(datatype),
      comm->comm->bootstrap()->getNranksPerNode(), comm->comm->bootstrap()->getNranks());
  if (!algo.isEmpty()) {
    std::unordered_map<std::string, std::shared_ptr<void>> extras = {
        {"scratch", comm->scratchBuffer_}, {"scratch_size", std::make_shared<size_t>(comm->scratchBufferSize_)}};
    return static_cast<ncclResult_t>(algo.launch(comm->comm, sendbuff, recvbuff, sendcount, datatype, stream, extras));
  }

  if (mscclppNcclDlopenSharedLib == true) {
    return mscclppNcclOps.AllGather(sendbuff, recvbuff, sendcount, datatype,
                                    *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm), stream);
  }

  WARN("No FallBack implementation for AllGather");
  return ncclInvalidUsage;
}

NCCL_API ncclResult_t ncclSend(const void* sendbuff, size_t count, ncclDataType_t datatype, int peer, ncclComm_t comm,
                               hipStream_t stream) {
  if (mscclppNcclDlopenSharedLib == true) {
    return mscclppNcclOps.Send(sendbuff, count, datatype, peer, *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm),
                               stream);
  }
  WARN("ncclSend is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclRecv(void* recvbuff, size_t count, ncclDataType_t datatype, int peer, ncclComm_t comm,
                               hipStream_t stream) {
  if (mscclppNcclDlopenSharedLib == true) {
    return mscclppNcclOps.Recv(recvbuff, count, datatype, peer, *reinterpret_cast<ncclComm_t*>(comm->mscclppNcclComm),
                               stream);
  }
  WARN("ncclRecv is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllToAll(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                   ncclComm_t comm, hipStream_t stream) {
  size_t bytes = count * ncclTypeSize(datatype);
  if (comm->worldSize == 1) {
    if (sendbuff != recvbuff) {
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, bytes, hipMemcpyDeviceToDevice, stream));
    }
    return ncclSuccess;
  }
  // TODO: implement this function
  WARN("ncclAllToAll is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllToAllv(const void* sendbuff, [[maybe_unused]] const size_t sendcounts[],
                                    const size_t sdispls[], void* recvbuff, const size_t recvcounts[],
                                    const size_t rdispls[], ncclDataType_t datatype, ncclComm_t comm,
                                    hipStream_t stream) {
  size_t bytes = recvcounts[0] * ncclTypeSize(datatype);
  if (comm->worldSize == 1) {
    MSCCLPP_CUDATHROW(hipMemcpyAsync((char*)recvbuff + rdispls[0] * ncclTypeSize(datatype),
                                      (const char*)sendbuff + sdispls[0] * ncclTypeSize(datatype), bytes,
                                      hipMemcpyDeviceToDevice, stream));
    return ncclSuccess;
  }
  WARN("ncclAllToAllv is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclGroupStart() {
  if (!tryLoadNcclSharedLib()) {
    WARN("Failed to load the shared library for nccl/rccl");
    return ncclInternalError;
  }
  if (mscclppNcclDlopenSharedLib == true) {
    return mscclppNcclOps.GroupStart();
  }
  WARN("ncclGroupStart is currently unavailable, return success");
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGroupEnd() {
  if (mscclppNcclDlopenSharedLib == true) {
    return mscclppNcclOps.GroupEnd();
  }
  WARN("ncclGroupEnd is currently unavailable, return success");
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGroupSimulateEnd(ncclSimInfo_t*) {
  // TODO: implement this function
  WARN("ncclGroupSimulateEnd is not implemented");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclCommRegister(const ncclComm_t, void*, size_t, void**) {
  // TODO: Implementation
  WARN("ncclCommRegister is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclCommDeregister(const ncclComm_t, void*) {
  // TODO: Implementation
  WARN("ncclCommDeregister is currently unavailable");
  return ncclInternalError;
}

ncclResult_t ncclMemAlloc(void** ptr, size_t size) {
  if (ptr == nullptr || size == 0) {
    WARN("ptr is nullptr or size is 0");
    return ncclInvalidArgument;
  }
  std::shared_ptr<char> sharedPtr;
  try {
    sharedPtr = mscclpp::GpuBuffer(size).memory();
    if (sharedPtr == nullptr) {
      WARN("Failed to allocate memory via ncclMemAlloc");
      return ncclSystemError;
    }
  } catch (const mscclpp::Error& e) {
    if (e.getErrorCode() == mscclpp::ErrorCode::InvalidUsage) {
      WARN("Invalid usage: %s", e.what());
      return ncclInvalidUsage;
    } else {
      WARN("Internal error: %s", e.what());
      return ncclInternalError;
    }
  } catch (const mscclpp::CudaError& e) {
    WARN("Cuda error: %s", e.what());
    return ncclUnhandledCudaError;
  } catch (const mscclpp::CuError& e) {
    WARN("Cu error: %s", e.what());
    return ncclUnhandledCudaError;
  } catch (const mscclpp::BaseError& e) {
    WARN("Base error: %s", e.what());
    return ncclInternalError;
  }
  ptrMap[sharedPtr.get()] = sharedPtr;

  // Return the pointer
  *ptr = sharedPtr.get();
  return ncclSuccess;
}

ncclResult_t ncclMemFree(void* ptr) {
  auto ptrIt = ptrMap.find(ptr);
  if (ptrIt != ptrMap.end()) {
    ptrMap.erase(ptrIt);
    return ncclSuccess;
  }

  // Pointer not found
  WARN("Pointer not found");
  return ncclInvalidUsage;
}
