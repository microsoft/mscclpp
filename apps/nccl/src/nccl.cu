// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <algorithm>
#include <filesystem>
#include <mscclpp/concurrency_device.hpp>
#include <mscclpp/core.hpp>
#include <mscclpp/executor.hpp>
#include <mscclpp/sm_channel.hpp>
#include <mscclpp/sm_channel_device.hpp>
#include <mscclpp/utils.hpp>
#include <sstream>
#include <unordered_map>
#include <vector>
#if defined(ENABLE_NPKIT)
#include <mscclpp/npkit/npkit.hpp>
#endif
#include "allgather.hpp"
#include "allreduce.hpp"
#include "broadcast.hpp"
#include "debug.h"
#include "nccl.h"

#define NCCL_API extern "C" __attribute__((visibility("default")))

#define CUDACHECK(cmd)                                                                      \
  do {                                                                                      \
    hipError_t e = cmd;                                                                    \
    if (e != hipSuccess) {                                                                 \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
      exit(EXIT_FAILURE);                                                                   \
    }                                                                                       \
  } while (0)

#define NUM_CHANNELS_PER_CONNECTION 64

// static const mscclpp::Transport IBs[] = {mscclpp::Transport::IB0, mscclpp::Transport::IB1, mscclpp::Transport::IB2,
//                             mscclpp::Transport::IB3, mscclpp::Transport::IB4, mscclpp::Transport::IB5,
//                             mscclpp::Transport::IB6, mscclpp::Transport::IB7};

// Declare the global map to store associations between raw pointer and shared pointer
static std::unordered_map<void*, std::shared_ptr<char>> ptrMap;

struct channelKey {
  const void* buff;
  size_t bytes;
  bool operator==(const channelKey& other) const { return buff == other.buff && bytes == other.bytes; }
};

struct planKey {
  size_t minMessageSize;
  size_t maxMessageSize;
  bool isInPlace;
};

struct executionPlanInstance {
  planKey key;
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
};

namespace std {
template <>
struct hash<channelKey> {
  std::size_t operator()(const channelKey& k) const {
    return std::hash<const void*>()(k.buff) ^ std::hash<size_t>()(k.bytes);
  }
};
}  // namespace std

struct ChannelInfo {
  std::vector<mscclpp::SmChannel> smChannels;
  std::shared_ptr<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
};

struct splitCommInfo {
  int color;
  int key;
  int originalRank;
};

struct ncclComm {
  std::shared_ptr<mscclpp::Communicator> comm;
  std::vector<std::shared_ptr<mscclpp::Connection>> connections;
  std::vector<std::shared_ptr<mscclpp::SmDevice2DeviceSemaphore>> smSemaphores;
  std::shared_ptr<mscclpp::Executor> executor;
  std::unordered_map<std::string, std::vector<executionPlanInstance>> executionPlans;

  std::unordered_map<channelKey, ChannelInfo> channelInInfos;
  std::unordered_map<channelKey, ChannelInfo> channelOutInfos;
  std::unordered_map<channelKey, ChannelInfo> channelScratchInfos;
  std::shared_ptr<char> scratchBuff;
  std::vector<mscclpp::RegisteredMemory> remoteScratchRegMemories;

  uint32_t numScratchBuff;
  uint32_t buffFlag;
};

static size_t ncclTypeSize(ncclDataType_t type) {
  switch (type) {
    case ncclInt8:
    case ncclUint8:
      return 1;
    case ncclFloat16:
      return 2;
    case ncclInt32:
    case ncclUint32:
      return 4;
    case ncclInt64:
    case ncclUint64:
      return 8;
    case ncclFloat32:
      return 4;
    case ncclFloat64:
      return 8;
#if defined(__CUDA_BF16_TYPES_EXIST__)
    case ncclBfloat16:
      return 2;
#endif  // defined(__CUDA_BF16_TYPES_EXIST__)
#if defined(__CUDA_FP8_TYPES_EXIST__)
    case ncclFp8E4M3:
    case ncclFp8E5M2:
      return 1;
#endif  // defined(__CUDA_FP8_TYPES_EXIST__)
    case ncclNumTypes:
      return 0;
  }
  return 0;
}

static mscclpp::Transport getTransport(int, int) {
  // if (rank / nRanksPerNode == peerRank / nRanksPerNode) {
  //   return mscclpp::Transport::CudaIpc;
  // } else {
  //   return IBs[rank % nRanksPerNode];
  // }
  return mscclpp::Transport::CudaIpc;
}

static std::vector<mscclpp::RegisteredMemory> setupRemoteMemories(std::shared_ptr<mscclpp::Communicator> comm, int rank,
                                                                  void* buff, size_t bytes,
                                                                  mscclpp::TransportFlags transport) {
  std::vector<mscclpp::RegisteredMemory> remoteMemories;
  mscclpp::RegisteredMemory memory = comm->registerMemory(buff, bytes, transport);
  std::vector<mscclpp::NonblockingFuture<mscclpp::RegisteredMemory>> remoteRegMemoryFutures;
  for (int i = 0; i < comm->bootstrap()->getNranks(); i++) {
    if (i == rank) continue;
    remoteRegMemoryFutures.push_back(comm->recvMemoryOnSetup(i, 0));
    comm->sendMemoryOnSetup(memory, i, 0);
  }
  comm->setup();
  std::transform(remoteRegMemoryFutures.begin(), remoteRegMemoryFutures.end(), std::back_inserter(remoteMemories),
                 [](const auto& future) { return future.get(); });
  return remoteMemories;
}

static std::vector<mscclpp::SmChannel> setupSmChannels(ncclComm_t comm,
                                                       const std::vector<mscclpp::RegisteredMemory>& remoteMemories,
                                                       void* src) {
  std::vector<mscclpp::SmChannel> channels;
  std::vector<std::shared_ptr<mscclpp::SmDevice2DeviceSemaphore>>& smSemaphores = comm->smSemaphores;
  size_t nConnections = comm->connections.size();
  for (size_t idx = 0; idx < NUM_CHANNELS_PER_CONNECTION; ++idx) {
    for (size_t cid = 0; cid < nConnections; ++cid) {
      if (comm->connections[cid]->transport() == mscclpp::Transport::CudaIpc) {
        channels.emplace_back(smSemaphores[idx * nConnections + cid], remoteMemories[cid], src, nullptr);
      }
    }
  }
  return channels;
}

static std::pair<std::string, executionPlanInstance> loadExecutionPlan(const std::string& filename) {
  std::shared_ptr<mscclpp::ExecutionPlan> plan = std::make_shared<mscclpp::ExecutionPlan>(filename);
  std::string collective = plan->collective();
  planKey key{plan->minMessageSize(), plan->maxMessageSize(), plan->isInPlace()};
  return std::make_pair(collective, executionPlanInstance{key, plan});
}

static std::shared_ptr<mscclpp::DeviceHandle<mscclpp::SmChannel>> setupSmChannelDeviceHandles(
    const std::vector<mscclpp::SmChannel>& smChannels) {
  std::vector<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
  std::transform(smChannels.begin(), smChannels.end(), std::back_inserter(smChannelDeviceHandles),
                 [](const mscclpp::SmChannel& smChannel) { return mscclpp::deviceHandle(smChannel); });
  std::shared_ptr<mscclpp::DeviceHandle<mscclpp::SmChannel>> ptr =
      mscclpp::detail::gpuCallocShared<mscclpp::DeviceHandle<mscclpp::SmChannel>>(smChannelDeviceHandles.size());
  mscclpp::gpuMemcpy<mscclpp::DeviceHandle<mscclpp::SmChannel>>(ptr.get(), smChannelDeviceHandles.data(),
                                                                smChannelDeviceHandles.size(), hipMemcpyHostToDevice);
  return ptr;
}

static ncclResult_t ncclAllReduceFallback(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                          ncclRedOp_t, ncclComm_t comm, hipStream_t stream) {
  // FallBack for single node
  if (comm->comm->bootstrap()->getNranks() != comm->comm->bootstrap()->getNranksPerNode()) {
    WARN("ncclAllReduceFallback is currently unavailable for multi-node");
    return ncclInvalidUsage;
  }

  // Checking if the parameters are valids
  if (sendbuff == nullptr || recvbuff == nullptr || count == 0 || ncclTypeSize(datatype) == 0 || comm == nullptr) {
    WARN(
        "One or more of the following conditions is met: sendbuff or recvbuff pointer is nullptr, count is 0, "
        "datatype is invalid, or comm is nullptr.");
    return ncclInvalidArgument;
  }

  // Declarating variables
  size_t sendBytes, recvBytes;
  hipDeviceptr_t sendBasePtr, recvBasePtr;
  MSCCLPP_CUTHROW(hipMemGetAddressRange(&sendBasePtr, &sendBytes, (hipDeviceptr_t)sendbuff));
  MSCCLPP_CUTHROW(hipMemGetAddressRange(&recvBasePtr, &recvBytes, (hipDeviceptr_t)recvbuff));
  size_t offsetIn = (char*)sendbuff - (char*)sendBasePtr;
  size_t offsetOut = (char*)recvbuff - (char*)recvBasePtr;
  uint32_t scratchBuffIdx = (++(comm->buffFlag)) % comm->numScratchBuff;
  size_t offsetScratch = (SCRATCH_SIZE / comm->numScratchBuff) * scratchBuffIdx;
  int rank = comm->comm->bootstrap()->getRank();
  channelKey sendKey{(void*)sendBasePtr, sendBytes};
  channelKey recvKey{(void*)recvBasePtr, recvBytes};
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels = nullptr;
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smOutChannels = nullptr;

  // Creating the channels
  if (count * ncclTypeSize(datatype) <= (1 << 20)) {
    auto sendIt = comm->channelScratchInfos.find(sendKey);
    if (sendIt == comm->channelScratchInfos.end()) {
      std::vector<mscclpp::SmChannel> channels =
          setupSmChannels(comm, comm->remoteScratchRegMemories, const_cast<void*>((void*)sendBasePtr));
      ChannelInfo channelInfo{channels, setupSmChannelDeviceHandles(channels)};
      sendIt = comm->channelScratchInfos.emplace(sendKey, channelInfo).first;
    }

    smChannels = sendIt->second.smChannelDeviceHandles.get();
  } else {
    std::vector<mscclpp::RegisteredMemory> remoteMemories;

    auto sendIt = comm->channelInInfos.find(sendKey);
    if (sendIt == comm->channelInInfos.end()) {
      std::vector<mscclpp::SmChannel> channels =
          setupSmChannels(comm, comm->remoteScratchRegMemories, const_cast<void*>((void*)sendBasePtr));
      ChannelInfo channelInfo{channels, setupSmChannelDeviceHandles(channels)};
      sendIt = comm->channelInInfos.emplace(sendKey, channelInfo).first;
    }

    auto recvIt = comm->channelOutInfos.find(recvKey);
    // if (recvIt == comm->channelOutInfos.end()) {
    {
      remoteMemories =
          setupRemoteMemories(comm->comm, rank, (void*)recvBasePtr, recvBytes, mscclpp::Transport::CudaIpc);
      std::vector<mscclpp::SmChannel> outChannels =
          setupSmChannels(comm, remoteMemories, const_cast<void*>((void*)recvBasePtr));
      ChannelInfo channelInfo{outChannels, setupSmChannelDeviceHandles(outChannels)};
      recvIt = comm->channelOutInfos.emplace(recvKey, channelInfo).first;
    }

    smChannels = sendIt->second.smChannelDeviceHandles.get();
    smOutChannels = recvIt->second.smChannelDeviceHandles.get();
  }

  switch (datatype) {
    case ncclFloat16:
      CUDACHECK(allreduce((half*)sendbuff, (half*)comm->scratchBuff.get(), (half*)recvbuff, smChannels, smOutChannels,
                          offsetIn, offsetOut, offsetScratch, rank, NRANKS_PER_NODE,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case ncclFloat32:
      CUDACHECK(allreduce((float*)sendbuff, (float*)comm->scratchBuff.get(), (float*)recvbuff, smChannels,
                          smOutChannels, offsetIn, offsetOut, offsetScratch, comm->comm->bootstrap()->getRank(),
                          NRANKS_PER_NODE, comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case ncclBfloat16:
      CUDACHECK(allreduce((__bfloat16*)sendbuff, (__bfloat16*)comm->scratchBuff.get(), (__bfloat16*)recvbuff,
                          smChannels, smOutChannels, offsetIn, offsetOut, offsetScratch, rank, NRANKS_PER_NODE,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case ncclInt32:
    case ncclUint32:
      CUDACHECK(allreduce((int*)sendbuff, (int*)comm->scratchBuff.get(), (int*)recvbuff, smChannels, smOutChannels,
                          offsetIn, offsetOut, offsetScratch, comm->comm->bootstrap()->getRank(), NRANKS_PER_NODE,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    default:
      WARN("datatype is invalid, datatype: %d", datatype);
      return ncclInvalidArgument;
  }
  // printf("allreduce rank %d, done\n", comm->comm->bootstrap()->getRank());
  return ncclSuccess;
}

static ncclResult_t ncclAllGatherFallback(const void* sendbuff, void* recvbuff, size_t sendcount,
                                          ncclDataType_t datatype, ncclComm_t comm, hipStream_t stream) {
  // return ncclSuccess;
  // return ncclInvalidArgument;
  // FallBack for single node
  if (comm->comm->bootstrap()->getNranks() != comm->comm->bootstrap()->getNranksPerNode()) {
    WARN("ncclAllGatherFallback is currently unavailable for multi-node");
    return ncclInvalidUsage;
  }

  // Checking if the parameters are valids
  size_t bytes = sendcount * ncclTypeSize(datatype);
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr) {
    WARN(
        "One or more of the following conditions is met: sendbuff or recvbuff pointer is nullptr, bytes is 0, "
        "or comm is nullptr.");
    return ncclInvalidArgument;
  }

  // printf("allgather rank %d, sendbuff: %p, recvbuff: %p, bytes: %zu\n", comm->comm->bootstrap()->getRank(), sendbuff,
  //        recvbuff, bytes);
  // Declarating variables
  size_t recvBytes;
  hipDeviceptr_t recvBasePtr;
  // size_t sendBytes;
  // hipDeviceptr_t sendBasePtr;
  MSCCLPP_CUTHROW(hipMemGetAddressRange(&recvBasePtr, &recvBytes, (hipDeviceptr_t)recvbuff));
  // MSCCLPP_CUTHROW(hipMemGetAddressRange(&sendBasePtr, &sendBytes, (hipDeviceptr_t)sendbuff));
  size_t offsetOut = (char*)recvbuff - (char*)recvBasePtr;
  channelKey recvKey{(void*)recvBasePtr, recvBytes};
  int rank = comm->comm->bootstrap()->getRank();
  int nRank = comm->comm->bootstrap()->getNranks();
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels = nullptr;

  // printf(
  //     "allgather rank %d, setupRemoteMemories start, recvBasePtr: %p, recvBytes %ld, sendBasePtr: %p, sendBytes %ld\n",
  //     comm->comm->bootstrap()->getRank(), recvBasePtr, recvBytes, sendBasePtr, sendBytes);

  auto it = comm->channelOutInfos.find(recvKey);
  // if (it == comm->channelOutInfos.end())
  {
    std::vector<mscclpp::RegisteredMemory> remoteMemories = setupRemoteMemories(
        comm->comm, rank, const_cast<void*>((void*)recvBasePtr), recvBytes, mscclpp::Transport::CudaIpc);
    // printf("allgather rank %d, setupRemoteMemories done, recvBasePtr: %p, recvBytes %ld\n",
    //        comm->comm->bootstrap()->getRank(), recvBasePtr, recvBytes);
    std::vector<mscclpp::SmChannel> channels =
        setupSmChannels(comm, remoteMemories, const_cast<void*>((void*)recvBasePtr));
    // printf("allgather rank %d, setupSmChannels done\n", comm->comm->bootstrap()->getRank());
    std::vector<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
    std::transform(channels.begin(), channels.end(), std::back_inserter(smChannelDeviceHandles),
                   [](const mscclpp::SmChannel& smChannel) { return mscclpp::deviceHandle(smChannel); });
    ChannelInfo channelInfo{channels, setupSmChannelDeviceHandles(channels)};
    it = comm->channelOutInfos.emplace(recvKey, channelInfo).first;
  }

  smChannels = it->second.smChannelDeviceHandles.get();
  if ((char*)sendbuff == (char*)recvbuff + rank * sendcount) {
    CUDACHECK(allgather<false>((int*)sendbuff, (int*)nullptr, (int*)recvbuff, smChannels, offsetOut, rank,
                               NRANKS_PER_NODE, nRank, bytes / sizeof(int), stream));
  } else {
    CUDACHECK(allgather<true>((int*)sendbuff, (int*)nullptr, (int*)recvbuff, smChannels, offsetOut, rank,
                              NRANKS_PER_NODE, nRank, bytes / sizeof(int), stream));
  }
  // printf("allgather rank %d, done\n", comm->comm->bootstrap()->getRank());
  return ncclSuccess;
}

static void ncclCommInitRankFallbackSingleNode(ncclComm* commPtr, std::shared_ptr<mscclpp::Communicator> mscclppComm,
                                               int rank) {
  std::vector<mscclpp::NonblockingFuture<std::shared_ptr<mscclpp::Connection>>> connectionFutures;

  for (int i = 0; i < mscclppComm->bootstrap()->getNranks(); i++) {
    if (i == rank) continue;
    mscclpp::Transport transport = getTransport(rank, i);
    connectionFutures.push_back(mscclppComm->connectOnSetup(i, 0, transport));
  }
  mscclppComm->setup();

  std::vector<std::shared_ptr<mscclpp::Connection>> connections;
  std::transform(connectionFutures.begin(), connectionFutures.end(), std::back_inserter(connections),
                 [](const auto& future) { return future.get(); });

  std::vector<std::shared_ptr<mscclpp::SmDevice2DeviceSemaphore>> smSemaphores;
  for (size_t idx = 0; idx < NUM_CHANNELS_PER_CONNECTION; ++idx) {
    for (size_t cid = 0; cid < connections.size(); ++cid) {
      if (connections[cid]->transport() == mscclpp::Transport::CudaIpc) {
        smSemaphores.emplace_back(
            std::make_shared<mscclpp::SmDevice2DeviceSemaphore>(*(mscclppComm), connections[cid]));
      }
    }
  }

  mscclppComm->setup();
  commPtr->connections = std::move(connections);
  commPtr->smSemaphores = std::move(smSemaphores);
  commPtr->buffFlag = 0;
  commPtr->numScratchBuff = 2;
  commPtr->scratchBuff = mscclpp::detail::gpuCallocShared<char>(SCRATCH_SIZE);
  commPtr->remoteScratchRegMemories =
      setupRemoteMemories(commPtr->comm, rank, commPtr->scratchBuff.get(), SCRATCH_SIZE, mscclpp::Transport::CudaIpc);
}

NCCL_API ncclResult_t ncclGetVersion(int* version) {
  if (version == nullptr) {
    WARN("version is nullptr");
    return ncclInvalidArgument;
  }
  *version = MSCCLPP_VERSION;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGetUniqueId(ncclUniqueId* uniqueId) {
  if (uniqueId == nullptr) {
    WARN("uniqueId is nullptr");
    return ncclInvalidArgument;
  }
  if (MSCCLPP_UNIQUE_ID_BYTES != NCCL_UNIQUE_ID_BYTES) return ncclInternalError;
  mscclpp::UniqueId id = mscclpp::TcpBootstrap::createUniqueId();
  memcpy(uniqueId, &id, sizeof(ncclUniqueId));
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommInitRankConfig(ncclComm_t* comm, int nranks, ncclUniqueId commId, int rank,
                                             ncclConfig_t*) {
  // TODO: implement config
  return ncclCommInitRank(comm, nranks, commId, rank);
}

NCCL_API ncclResult_t ncclCommInitRank(ncclComm_t* comm, int nranks, ncclUniqueId commId, int rank) {
  if (comm == nullptr) {
    WARN("comm is nullptr");
    return ncclInvalidArgument;
  }
  if (nranks < 0 || rank < 0 || rank >= nranks) {
    WARN("nranks is %d, rank is %d", nranks, rank);
    return ncclInvalidArgument;
  }
  std::shared_ptr<mscclpp::TcpBootstrap> bootstrap = std::make_shared<mscclpp::TcpBootstrap>(rank, nranks);
  mscclpp::UniqueId id;
  memcpy(id.data(), &commId, sizeof(ncclUniqueId));
  bootstrap->initialize(id);
  std::shared_ptr<mscclpp::Communicator> mscclppComm = std::make_shared<mscclpp::Communicator>(bootstrap);
  ncclComm* commPtr = new ncclComm();

  commPtr->comm = mscclppComm;
  commPtr->executor = std::make_shared<mscclpp::Executor>(mscclppComm);

  // FallBack for single node
  if (mscclppComm->bootstrap()->getNranks() == mscclppComm->bootstrap()->getNranksPerNode())
    ncclCommInitRankFallbackSingleNode(commPtr, mscclppComm, rank);

  if (getenv("MSCCLPP_EXECUTION_PLAN_DIR")) {
    std::string collectiveDir = getenv("MSCCLPP_EXECUTION_PLAN_DIR");
    if (!std::filesystem::is_directory(collectiveDir)) {
      WARN("The value of the environment variable %s is not a directory", collectiveDir.c_str());
      return ncclInvalidArgument;
    }
    for (const auto& entry : std::filesystem::directory_iterator(collectiveDir)) {
      if (entry.is_regular_file()) {
        auto plan = loadExecutionPlan(entry.path());
        commPtr->executionPlans[plan.first].push_back(plan.second);
      }
    }
  }

  *comm = commPtr;
#if defined(ENABLE_NPKIT)
  const char* npkitDumpDir = getenv("NPKIT_DUMP_DIR");
  if (npkitDumpDir != nullptr) {
    NpKit::Init(rank);
  }
#endif
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommInitAll(ncclComm_t*, int, const int*) {
  // TODO: implement this function
  WARN("ncclCommInitAll is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclCommFinalize(ncclComm_t comm) {
  comm->comm->bootstrap()->barrier();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommDestroy(ncclComm_t comm) {
  if (comm == nullptr) {
    WARN("comm is nullptr");
    return ncclInvalidArgument;
  }
#if defined(ENABLE_NPKIT)
  const char* npkitDumpDir = getenv("NPKIT_DUMP_DIR");
  if (npkitDumpDir != nullptr) {
    NpKit::Dump(npkitDumpDir);
    NpKit::Shutdown();
  }
#endif
  delete comm;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommAbort(ncclComm_t) {
  // TODO: implement this function
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommSplit(ncclComm_t comm, int color, int key, ncclComm_t* newcomm, ncclConfig_t*) {
  *newcomm = NCCL_COMM_NULL;
  int nRanks = comm->comm->bootstrap()->getNranks();
  int rank = comm->comm->bootstrap()->getRank();
  splitCommInfo info{color, key, comm->comm->bootstrap()->getRank()};
  std::vector<splitCommInfo> infos(nRanks);
  infos[rank] = info;
  comm->comm->bootstrap()->allGather(infos.data(), sizeof(splitCommInfo));
  comm->comm->bootstrap()->barrier();
  std::vector<splitCommInfo> group;
  std::copy_if(infos.begin(), infos.end(), std::back_inserter(group),
               [color](const splitCommInfo& info) { return info.color == color; });
  std::sort(group.begin(), group.end(), [](const splitCommInfo& a, const splitCommInfo& b) { return a.key < b.key; });
  int newRank = std::distance(group.begin(),
                              std::find_if(group.begin(), group.end(),
                                           [rank](const splitCommInfo& info) { return info.originalRank == rank; }));
  int groupSize = group.size();
  ncclUniqueId uniqueId;
  if (newRank == 0) {
    ncclGetUniqueId(&uniqueId);
  }
  std::vector<ncclUniqueId> uniqueIds(nRanks);
  uniqueIds[rank] = uniqueId;
  comm->comm->bootstrap()->allGather(uniqueIds.data(), sizeof(ncclUniqueId));
  comm->comm->bootstrap()->barrier();
  uniqueId = uniqueIds[group.front().originalRank];
  if (color == NCCL_SPLIT_NOCOLOR) {
    return ncclSuccess;
  }
  return ncclCommInitRankConfig(newcomm, groupSize, uniqueId, newRank, nullptr);
}

NCCL_API const char* ncclGetErrorString(ncclResult_t result) {
  switch (result) {
    case ncclSuccess:
      return "no error";
    case ncclUnhandledCudaError:
      return "unhandled cuda error (run with MSCCLPP_DEBUG=INFO for details)";
    case ncclSystemError:
      return "unhandled system error (run with MSCCLPP_DEBUG=INFO for details)";
    case ncclInternalError:
      return "internal error (run with MSCCLPP_DEBUG=WARN for details)";
    case ncclInvalidArgument:
      return "invalid argument (run with MSCCLPP_DEBUG=WARN for details)";
    case ncclInvalidUsage:
      return "invalid usage (run with MSCCLPP_DEBUG=WARN for details)";
    case ncclRemoteError:
      return "remote process exited or there was a network error";
    case ncclInProgress:
      return "NCCL operation in progress";
    default:
      return "unknown result code";
  }
}

NCCL_API const char* ncclGetLastError(ncclComm_t) {
  // TODO: implement this function
  return "";
}

NCCL_API ncclResult_t ncclCommGetAsyncError(ncclComm_t, ncclResult_t* asyncError) {
  if (asyncError == nullptr) {
    WARN("asyncError is nullptr");
    return ncclInvalidArgument;
  }
  *asyncError = ncclSuccess;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommCount(const ncclComm_t comm, int* count) {
  if (comm == nullptr || count == nullptr) {
    WARN("comm is nullptr or count is nullptr");
    return ncclInvalidArgument;
  }
  *count = comm->comm->bootstrap()->getNranks();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommCuDevice(const ncclComm_t comm, int* device) {
  if (comm == nullptr || device == nullptr) {
    WARN("comm is nullptr or device is nullptr");
    return ncclInvalidArgument;
  }
  *device = comm->comm->bootstrap()->getRank();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommUserRank(const ncclComm_t comm, int* rank) {
  if (comm == nullptr || rank == nullptr) {
    WARN("comm is nullptr or rank is nullptr");
    return ncclInvalidArgument;
  }
  *rank = comm->comm->bootstrap()->getRank();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclRedOpCreatePreMulSum(ncclRedOp_t*, void*, ncclDataType_t, ncclScalarResidence_t, ncclComm_t) {
  // TODO: implement this function
  WARN("ncclRedOpCreatePreMulSum is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclRedOpDestroy(ncclRedOp_t, ncclComm_t) {
  // TODO: implement this function
  WARN("ncclRedOpDestroy is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclReduce(const void*, void*, size_t, ncclDataType_t, ncclRedOp_t, int, ncclComm_t,
                                 hipStream_t) {
  // TODO: implement this function
  WARN("ncclReduce is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclBcast(void* buff, size_t count, ncclDataType_t datatype, int root, ncclComm_t comm,
                                hipStream_t stream) {
  return ncclBroadcast(buff, buff, count, datatype, root, comm, stream);
}

NCCL_API ncclResult_t ncclBroadcastFallback(const void* sendbuff, void* recvbuff, size_t sendcount,
                                            ncclDataType_t datatype, int root, ncclComm_t comm, hipStream_t stream) {
  size_t bytes = sendcount * ncclTypeSize(datatype);
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr) {
    WARN(
        "One or more of the following conditions is met: sendbuff or recvbuff pointer is nullptr, bytes is 0, "
        "or comm is nullptr.");
    return ncclInvalidArgument;
  }

  // Declarating variables
  size_t recvBytes;
  hipDeviceptr_t recvBasePtr;
  MSCCLPP_CUTHROW(hipMemGetAddressRange(&recvBasePtr, &recvBytes, (hipDeviceptr_t)recvbuff));
  // size_t offsetOut = (char*)recvbuff - (char*)recvBasePtr;
  size_t offsetOut = 0;
  // channelKey recvKey{(void*)recvBasePtr, recvBytes};
  channelKey recvKey{(void*)0x0, 0};  // Just create the channel once.
  int rank = comm->comm->bootstrap()->getRank();
  int nRank = comm->comm->bootstrap()->getNranks();
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels = nullptr;

  auto it = comm->channelOutInfos.find(recvKey);
  if (it == comm->channelOutInfos.end()) {
    // std::vector<mscclpp::RegisteredMemory> remoteMemories = setupRemoteMemories(
    //     comm->comm, rank, const_cast<void*>((void*)recvBasePtr), recvBytes, mscclpp::Transport::CudaIpc);
    // std::vector<mscclpp::SmChannel> channels =
    //     setupSmChannels(comm, remoteMemories, const_cast<void*>((void*)recvBasePtr));
    std::vector<mscclpp::SmChannel> channels =
        setupSmChannels(comm, comm->remoteScratchRegMemories, const_cast<void*>((void*)recvBasePtr));
    std::vector<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
    std::transform(channels.begin(), channels.end(), std::back_inserter(smChannelDeviceHandles),
                   [](const mscclpp::SmChannel& smChannel) { return mscclpp::deviceHandle(smChannel); });
    ChannelInfo channelInfo{channels, setupSmChannelDeviceHandles(channels)};
    it = comm->channelOutInfos.emplace(recvKey, channelInfo).first;
  }

  smChannels = it->second.smChannelDeviceHandles.get();
  if ((char*)sendbuff == (char*)recvbuff) {
    CUDACHECK(broadcast<false>((int*)sendbuff, (int*)comm->scratchBuff.get(), (int*)recvbuff, smChannels, offsetOut,
                               rank, NRANKS_PER_NODE, root, nRank, bytes / sizeof(int), stream));
  } else {
    CUDACHECK(broadcast<true>((int*)sendbuff, (int*)comm->scratchBuff.get(), (int*)recvbuff, smChannels, offsetOut,
                              rank, NRANKS_PER_NODE, root, nRank, bytes / sizeof(int), stream));
  }

  return ncclSuccess;
}

NCCL_API ncclResult_t ncclBroadcast(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                    int root, ncclComm_t comm, hipStream_t stream) {
  size_t bytes = count * ncclTypeSize(datatype);
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr) {
    WARN(
        "One or more of the following conditions is met: sendbuff or recvbuff pointer is nullptr, bytes is 0, "
        "or comm is nullptr.");
    return ncclInvalidArgument;
  }

  int rank = comm->comm->bootstrap()->getRank();

  std::vector<executionPlanInstance>& plans = comm->executionPlans["broadcast"];
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
  void* basePtr = (char*)sendbuff;
  bool inPlace = basePtr == recvbuff;
  const size_t totalBytes = bytes;
  for (const auto& p : plans) {
    if (totalBytes >= p.key.minMessageSize && totalBytes < p.key.maxMessageSize && inPlace == p.key.isInPlace) {
      plan = p.plan;
      break;
    }
  }

  if (plan == nullptr) return ncclBroadcastFallback(sendbuff, recvbuff, count, datatype, root, comm, stream);

  switch (datatype) {
    case ncclFloat16:
      comm->executor->execute(rank, (half*)sendbuff, (half*)recvbuff, bytes, bytes, mscclpp::DataType::FLOAT16, *plan,
                              stream);
      break;
    case ncclFloat32:
      comm->executor->execute(rank, (float*)sendbuff, (float*)recvbuff, bytes, bytes, mscclpp::DataType::FLOAT32, *plan,
                              stream);
      break;
    case ncclBfloat16:
      comm->executor->execute(rank, (__bfloat16*)sendbuff, (__bfloat16*)recvbuff, bytes, bytes,
                              mscclpp::DataType::BFLOAT16, *plan, stream);
      break;
    case ncclInt32:
    case ncclUint32:
      comm->executor->execute(rank, (int*)sendbuff, (int*)recvbuff, bytes, bytes, mscclpp::DataType::UINT32, *plan,
                              stream);
      break;
    default:
      WARN("datatype is invalid");
      return ncclInvalidArgument;
  }

  return ncclSuccess;
}

NCCL_API ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                    ncclRedOp_t reductionOperation, ncclComm_t comm, hipStream_t stream) {
  // Checking if the parameters are valids
  if (sendbuff == nullptr || recvbuff == nullptr || count == 0 || ncclTypeSize(datatype) == 0 || comm == nullptr) {
    WARN(
        "One or more of the following conditions is met: sendbuff or recvbuff pointer is nullptr, count is 0, "
        "datatype is invalid, or comm is nullptr.");
    return ncclInvalidArgument;
  }

  // Declarating variables
  size_t bytes = count * ncclTypeSize(datatype);
  int rank = comm->comm->bootstrap()->getRank();

  std::vector<executionPlanInstance>& plans = comm->executionPlans["allreduce"];
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
  bool inPlace = sendbuff == recvbuff;
  for (const auto& p : plans) {
    if (bytes >= p.key.minMessageSize && bytes < p.key.maxMessageSize && inPlace == p.key.isInPlace) {
      plan = p.plan;
      break;
    }
  }

  if (plan == nullptr)
    return ncclAllReduceFallback(sendbuff, recvbuff, count, datatype, reductionOperation, comm, stream);

  switch (datatype) {
    case ncclFloat16:
      comm->executor->execute(rank, (half*)sendbuff, (half*)recvbuff, bytes, bytes, mscclpp::DataType::FLOAT16, *plan,
                              stream, mscclpp::PacketType::LL8);
      break;
    case ncclFloat32:
      comm->executor->execute(rank, (float*)sendbuff, (float*)recvbuff, bytes, bytes, mscclpp::DataType::FLOAT32, *plan,
                              stream, mscclpp::PacketType::LL8);
      break;
    case ncclBfloat16:
      comm->executor->execute(rank, (__bfloat16*)sendbuff, (__bfloat16*)recvbuff, bytes, bytes,
                              mscclpp::DataType::BFLOAT16, *plan, stream, mscclpp::PacketType::LL8);
      break;
    case ncclInt32:
    case ncclUint32:
      comm->executor->execute(rank, (int*)sendbuff, (int*)recvbuff, bytes, bytes, mscclpp::DataType::UINT32, *plan,
                              stream, mscclpp::PacketType::LL8);
      break;
    default:
      WARN("datatype is invalid");
      return ncclInvalidArgument;
  }

  return ncclSuccess;
}

NCCL_API ncclResult_t ncclReduceScatter(const void*, void*, size_t, ncclDataType_t, ncclRedOp_t, ncclComm_t,
                                        hipStream_t) {
  // TODO: implement this function
  WARN("ncclReduceScatter is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllGather(const void* sendbuff, void* recvbuff, size_t sendcount, ncclDataType_t datatype,
                                    ncclComm_t comm, hipStream_t stream) {
  size_t bytes = sendcount * ncclTypeSize(datatype);
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr) {
    WARN(
        "One or more of the following conditions is met: sendbuff or recvbuff pointer is nullptr, bytes is 0, "
        "or comm is nullptr.");
    return ncclInvalidArgument;
  }

  int rank = comm->comm->bootstrap()->getRank();
  int nRank = comm->comm->bootstrap()->getNranks();

  std::vector<executionPlanInstance>& plans = comm->executionPlans["allgather"];
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
  void* basePtr = (char*)sendbuff - rank * bytes;
  bool inPlace = basePtr == recvbuff;
  const size_t totalBytes = bytes * nRank;
  for (const auto& p : plans) {
    if (totalBytes >= p.key.minMessageSize && totalBytes < p.key.maxMessageSize && inPlace == p.key.isInPlace) {
      plan = p.plan;
      break;
    }
  }
  if (plan == nullptr) return ncclAllGatherFallback(sendbuff, recvbuff, sendcount, datatype, comm, stream);

  switch (datatype) {
    case ncclFloat16:
      comm->executor->execute(rank, (half*)sendbuff, (half*)recvbuff, bytes, bytes * nRank, mscclpp::DataType::FLOAT16,
                              *plan, stream);
      break;
    case ncclFloat32:
      comm->executor->execute(rank, (float*)sendbuff, (float*)recvbuff, bytes, bytes * nRank,
                              mscclpp::DataType::FLOAT32, *plan, stream);
      break;
    case ncclBfloat16:
      comm->executor->execute(rank, (__bfloat16*)sendbuff, (__bfloat16*)recvbuff, bytes, bytes * nRank,
                              mscclpp::DataType::BFLOAT16, *plan, stream);
      break;
    case ncclInt32:
    case ncclUint32:
      comm->executor->execute(rank, (int*)sendbuff, (int*)recvbuff, bytes, bytes * nRank, mscclpp::DataType::UINT32,
                              *plan, stream);
      break;
    default:
      WARN("datatype is invalid");
      return ncclInvalidArgument;
  }

  return ncclSuccess;
}

NCCL_API ncclResult_t ncclSend(const void*, size_t, ncclDataType_t, int, ncclComm_t, hipStream_t) {
  // TODO: implement this function
  WARN("ncclSend is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclRecv(void*, size_t, ncclDataType_t, int, ncclComm_t, hipStream_t) {
  // TODO: implement this function
  WARN("ncclRecv is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllToAll(const void*, void*, size_t, ncclDataType_t, ncclComm_t, hipStream_t) {
  // TODO: implement this function
  WARN("ncclAllToAll is currently unavailable");
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclGroupStart() {
  // Do nothing
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGroupEnd() {
  // Do nothing
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommRegister(const ncclComm_t, void*, size_t, void**) {
  // TODO: Implementation
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommDeregister(const ncclComm_t, void*) {
  // TODO: Implementation
  return ncclSuccess;
}

ncclResult_t ncclMemAlloc(void** ptr, size_t size) {
  if (ptr == nullptr || size == 0) {
    WARN("ptr is nullptr or size is 0");
    return ncclInvalidArgument;
  }
  std::shared_ptr<char> sharedPtr;
  try {
    sharedPtr = mscclpp::GpuBuffer(size).memory();
    if (sharedPtr == nullptr) {
      INFO(MSCCLPP_ALLOC, "Failed to allocate memory");
      return ncclSystemError;
    }
  } catch (const mscclpp::Error& e) {
    if (e.getErrorCode() == mscclpp::ErrorCode::InvalidUsage) {
      WARN("Invalid usage: %s", e.what());
      return ncclInvalidUsage;
    } else {
      WARN("Internal error: %s", e.what());
      return ncclInternalError;
    }
  } catch (const mscclpp::CudaError& e) {
    INFO(MSCCLPP_ALLOC, "Cuda error: %s", e.what());
    return ncclUnhandledCudaError;
  } catch (const mscclpp::CuError& e) {
    INFO(MSCCLPP_ALLOC, "Cu error: %s", e.what());
    return ncclUnhandledCudaError;
  } catch (const mscclpp::BaseError& e) {
    WARN("Base error: %s", e.what());
    return ncclInternalError;
  }
  ptrMap[sharedPtr.get()] = sharedPtr;

  // Return the pointer
  *ptr = sharedPtr.get();
  return ncclSuccess;
}

ncclResult_t ncclMemFree(void* ptr) {
  auto ptrIt = ptrMap.find(ptr);
  if (ptrIt != ptrMap.end()) {
    ptrMap.erase(ptrIt);
    return ncclSuccess;
  }

  // Pointer not found
  WARN("Pointer not found");
  return ncclInvalidUsage;
}
