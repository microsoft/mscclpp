// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <algorithm>
#include <filesystem>
#include <mscclpp/concurrency_device.hpp>
#include <mscclpp/core.hpp>
#include <mscclpp/executor.hpp>
#include <mscclpp/sm_channel.hpp>
#include <mscclpp/sm_channel_device.hpp>
#include <mscclpp/utils.hpp>
#include <sstream>
#include <unordered_map>
#include <vector>

#include "allgather.hpp"
#include "allreduce.hpp"
#include "nccl.h"

#define NCCL_API extern "C" __attribute__((visibility("default")))

#define CUDACHECK(cmd)                                                                      \
  do {                                                                                      \
    hipError_t e = cmd;                                                                    \
    if (e != hipSuccess) {                                                                 \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
      exit(EXIT_FAILURE);                                                                   \
    }                                                                                       \
  } while (0)

#define NUM_CHANNELS_PER_CONNECTION 64

// static const mscclpp::Transport IBs[] = {mscclpp::Transport::IB0, mscclpp::Transport::IB1, mscclpp::Transport::IB2,
//                             mscclpp::Transport::IB3, mscclpp::Transport::IB4, mscclpp::Transport::IB5,
//                             mscclpp::Transport::IB6, mscclpp::Transport::IB7};

// Declare the global map to store associations between raw pointer and shared pointer
static std::unordered_map<void*, std::shared_ptr<char>> ptrMap;

struct channelKey {
  const void* buff;
  size_t bytes;
  bool operator==(const channelKey& other) const { return buff == other.buff && bytes == other.bytes; }
};

struct planKey {
  size_t minMessageSize;
  size_t maxMessageSize;
  bool isInPlace;
};

struct executionPlanInstance {
  planKey key;
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
};

namespace std {
template <>
struct hash<channelKey> {
  std::size_t operator()(const channelKey& k) const {
    return std::hash<const void*>()(k.buff) ^ std::hash<size_t>()(k.bytes);
  }
};
}  // namespace std

struct ChannelInfo {
  std::vector<mscclpp::SmChannel> smChannels;
  std::shared_ptr<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
};

struct ncclComm {
  std::shared_ptr<mscclpp::Communicator> comm;
  std::vector<std::shared_ptr<mscclpp::Connection>> connections;
  std::vector<std::shared_ptr<mscclpp::SmDevice2DeviceSemaphore>> smSemaphores;
  std::shared_ptr<mscclpp::Executor> executor;
  std::unordered_map<std::string, std::vector<executionPlanInstance>> executionPlans;

  std::unordered_map<channelKey, ChannelInfo> channelInInfos;
  std::unordered_map<channelKey, ChannelInfo> channelOutInfos;
  std::unordered_map<channelKey, ChannelInfo> channelScratchInfos;
  std::shared_ptr<char> scratchBuff;
  std::vector<mscclpp::RegisteredMemory> remoteScratchRegMemories;

  uint32_t numScratchBuff;
  uint32_t buffFlag;
};

static size_t ncclTypeSize(ncclDataType_t type) {
  switch (type) {
    case ncclInt8:
    case ncclUint8:
      return 1;
    case ncclFloat16:
      return 2;
    case ncclInt32:
    case ncclUint32:
      return 4;
    case ncclInt64:
    case ncclUint64:
      return 8;
    case ncclFloat32:
      return 4;
    case ncclFloat64:
      return 8;
#if defined(__CUDA_BF16_TYPES_EXIST__)
    case ncclBfloat16:
      return 2;
#endif  // defined(__CUDA_BF16_TYPES_EXIST__)
#if defined(__CUDA_FP8_TYPES_EXIST__)
    case ncclFp8E4M3:
    case ncclFp8E5M2:
      return 1;
#endif  // defined(__CUDA_FP8_TYPES_EXIST__)
    case ncclNumTypes:
      return 0;
  }
  return 0;
}

static double parseSize(const char* value) {
  std::string valueStr(value);
  std::istringstream iss(valueStr);
  long long int units;
  double size;
  char size_lit = 0;

  if (iss >> size) {
    iss >> std::ws;  // eat whitespace
    iss >> size_lit;
  } else {
    return -1.0;
  }

  if (size_lit != 0 && !std::isspace(size_lit)) {
    switch (size_lit) {
      case 'G':
      case 'g':
        units = 1024 * 1024 * 1024;
        break;
      case 'M':
      case 'm':
        units = 1024 * 1024;
        break;
      case 'K':
      case 'k':
        units = 1024;
        break;
      default:
        return -1.0;
    };
  } else {
    units = 1;
  }
  return size * units;
}

static mscclpp::Transport getTransport(int, int) {
  // if (rank / nRanksPerNode == peerRank / nRanksPerNode) {
  //   return mscclpp::Transport::CudaIpc;
  // } else {
  //   return IBs[rank % nRanksPerNode];
  // }
  return mscclpp::Transport::CudaIpc;
}

static std::vector<mscclpp::RegisteredMemory> setupRemoteMemories(std::shared_ptr<mscclpp::Communicator> comm, int rank,
                                                                  void* buff, size_t bytes,
                                                                  mscclpp::TransportFlags transport) {
  std::vector<mscclpp::RegisteredMemory> remoteMemories;
  mscclpp::RegisteredMemory memory = comm->registerMemory(buff, bytes, transport);
  std::vector<mscclpp::NonblockingFuture<mscclpp::RegisteredMemory>> remoteRegMemoryFutures;
  for (int i = 0; i < comm->bootstrap()->getNranks(); i++) {
    if (i == rank) continue;
    remoteRegMemoryFutures.push_back(comm->recvMemoryOnSetup(i, 0));
    comm->sendMemoryOnSetup(memory, i, 0);
  }
  comm->setup();
  std::transform(remoteRegMemoryFutures.begin(), remoteRegMemoryFutures.end(), std::back_inserter(remoteMemories),
                 [](const auto& future) { return future.get(); });
  return remoteMemories;
}

static std::vector<mscclpp::SmChannel> setupSmChannels(ncclComm_t comm,
                                                       const std::vector<mscclpp::RegisteredMemory>& remoteMemories,
                                                       void* src) {
  std::vector<mscclpp::SmChannel> channels;
  std::vector<std::shared_ptr<mscclpp::SmDevice2DeviceSemaphore>>& smSemaphores = comm->smSemaphores;
  size_t nConnections = comm->connections.size();
  for (size_t idx = 0; idx < NUM_CHANNELS_PER_CONNECTION; ++idx) {
    for (size_t cid = 0; cid < nConnections; ++cid) {
      if (comm->connections[cid]->transport() == mscclpp::Transport::CudaIpc) {
        channels.emplace_back(smSemaphores[idx * nConnections + cid], remoteMemories[cid], src, nullptr);
      }
    }
  }
  return channels;
}

static std::pair<std::string, executionPlanInstance> loadExecutionPlan(const std::string& filename) {
  std::shared_ptr<mscclpp::ExecutionPlan> plan = std::make_shared<mscclpp::ExecutionPlan>(filename);
  std::string collective = plan->collective();
  planKey key{plan->minMessageSize(), plan->maxMessageSize(), plan->isInPlace()};
  return std::make_pair(collective, executionPlanInstance{key, plan});
}

static std::shared_ptr<mscclpp::DeviceHandle<mscclpp::SmChannel>> setupSmChannelDeviceHandles(
    const std::vector<mscclpp::SmChannel>& smChannels) {
  std::vector<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
  std::transform(smChannels.begin(), smChannels.end(), std::back_inserter(smChannelDeviceHandles),
                 [](const mscclpp::SmChannel& smChannel) { return mscclpp::deviceHandle(smChannel); });
  std::shared_ptr<mscclpp::DeviceHandle<mscclpp::SmChannel>> ptr =
      mscclpp::allocSharedCuda<mscclpp::DeviceHandle<mscclpp::SmChannel>>(smChannelDeviceHandles.size());
  mscclpp::memcpyCuda<mscclpp::DeviceHandle<mscclpp::SmChannel>>(ptr.get(), smChannelDeviceHandles.data(),
                                                                 smChannelDeviceHandles.size(), hipMemcpyHostToDevice);
  return ptr;
}

static ncclResult_t ncclAllReduceFallback(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                          ncclRedOp_t, ncclComm_t comm, hipStream_t stream) {
  // Checking if the parameters are valids
  if (sendbuff == nullptr || recvbuff == nullptr || count == 0 || ncclTypeSize(datatype) == 0 || comm == nullptr)
    return ncclInvalidArgument;

  // Declarating variables
  size_t sendBytes, recvBytes;
  hipDeviceptr_t sendBasePtr, recvBasePtr;
  MSCCLPP_CUTHROW(hipMemGetAddressRange(&sendBasePtr, &sendBytes, (hipDeviceptr_t)sendbuff));
  MSCCLPP_CUTHROW(hipMemGetAddressRange(&recvBasePtr, &recvBytes, (hipDeviceptr_t)recvbuff));
  size_t offsetIn = (char*)sendbuff - (char*)sendBasePtr;
  size_t offsetOut = (char*)recvbuff - (char*)recvBasePtr;
  uint32_t scratchBuffIdx = (++(comm->buffFlag)) % comm->numScratchBuff;
  size_t offsetScratch = (SCRATCH_SIZE / comm->numScratchBuff) * scratchBuffIdx;
  int rank = comm->comm->bootstrap()->getRank();
  channelKey sendKey{(void*)sendBasePtr, sendBytes};
  channelKey recvKey{(void*)recvBasePtr, recvBytes};
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels = nullptr;
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smOutChannels = nullptr;

  // Creating the channels
  if (count * ncclTypeSize(datatype) <= (1 << 20)) {
    auto sendIt = comm->channelScratchInfos.find(sendKey);
    if (sendIt == comm->channelScratchInfos.end()) {
      std::vector<mscclpp::SmChannel> channels =
          setupSmChannels(comm, comm->remoteScratchRegMemories, const_cast<void*>((void*)sendBasePtr));
      ChannelInfo channelInfo{channels, setupSmChannelDeviceHandles(channels)};
      sendIt = comm->channelScratchInfos.emplace(sendKey, channelInfo).first;
    }

    smChannels = sendIt->second.smChannelDeviceHandles.get();
  } else {
    std::vector<mscclpp::RegisteredMemory> remoteMemories;

    auto sendIt = comm->channelInInfos.find(sendKey);
    if (sendIt == comm->channelInInfos.end()) {
      std::vector<mscclpp::SmChannel> channels =
          setupSmChannels(comm, comm->remoteScratchRegMemories, const_cast<void*>((void*)sendBasePtr));
      ChannelInfo channelInfo{channels, setupSmChannelDeviceHandles(channels)};
      sendIt = comm->channelInInfos.emplace(sendKey, channelInfo).first;
    }

    auto recvIt = comm->channelOutInfos.find(recvKey);
    if (recvIt == comm->channelOutInfos.end()) {
      remoteMemories =
          setupRemoteMemories(comm->comm, rank, (void*)recvBasePtr, recvBytes, mscclpp::Transport::CudaIpc);
      std::vector<mscclpp::SmChannel> outChannels =
          setupSmChannels(comm, remoteMemories, const_cast<void*>((void*)recvBasePtr));
      ChannelInfo channelInfo{outChannels, setupSmChannelDeviceHandles(outChannels)};
      recvIt = comm->channelOutInfos.emplace(recvKey, channelInfo).first;
    }

    smChannels = sendIt->second.smChannelDeviceHandles.get();
    smOutChannels = recvIt->second.smChannelDeviceHandles.get();
  }

  switch (datatype) {
    case ncclFloat16:
      CUDACHECK(allreduce((half*)sendbuff, (half*)comm->scratchBuff.get(), (half*)recvbuff, smChannels, smOutChannels,
                          offsetIn, offsetOut, offsetScratch, rank, NRANKS_PER_NODE,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case ncclFloat32:
      CUDACHECK(allreduce((float*)sendbuff, (float*)comm->scratchBuff.get(), (float*)recvbuff, smChannels,
                          smOutChannels, offsetIn, offsetOut, offsetScratch, comm->comm->bootstrap()->getRank(),
                          NRANKS_PER_NODE, comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case ncclBfloat16:
      CUDACHECK(allreduce((__bfloat16*)sendbuff, (__bfloat16*)comm->scratchBuff.get(), (__bfloat16*)recvbuff,
                          smChannels, smOutChannels, offsetIn, offsetOut, offsetScratch, rank, NRANKS_PER_NODE,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case ncclInt32:
    case ncclUint32:
      CUDACHECK(allreduce((int*)sendbuff, (int*)comm->scratchBuff.get(), (int*)recvbuff, smChannels, smOutChannels,
                          offsetIn, offsetOut, offsetScratch, comm->comm->bootstrap()->getRank(), NRANKS_PER_NODE,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    default:
      return ncclInvalidArgument;
  }
  return ncclSuccess;
}

static ncclResult_t ncclAllGatherFallback(const void* sendbuff, void* recvbuff, size_t sendcount,
                                          ncclDataType_t datatype, ncclComm_t comm, hipStream_t stream) {
  size_t bytes = sendcount * ncclTypeSize(datatype);
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr) return ncclInvalidArgument;

  // Declarating variables
  size_t recvBytes;
  hipDeviceptr_t recvBasePtr;
  MSCCLPP_CUTHROW(hipMemGetAddressRange(&recvBasePtr, &recvBytes, (hipDeviceptr_t)recvbuff));
  size_t offsetOut = (char*)recvbuff - (char*)recvBasePtr;
  channelKey recvKey{(void*)recvBasePtr, recvBytes};
  int rank = comm->comm->bootstrap()->getRank();
  int nRank = comm->comm->bootstrap()->getNranks();
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels = nullptr;

  auto it = comm->channelOutInfos.find(recvKey);
  if (it == comm->channelOutInfos.end()) {
    std::vector<mscclpp::RegisteredMemory> remoteMemories = setupRemoteMemories(
        comm->comm, rank, const_cast<void*>((void*)recvBasePtr), recvBytes, mscclpp::Transport::CudaIpc);
    std::vector<mscclpp::SmChannel> channels =
        setupSmChannels(comm, remoteMemories, const_cast<void*>((void*)recvBasePtr));
    std::vector<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
    std::transform(channels.begin(), channels.end(), std::back_inserter(smChannelDeviceHandles),
                   [](const mscclpp::SmChannel& smChannel) { return mscclpp::deviceHandle(smChannel); });
    ChannelInfo channelInfo{channels, setupSmChannelDeviceHandles(channels)};
    it = comm->channelOutInfos.emplace(recvKey, channelInfo).first;
  }

  smChannels = it->second.smChannelDeviceHandles.get();
  if ((char*)sendbuff == (char*)recvbuff + rank * sendcount) {
    CUDACHECK(allgather<false>((int*)sendbuff, (int*)nullptr, (int*)recvbuff, smChannels, offsetOut, rank,
                               NRANKS_PER_NODE, nRank, bytes / sizeof(int), stream));
  } else {
    CUDACHECK(allgather<true>((int*)sendbuff, (int*)nullptr, (int*)recvbuff, smChannels, offsetOut, rank,
                              NRANKS_PER_NODE, nRank, bytes / sizeof(int), stream));
  }

  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGetVersion(int* version) {
  if (version == nullptr) return ncclInvalidArgument;
  *version = MSCCLPP_VERSION;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGetUniqueId(ncclUniqueId* uniqueId) {
  if (uniqueId == nullptr) return ncclInvalidArgument;
  if (MSCCLPP_UNIQUE_ID_BYTES != NCCL_UNIQUE_ID_BYTES) return ncclInternalError;
  mscclpp::UniqueId id = mscclpp::TcpBootstrap::createUniqueId();
  memcpy(uniqueId, &id, sizeof(ncclUniqueId));
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommInitRankConfig(ncclComm_t* comm, int nranks, ncclUniqueId commId, int rank,
                                             ncclConfig_t*) {
  // TODO: implement config
  return ncclCommInitRank(comm, nranks, commId, rank);
}

NCCL_API ncclResult_t ncclCommInitRank(ncclComm_t* comm, int nranks, ncclUniqueId commId, int rank) {
  if (comm == nullptr) return ncclInvalidArgument;
  if (nranks < 0 || rank < 0 || rank >= nranks) return ncclInvalidArgument;
  std::shared_ptr<mscclpp::TcpBootstrap> bootstrap = std::make_shared<mscclpp::TcpBootstrap>(rank, nranks);
  mscclpp::UniqueId id;
  memcpy(id.data(), &commId, sizeof(ncclUniqueId));
  bootstrap->initialize(id);
  std::shared_ptr<mscclpp::Communicator> mscclppComm = std::make_shared<mscclpp::Communicator>(bootstrap);
  std::vector<mscclpp::NonblockingFuture<std::shared_ptr<mscclpp::Connection>>> connectionFutures;

  for (int i = 0; i < mscclppComm->bootstrap()->getNranks(); i++) {
    if (i == rank) continue;
    mscclpp::Transport transport = getTransport(rank, i);
    connectionFutures.push_back(mscclppComm->connectOnSetup(i, 0, transport));
  }
  mscclppComm->setup();

  std::vector<std::shared_ptr<mscclpp::Connection>> connections;
  std::transform(connectionFutures.begin(), connectionFutures.end(), std::back_inserter(connections),
                 [](const auto& future) { return future.get(); });

  std::vector<std::shared_ptr<mscclpp::SmDevice2DeviceSemaphore>> smSemaphores;
  for (size_t idx = 0; idx < NUM_CHANNELS_PER_CONNECTION; ++idx) {
    for (size_t cid = 0; cid < connections.size(); ++cid) {
      if (connections[cid]->transport() == mscclpp::Transport::CudaIpc) {
        smSemaphores.emplace_back(
            std::make_shared<mscclpp::SmDevice2DeviceSemaphore>(*(mscclppComm), connections[cid]));
      }
    }
  }
  mscclppComm->setup();

  ncclComm* commPtr = new ncclComm();
  commPtr->comm = mscclppComm;
  commPtr->connections = std::move(connections);
  commPtr->smSemaphores = std::move(smSemaphores);
  commPtr->buffFlag = 0;
  commPtr->numScratchBuff = 2;
  commPtr->scratchBuff = mscclpp::allocExtSharedCuda<char>(SCRATCH_SIZE);
  commPtr->remoteScratchRegMemories =
      setupRemoteMemories(commPtr->comm, rank, commPtr->scratchBuff.get(), SCRATCH_SIZE, mscclpp::Transport::CudaIpc);
  commPtr->executor = std::make_shared<mscclpp::Executor>(mscclppComm);

  if (getenv("MSCCLPP_EXECUTION_PLAN_DIR")) {
    std::string collectiveDir = getenv("MSCCLPP_EXECUTION_PLAN_DIR");
    if (!std::filesystem::is_directory(collectiveDir)) {
      return ncclInvalidArgument;
    }
    for (const auto& entry : std::filesystem::directory_iterator(collectiveDir)) {
      if (entry.is_regular_file()) {
        auto plan = loadExecutionPlan(entry.path());
        commPtr->executionPlans[plan.first].push_back(plan.second);
      }
    }
  }

  *comm = commPtr;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommInitAll(ncclComm_t*, int, const int*) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclCommFinalize(ncclComm_t comm) {
  comm->comm->bootstrap()->barrier();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommDestroy(ncclComm_t comm) {
  if (comm == nullptr) return ncclInvalidArgument;
  delete comm;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommAbort(ncclComm_t) {
  // TODO: implement this function
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommSplit(ncclComm_t, int, int, ncclComm_t*, ncclConfig_t*) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API const char* ncclGetErrorString(ncclResult_t result) {
  switch (result) {
    case ncclSuccess:
      return "no error";
    case ncclUnhandledCudaError:
      return "unhandled cuda error (run with NCCL_DEBUG=INFO for details)";
    case ncclSystemError:
      return "unhandled system error (run with NCCL_DEBUG=INFO for details)";
    case ncclInternalError:
      return "internal error - please report this issue to the NCCL developers";
    case ncclInvalidArgument:
      return "invalid argument (run with NCCL_DEBUG=WARN for details)";
    case ncclInvalidUsage:
      return "invalid usage (run with NCCL_DEBUG=WARN for details)";
    case ncclRemoteError:
      return "remote process exited or there was a network error";
    case ncclInProgress:
      return "NCCL operation in progress";
    default:
      return "unknown result code";
  }
}

NCCL_API const char* ncclGetLastError(ncclComm_t) {
  // TODO: implement this function
  return "";
}

NCCL_API ncclResult_t ncclCommGetAsyncError(ncclComm_t, ncclResult_t* asyncError) {
  if (asyncError == nullptr) return ncclInvalidArgument;
  *asyncError = ncclSuccess;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommCount(const ncclComm_t comm, int* count) {
  if (comm == nullptr || count == nullptr) return ncclInvalidArgument;
  *count = comm->comm->bootstrap()->getNranks();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommCuDevice(const ncclComm_t comm, int* device) {
  if (comm == nullptr || device == nullptr) return ncclInvalidArgument;
  *device = comm->comm->bootstrap()->getRank();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommUserRank(const ncclComm_t comm, int* rank) {
  if (comm == nullptr || rank == nullptr) return ncclInvalidArgument;
  *rank = comm->comm->bootstrap()->getRank();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclRedOpCreatePreMulSum(ncclRedOp_t*, void*, ncclDataType_t, ncclScalarResidence_t, ncclComm_t) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclRedOpDestroy(ncclRedOp_t, ncclComm_t) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclReduce(const void*, void*, size_t, ncclDataType_t, ncclRedOp_t, int, ncclComm_t,
                                 hipStream_t) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclBcast(void*, size_t, ncclDataType_t, int, ncclComm_t, hipStream_t) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclBroadcast(const void*, void*, size_t, ncclDataType_t, int, ncclComm_t, hipStream_t) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                    ncclRedOp_t reductionOperation, ncclComm_t comm, hipStream_t stream) {
  // Checking if the parameters are valids
  if (sendbuff == nullptr || recvbuff == nullptr || count == 0 || ncclTypeSize(datatype) == 0 || comm == nullptr)
    return ncclInvalidArgument;

  // Declarating variables
  size_t bytes = count * ncclTypeSize(datatype);
  int rank = comm->comm->bootstrap()->getRank();

  std::vector<executionPlanInstance>& plans = comm->executionPlans["allreduce"];
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
  bool inPlace = sendbuff == recvbuff;
  for (const auto& p : plans) {
    if (bytes >= p.key.minMessageSize && bytes < p.key.maxMessageSize && inPlace == p.key.isInPlace) {
      plan = p.plan;
      break;
    }
  }

  if (plan == nullptr)
    return ncclAllReduceFallback(sendbuff, recvbuff, count, datatype, reductionOperation, comm, stream);

  switch (datatype) {
    case ncclFloat16:
      comm->executor->execute(rank, (half*)sendbuff, (half*)recvbuff, bytes, bytes, mscclpp::DataType::FLOAT16, *plan,
                              stream, mscclpp::PacketType::LL8);
      break;
    case ncclFloat32:
      comm->executor->execute(rank, (float*)sendbuff, (float*)recvbuff, bytes, bytes, mscclpp::DataType::FLOAT32, *plan,
                              stream, mscclpp::PacketType::LL8);
      break;
    case ncclBfloat16:
      comm->executor->execute(rank, (__bfloat16*)sendbuff, (__bfloat16*)recvbuff, bytes, bytes,
                              mscclpp::DataType::BFLOAT16, *plan, stream, mscclpp::PacketType::LL8);
      break;
    case ncclInt32:
    case ncclUint32:
      comm->executor->execute(rank, (int*)sendbuff, (int*)recvbuff, bytes, bytes, mscclpp::DataType::UINT32, *plan,
                              stream, mscclpp::PacketType::LL8);
      break;
    default:
      return ncclInvalidArgument;
  }

  return ncclSuccess;
}

NCCL_API ncclResult_t ncclReduceScatter(const void*, void*, size_t, ncclDataType_t, ncclRedOp_t, ncclComm_t,
                                        hipStream_t) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllGather(const void* sendbuff, void* recvbuff, size_t sendcount, ncclDataType_t datatype,
                                    ncclComm_t comm, hipStream_t stream) {
  size_t bytes = sendcount * ncclTypeSize(datatype);
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr) return ncclInvalidArgument;

  int rank = comm->comm->bootstrap()->getRank();
  int nRank = comm->comm->bootstrap()->getNranks();

  std::vector<executionPlanInstance>& plans = comm->executionPlans["allgather"];
  std::shared_ptr<mscclpp::ExecutionPlan> plan;
  void* basePtr = (char*)sendbuff - rank * bytes;
  bool inPlace = basePtr == recvbuff;
  const size_t totalBytes = bytes * nRank;
  for (const auto& p : plans) {
    if (totalBytes >= p.key.minMessageSize && totalBytes < p.key.maxMessageSize && inPlace == p.key.isInPlace) {
      plan = p.plan;
      break;
    }
  }
  if (plan == nullptr) return ncclAllGatherFallback(sendbuff, recvbuff, sendcount, datatype, comm, stream);

  switch (datatype) {
    case ncclFloat16:
      comm->executor->execute(rank, (half*)sendbuff, (half*)recvbuff, bytes, bytes * nRank, mscclpp::DataType::FLOAT16,
                              *plan, stream);
      break;
    case ncclFloat32:
      comm->executor->execute(rank, (float*)sendbuff, (float*)recvbuff, bytes, bytes * nRank,
                              mscclpp::DataType::FLOAT32, *plan, stream);
      break;
    case ncclBfloat16:
      comm->executor->execute(rank, (__bfloat16*)sendbuff, (__bfloat16*)recvbuff, bytes, bytes * nRank,
                              mscclpp::DataType::BFLOAT16, *plan, stream);
      break;
    case ncclInt32:
    case ncclUint32:
      comm->executor->execute(rank, (int*)sendbuff, (int*)recvbuff, bytes, bytes * nRank, mscclpp::DataType::UINT32,
                              *plan, stream);
      break;
    default:
      return ncclInvalidArgument;
  }

  return ncclSuccess;
}

NCCL_API ncclResult_t ncclSend(const void*, size_t, ncclDataType_t, int, ncclComm_t, hipStream_t) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclRecv(void*, size_t, ncclDataType_t, int, ncclComm_t, hipStream_t) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllToAll(const void*, void*, size_t, ncclDataType_t, ncclComm_t, hipStream_t) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclGroupStart() {
  // Do nothing
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGroupEnd() {
  // Do nothing
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommRegister(const ncclComm_t, void*, size_t, void**) {
  // TODO: Implementation
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommDeregister(const ncclComm_t, void*) {
  // TODO: Implementation
  return ncclSuccess;
}

ncclResult_t ncclMemAlloc(void** ptr, size_t size) {
  // Allocate memory using mscclpp::allocSharedPhysicalCuda
  if (ptr == nullptr || size == 0) {
    return ncclInvalidArgument;
  }
  std::shared_ptr<char> sharedPtr;
  try {
    if (mscclpp::isNvlsSupported()) {
      sharedPtr = mscclpp::allocSharedPhysicalCuda<char>(size);
    } else {
      sharedPtr = mscclpp::allocExtSharedCuda<char>(size);
    }
    if (sharedPtr == nullptr) {
      return ncclSystemError;
    }
  } catch (const mscclpp::Error& e) {
    if (e.getErrorCode() == mscclpp::ErrorCode::InvalidUsage) {
      return ncclInvalidUsage;
    } else {
      return ncclInternalError;
    }
  } catch (const mscclpp::CudaError& e) {
    return ncclUnhandledCudaError;
  } catch (const mscclpp::CuError& e) {
    return ncclUnhandledCudaError;
  } catch (const mscclpp::BaseError& e) {
    return ncclInternalError;
  }
  ptrMap[sharedPtr.get()] = sharedPtr;

  // Return the pointer
  *ptr = sharedPtr.get();
  return ncclSuccess;
}

ncclResult_t ncclMemFree(void* ptr) {
  auto ptrIt = ptrMap.find(ptr);
  if (ptrIt != ptrMap.end()) {
    ptrMap.erase(ptrIt);
    return ncclSuccess;
  }

  // Pointer not found
  return ncclInvalidUsage;
}
