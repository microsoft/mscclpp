// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include "execution_kernel.hpp"

#if defined(MSCCLPP_DEVICE_CUDA)
namespace mscclpp {

template <typename PacketType>
void ExecutionKernel::launchKernel(int rank, int nthreadblocks, int nthreads, void* src, void* dst, void* scratch,
                                   size_t scratchSize, DataType dataType, DeviceExecutionPlan* plan,
                                   size_t sharedMemSize, hipStream_t stream, uint32_t flag) {
  switch (dataType) {
    case DataType::INT32:
      executionKernel<int32_t, PacketType><<<nthreadblocks, nthreads, sharedMemSize, stream>>>(
          rank, (int32_t*)src, (int32_t*)dst, (int32_t*)scratch, scratchSize, plan, flag
#if defined(ENABLE_NPKIT)
          ,
          NpKit::GetGpuEventCollectContexts(), NpKit::GetCpuTimestamp());
#else
      );
#endif
      break;
    case DataType::UINT32:
      executionKernel<uint32_t><<<nthreadblocks, nthreads, sharedMemSize, stream>>>(
          rank, (uint32_t*)src, (uint32_t*)dst, (uint32_t*)scratch, scratchSize, plan, flag
#if defined(ENABLE_NPKIT)
          ,
          NpKit::GetGpuEventCollectContexts(), NpKit::GetCpuTimestamp());
#else
      );
#endif
      break;
    case DataType::FLOAT16:
      executionKernel<half><<<nthreadblocks, nthreads, sharedMemSize, stream>>>(
          rank, (half*)src, (half*)dst, (half*)scratch, scratchSize, plan, flag
#if defined(ENABLE_NPKIT)
          ,
          NpKit::GetGpuEventCollectContexts(), NpKit::GetCpuTimestamp());
#else
      );
#endif
      break;
    case DataType::FLOAT32:
      executionKernel<float><<<nthreadblocks, nthreads, sharedMemSize, stream>>>(
          rank, (float*)src, (float*)dst, (float*)scratch, scratchSize, plan, flag
#if defined(ENABLE_NPKIT)
          ,
          NpKit::GetGpuEventCollectContexts(), NpKit::GetCpuTimestamp());
#else
      );
#endif
      break;
    case DataType::BFLOAT16:
      executionKernel<__bfloat16><<<nthreadblocks, nthreads, sharedMemSize, stream>>>(
          rank, (__bfloat16*)src, (__bfloat16*)dst, (__bfloat16*)scratch, scratchSize, plan, flag
#if defined(ENABLE_NPKIT)
          ,
          NpKit::GetGpuEventCollectContexts(), NpKit::GetCpuTimestamp());
#else
      );
#endif
      break;
  }
}

template void ExecutionKernel::launchKernel<LL16Packet>(int rank, int nthreadblocks, int nthreads, void* src, void* dst,
                                                        void* scratch, size_t scratchSize, DataType dataType,
                                                        DeviceExecutionPlan* plan, size_t sharedMemSize,
                                                        hipStream_t stream, uint32_t flag);
template void ExecutionKernel::launchKernel<LL8Packet>(int rank, int nthreadblocks, int nthreads, void* src, void* dst,
                                                       void* scratch, size_t scratchSize, DataType dataType,
                                                       DeviceExecutionPlan* plan, size_t sharedMemSize,
                                                       hipStream_t stream, uint32_t flag);
}  // namespace mscclpp
#endif
