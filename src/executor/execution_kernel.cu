// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include "execution_kernel.hpp"

namespace mscclpp {
void ExecutionKernel::launchKernel(int rank, int nthreadblocks, int nthreads, void* src, void* dst, void* scratch,
                                   DataType dataType, DeviceExecutionPlan* plan, size_t sharedMemSize,
                                   hipStream_t stream) {
  switch (dataType) {
    case DataType::INT32:
      executionKernel<int32_t><<<nthreadblocks, nthreads, sharedMemSize, stream>>>(rank, (int32_t*)src, (int32_t*)dst,
                                                                                   (int32_t*)scratch, plan);
      break;
    case DataType::UINT32:
      executionKernel<uint32_t><<<nthreadblocks, nthreads, sharedMemSize, stream>>>(
          rank, (uint32_t*)src, (uint32_t*)dst, (uint32_t*)scratch, plan);
      break;
    case DataType::FLOAT16:
      executionKernel<half>
          <<<nthreadblocks, nthreads, sharedMemSize, stream>>>(rank, (half*)src, (half*)dst, (half*)scratch, plan);
      break;
    case DataType::FLOAT32:
      executionKernel<float>
          <<<nthreadblocks, nthreads, sharedMemSize, stream>>>(rank, (float*)src, (float*)dst, (float*)scratch, plan);
      break;
  }
}
}  // namespace mscclpp
