#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <gtest/gtest.h>

#include <mscclpp/fifo.hpp>
#include <mscclpp/gpu_utils.hpp>
#include <mscclpp/numa.hpp>
#include <mscclpp/utils.hpp>

#define ITER 10000  // should be larger than the FIFO size for proper testing

__constant__ mscclpp::FifoDeviceHandle gFifoTestFifoDeviceHandle;
__global__ void kernelFifoTest() {
  if (threadIdx.x + blockIdx.x * blockDim.x != 0) return;

  mscclpp::FifoDeviceHandle& fifo = gFifoTestFifoDeviceHandle;
  mscclpp::ProxyTrigger trigger;
  for (uint64_t i = 1; i < ITER + 1; ++i) {
    trigger.fst = i;
    trigger.snd = i;
    uint64_t curFifoHead = fifo.push(trigger);
    if (i % fifo.size == 0) {
      fifo.sync(curFifoHead);
    }
  }
}

TEST(FifoTest, Fifo) {
  int cudaNum;
  MSCCLPP_CUDATHROW(hipGetDevice(&cudaNum));
  int numaNode = mscclpp::getDeviceNumaNode(cudaNum);
  mscclpp::numaBind(numaNode);

  mscclpp::Fifo hostFifo;
  if (hostFifo.size() >= ITER) {
    FAIL() << "ITER is too small for proper testing.";
  }

  mscclpp::FifoDeviceHandle devFifo = hostFifo.deviceHandle();
  MSCCLPP_CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(gFifoTestFifoDeviceHandle), &devFifo, sizeof(devFifo)));

  kernelFifoTest<<<1, 1>>>();
  MSCCLPP_CUDATHROW(hipGetLastError());

  mscclpp::ProxyTrigger trigger;
  trigger.fst = 0;
  trigger.snd = 0;

  uint64_t spin = 0;
  uint64_t flushCnt = 0;
  mscclpp::Timer timer(3);
  for (uint64_t i = 0; i < ITER; ++i) {
    trigger = hostFifo.poll();
    while (trigger.fst == 0 || trigger.snd == 0) {
      trigger = hostFifo.poll();

      if (spin++ > 1000000) {
        FAIL() << "Polling is stuck.";
      }
    }
    // see `src/proxy.cc` for the reason of this line
    trigger.snd ^= ((uint64_t)1 << (uint64_t)63);
    ASSERT_TRUE(trigger.fst == (i + 1));
    ASSERT_TRUE(trigger.snd == (i + 1));
    hostFifo.pop();
    if ((++flushCnt % hostFifo.size()) == 0) {
      hostFifo.flushTail();
    }
    spin = 0;
  }
  hostFifo.flushTail(true);

  std::stringstream ss;
  ss << "FifoTest.Fifo: " << (float)timer.elapsed() / ITER << " us/iter\n";
  std::cout << ss.str();

  MSCCLPP_CUDATHROW(hipDeviceSynchronize());
}
