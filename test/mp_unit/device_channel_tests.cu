#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/concurrency.hpp>

#include "mp_unit_tests.hpp"

void DeviceChannelOneToOneTest::SetUp() {
  // Use only two ranks
  setNumRanksToUse(2);
  CommunicatorTestBase::SetUp();
  channelService = std::make_shared<mscclpp::ProxyService>(*communicator.get());
}

void DeviceChannelOneToOneTest::TearDown() { CommunicatorTestBase::TearDown(); }

void DeviceChannelOneToOneTest::setupMeshConnections(std::vector<mscclpp::SimpleProxyChannel>& devChannels,
                                                     bool useIbOnly, void* sendBuff, size_t sendBuffBytes,
                                                     void* recvBuff, size_t recvBuffBytes) {
  const int rank = communicator->bootstrapper()->getRank();
  const int worldSize = communicator->bootstrapper()->getNranks();
  const bool isInPlace = (recvBuff == nullptr);
  mscclpp::TransportFlags transport = (useIbOnly) ? ibTransport : (mscclpp::Transport::CudaIpc | ibTransport);

  mscclpp::RegisteredMemory sendBufRegMem = communicator->registerMemory(sendBuff, sendBuffBytes, transport);
  mscclpp::RegisteredMemory recvBufRegMem;
  if (!isInPlace) {
    recvBufRegMem = communicator->registerMemory(recvBuff, recvBuffBytes, transport);
  }

  for (int r = 0; r < worldSize; r++) {
    if (r == rank) {
      continue;
    }
    std::shared_ptr<mscclpp::Connection> conn;
    if ((rankToNode(r) == rankToNode(gEnv->rank)) && !useIbOnly) {
      conn = communicator->connectOnSetup(r, 0, mscclpp::Transport::CudaIpc);
    } else {
      conn = communicator->connectOnSetup(r, 0, ibTransport);
    }
    connections[r] = conn;

    if (isInPlace) {
      communicator->sendMemoryOnSetup(sendBufRegMem, r, 0);
    } else {
      communicator->sendMemoryOnSetup(recvBufRegMem, r, 0);
    }
    auto remoteMemory = communicator->recvMemoryOnSetup(r, 0);

    communicator->setup();

    mscclpp::SemaphoreId cid = channelService->addSemaphore(conn);
    communicator->setup();

    devChannels.emplace_back(channelService->deviceChannel(cid), channelService->addMemory(remoteMemory.get()),
                             channelService->addMemory(sendBufRegMem));
  }
}

__constant__ mscclpp::SimpleProxyChannel gChannelOneToOneTestConstDevChans;

__global__ void kernelDevicePingPong(int* buff, int rank, int nElem, int* ret) {
  mscclpp::SimpleProxyChannel& devChan = gChannelOneToOneTestConstDevChans;
  volatile int* sendBuff = (volatile int*)buff;
  int nTries = 1000;
  int flusher = 0;
  int rank1Offset = 10000000;
  for (int i = 0; i < nTries; i++) {
    if (rank == 0) {
      if (i > 0) {
        if (threadIdx.x == 0) devChan.wait();
        __syncthreads();
        for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
          if (sendBuff[j] != rank1Offset + i - 1 + j) {
            // printf("rank 0 ERROR: sendBuff[%d] = %d, expected %d\n", j, sendBuff[j], rank1Offset + i - 1 + j);
            *ret = 1;
            break;
          }
        }
      }
      for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
        sendBuff[j] = i + j;
      }
      __syncthreads();
      // __threadfence_system(); // not necessary if we make sendBuff volatile
      if (threadIdx.x == 0) devChan.putWithSignal(0, nElem * sizeof(int));
    }
    if (rank == 1) {
      if (threadIdx.x == 0) devChan.wait();
      __syncthreads();
      for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
        if (sendBuff[j] != i + j) {
          // printf("rank 1 ERROR: sendBuff[%d] = %d, expected %d\n", j, sendBuff[j], i + j);
          *ret = 1;
          break;
        }
      }
      if (i < nTries - 1) {
        for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
          sendBuff[j] = rank1Offset + i + j;
        }
        __syncthreads();
        // __threadfence_system(); // not necessary if we make sendBuff volatile
        if (threadIdx.x == 0) devChan.putWithSignal(0, nElem * sizeof(int));
      }
    }
    flusher++;
    if (flusher == 100) {
      if (threadIdx.x == 0) devChan.flush();
      flusher = 0;
    }
  }
}

TEST_F(DeviceChannelOneToOneTest, PingPongIb) {
  if (gEnv->rank >= numRanksToUse) return;

  const int nElem = 4 * 1024 * 1024;

  std::vector<mscclpp::SimpleProxyChannel> devChannels;
  std::shared_ptr<int> buff = mscclpp::allocSharedCuda<int>(nElem);
  setupMeshConnections(devChannels, true, buff.get(), nElem * sizeof(int));

  ASSERT_EQ(devChannels.size(), 1);
  MSCCLPP_CUDATHROW(
      hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestConstDevChans), devChannels.data(), sizeof(mscclpp::SimpleProxyChannel)));

  channelService->startProxy();

  std::shared_ptr<int> ret = mscclpp::makeSharedCudaHost<int>(0);

  kernelDevicePingPong<<<1, 1024>>>(buff.get(), gEnv->rank, 1, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  kernelDevicePingPong<<<1, 1024>>>(buff.get(), gEnv->rank, 1024, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  kernelDevicePingPong<<<1, 1024>>>(buff.get(), gEnv->rank, 1024 * 1024, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  kernelDevicePingPong<<<1, 1024>>>(buff.get(), gEnv->rank, 4 * 1024 * 1024, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  channelService->stopProxy();
}

__device__ mscclpp::DeviceSyncer gChannelOneToOneTestDevChansSyncer;

template <bool CheckCorrectness>
__global__ void kernelDeviceLLPingPong(int* buff, mscclpp::LLPacket* putPktBuf, mscclpp::LLPacket* getPktBuf, int rank,
                                       int nElem, int nTries, int* ret) {
  if (rank > 1) return;

  mscclpp::SimpleProxyChannel& devChan = gChannelOneToOneTestConstDevChans;
  volatile int* buffPtr = (volatile int*)buff;
  int putOffset = (rank == 0) ? 0 : 10000000;
  int getOffset = (rank == 0) ? 10000000 : 0;
  int threadId = threadIdx.x + blockIdx.x * blockDim.x;
  int numThreads = blockDim.x * gridDim.x;
  int flusher = 0;
  const size_t nPkt = nElem / 2;
  for (int i = 0; i < nTries; i++) {
    uint64_t flag = (uint64_t)i + 1;

    // rank=0: 0, 1, 0, 1, ...
    // rank=1: 1, 0, 1, 0, ...
    if ((rank ^ (i & 1)) == 0) {
      if (CheckCorrectness) {
        // If each thread writes 8 bytes at once, we don't need a barrier before putPackets().
        for (int j = threadId; j < nPkt; j += numThreads) {
          buffPtr[2 * j] = putOffset + i + 2 * j;
          buffPtr[2 * j + 1] = putOffset + i + 2 * j + 1;
        }
        // __syncthreads();
      }
      mscclpp::putPackets(putPktBuf, 0, buff, 0, nElem * sizeof(int), threadId, numThreads, flag);
      gChannelOneToOneTestDevChansSyncer.sync(gridDim.x);
      if (threadId == 0) {
        // Send data from the local putPacketBuffer to the remote getPacketBuffer
        devChan.put(0, nPkt * sizeof(mscclpp::LLPacket));
      }
      flusher++;
      if (flusher == 64) {
        if (threadId == 0) devChan.flush();
        flusher = 0;
      }
    } else {
      mscclpp::getPackets(buff, 0, getPktBuf, 0, nElem * sizeof(int), threadId, numThreads, flag);
      if (CheckCorrectness) {
        // If each thread reads 8 bytes at once, we don't need a barrier after getPackets().
        // __syncthreads();
        for (int j = threadId; j < nPkt; j += numThreads) {
          if (buffPtr[2 * j] != getOffset + i + 2 * j) {
            // printf("ERROR: rank = %d, buffPtr[%d] = %d, expected %d. Skipping following errors\n", rank, 2 * j,
            //        buffPtr[2 * j], getOffset + i + 2 * j);
            *ret = 1;
            break;
          }
          if (buffPtr[2 * j + 1] != getOffset + i + 2 * j + 1) {
            // printf("ERROR: rank = %d, buffPtr[%d] = %d, expected %d. Skipping following errors\n", rank, 2 * j + 1,
            //        buffPtr[2 * j + 1], getOffset + i + 2 * j + 1);
            *ret = 1;
            break;
          }
        }
      }
      // Make sure all threads are done in this iteration
      gChannelOneToOneTestDevChansSyncer.sync(gridDim.x);
    }
  }
}

void DeviceChannelOneToOneTest::testPacketPingPong(bool useIbOnly) {
  if (gEnv->rank >= numRanksToUse) return;

  const int nElem = 4 * 1024 * 1024;

  std::vector<mscclpp::SimpleProxyChannel> devChannels;
  std::shared_ptr<int> buff = mscclpp::allocSharedCuda<int>(nElem);

  const size_t nPacket = (nElem * sizeof(int) + sizeof(uint64_t) - 1) / sizeof(uint64_t);
  auto putPacketBuffer = mscclpp::allocSharedCuda<mscclpp::LLPacket>(nPacket);
  auto getPacketBuffer = mscclpp::allocSharedCuda<mscclpp::LLPacket>(nPacket);

  setupMeshConnections(devChannels, useIbOnly, putPacketBuffer.get(), nPacket * sizeof(mscclpp::LLPacket),
                       getPacketBuffer.get(), nPacket * sizeof(mscclpp::LLPacket));

  ASSERT_EQ(devChannels.size(), 1);
  MSCCLPP_CUDATHROW(
      hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestConstDevChans), devChannels.data(), sizeof(mscclpp::SimpleProxyChannel)));

  mscclpp::DeviceSyncer syncer = {};
  MSCCLPP_CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestDevChansSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));

  channelService->startProxy();

  std::shared_ptr<int> ret = mscclpp::makeSharedCudaHost<int>(0);

  const int nTries = 1000;

  // The least nelem is 2 for packet ping pong
  kernelDeviceLLPingPong<true>
      <<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank, 2, nTries, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);
  *ret = 0;

  kernelDeviceLLPingPong<true>
      <<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank, 1024, nTries, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);
  *ret = 0;

  kernelDeviceLLPingPong<true><<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank,
                                            1024 * 1024, nTries, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);
  *ret = 0;

  kernelDeviceLLPingPong<true><<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank,
                                            4 * 1024 * 1024, nTries, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  communicator->bootstrapper()->barrier();

  channelService->stopProxy();
}

void DeviceChannelOneToOneTest::testPacketPingPongPerf(bool useIbOnly) {
  if (gEnv->rank >= numRanksToUse) return;

  const int nElem = 4 * 1024 * 1024;

  std::vector<mscclpp::SimpleProxyChannel> devChannels;
  std::shared_ptr<int> buff = mscclpp::allocSharedCuda<int>(nElem);

  const size_t nPacket = (nElem * sizeof(int) + sizeof(uint64_t) - 1) / sizeof(uint64_t);
  auto putPacketBuffer = mscclpp::allocSharedCuda<mscclpp::LLPacket>(nPacket);
  auto getPacketBuffer = mscclpp::allocSharedCuda<mscclpp::LLPacket>(nPacket);

  setupMeshConnections(devChannels, useIbOnly, putPacketBuffer.get(), nPacket * sizeof(mscclpp::LLPacket),
                       getPacketBuffer.get(), nPacket * sizeof(mscclpp::LLPacket));

  ASSERT_EQ(devChannels.size(), 1);
  MSCCLPP_CUDATHROW(
      hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestConstDevChans), devChannels.data(), sizeof(mscclpp::SimpleProxyChannel)));

  mscclpp::DeviceSyncer syncer = {};
  MSCCLPP_CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestDevChansSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));

  channelService->startProxy();

  auto* testInfo = ::testing::UnitTest::GetInstance()->current_test_info();
  const std::string testName = std::string(testInfo->test_suite_name()) + "." + std::string(testInfo->name());
  const int nTries = 1000;

  // Warm-up
  kernelDeviceLLPingPong<false>
      <<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank, 2, nTries, nullptr);
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  communicator->bootstrapper()->barrier();

  // Measure latency
  mscclpp::Timer timer;
  kernelDeviceLLPingPong<false>
      <<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank, 2, nTries, nullptr);
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  communicator->bootstrapper()->barrier();

  if (gEnv->rank == 0) {
    std::cout << testName << ": " << std::setprecision(4) << (float)timer.elapsed() / (float)nTries << " us/iter\n";
  }

  channelService->stopProxy();
}

TEST_F(DeviceChannelOneToOneTest, PacketPingPong) { testPacketPingPong(false); }

TEST_F(DeviceChannelOneToOneTest, PacketPingPongIb) { testPacketPingPong(true); }

TEST_F(DeviceChannelOneToOneTest, PacketPingPongPerf) { testPacketPingPongPerf(false); }

TEST_F(DeviceChannelOneToOneTest, PacketPingPongPerfIb) { testPacketPingPongPerf(true); }
