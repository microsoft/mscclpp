#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <algorithm>
#include <mscclpp/switch_channel.hpp>
#include <mscclpp/switch_channel_device.hpp>

#include "mp_unit_tests.hpp"

void SwitchChannelTest::SetUp() {
  // Need at least two ranks within a node
  if (gEnv->nRanksPerNode < 2) {
    GTEST_SKIP();
  }
  if (!mscclpp::isNvlsSupported()) {
    GTEST_SKIP();
  }
  // Use only two ranks
  setNumRanksToUse(2);
  CommunicatorTestBase::SetUp();
}

void SwitchChannelTest::TearDown() { CommunicatorTestBase::TearDown(); }

__constant__ mscclpp::SwitchChannelDeviceHandle gConstSwitchChan;

__global__ void kernelSwitchReduce() {
#if (CUDA_NVLS_API_AVAILABLE) && (__CUDA_ARCH__ >= 900)
  auto val = gConstSwitchChan.reduce<mscclpp::f32x1>(0);
  gConstSwitchChan.broadcast(0, val);
#endif  // (CUDA_NVLS_API_AVAILABLE) && (__CUDA_ARCH__ >= 900)
}

TEST_F(SwitchChannelTest, SimpleAllReduce) {
  if (gEnv->rank >= numRanksToUse) return;

  std::vector<int> ranks;
  for (int i = 0; i < numRanksToUse; i++) {
    ranks.push_back(i);
  }

  auto buffer = mscclpp::GpuBuffer<float>(1024);
  float data = gEnv->rank + 1.0f;
  MSCCLPP_CUDATHROW(hipMemcpy(buffer.data(), &data, sizeof(data), hipMemcpyHostToDevice));

  auto nvlsConnection = mscclpp::connectNvlsCollective(communicator, ranks, 1024);
  auto switchChannel = nvlsConnection->bindAllocatedMemory(hipDeviceptr_t(buffer.data()), 1024);
  auto deviceHandle = switchChannel.deviceHandle();

  MSCCLPP_CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(gConstSwitchChan), &deviceHandle, sizeof(deviceHandle)));
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  communicator->bootstrap()->barrier();

  if (gEnv->rank == 0) {
    kernelSwitchReduce<<<1, 1>>>();
    MSCCLPP_CUDATHROW(hipGetLastError());
    MSCCLPP_CUDATHROW(hipDeviceSynchronize());
  }
  communicator->bootstrap()->barrier();

  float result;
  MSCCLPP_CUDATHROW(hipMemcpy(&result, buffer.data(), sizeof(result), hipMemcpyDeviceToHost));

  float expected = 0.0f;
  for (int i = 0; i < numRanksToUse; i++) {
    expected += i + 1.0f;
  }
  ASSERT_EQ(result, expected) << "Expected " << expected << " but got " << result << " for rank " << gEnv->rank;
}
