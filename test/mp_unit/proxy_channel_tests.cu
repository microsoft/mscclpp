#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/concurrency.hpp>

#include "mp_unit_tests.hpp"

void ProxyChannelOneToOneTest::SetUp() {
  // Use only two ranks
  setNumRanksToUse(2);
  CommunicatorTestBase::SetUp();
  proxyService = std::make_shared<mscclpp::ProxyService>();
}

void ProxyChannelOneToOneTest::TearDown() { CommunicatorTestBase::TearDown(); }

void ProxyChannelOneToOneTest::setupMeshConnections(std::vector<mscclpp::SimpleProxyChannel>& proxyChannels,
                                                    bool useIbOnly, void* sendBuff, size_t sendBuffBytes,
                                                    void* recvBuff, size_t recvBuffBytes) {
  setupMeshConnections(proxyChannels, useIbOnly, sendBuff, sendBuffBytes, sendBuffBytes, recvBuff, recvBuffBytes);
}

void ProxyChannelOneToOneTest::setupMeshConnections(std::vector<mscclpp::SimpleProxyChannel>& proxyChannels,
                                                    bool useIbOnly, void* sendBuff, size_t sendBuffBytes, size_t pitch,
                                                    void* recvBuff, size_t recvBuffBytes) {
  const int rank = communicator->bootstrap()->getRank();
  const int worldSize = communicator->bootstrap()->getNranks();
  const bool isInPlace = (recvBuff == nullptr);
  mscclpp::TransportFlags transport = (useIbOnly) ? ibTransport : (mscclpp::Transport::CudaIpc | ibTransport);

  mscclpp::RegisteredMemory sendBufRegMem = communicator->registerMemory(sendBuff, sendBuffBytes, transport);
  mscclpp::RegisteredMemory recvBufRegMem;
  if (!isInPlace) {
    recvBufRegMem = communicator->registerMemory(recvBuff, recvBuffBytes, transport);
  }

  for (int r = 0; r < worldSize; r++) {
    if (r == rank) {
      continue;
    }
    std::shared_ptr<mscclpp::Connection> conn;
    if ((rankToNode(r) == rankToNode(gEnv->rank)) && !useIbOnly) {
      conn = communicator->connectOnSetup(r, 0, mscclpp::Transport::CudaIpc);
    } else {
      conn = communicator->connectOnSetup(r, 0, ibTransport);
    }
    connections[r] = conn;

    if (isInPlace) {
      communicator->sendMemoryOnSetup(sendBufRegMem, r, 0);
    } else {
      communicator->sendMemoryOnSetup(recvBufRegMem, r, 0);
    }
    auto remoteMemory = communicator->recvMemoryOnSetup(r, 0);

    communicator->setup();

    mscclpp::SemaphoreId cid;
    if (sendBuffBytes == pitch) {
      cid = proxyService->buildAndAddSemaphore(*communicator, conn);
    } else {
      cid = proxyService->buildAndAddSemaphore(*communicator, conn, std::pair<size_t, size_t>(pitch, pitch));
    }
    communicator->setup();

    proxyChannels.emplace_back(proxyService->proxyChannel(cid), proxyService->addMemory(remoteMemory.get()),
                               proxyService->addMemory(sendBufRegMem));
  }
}

__constant__ DeviceHandle<mscclpp::SimpleProxyChannel> gChannelOneToOneTestConstProxyChans;

__device__ size_t getTileElementOffset(int elementId, int width, int rowIndex, int colIndex, int nElemPerPitch) {
  int rowIndexInTile = elementId / width;
  int colIndexInTile = elementId % width;
  return (rowIndex + rowIndexInTile) * nElemPerPitch + (colIndex + colIndexInTile);
}

__global__ void kernelProxyTilePingPong(int* buff, int rank, int pitch, int rowIndex, int colIndex, int width,
                                        int height, int* ret) {
  DeviceHandle<mscclpp::SimpleProxyChannel>& proxyChan = gChannelOneToOneTestConstProxyChans;
  volatile int* sendBuff = (volatile int*)buff;
  int nTries = 1000;
  int flusher = 0;
  size_t offset = rowIndex * pitch + colIndex * sizeof(int);
  size_t nElem = width * height;
  size_t nElemPerPitch = pitch / sizeof(int);
  for (int i = 0; i < nTries; i++) {
    if (rank == 0) {
      if (i > 0) {
        if (threadIdx.x == 0) proxyChan.wait();
        __syncthreads();
        for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
          size_t tileOffset = getTileElementOffset(j, width, rowIndex, colIndex, nElemPerPitch);
          if (sendBuff[tileOffset] != offset + i - 1 + j) {
            // printf("rank 0 ERROR: sendBuff[%d] = %d, expected %d\n", j, sendBuff[j], rank1Offset + i - 1 + j);
            *ret = 1;
            break;
          }
        }
      }
      for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
        size_t tileOffset = getTileElementOffset(j, width, rowIndex, colIndex, nElemPerPitch);
        sendBuff[tileOffset] = i + j;
      }
      __syncthreads();
      // __threadfence_system(); // not necessary if we make sendBuff volatile
      if (threadIdx.x == 0) proxyChan.put2DWithSignal(offset, width * sizeof(int), height);
    }
    if (rank == 1) {
      if (threadIdx.x == 0) proxyChan.wait();
      __syncthreads();
      for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
        size_t tileOffset = getTileElementOffset(j, width, rowIndex, colIndex, nElemPerPitch);
        if (sendBuff[tileOffset] != i + j) {
          // printf("rank 1 ERROR: sendBuff[%d] = %d, expected %d\n", j, sendBuff[j], i + j);
          *ret = 1;
          break;
        }
      }
      if (i < nTries - 1) {
        for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
          size_t tileOffset = getTileElementOffset(j, width, rowIndex, colIndex, nElemPerPitch);
          sendBuff[tileOffset] = offset + i + j;
        }
        __syncthreads();
        // __threadfence_system(); // not necessary if we make sendBuff volatile
        if (threadIdx.x == 0) proxyChan.put2DWithSignal(offset, width * sizeof(int), height);
      }
    }
    flusher++;
    if (flusher == 100) {
      if (threadIdx.x == 0) proxyChan.flush();
      flusher = 0;
    }
  }
}

__global__ void kernelProxyPingPong(int* buff, int rank, int nElem, int* ret) {
  DeviceHandle<mscclpp::SimpleProxyChannel>& proxyChan = gChannelOneToOneTestConstProxyChans;
  volatile int* sendBuff = (volatile int*)buff;
  int nTries = 1000;
  int flusher = 0;
  int rank1Offset = 10000000;
  for (int i = 0; i < nTries; i++) {
    if (rank == 0) {
      if (i > 0) {
        if (threadIdx.x == 0) proxyChan.wait();
        __syncthreads();
        for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
          if (sendBuff[j] != rank1Offset + i - 1 + j) {
            // printf("rank 0 ERROR: sendBuff[%d] = %d, expected %d\n", j, sendBuff[j], rank1Offset + i - 1 + j);
            *ret = 1;
            break;
          }
        }
      }
      for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
        sendBuff[j] = i + j;
      }
      __syncthreads();
      // __threadfence_system(); // not necessary if we make sendBuff volatile
      if (threadIdx.x == 0) proxyChan.putWithSignal(0, nElem * sizeof(int));
    }
    if (rank == 1) {
      if (threadIdx.x == 0) proxyChan.wait();
      __syncthreads();
      for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
        if (sendBuff[j] != i + j) {
          // printf("rank 1 ERROR: sendBuff[%d] = %d, expected %d\n", j, sendBuff[j], i + j);
          *ret = 1;
          break;
        }
      }
      if (i < nTries - 1) {
        for (int j = threadIdx.x; j < nElem; j += blockDim.x) {
          sendBuff[j] = rank1Offset + i + j;
        }
        __syncthreads();
        // __threadfence_system(); // not necessary if we make sendBuff volatile
        if (threadIdx.x == 0) proxyChan.putWithSignal(0, nElem * sizeof(int));
      }
    }
    flusher++;
    if (flusher == 100) {
      if (threadIdx.x == 0) proxyChan.flush();
      flusher = 0;
    }
  }
}

TEST_F(ProxyChannelOneToOneTest, PingPongIb) {
  if (gEnv->rank >= numRanksToUse) return;

  const int nElem = 4 * 1024 * 1024;

  std::vector<mscclpp::SimpleProxyChannel> proxyChannels;
  std::shared_ptr<int> buff = mscclpp::allocSharedCuda<int>(nElem);
  setupMeshConnections(proxyChannels, true, buff.get(), nElem * sizeof(int));

  std::vector<DeviceHandle<mscclpp::SimpleProxyChannel>> proxyChannelHandles;
  for (auto& ch : proxyChannels) proxyChannelHandles.push_back(ch.deviceHandle());

  ASSERT_EQ(proxyChannels.size(), 1);
  MSCCLPP_CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestConstProxyChans), proxyChannelHandles.data(),
                                       sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>)));

  proxyService->startProxy();

  std::shared_ptr<int> ret = mscclpp::makeSharedCudaHost<int>(0);

  kernelProxyPingPong<<<1, 1024>>>(buff.get(), gEnv->rank, 1, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  kernelProxyPingPong<<<1, 1024>>>(buff.get(), gEnv->rank, 1024, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  kernelProxyPingPong<<<1, 1024>>>(buff.get(), gEnv->rank, 1024 * 1024, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  kernelProxyPingPong<<<1, 1024>>>(buff.get(), gEnv->rank, 4 * 1024 * 1024, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  proxyService->stopProxy();
}

TEST_F(ProxyChannelOneToOneTest, PingPongTile) {
  if (gEnv->rank >= numRanksToUse) return;
  if (gEnv->worldSize > gEnv->nRanksPerNode) {
    // tile write only support single node
    GTEST_SKIP();
  }

  const int nElem = 4 * 1024 * 1024;

  std::vector<mscclpp::SimpleProxyChannel> proxyChannels;
  std::shared_ptr<int> buff = mscclpp::allocSharedCuda<int>(nElem);
  const int pitchSize = 512;  // the buff tile is 8192x128
  setupMeshConnections(proxyChannels, false, buff.get(), nElem * sizeof(int), pitchSize);

  ASSERT_EQ(proxyChannels.size(), 1);
  MSCCLPP_CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestConstProxyChans), proxyChannels.data(),
                                       sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>)));

  proxyService->startProxy();

  std::shared_ptr<int> ret = mscclpp::makeSharedCudaHost<int>(0);

  kernelProxyTilePingPong<<<1, 1024>>>(buff.get(), gEnv->rank, pitchSize, 0, 0, 1, 1, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  kernelProxyTilePingPong<<<1, 1024>>>(buff.get(), gEnv->rank, pitchSize, 128, 32, 64, 64, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  kernelProxyTilePingPong<<<1, 1024>>>(buff.get(), gEnv->rank, pitchSize, 16, 16, 1, 8192, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  kernelProxyTilePingPong<<<1, 1024>>>(buff.get(), gEnv->rank, pitchSize, 5, 0, 128, 1, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  kernelProxyTilePingPong<<<1, 1024>>>(buff.get(), gEnv->rank, pitchSize, 0, 0, 128, 8192, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);
}

__device__ mscclpp::DeviceSyncer gChannelOneToOneTestProxyChansSyncer;

template <bool CheckCorrectness>
__global__ void kernelProxyLLPingPong(int* buff, mscclpp::LLPacket* putPktBuf, mscclpp::LLPacket* getPktBuf, int rank,
                                      int nElem, int nTries, int* ret) {
  if (rank > 1) return;

  DeviceHandle<mscclpp::SimpleProxyChannel>& proxyChan = gChannelOneToOneTestConstProxyChans;
  volatile int* buffPtr = (volatile int*)buff;
  int putOffset = (rank == 0) ? 0 : 10000000;
  int getOffset = (rank == 0) ? 10000000 : 0;
  int threadId = threadIdx.x + blockIdx.x * blockDim.x;
  int numThreads = blockDim.x * gridDim.x;
  int flusher = 0;
  const size_t nPkt = nElem / 2;
  for (int i = 0; i < nTries; i++) {
    uint64_t flag = (uint64_t)i + 1;

    // rank=0: 0, 1, 0, 1, ...
    // rank=1: 1, 0, 1, 0, ...
    if ((rank ^ (i & 1)) == 0) {
      if (CheckCorrectness) {
        // If each thread writes 8 bytes at once, we don't need a barrier before putPackets().
        for (int j = threadId; j < nPkt; j += numThreads) {
          buffPtr[2 * j] = putOffset + i + 2 * j;
          buffPtr[2 * j + 1] = putOffset + i + 2 * j + 1;
        }
        // __syncthreads();
      }
      mscclpp::putPackets(putPktBuf, 0, buff, 0, nElem * sizeof(int), threadId, numThreads, flag);
      gChannelOneToOneTestProxyChansSyncer.sync(gridDim.x);
      if (threadId == 0) {
        // Send data from the local putPacketBuffer to the remote getPacketBuffer
        proxyChan.put(0, nPkt * sizeof(mscclpp::LLPacket));
      }
      flusher++;
      if (flusher == 64) {
        if (threadId == 0) proxyChan.flush();
        flusher = 0;
      }
    } else {
      mscclpp::getPackets(buff, 0, getPktBuf, 0, nElem * sizeof(int), threadId, numThreads, flag);
      if (CheckCorrectness) {
        // If each thread reads 8 bytes at once, we don't need a barrier after getPackets().
        // __syncthreads();
        for (int j = threadId; j < nPkt; j += numThreads) {
          if (buffPtr[2 * j] != getOffset + i + 2 * j) {
            // printf("ERROR: rank = %d, buffPtr[%d] = %d, expected %d. Skipping following errors\n", rank, 2 * j,
            //        buffPtr[2 * j], getOffset + i + 2 * j);
            *ret = 1;
            break;
          }
          if (buffPtr[2 * j + 1] != getOffset + i + 2 * j + 1) {
            // printf("ERROR: rank = %d, buffPtr[%d] = %d, expected %d. Skipping following errors\n", rank, 2 * j + 1,
            //        buffPtr[2 * j + 1], getOffset + i + 2 * j + 1);
            *ret = 1;
            break;
          }
        }
      }
      // Make sure all threads are done in this iteration
      gChannelOneToOneTestProxyChansSyncer.sync(gridDim.x);
    }
  }
}

void ProxyChannelOneToOneTest::testPacketPingPong(bool useIbOnly) {
  if (gEnv->rank >= numRanksToUse) return;

  const int nElem = 4 * 1024 * 1024;

  std::vector<mscclpp::SimpleProxyChannel> proxyChannels;
  std::shared_ptr<int> buff = mscclpp::allocSharedCuda<int>(nElem);

  const size_t nPacket = (nElem * sizeof(int) + sizeof(uint64_t) - 1) / sizeof(uint64_t);
  auto putPacketBuffer = mscclpp::allocSharedCuda<mscclpp::LLPacket>(nPacket);
  auto getPacketBuffer = mscclpp::allocSharedCuda<mscclpp::LLPacket>(nPacket);

  setupMeshConnections(proxyChannels, useIbOnly, putPacketBuffer.get(), nPacket * sizeof(mscclpp::LLPacket),
                       getPacketBuffer.get(), nPacket * sizeof(mscclpp::LLPacket));

  ASSERT_EQ(proxyChannels.size(), 1);

  std::vector<DeviceHandle<mscclpp::SimpleProxyChannel>> proxyChannelHandles;
  for (auto& proxyChannel : proxyChannels) {
    proxyChannelHandles.push_back(proxyChannel.deviceHandle());
  }

  MSCCLPP_CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestConstProxyChans), proxyChannelHandles.data(),
                                       sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>)));

  mscclpp::DeviceSyncer syncer = {};
  MSCCLPP_CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestProxyChansSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));

  proxyService->startProxy();

  std::shared_ptr<int> ret = mscclpp::makeSharedCudaHost<int>(0);

  const int nTries = 1000;

  // The least nelem is 2 for packet ping pong
  kernelProxyLLPingPong<true>
      <<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank, 2, nTries, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);
  *ret = 0;

  kernelProxyLLPingPong<true>
      <<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank, 1024, nTries, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);
  *ret = 0;

  kernelProxyLLPingPong<true><<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank,
                                           1024 * 1024, nTries, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);
  *ret = 0;

  kernelProxyLLPingPong<true><<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank,
                                           4 * 1024 * 1024, nTries, ret.get());
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  EXPECT_EQ(*ret, 0);

  communicator->bootstrap()->barrier();

  proxyService->stopProxy();
}

void ProxyChannelOneToOneTest::testPacketPingPongPerf(bool useIbOnly) {
  if (gEnv->rank >= numRanksToUse) return;

  const int nElem = 4 * 1024 * 1024;

  std::vector<mscclpp::SimpleProxyChannel> proxyChannels;
  std::shared_ptr<int> buff = mscclpp::allocSharedCuda<int>(nElem);

  const size_t nPacket = (nElem * sizeof(int) + sizeof(uint64_t) - 1) / sizeof(uint64_t);
  auto putPacketBuffer = mscclpp::allocSharedCuda<mscclpp::LLPacket>(nPacket);
  auto getPacketBuffer = mscclpp::allocSharedCuda<mscclpp::LLPacket>(nPacket);

  setupMeshConnections(proxyChannels, useIbOnly, putPacketBuffer.get(), nPacket * sizeof(mscclpp::LLPacket),
                       getPacketBuffer.get(), nPacket * sizeof(mscclpp::LLPacket));

  ASSERT_EQ(proxyChannels.size(), 1);

  std::vector<DeviceHandle<mscclpp::SimpleProxyChannel>> proxyChannelHandles;
  for (auto& proxyChannel : proxyChannels) {
    proxyChannelHandles.push_back(proxyChannel.deviceHandle());
  }

  MSCCLPP_CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestConstProxyChans), proxyChannelHandles.data(),
                                       sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>)));

  mscclpp::DeviceSyncer syncer = {};
  MSCCLPP_CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(gChannelOneToOneTestProxyChansSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));

  proxyService->startProxy();

  auto* testInfo = ::testing::UnitTest::GetInstance()->current_test_info();
  const std::string testName = std::string(testInfo->test_suite_name()) + "." + std::string(testInfo->name());
  const int nTries = 1000;

  // Warm-up
  kernelProxyLLPingPong<false>
      <<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank, 2, nTries, nullptr);
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  communicator->bootstrap()->barrier();

  // Measure latency
  mscclpp::Timer timer;
  kernelProxyLLPingPong<false>
      <<<1, 1024>>>(buff.get(), putPacketBuffer.get(), getPacketBuffer.get(), gEnv->rank, 2, nTries, nullptr);
  MSCCLPP_CUDATHROW(hipDeviceSynchronize());

  communicator->bootstrap()->barrier();

  if (gEnv->rank == 0) {
    std::cout << testName << ": " << std::setprecision(4) << (float)timer.elapsed() / (float)nTries << " us/iter\n";
  }

  proxyService->stopProxy();
}

TEST_F(ProxyChannelOneToOneTest, PacketPingPong) { testPacketPingPong(false); }

TEST_F(ProxyChannelOneToOneTest, PacketPingPongIb) { testPacketPingPong(true); }

TEST_F(ProxyChannelOneToOneTest, PacketPingPongPerf) { testPacketPingPongPerf(false); }

TEST_F(ProxyChannelOneToOneTest, PacketPingPongPerfIb) { testPacketPingPongPerf(true); }
