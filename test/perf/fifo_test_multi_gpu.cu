#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <getopt.h>

#include <iostream>
#include <map>
#include <memory>
#include <mscclpp/fifo.hpp>
#include <mscclpp/gpu_utils.hpp>
#include <mscclpp/numa.hpp>
#include <mscclpp/port_channel.hpp>
#include <mscclpp/port_channel_device.hpp>
#include <mscclpp/proxy.hpp>
#include <sstream>
#include <stdexcept>

#include "framework.hpp"

using namespace mscclpp::test;

// Constants for trigger calculation
constexpr int MIN_TRIGGERS = 1000;
constexpr int TRIGGERS_PER_FIFO_SIZE = 10;

__constant__ mscclpp::FifoDeviceHandle gFifoDeviceHandle;
__constant__ mscclpp::PortChannelDeviceHandle gPortChannel;

struct MultiGpuTestConfig {
  int fifoSize;
  int numGpus;    // Total number of GPUs
  int numGroups;  // Number of groups
  std::vector<int> parallelismLevels;

  MultiGpuTestConfig(int size, int gpus, int groups, const std::vector<int>& parallel = {64, 128, 256, 512})
      : fifoSize(size), numGpus(gpus), numGroups(groups), parallelismLevels(parallel) {
    if (numGpus % numGroups != 0) {
      throw std::invalid_argument("Number of GPUs must be divisible by number of groups");
    }
  }

  int getGroupSize() const { return numGpus / numGroups; }
  int getGroupIndex(int rank) const { return rank / getGroupSize(); }
  int getLocalRankInGroup(int rank) const { return rank % getGroupSize(); }

  // Get all ranks that participate in cross-group signaling (local rank 0 from each group)
  std::vector<int> getCrossGroupSignalingRanks() const {
    std::vector<int> signalingRanks;
    for (int group = 0; group < numGroups; group++) {
      int localRank0 = group * getGroupSize();  // First rank in each group
      signalingRanks.push_back(localRank0);
    }
    return signalingRanks;
  }

  // Check if this rank should participate in cross-group signaling
  bool shouldParticipateInSignaling(int rank) const {
    return getLocalRankInGroup(rank) == 0;  // Only local rank 0 in each group participates
  }
};

// Enhanced kernels for multi-GPU signaling
__global__ void kernelMultiGpuSignalSend(mscclpp::PortChannelDeviceHandle* portHandles, int numPeers, int numParallel) {
  int tid = threadIdx.x;

  // Each thread sends signals to all peers
  if (tid < numParallel) {
    for (int peer = 0; peer < numPeers; peer++) {
      portHandles[peer].signal();
    }
  }
}

__global__ void kernelMultiGpuSignalWait(mscclpp::PortChannelDeviceHandle* portHandles, int numPeers, int numParallel) {
  int tid = threadIdx.x;

  // Each thread waits for signals from all peers
  if (tid < numParallel) {
    for (int peer = 0; peer < numPeers; peer++) {
      portHandles[peer].wait();
    }
  }
}

static void setupCuda(int& cudaDevice, int& numaNode) {
  utils::CUDA_CHECK(hipGetDevice(&cudaDevice));
  numaNode = mscclpp::getDeviceNumaNode(cudaDevice);
  mscclpp::numaBind(numaNode);
}

// Enhanced performance measurement function
std::tuple<double, double, int> runMultiGpuKernelVariant(
    std::unique_ptr<mscclpp::Fifo>& hostFifo, hipStream_t stream, int numParallel, int rank,
    const std::vector<mscclpp::PortChannelDeviceHandle>& sendPortHandles,
    const std::vector<mscclpp::PortChannelDeviceHandle>& recvPortHandles, const MultiGpuTestConfig& config) {
  // Calculate triggers based on FIFO size, but respect the limit
  const int maxParallel = std::min(numParallel, config.fifoSize);
  const int numTriggers = std::max(MIN_TRIGGERS, static_cast<int>(hostFifo->size() * TRIGGERS_PER_FIFO_SIZE));

  // Configure kernel launch parameters
  int threadsPerBlock = std::min(maxParallel, 256);
  int threadBlocks = (maxParallel + threadsPerBlock - 1) / threadsPerBlock;

  // Copy port handles to device memory
  mscclpp::PortChannelDeviceHandle* d_sendHandles = nullptr;
  mscclpp::PortChannelDeviceHandle* d_recvHandles = nullptr;

  if (!sendPortHandles.empty()) {
    utils::CUDA_CHECK(hipMalloc(&d_sendHandles, sendPortHandles.size() * sizeof(mscclpp::PortChannelDeviceHandle)));
    utils::CUDA_CHECK(hipMemcpy(d_sendHandles, sendPortHandles.data(),
                                 sendPortHandles.size() * sizeof(mscclpp::PortChannelDeviceHandle),
                                 hipMemcpyHostToDevice));
  }

  if (!recvPortHandles.empty()) {
    utils::CUDA_CHECK(hipMalloc(&d_recvHandles, recvPortHandles.size() * sizeof(mscclpp::PortChannelDeviceHandle)));
    utils::CUDA_CHECK(hipMemcpy(d_recvHandles, recvPortHandles.data(),
                                 recvPortHandles.size() * sizeof(mscclpp::PortChannelDeviceHandle),
                                 hipMemcpyHostToDevice));
  }

  // Benchmark
  utils::Timer timer;
  timer.start();

  bool shouldSignal = config.shouldParticipateInSignaling(rank);

  if (shouldSignal) {
    // Launch signaling kernels
    if (!sendPortHandles.empty()) {
      kernelMultiGpuSignalSend<<<threadBlocks, threadsPerBlock, 0, stream>>>(d_sendHandles, sendPortHandles.size(),
                                                                             maxParallel);
      utils::CUDA_CHECK(hipGetLastError());
    }

    // Launch waiting kernels
    if (!recvPortHandles.empty()) {
      kernelMultiGpuSignalWait<<<threadBlocks, threadsPerBlock, 0, stream>>>(d_recvHandles, recvPortHandles.size(),
                                                                             maxParallel);
      utils::CUDA_CHECK(hipGetLastError());
    }
  }

  utils::CUDA_CHECK(hipStreamSynchronize(stream));
  timer.stop();

  // Cleanup device memory
  if (d_sendHandles) hipFree(d_sendHandles);
  if (d_recvHandles) hipFree(d_recvHandles);

  const int totalSignals = numTriggers * maxParallel * (sendPortHandles.size() + recvPortHandles.size());
  double throughput = totalSignals / timer.elapsedSeconds();
  double duration_us = timer.elapsedMicroseconds();

  utils::CUDA_CHECK(hipDeviceSynchronize());

  return {throughput, duration_us, totalSignals};
}

// Main multi-GPU test function
void runMultiGpuTest(const MultiGpuTestConfig& config, const mscclpp::test::TestContext& context) {
  int rank = context.rank;
  int worldSize = context.size;
  auto communicator = context.communicator;
  auto bootstrap = context.bootstrap;

  if (worldSize != config.numGpus) {
    throw std::invalid_argument("World size must match number of GPUs in config");
  }

  // Set the device for this process
  hipSetDevice(rank);

  // Setup transport
  mscclpp::TransportFlags transport = mscclpp::Transport::CudaIpc;
  std::vector<mscclpp::Transport> ibTransports{
      mscclpp::Transport::IB0, mscclpp::Transport::IB1, mscclpp::Transport::IB2, mscclpp::Transport::IB3,
      mscclpp::Transport::IB4, mscclpp::Transport::IB5, mscclpp::Transport::IB6, mscclpp::Transport::IB7};
  std::vector<std::shared_ptr<mscclpp::Connection>> connections;

  // Only create connections for GPUs that need to communicate
  if (config.shouldParticipateInSignaling(rank)) {
    mscclpp::Transport selectedTransport = ibTransports[config.getGroupIndex(rank) % ibTransports.size()];
    transport |= selectedTransport;

    // Get all ranks that participate in cross-group signaling
    auto signalingRanks = config.getCrossGroupSignalingRanks();

    for (int peerRank : signalingRanks) {
      if (peerRank != rank) {
        connections.push_back(communicator->connect(selectedTransport, peerRank).get());
      }
    }
  }

  // Wait for all connections to be established
  bootstrap->barrier();

  // Create and start proxy service
  auto proxyService = std::make_shared<mscclpp::ProxyService>(config.fifoSize);
  proxyService->startProxy();

  // Setup semaphore flags
  uint64_t* localSemaphoreFlag;
  hipMalloc(&localSemaphoreFlag, sizeof(uint64_t));
  hipMemset(localSemaphoreFlag, 0, sizeof(uint64_t));
  auto localFlagRegmem = communicator->registerMemory(localSemaphoreFlag, sizeof(uint64_t), transport);

  int cudaDevice, numaNode;
  setupCuda(cudaDevice, numaNode);

  // Create FIFO
  auto hostFifo = std::make_unique<mscclpp::Fifo>(config.fifoSize);

  hipStream_t stream;
  utils::CUDA_CHECK(hipStreamCreate(&stream));

  // Setup port channels for communication
  std::vector<mscclpp::PortChannelDeviceHandle> sendPortHandles;
  std::vector<mscclpp::PortChannelDeviceHandle> recvPortHandles;

  if (config.shouldParticipateInSignaling(rank)) {
    // Get all ranks that participate in cross-group signaling
    auto signalingRanks = config.getCrossGroupSignalingRanks();
    int connIndex = 0;

    for (int peerRank : signalingRanks) {
      if (peerRank != rank && connIndex < connections.size()) {
        auto connection = connections[connIndex++];
        auto semaphoreId = proxyService->buildAndAddSemaphore(*communicator, connection);

        // Create port channels for bidirectional communication
        auto sendPortChannel = proxyService->portChannel(semaphoreId, proxyService->addMemory(localFlagRegmem),
                                                         proxyService->addMemory(localFlagRegmem));
        auto recvPortChannel = proxyService->portChannel(semaphoreId, proxyService->addMemory(localFlagRegmem),
                                                         proxyService->addMemory(localFlagRegmem));

        sendPortHandles.push_back(sendPortChannel.deviceHandle());
        recvPortHandles.push_back(recvPortChannel.deviceHandle());
      }
    }
  }

  // Create test name
  std::string testName = "MultiGpuTest_GPUs" + std::to_string(config.numGpus) + "_Groups" +
                         std::to_string(config.numGroups) + "_FifoSize" + std::to_string(config.fifoSize);

  // Print test configuration
  if (utils::isMainRank()) {
    std::cout << "Running Multi-GPU test: " << config.numGpus << " GPUs, " << config.numGroups
              << " groups, FIFO size=" << config.fifoSize << std::endl;

    // Print which ranks participate in cross-group signaling
    auto signalingRanks = config.getCrossGroupSignalingRanks();
    std::cout << "Cross-group signaling participants: ";
    for (size_t i = 0; i < signalingRanks.size(); ++i) {
      if (i > 0) std::cout << ", ";
      std::cout << "rank " << signalingRanks[i] << " (group " << config.getGroupIndex(signalingRanks[i]) << ")";
    }
    std::cout << std::endl;
  }

  nlohmann::ordered_json combinedMetrics;

  // Run tests for different parallelism levels
  for (int numParallel : config.parallelismLevels) {
    // Ensure parallelism doesn't exceed FIFO size
    int effectiveParallel = std::min(numParallel, config.fifoSize);

    // Add synchronization before each test iteration
    MPI_Barrier(MPI_COMM_WORLD);

    if (config.shouldParticipateInSignaling(rank)) {
      auto [throughput, duration, totalSignals] =
          runMultiGpuKernelVariant(hostFifo, stream, effectiveParallel, rank, sendPortHandles, recvPortHandles, config);

      std::string prefix = "p" + std::to_string(effectiveParallel) + "_";
      combinedMetrics[prefix + "throughput_signals_per_sec"] = double(int(throughput * 10)) / 10.0;
      combinedMetrics[prefix + "duration_us"] = duration;
      combinedMetrics[prefix + "total_signals"] = totalSignals;
      combinedMetrics[prefix + "participating_gpus"] = config.numGpus;
    }

    // Add synchronization after each test iteration
    MPI_Barrier(MPI_COMM_WORLD);
  }

  // Record results
  std::map<std::string, std::string> testParams;
  testParams["num_gpus"] = std::to_string(config.numGpus);
  testParams["num_groups"] = std::to_string(config.numGroups);
  testParams["group_size"] = std::to_string(config.getGroupSize());
  testParams["fifo_size"] = std::to_string(config.fifoSize);
  testParams["participating_in_signaling"] = config.shouldParticipateInSignaling(rank) ? "true" : "false";

  // Add information about cross-group signaling ranks
  if (config.shouldParticipateInSignaling(rank)) {
    auto signalingRanks = config.getCrossGroupSignalingRanks();
    std::stringstream ss;
    for (size_t i = 0; i < signalingRanks.size(); ++i) {
      if (i > 0) ss << ",";
      ss << signalingRanks[i];
    }
    testParams["cross_group_signaling_ranks"] = ss.str();
  }

  utils::recordResult(testName, "multi_gpu_signaling", combinedMetrics, testParams);

  // Cleanup
  utils::CUDA_CHECK(hipStreamDestroy(stream));
  hipFree(localSemaphoreFlag);
  proxyService->stopProxy();
}

void runAllMultiGpuTests(const mscclpp::test::TestContext& context) {
  std::vector<MultiGpuTestConfig> configs = {
      // 8 GPUs, 2 groups (4 GPUs per group) - local rank 0 participates in signaling
      MultiGpuTestConfig(512, 8, 2, {1, 8, 64, 128, 256, 512}),

      // 8 GPUs, 4 groups (2 GPUs per group) - local rank 0 participates in signaling
      MultiGpuTestConfig(512, 8, 4, {1, 8, 64, 128, 256, 512}),

      // 8 GPUs, 8 groups (1 GPU per group) - local rank 0 participates in signaling
      MultiGpuTestConfig(512, 8, 8, {1, 8, 64, 128, 256, 512}),
  };

  for (const auto& config : configs) {
    // Only run if we have the right number of GPUs
    if (context.size == config.numGpus) {
      runMultiGpuTest(config, context);
    }
  }
}

static void printUsage(char* argv0) {
  std::stringstream ss;
  ss << "Usage: " << argv0 << " [OPTIONS]\n"
     << "\n"
     << "Options:\n"
     << "  -o, --output-format FORMAT   Output format: human or json (default: human)\n"
     << "  -f, --output-file FILE       JSON output file path (default: report.jsonl)\n"
     << "  -v, --verbose                Increase verbosity\n"
     << "  -h, --help                   Show this help message\n";
  std::cout << ss.str();
}

int main(int argc, char* argv[]) {
  std::string outputFormat = "human";
  std::string outputFile = "report.jsonl";
  bool verbose = false;

  static struct option longOptions[] = {{"output-format", required_argument, 0, 'o'},
                                        {"output-file", required_argument, 0, 'f'},
                                        {"verbose", no_argument, 0, 'v'},
                                        {"help", no_argument, 0, 'h'},
                                        {0, 0, 0, 0}};

  int c;
  while ((c = getopt_long(argc, argv, "o:f:vh", longOptions, nullptr)) != -1) {
    switch (c) {
      case 'o':
        outputFormat = optarg;
        break;
      case 'f':
        outputFile = optarg;
        break;
      case 'v':
        verbose = true;
        break;
      case 'h':
        printUsage(argv[0]);
        return 0;
      default:
        printUsage(argv[0]);
        return 1;
    }
  }

  std::vector<std::tuple<std::string, std::string, std::function<void(const mscclpp::test::TestContext&)>>> tests = {
      {"AllMultiGpuTests", "Multi-GPU signaling tests with configurable groups", runAllMultiGpuTests}};

  int result = utils::runMultipleTests(argc, argv, tests);

  if (utils::isMainRank()) {
    if (outputFormat == "json") {
      utils::writeResultsToFile(outputFile);
    } else {
      utils::printResults(verbose);
    }
  }

  utils::cleanupMPI();

  return result;
}
