#include "hip/hip_runtime.h"
#include <cassert>
#include <mscclpp/concurrency.hpp>
#include <vector>

#include "common.hpp"

#define ALIGN 4
#define BLOCKS_PER_PEER 1

__constant__ mscclpp::channel::SimpleDeviceChannel constDevFstRoundChans[16];
__constant__ mscclpp::channel::SimpleDeviceChannel constDevSndRoundChans[16];

__constant__ mscclpp::channel::SmChannel constSmChans[16];
__device__ mscclpp::channel::SimpleSmDeviceChannel globalSmDevChans[16];

// TODO(chhwang): need an interface for this.
static void* inputBuff = nullptr;
static void* resultBuff = nullptr;
static void* scratchBuff = nullptr;
static void* putPacketBuff = nullptr;
static void* getPacketBuff = nullptr;

struct Chunk {
  size_t offset;
  size_t size;
};

__host__ __device__ Chunk getChunk(size_t dataCount, size_t numChunks, size_t chunkIdx) {
  size_t remainder = dataCount % numChunks;
  size_t smallChunkSize = dataCount / numChunks;
  size_t largeChunkSize = smallChunkSize + 1;
  size_t numRemainedLargeChunks = chunkIdx < remainder ? remainder - chunkIdx : 0;
  size_t offset = (remainder - numRemainedLargeChunks) * largeChunkSize +
                  (chunkIdx > remainder ? chunkIdx - remainder : 0) * smallChunkSize;
  return Chunk{offset, chunkIdx < remainder ? largeChunkSize : smallChunkSize};
}

__forceinline__ __device__ void vectorSum(int* dst, int* src, size_t nElem) {
  size_t nInt4 = nElem / 4;
  size_t nLastInts = nElem % 4;
  int4* dst4 = (int4*)dst;
  int4* src4 = (int4*)src;
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < nInt4; i += blockDim.x * gridDim.x) {
    dst4[i].w += src4[i].w;
    dst4[i].x += src4[i].x;
    dst4[i].y += src4[i].y;
    dst4[i].z += src4[i].z;
  }
  if (nLastInts > 0) {
    int* dstLast = dst + nInt4 * 4;
    int* srcLast = src + nInt4 * 4;
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < nLastInts; i += blockDim.x * gridDim.x) {
      dstLast[i] += srcLast[i];
    }
  }
}

__device__ void vectorSumSingleBlock(int* dst, int* src, size_t nElem) {
  for (int i = threadIdx.x; i < nElem; i += blockDim.x) {
    dst[i] += src[i];
  }
}

__device__ mscclpp::DeviceSyncer deviceSyncer;

__device__ void allreduce0(int* buff, int* scratch, int rank, int worldSize, size_t nelems, size_t scratchDataCount) {
  int peerId = blockIdx.x / BLOCKS_PER_PEER;
  int isComm = (threadIdx.x == 0) && (blockIdx.x % BLOCKS_PER_PEER == 0);
  int remoteRank = (peerId < rank) ? peerId : peerId + 1;

  // 1st communication phase: send data to the scratch buffer of the peer associated with this block
  mscclpp::channel::SimpleDeviceChannel& devFstRoundChan = constDevFstRoundChans[peerId];
  Chunk toPeerChunk = getChunk(nelems, worldSize, remoteRank);
  // Now we need to figure out the offset of this chunk in the scratch buffer of the destination.
  // The destination will have allocated a scratch buffer of size numPeers() * toPeerChunk.size and
  // inside that each of the destination's peers send to the nth chunk, where n is the index of the
  // source peer from the destination's perspective.
  size_t dstOffset = (rank < remoteRank ? rank : rank - 1) * toPeerChunk.size;
  if (isComm) {
    // Write data to the peer
    devFstRoundChan.putWithSignalAndFlush(dstOffset * sizeof(int), toPeerChunk.offset * sizeof(int),
                                          toPeerChunk.size * sizeof(int));
    // Wait for data from the peer
    devFstRoundChan.wait();
  }

  deviceSyncer.sync(gridDim.x);

  // Local reduction: every block reduces a slice of each chunk in the scratch buffer into the user buffer
  mscclpp::channel::SimpleDeviceChannel& devSndRoundChan = constDevSndRoundChans[peerId];
  Chunk rankChunk = getChunk(nelems, worldSize, rank);
  int* chunk = buff + rankChunk.offset;
  int numPeers = gridDim.x / BLOCKS_PER_PEER;
  int numBlocks = gridDim.x;
  Chunk blockUserChunk = getChunk(rankChunk.size, numBlocks, blockIdx.x);
  size_t scratchDataCountPerPeer = scratchDataCount / numPeers;
  Chunk blockScratchChunk = getChunk(scratchDataCountPerPeer, numBlocks, blockIdx.x);
  for (int peerIdx = 0; peerIdx < numPeers; ++peerIdx) {
    int* scratchChunk = scratch + peerIdx * scratchDataCountPerPeer;
    vectorSumSingleBlock(chunk + blockUserChunk.offset, scratchChunk + blockScratchChunk.offset,
                         blockScratchChunk.size);
  }

  deviceSyncer.sync(gridDim.x);

  // 2nd communication phase: send the now reduced data between the user buffers
  Chunk collectionChunk = getChunk(nelems, worldSize, rank);
  if (isComm) {
    // Write data to the peer
    devSndRoundChan.putWithSignalAndFlush(collectionChunk.offset * sizeof(int), collectionChunk.offset * sizeof(int),
                                          collectionChunk.size * sizeof(int));
    // Wait for data from the peer
    devSndRoundChan.wait();
  }
}

__device__ void allreduce1(int* buff, int* scratch, int rank, int worldSize, size_t nelems, size_t scratchDataCount) {
  int isComm = (threadIdx.x == 0) && (blockIdx.x == 0);
  int remoteSendRank = (rank + 1) % worldSize;
  int remoteRecvRank = (rank + worldSize - 1) % worldSize;
  int peerSendId = (remoteSendRank < rank) ? remoteSendRank : remoteSendRank - 1;
  int peerRecvId = (remoteRecvRank < rank) ? remoteRecvRank : remoteRecvRank - 1;

  mscclpp::channel::SimpleDeviceChannel& devFstSendChan = constDevFstRoundChans[peerSendId];
  mscclpp::channel::SimpleDeviceChannel& devFstRecvChan = constDevFstRoundChans[peerRecvId];
  mscclpp::channel::SimpleDeviceChannel& devSndSendChan = constDevSndRoundChans[peerSendId];
  mscclpp::channel::SimpleDeviceChannel& devSndRecvChan = constDevSndRoundChans[peerRecvId];

  // Step 1
  size_t chunkIndex = (rank + worldSize - 1) % worldSize;
  size_t chunkNelem = nelems / worldSize;
  size_t offset = chunkIndex * chunkNelem * sizeof(int);
  if (isComm) {
    if (chunkNelem > 1) {
      devFstSendChan.putWithSignal(offset, chunkNelem / 2 * sizeof(int));
    }
  }

  // Step 2 ~ Step n-1
  for (int step = 2; step < worldSize; ++step) {
    if (isComm) {
      if (chunkNelem > 1) {
        devFstRecvChan.wait();
        devFstSendChan.flush();
      }
      devFstSendChan.putWithSignal(offset + chunkNelem / 2 * sizeof(int), (chunkNelem - chunkNelem / 2) * sizeof(int));
    }
    deviceSyncer.sync(gridDim.x);

    // Reduce
    chunkIndex = (rank + worldSize - step) % worldSize;
    offset = chunkIndex * chunkNelem * sizeof(int);
    int* dst = (int*)((char*)buff + offset);
    int* src = (int*)((char*)scratch + offset);
    vectorSum(dst, src, chunkNelem / 2);

    if (isComm) {
      devFstRecvChan.wait();
      devFstSendChan.flush();
      if (chunkNelem > 1) {
        devFstSendChan.putWithSignal(offset, chunkNelem / 2 * sizeof(int));
      }
    }
    deviceSyncer.sync(gridDim.x);

    dst += chunkNelem / 2;
    src += chunkNelem / 2;
    vectorSum(dst, src, chunkNelem - chunkNelem / 2);
  }

  // Step n
  if (isComm) {
    if (chunkNelem > 1) {
      devFstRecvChan.wait();
      devFstSendChan.flush();
    }
    devFstSendChan.putWithSignal(offset + chunkNelem / 2 * sizeof(int), (chunkNelem - chunkNelem / 2) * sizeof(int));
  }
  deviceSyncer.sync(gridDim.x);

  offset = rank * chunkNelem * sizeof(int);
  int* dst = (int*)((char*)buff + offset);
  int* src = (int*)((char*)scratch + offset);
  vectorSum(dst, src, chunkNelem / 2);

  if (isComm) {
    devFstRecvChan.wait();
    devFstSendChan.flush();
    if (chunkNelem > 1) {
      devSndSendChan.putWithSignal(offset, chunkNelem / 2 * sizeof(int));
    }
  }
  deviceSyncer.sync(gridDim.x);

  dst += chunkNelem / 2;
  src += chunkNelem / 2;
  vectorSum(dst, src, chunkNelem - chunkNelem / 2);

  if (isComm) {
    if (chunkNelem > 1) {
      devSndRecvChan.wait();
      devSndSendChan.flush();
    }
    devSndSendChan.putWithSignalAndFlush(offset + chunkNelem / 2 * sizeof(int),
                                         (chunkNelem - chunkNelem / 2) * sizeof(int));
  }

  // Step n+1 ~ Step 2n-2
  for (int i = 1; i < worldSize - 1; ++i) {
    if (isComm) {
      devSndRecvChan.wait();
    }
    deviceSyncer.sync(gridDim.x);

    // Copy
    chunkIndex = (rank + worldSize - i) % worldSize;
    if (isComm) {
      devSndSendChan.putWithSignalAndFlush(chunkIndex * chunkNelem * sizeof(int), chunkNelem * sizeof(int));
    }
  }

  // Final receive
  if (isComm) {
    devSndRecvChan.wait();
  }
}

__device__ void allreduce2(int* buff, void* putPktBuf, void* getPktBuf, void* result, int rank, int nRanksPerNode,
                           int worldSize, size_t nelems) {
  int chanIdx = blockIdx.x / BLOCKS_PER_PEER;
  // int numPeers = worldSize - 1;
  size_t nPkts = nelems / 2;  // 2 elems per packet, assume nelems is even
  size_t pktBytes = nPkts * sizeof(mscclpp::channel::ChannelPacket);
  mscclpp::channel::SmChannel smChan = constSmChans[chanIdx];
  uint32_t flag = (uint32_t)smChan.epochGetLocal() + 1;  // +1 as flag should be non-zero
  size_t srcOffset =
      ((blockIdx.x % BLOCKS_PER_PEER) * nelems * sizeof(int) / BLOCKS_PER_PEER);  // offset for this block
  size_t dstOffset = ((flag & 1) ? 0 : pktBytes * nRanksPerNode) +                // double buffering
                     ((chanIdx < rank ? rank - 1 : rank) * pktBytes) +            // offset for this rank
                     (srcOffset * 2);  // offset for this block: twice of srcOffset because 2 elems per packet

  smChan.putPacket(dstOffset, srcOffset, nelems / BLOCKS_PER_PEER * sizeof(int), threadIdx.x, blockDim.x, flag);

  int2* src = (int2*)buff;
  int2* res = (int2*)result;  // cumulate into here
  mscclpp::channel::ChannelPacket* tmpPtr =
      (mscclpp::channel::ChannelPacket*)getPktBuf + ((flag & 1) ? 0 : nPkts * nRanksPerNode);  // double buffering
  for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nPkts; idx += blockDim.x * gridDim.x) {
    int x = 0;
    int y = 0;
    for (int peerIdx = 0; peerIdx < nRanksPerNode / 2; ++peerIdx) {
      mscclpp::channel::ChannelPacket* pkt0 = tmpPtr + 2 * peerIdx * nPkts;
      mscclpp::channel::ChannelPacket* pkt1 = tmpPtr + (2 * peerIdx + 1) * nPkts;
      uint2 data0 = pkt0[idx].read(flag);
      uint2 data1 = pkt1[idx].read(flag);
      x += (int)data0.x;
      y += (int)data0.y;
      x += (int)data1.x;
      y += (int)data1.y;
    }
    if (nRanksPerNode & 1) {
      mscclpp::channel::ChannelPacket* pkt = tmpPtr + (nRanksPerNode - 1) * nPkts;
      uint2 data = pkt[idx].read(flag);
      x += (int)data.x;
      y += (int)data.y;
    }
    res[idx].x = src[idx].x + x;
    res[idx].y = src[idx].y + y;
  }

  if (threadIdx.x == 0 && (blockIdx.x % BLOCKS_PER_PEER) == 0) {
    smChan.epochIncrement();
  }
}

__global__ void kernel(void* buff, void* scratch, void* result, void* putPktBuf, void* getPktBuf, int rank,
                       int nRanksPerNode, int worldSize, size_t nelems, size_t scratchDataCount, int kernel) {
  if (kernel == 0)
    allreduce0((int*)buff, (int*)scratch, rank, worldSize, nelems, scratchDataCount);
  else if (kernel == 1)
    allreduce1((int*)buff, (int*)scratch, rank, worldSize, nelems, scratchDataCount);
  else if (kernel == 2)
    allreduce2((int*)buff, putPktBuf, getPktBuf, result, rank, nRanksPerNode, worldSize, nelems);
}

class AllReduceTestColl : public BaseTestColl {
 public:
  AllReduceTestColl() = default;
  ~AllReduceTestColl() = default;

  void runColl(const TestArgs& args, hipStream_t stream) override;
  void initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) override;
  void getBw(const double deltaSec, double& algBw /*OUT*/, double& busBw /*OUT*/) override;
  void setupCollTest(size_t size) override;
};

void AllReduceTestColl::runColl(const TestArgs& args, hipStream_t stream) {
  const int worldSize = args.totalRanks;
  const int rank = args.rank;
  const int kernelNum = args.kernelNum;
  const int nPeers = worldSize - 1;
  const Chunk chunk = getChunk(paramCount_, worldSize, rank);
  const size_t scratchDataCount = chunk.size * nPeers;
  int nBlocks;
  if (kernelNum == 0) {
    nBlocks = nPeers * BLOCKS_PER_PEER;
  } else if (kernelNum == 1) {
    nBlocks = 24;
  } else {
    nBlocks = (args.nRanksPerNode - 1) * BLOCKS_PER_PEER;
  }
  kernel<<<nBlocks, 1024, 0, stream>>>(inputBuff, scratchBuff, resultBuff, putPacketBuff, getPacketBuff, rank,
                                       args.nRanksPerNode, worldSize, paramCount_, scratchDataCount, kernelNum);
}

void AllReduceTestColl::initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) {
  assert(sendBuff.size() == 1);
  const int rank = args.rank;
  const int worldSize = args.totalRanks;
  std::vector<int> dataHost(std::max(sendCount_, recvCount_), rank);
  CUDATHROW(hipMemcpy(sendBuff[0], dataHost.data(), sendCount_ * typeSize_, hipMemcpyHostToDevice));

  for (size_t i = 0; i < recvCount_; i++) {
    dataHost[i] = worldSize * (worldSize - 1) / 2;
  }
  std::memcpy(expectedBuff, dataHost.data(), recvCount_ * typeSize_);
}

void AllReduceTestColl::getBw(const double deltaSec, double& algBw /*OUT*/, double& busBw /*OUT*/) {
  double baseBw = (double)(paramCount_ * typeSize_) / 1.0E9 / deltaSec;
  algBw = baseBw;
  double factor = (2 * (double)(worldSize_ - 1)) / ((double)worldSize_);
  busBw = baseBw * factor;
}

void AllReduceTestColl::setupCollTest(size_t size) {
  size_t count = size / typeSize_;
  size_t base = (count / ALIGN) * ALIGN;
  sendCount_ = base;
  recvCount_ = base;
  paramCount_ = base;
  recvCount_ = base;
  expectedCount_ = base;

  mscclpp::DeviceSyncer syncer = {};
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(deviceSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));
}

class AllReduceTestEngine : public BaseTestEngine {
 public:
  AllReduceTestEngine(const TestArgs& args);
  ~AllReduceTestEngine() = default;

  void allocateBuffer() override;
  void setupConnections() override;

  bool isUsePacket() const;
  bool isInPlace() const;

  std::vector<void*> getSendBuff() override;
  void* getRecvBuff() override;
  void* getScratchBuff() override;

 private:
  void* getExpectedBuff() override;

  std::shared_ptr<int> inputBuff_;
  std::shared_ptr<int> scratchBuff_;
  std::shared_ptr<int> resultBuff_;
  std::shared_ptr<mscclpp::channel::ChannelPacket> putPacketBuff_;
  std::shared_ptr<mscclpp::channel::ChannelPacket> getPacketBuff_;
  std::shared_ptr<int[]> expectedBuff_;
};

AllReduceTestEngine::AllReduceTestEngine(const TestArgs& args) : BaseTestEngine(args) { inPlace_ = isInPlace(); }

bool AllReduceTestEngine::isUsePacket() const { return (args_.kernelNum == 2); }

bool AllReduceTestEngine::isInPlace() const { return (args_.kernelNum != 2); }

void AllReduceTestEngine::allocateBuffer() {
  inputBuff_ = mscclpp::allocSharedCuda<int>(args_.maxBytes / sizeof(int));
  resultBuff_ = mscclpp::allocSharedCuda<int>(args_.maxBytes / sizeof(int));
  inputBuff = inputBuff_.get();
  resultBuff = resultBuff_.get();

  if (args_.kernelNum == 0 || args_.kernelNum == 1) {
    scratchBuff_ = mscclpp::allocSharedCuda<int>(args_.maxBytes / sizeof(int));
    scratchBuff = scratchBuff_.get();
  } else if (args_.kernelNum == 2) {
    const size_t nPacket = (args_.maxBytes + sizeof(uint64_t) - 1) / sizeof(uint64_t);
    // 2x for double-buffering
    putPacketBuff_ = mscclpp::allocSharedCuda<mscclpp::channel::ChannelPacket>(nPacket * (args_.nRanksPerNode - 1) * 2);
    getPacketBuff_ = mscclpp::allocSharedCuda<mscclpp::channel::ChannelPacket>(nPacket * (args_.nRanksPerNode - 1) * 2);
    putPacketBuff = putPacketBuff_.get();
    getPacketBuff = getPacketBuff_.get();
  } else {
    throw std::runtime_error("Invalid kernel number");
  }

  expectedBuff_ = std::shared_ptr<int[]>(new int[args_.maxBytes / sizeof(int)]);
}

void AllReduceTestEngine::setupConnections() {
  if (isUsePacket()) {
    std::vector<mscclpp::channel::SmChannel> smChannels;
    std::vector<mscclpp::channel::SimpleSmDeviceChannel> smDevChannels;

    const size_t nPacket = (args_.maxBytes + sizeof(uint64_t) - 1) / sizeof(uint64_t);
    const size_t packetBuffBytes = nPacket * (args_.nRanksPerNode - 1) * 2 * sizeof(mscclpp::channel::ChannelPacket);
    setupMeshConnections(smChannels, smDevChannels, inputBuff_.get(), args_.maxBytes, putPacketBuff_.get(),
                         packetBuffBytes, getPacketBuff_.get(), packetBuffBytes);

    assert(smChannels.size() < sizeof(constSmChans) / sizeof(mscclpp::channel::SmChannel));
    assert(smDevChannels.size() < sizeof(globalSmDevChans) / sizeof(mscclpp::channel::SimpleSmDeviceChannel));
    CUDATHROW(
        hipMemcpyToSymbol(HIP_SYMBOL(constSmChans), smChannels.data(), sizeof(mscclpp::channel::SmChannel) * smChannels.size()));
    CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(globalSmDevChans), smDevChannels.data(),
                                 sizeof(mscclpp::channel::SimpleSmDeviceChannel) * smDevChannels.size()));
  } else {
    std::vector<mscclpp::channel::SimpleDeviceChannel> fstRoundChannels;
    std::vector<mscclpp::channel::SimpleDeviceChannel> sndRoundChannels;

    // Send data from local sendBuff to remote scratchBuff (out-of-place)
    setupMeshConnections(fstRoundChannels, inputBuff_.get(), args_.maxBytes, scratchBuff_.get(), args_.maxBytes);
    assert(fstRoundChannels.size() < sizeof(constDevFstRoundChans) / sizeof(mscclpp::channel::SimpleDeviceChannel));
    CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constDevFstRoundChans), fstRoundChannels.data(),
                                 sizeof(mscclpp::channel::SimpleDeviceChannel) * fstRoundChannels.size()));

    // Send data from local sendBuff to remote sendBuff (in-place)
    setupMeshConnections(sndRoundChannels, inputBuff_.get(), args_.maxBytes);
    assert(sndRoundChannels.size() < sizeof(constDevSndRoundChans) / sizeof(mscclpp::channel::SimpleDeviceChannel));
    CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constDevSndRoundChans), sndRoundChannels.data(),
                                 sizeof(mscclpp::channel::SimpleDeviceChannel) * sndRoundChannels.size()));
  }
}

std::vector<void*> AllReduceTestEngine::getSendBuff() { return {inputBuff_.get()}; }

void* AllReduceTestEngine::getExpectedBuff() { return expectedBuff_.get(); }

void* AllReduceTestEngine::getRecvBuff() { return isInPlace() ? inputBuff_.get() : resultBuff_.get(); }

void* AllReduceTestEngine::getScratchBuff() { return scratchBuff_.get(); }

std::shared_ptr<BaseTestEngine> getTestEngine(const TestArgs& args) {
  return std::make_shared<AllReduceTestEngine>(args);
}

std::shared_ptr<BaseTestColl> getTestColl() { return std::make_shared<AllReduceTestColl>(); }
