#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <algorithm>
#include <mscclpp/concurrency_device.hpp>
#include <mscclpp/packet_device.hpp>
#include <vector>

#include "common.hpp"

#define BLOCKS_PER_PEER 1

template <class T>
using DeviceHandle = mscclpp::DeviceHandle<T>;
__constant__ DeviceHandle<mscclpp::SimpleProxyChannel> constDevFstRoundChans[16];
__constant__ DeviceHandle<mscclpp::SimpleProxyChannel> constDevSndRoundChans[16];

__constant__ DeviceHandle<mscclpp::SmChannel> constSmInPlaceChans[8];
__constant__ DeviceHandle<mscclpp::SmChannel> constSmOutOfPlaceChans[8];
__constant__ DeviceHandle<mscclpp::SmChannel> constSmOutOfPlaceGetChans[8];
__device__ uint64_t globalFlag;

// TODO(chhwang): need an interface for this.
static void* inputBuff = nullptr;
static void* resultBuff = nullptr;
static void* scratchBuff = nullptr;
static void* scratchPacketBuff = nullptr;
static void* putPacketBuff = nullptr;
static void* getPacketBuff = nullptr;

struct Chunk {
  size_t offset;
  size_t size;
};

__host__ __device__ Chunk getChunk(size_t dataCount, size_t numChunks, size_t chunkIdx) {
  size_t remainder = dataCount % numChunks;
  size_t smallChunkSize = dataCount / numChunks;
  size_t largeChunkSize = smallChunkSize + 1;
  size_t numRemainedLargeChunks = chunkIdx < remainder ? remainder - chunkIdx : 0;
  size_t offset = (remainder - numRemainedLargeChunks) * largeChunkSize +
                  (chunkIdx > remainder ? chunkIdx - remainder : 0) * smallChunkSize;
  return Chunk{offset, chunkIdx < remainder ? largeChunkSize : smallChunkSize};
}

__forceinline__ __device__ void vectorSum(int* dst, int* src, size_t nElem, int blockId, int nBlocks) {
  size_t nInt4 = nElem / 4;
  size_t nLastInts = nElem % 4;
  int4* dst4 = (int4*)dst;
  int4* src4 = (int4*)src;
  for (size_t i = threadIdx.x + blockId * blockDim.x; i < nInt4; i += blockDim.x * nBlocks) {
    dst4[i].w += src4[i].w;
    dst4[i].x += src4[i].x;
    dst4[i].y += src4[i].y;
    dst4[i].z += src4[i].z;
  }
  if (nLastInts > 0) {
    int* dstLast = dst + nInt4 * 4;
    int* srcLast = src + nInt4 * 4;
    for (size_t i = threadIdx.x + blockId * blockDim.x; i < nLastInts; i += blockDim.x * nBlocks) {
      dstLast[i] += srcLast[i];
    }
  }
}

__forceinline__ __device__ void vectorSum(int* dst, int* src, size_t nElem) {
  vectorSum(dst, src, nElem, blockIdx.x, gridDim.x);
}

__device__ void vectorSumSingleBlock(int* dst, int* src, size_t nElem) {
  for (size_t i = threadIdx.x; i < nElem; i += blockDim.x) {
    dst[i] += src[i];
  }
}

__device__ mscclpp::DeviceSyncer deviceSyncer;
__device__ mscclpp::DeviceSyncer allGatherDeviceSyncer;
__device__ mscclpp::DeviceSyncer reduceScatterDeviceSyncer;
__device__ mscclpp::DeviceSyncer ibDeviceSyncer;

__device__ void localReduceScatter(int* buff, int* scratch, int rank, int nRanksPerNode, int startChunkIndex,
                                   size_t offsetInChunk, size_t chunkSize, size_t nelems) {
  if (nRanksPerNode == 1) {
    return;
  }
  int isComm = (threadIdx.x == 0) && (blockIdx.x == 0);
  int startRankInNode = (rank / nRanksPerNode) * nRanksPerNode;
  int rankIndexInNode = rank % nRanksPerNode;

  for (int i = 1; i < nRanksPerNode; ++i) {
    int remoteSendToRank = (rank + i) % nRanksPerNode + startRankInNode;
    int remoteRecvFromRank = (rank + nRanksPerNode - i) % nRanksPerNode + startRankInNode;
    int peerSendId = (remoteSendToRank < rank) ? remoteSendToRank : remoteSendToRank - 1;
    int peerRecvId = (remoteRecvFromRank < rank) ? remoteRecvFromRank : remoteRecvFromRank - 1;

    DeviceHandle<mscclpp::SimpleProxyChannel>& devFstSendChan = constDevFstRoundChans[peerSendId];
    DeviceHandle<mscclpp::SimpleProxyChannel>& devFstRecvChan = constDevFstRoundChans[peerRecvId];
    size_t srcOffset =
        (((rankIndexInNode + i) % nRanksPerNode + startChunkIndex) * chunkSize + offsetInChunk) * sizeof(int);
    size_t dstOffset = rank * chunkSize * sizeof(int);

    if (i == 1) {
      if (isComm) {
        devFstSendChan.putWithSignal(dstOffset, srcOffset, nelems * sizeof(int));
      }
    } else {
      int pre = i - 1;
      int preRemoteRecvFromRank = (rank + nRanksPerNode - pre) % nRanksPerNode + startRankInNode;
      int prePeerRecvId = (preRemoteRecvFromRank < rank) ? preRemoteRecvFromRank : preRemoteRecvFromRank - 1;

      // overlap communication and computation
      DeviceHandle<mscclpp::SimpleProxyChannel>& preDevFstRecvChan = constDevFstRoundChans[prePeerRecvId];
      if (isComm) {
        preDevFstRecvChan.wait();
        devFstSendChan.putWithSignal(dstOffset, srcOffset, nelems * sizeof(int));
      }

      deviceSyncer.sync(gridDim.x);
      size_t offset = ((startChunkIndex + rankIndexInNode) * chunkSize + offsetInChunk) * sizeof(int);
      size_t scratchOffset = preRemoteRecvFromRank * chunkSize * sizeof(int);
      int* dst = (int*)((char*)buff + offset);
      int* src = (int*)((char*)scratch + scratchOffset);
      vectorSum(dst, src, nelems);
    }
    // for last iteration, wait for the last send
    if (i == nRanksPerNode - 1) {
      if (isComm) {
        devFstRecvChan.wait();
      }
      deviceSyncer.sync(gridDim.x);
      size_t offset = ((startChunkIndex + rankIndexInNode) * chunkSize + offsetInChunk) * sizeof(int);
      size_t scratchOffset = remoteRecvFromRank * chunkSize * sizeof(int);
      int* dst = (int*)((char*)buff + offset);
      int* src = (int*)((char*)scratch + scratchOffset);
      vectorSum(dst, src, nelems);
    }
  }
}

__device__ void reduceScatter(int* buff, int* scratch, int rank, int nRanksPerNode, int worldSize,
                              size_t nelems  // must be divisible by 3
) {
  // this reduce-scatter algorithm works as follows:
  // Step 1: each node does a local reduce-scatter on peer node data chunks with 1/pipeline portion of chunk data. For
  // example, 2 nodes and each node has 2 ranks. rank 0 and rank 1 perform reduce-scatter on chunk 2 and chunk 3, with
  // 1/pipeline portion of the data.
  // Step 2: each node does a local reduce-scatter on peers data chunks with (pipeline-1)/pipeline portion of chunk
  // data. Meanwhile, exchange the reduced data of the previous step with its cross-node neighbor (same local rank
  // number on the other node) via IB. Then performs a reduce operation.
  // Step 3:  each node does a local reduce-scatter on local ranks, meanwhile exchange the reduced data of the previous
  // step with its cross-node neighbor (same local rank number on the other node) via IB. Then performs a reduce
  // operation.
  int pipelineSize = 3;
  const size_t chunkSize = nelems / worldSize;
  int peerRank = (rank + nRanksPerNode) % worldSize;
  int peerNodeId = peerRank / nRanksPerNode;
  int isComm = (threadIdx.x == 0) && (blockIdx.x == 0);
  int peer = (peerRank < rank) ? peerRank : peerRank - 1;
  DeviceHandle<mscclpp::SimpleProxyChannel>& proxyChan = constDevFstRoundChans[peer];
  if (peerNodeId == rank / nRanksPerNode) {
    localReduceScatter(buff, scratch, rank, nRanksPerNode, 0, 0, chunkSize, chunkSize);
    return;
  }

  // step 1: local reduce
  int startChunkIndex = peerNodeId * nRanksPerNode;
  localReduceScatter(buff, scratch, rank, nRanksPerNode, startChunkIndex, 0, chunkSize, chunkSize / pipelineSize);
  deviceSyncer.sync(gridDim.x);

  // step 2: local reduce and exchange data with neighbor
  if (isComm) {
    size_t offset = (peerRank * chunkSize) * sizeof(int);
    // opposite side
    proxyChan.putWithSignal(offset, (chunkSize / pipelineSize * sizeof(int)));
  }
  localReduceScatter(buff, scratch, rank, nRanksPerNode, startChunkIndex, chunkSize / pipelineSize, chunkSize,
                     2 * chunkSize / pipelineSize);
  if (isComm) {
    proxyChan.wait();
  }
  deviceSyncer.sync(gridDim.x);
  // reduce data received from peer to related rank
  size_t offset = rank * chunkSize * sizeof(int);
  int* dst = (int*)((char*)buff + offset);
  int* src = (int*)((char*)scratch + offset);
  vectorSum(dst, src, chunkSize / pipelineSize);
  if (isComm) {
    proxyChan.flush();
  }
  deviceSyncer.sync(gridDim.x);

  // step 3: local reduce and exchange data with neighbor
  startChunkIndex = (rank / nRanksPerNode) * nRanksPerNode;
  if (isComm) {
    size_t offset = (peerRank * chunkSize + chunkSize / pipelineSize) * sizeof(int);
    proxyChan.putWithSignal(offset, (pipelineSize - 1) * chunkSize / pipelineSize * sizeof(int));
  }
  localReduceScatter(buff, scratch, rank, nRanksPerNode, startChunkIndex, 0, chunkSize, chunkSize);
  if (isComm) {
    proxyChan.wait();
  }
  deviceSyncer.sync(gridDim.x);
  // reduce to related rank
  offset = (rank * chunkSize + chunkSize / pipelineSize) * sizeof(int);
  dst = (int*)((char*)buff + offset);
  src = (int*)((char*)scratch + offset);
  vectorSum(dst, src, 2 * chunkSize / pipelineSize);
  if (isComm) {
    proxyChan.flush();
  }
}

// Run with a single thread only.
__device__ void localAllGather(int rank, int nRanksPerNode, uint64_t offset, uint64_t size) {
  // this allgather algorithm works as follows:
  // Step 1: GPU rank i sends data to GPU rank (i+1) % nranksPerNode
  // and waits for data from GPU rank (i-1) % nranksPerNode
  // Step 2: GPU rank i sends data to GPU rank (i+2) % nranksPerNode
  // ...
  // This order is much better for DMA engine for NVLinks
  if (nRanksPerNode == 1) return;

  int startRankInNode = (rank / nRanksPerNode) * nRanksPerNode;
  for (int i = 1; i < nRanksPerNode; i++) {
    int remoteSendToRank = (rank + i) % nRanksPerNode + startRankInNode;
    int remoteRecvFromRank = (rank + nRanksPerNode - i) % nRanksPerNode + startRankInNode;
    int peerSendId = (remoteSendToRank < rank) ? remoteSendToRank : remoteSendToRank - 1;
    int peerRecvId = (remoteRecvFromRank < rank) ? remoteRecvFromRank : remoteRecvFromRank - 1;

    DeviceHandle<mscclpp::SimpleProxyChannel>& devSendChan = constDevSndRoundChans[peerSendId];
    DeviceHandle<mscclpp::SimpleProxyChannel>& devRecvChan = constDevSndRoundChans[peerRecvId];
    // wait for the data from GPU (rank-i) % nranksPerNode to arrive
    devSendChan.putWithSignal(offset, size);
    devRecvChan.wait();
  }
}

// Run with a single thread only.
__device__ void allGather(int rank, int worldSize, int nRanksPerNode, size_t nelemsPerGPU) {
  // this allgather is a pipelined and hierarchical one and only works for two nodes
  // it is implemented as follows:
  // Step 1: each node does a local allgather and concurrently,
  // local GPU i exchange (piplineSize-1)/pipelineSize portion of their data with
  // its cross-node neighbor (local GPU i on the other node) via IB
  // Step 2: each node does a local allgather again with the data just received from its
  // cross-node neighbor in step 1, and concurrently, exchange the rest of the data with
  // its cross-node neighbor
  // Step 3: each node does a local allgather for the last time with the rest of the data

  int pipelineSize = 3;
  int peerRank = (rank + nRanksPerNode) % worldSize;
  int peerNodeId = peerRank / nRanksPerNode;
  int peer = (peerRank < rank) ? peerRank : peerRank - 1;
  DeviceHandle<mscclpp::SimpleProxyChannel>& proxyChan = constDevSndRoundChans[peer];

  if (peerNodeId == rank / nRanksPerNode) {
    localAllGather(rank, nRanksPerNode, rank * nelemsPerGPU * sizeof(int), nelemsPerGPU * sizeof(int));
    return;
  }

  // Step 1
  proxyChan.putWithSignal(rank * nelemsPerGPU * sizeof(int),
                          (nelemsPerGPU * (pipelineSize - 1)) / pipelineSize * sizeof(int));
  localAllGather(rank, nRanksPerNode, rank * nelemsPerGPU * sizeof(int), nelemsPerGPU * sizeof(int));
  proxyChan.wait();
  proxyChan.flush();
  // Step 2
  proxyChan.putWithSignal((rank * nelemsPerGPU + (nelemsPerGPU * (pipelineSize - 1)) / pipelineSize) * sizeof(int),
                          nelemsPerGPU / pipelineSize * sizeof(int));
  localAllGather(rank, nRanksPerNode, peerRank * nelemsPerGPU * sizeof(int),
                 (nelemsPerGPU * (pipelineSize - 1)) / pipelineSize * sizeof(int));
  proxyChan.wait();
  proxyChan.flush();
  // Step 3
  localAllGather(rank, nRanksPerNode,
                 (peerRank * nelemsPerGPU + (nelemsPerGPU * (pipelineSize - 1)) / pipelineSize) * sizeof(int),
                 nelemsPerGPU / pipelineSize * sizeof(int));
}

__device__ void localReduceScatterSm(int* buff, int rank, int nRanksPerNode, int startChunkIndex, size_t offsetInChunk,
                                     size_t chunkSize, size_t nelems, int nBlocks) {
  if (nRanksPerNode == 1) return;
  if ((int)blockIdx.x >= nBlocks) return;
  const int nPeer = nRanksPerNode - 1;
  DeviceHandle<mscclpp::SmChannel>* smChans = constSmOutOfPlaceGetChans;

  const size_t localRankIndexInNode = rank % nRanksPerNode;
  const size_t indexOffset = ((localRankIndexInNode + startChunkIndex) * chunkSize + offsetInChunk);
  const size_t indexOffset4 = indexOffset / 4;

  int4* buff4 = (int4*)buff;

  for (int peerIdx = threadIdx.x + blockIdx.x * blockDim.x; peerIdx < nPeer; peerIdx += blockDim.x * nBlocks) {
    smChans[peerIdx].signal();
  }
  for (int peerIdx = threadIdx.x + blockIdx.x * blockDim.x; peerIdx < nPeer; peerIdx += blockDim.x * nBlocks) {
    smChans[peerIdx].wait();
  }
  reduceScatterDeviceSyncer.sync(nBlocks);

  const size_t nInt4 = nelems / 4;
  for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nInt4; idx += blockDim.x * nBlocks) {
    int4 sum = make_int4(0, 0, 0, 0);

    for (int peerIdx = 0; peerIdx < nPeer; peerIdx++) {
      int4 val = smChans[peerIdx].read<int4>(indexOffset4 + idx);
      sum.w += val.w;
      sum.x += val.x;
      sum.y += val.y;
      sum.z += val.z;
    }
    buff4[indexOffset4 + idx].w += sum.w;
    buff4[indexOffset4 + idx].x += sum.x;
    buff4[indexOffset4 + idx].y += sum.y;
    buff4[indexOffset4 + idx].z += sum.z;
  }

  const size_t nLastInts = nelems % 4;
  for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nLastInts; idx += blockDim.x * nBlocks) {
    int sum = 0;
    for (int peerIdx = 0; peerIdx < nPeer; peerIdx++) {
      int val = smChans[peerIdx].read<int>(indexOffset + nInt4 * 4 + idx);
      sum += val;
    }
    buff[indexOffset + nInt4 * 4 + idx] += sum;
  }
}

__device__ void localReduceScatterSm2(int* buff, int rank, int nRanksPerNode, size_t chunkSize, size_t nelems,
                                      int nBlocks) {
  if (nRanksPerNode == 1) return;
  if ((int)blockIdx.x >= nBlocks) return;
  const int nPeer = nRanksPerNode - 1;
  DeviceHandle<mscclpp::SmChannel>* smChans = constSmOutOfPlaceGetChans;

  const size_t localRankIndexInNode = rank % nRanksPerNode;
  const size_t indexOffset = localRankIndexInNode * chunkSize;
  const size_t indexOffset4 = indexOffset / 4;

  int4* buff4 = (int4*)buff;

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < nPeer) {
    smChans[tid].signal();
  }
  const int waitStart = nBlocks * blockDim.x - nPeer;
  if (tid >= waitStart && tid < (int)(nBlocks * blockDim.x)) {
    smChans[tid - waitStart].wait();
  }
  reduceScatterDeviceSyncer.sync(nBlocks);

  const size_t nInt4 = nelems / 4;
  for (int index = 0; index < nPeer; ++index) {
    int4 val;
    int peerIdx = (index + localRankIndexInNode) % nPeer;
    for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nInt4; idx += blockDim.x * nBlocks) {
      val = smChans[peerIdx].read<int4>(indexOffset4 + idx);
      buff4[indexOffset4 + idx].w += val.w;
      buff4[indexOffset4 + idx].x += val.x;
      buff4[indexOffset4 + idx].y += val.y;
      buff4[indexOffset4 + idx].z += val.z;
    }
  }

  const size_t nLastInts = nelems % 4;
  for (int peerIdx = 0; peerIdx < nPeer; peerIdx++) {
    for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nLastInts; idx += blockDim.x * nBlocks) {
      int val = smChans[(localRankIndexInNode + peerIdx) % nPeer].read<int>(indexOffset + nInt4 * 4 + idx);
      buff[indexOffset + nInt4 * 4 + idx] += val;
    }
  }
}

__device__ void localReduceScatterSm3(int* buff, int rank, int nRanksPerNode, size_t chunkSize, size_t nelems,
                                      int nBlocks) {
  if (nRanksPerNode == 1) return;
  if ((int)blockIdx.x >= nBlocks) return;
  const int nPeer = nRanksPerNode - 1;
  DeviceHandle<mscclpp::SmChannel>* smChans = constSmOutOfPlaceGetChans;

  const size_t localRankIndexInNode = rank % nRanksPerNode;
  const size_t indexOffset = localRankIndexInNode * chunkSize;
  const size_t indexOffset4 = indexOffset / 4;

  int4* buff4 = (int4*)buff;

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < nPeer) {
    smChans[tid].signal();
  }
  const int waitStart = nBlocks * blockDim.x - nPeer;
  if (tid >= waitStart && tid < (int)(nBlocks * blockDim.x)) {
    smChans[tid - waitStart].wait();
  }
  reduceScatterDeviceSyncer.sync(nBlocks);

  const size_t nInt4 = nelems / 4;

  size_t base = 0;
  const size_t unitNInt4 = blockDim.x * nBlocks;
  for (; base + unitNInt4 < nInt4; base += unitNInt4) {
    for (int index = 0; index < nPeer; ++index) {
      int4 val;
      int peerIdx = (index + localRankIndexInNode) % nPeer;
      for (size_t idx = base + threadIdx.x + blockIdx.x * blockDim.x; idx < base + unitNInt4;
           idx += blockDim.x * nBlocks) {
        val = smChans[peerIdx].read<int4>(indexOffset4 + idx);
        buff4[indexOffset4 + idx].w += val.w;
        buff4[indexOffset4 + idx].x += val.x;
        buff4[indexOffset4 + idx].y += val.y;
        buff4[indexOffset4 + idx].z += val.z;
      }
    }
  }
  for (int index = 0; index < nPeer; ++index) {
    int4 val;
    int peerIdx = (index + localRankIndexInNode) % nPeer;
    for (size_t idx = base + threadIdx.x + blockIdx.x * blockDim.x; idx < nInt4; idx += blockDim.x * nBlocks) {
      val = smChans[peerIdx].read<int4>(indexOffset4 + idx);
      buff4[indexOffset4 + idx].w += val.w;
      buff4[indexOffset4 + idx].x += val.x;
      buff4[indexOffset4 + idx].y += val.y;
      buff4[indexOffset4 + idx].z += val.z;
    }
  }

  const size_t nLastInts = nelems % 4;
  for (int peerIdx = 0; peerIdx < nPeer; peerIdx++) {
    for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nLastInts; idx += blockDim.x * nBlocks) {
      int val = smChans[(localRankIndexInNode + peerIdx) % nPeer].read<int>(indexOffset + nInt4 * 4 + idx);
      buff[indexOffset + nInt4 * 4 + idx] += val;
    }
  }
}

__device__ void reduceScatterSm(int* buff, int* scratch, int rank, int nRanksPerNode, int worldSize,
                                size_t nelems  // must be divisible by 3
) {
  // this reduce-scatter algorithm works as follows:
  // Step 1: each node does a local reduce-scatter on peer node data chunks with 1/pipeline portion of chunk data. For
  // example, 2 nodes and each node has 2 ranks. rank 0 and rank 1 perform reduce-scatter on chunk 2 and chunk 3, with
  // 1/pipeline portion of the data.
  // Step 2: each node does a local reduce-scatter on peers data chunks with (pipeline-1)/pipeline portion of chunk
  // data. Meanwhile, exchange the reduced data of the previous step with its cross-node neighbor (same local rank
  // number on the other node) via IB. Then performs a reduce operation.
  // Step 3:  each node does a local reduce-scatter on local ranks, meanwhile exchange the reduced data of the previous
  // step with its cross-node neighbor (same local rank number on the other node) via IB. Then performs a reduce
  // operation.
  int pipelineSize = 3;
  float nBlocksForReduceScatterRatio = 0.8;
  const size_t chunkSize = nelems / worldSize;
  const int peerRank = (rank + nRanksPerNode) % worldSize;
  int peerNodeId = peerRank / nRanksPerNode;
  int nBlocksForReduceScatter =
      (int)(nBlocksForReduceScatterRatio * gridDim.x) / (nRanksPerNode - 1) * (nRanksPerNode - 1);
  int isComm = (threadIdx.x == 0) && ((int)blockIdx.x == nBlocksForReduceScatter);
  int peer = (peerRank < rank) ? peerRank : peerRank - 1;
  int nBlocksRemain = gridDim.x - nBlocksForReduceScatter;
  DeviceHandle<mscclpp::SimpleProxyChannel>& proxyChan = constDevFstRoundChans[peer];
  if (peerNodeId == rank / nRanksPerNode) {
    localReduceScatterSm(buff, rank, nRanksPerNode, 0, 0, chunkSize, chunkSize, gridDim.x);
    return;
  }

  // step 1: local reduce
  int startChunkIndex = peerNodeId * nRanksPerNode;
  localReduceScatterSm(buff, rank, nRanksPerNode, startChunkIndex, 0, chunkSize, chunkSize / pipelineSize,
                       nBlocksForReduceScatter);
  deviceSyncer.sync(gridDim.x);

  // step 2: local reduce and exchange data with neighbor
  if (isComm) {
    size_t offset = (peerRank * chunkSize) * sizeof(int);
    // opposite side
    proxyChan.putWithSignal(offset, (chunkSize / pipelineSize * sizeof(int)));
  }
  localReduceScatterSm(buff, rank, nRanksPerNode, startChunkIndex, chunkSize / pipelineSize, chunkSize,
                       2 * chunkSize / pipelineSize, nBlocksForReduceScatter);
  if (isComm) {
    proxyChan.wait();
  }
  if ((int)blockIdx.x >= nBlocksForReduceScatter) {
    ibDeviceSyncer.sync(nBlocksRemain);
    // reduce data received from peer to related rank
    size_t offset = rank * chunkSize * sizeof(int);
    int* dst = (int*)((char*)buff + offset);
    int* src = (int*)((char*)scratch + offset);
    vectorSum(dst, src, chunkSize / pipelineSize, blockIdx.x - nBlocksForReduceScatter, nBlocksRemain);
  }
  if (isComm) {
    proxyChan.flush();
  }
  deviceSyncer.sync(gridDim.x);

  // step 3: local reduce and exchange data with neighbor
  startChunkIndex = (rank / nRanksPerNode) * nRanksPerNode;
  if (isComm) {
    size_t offset = (peerRank * chunkSize + chunkSize / pipelineSize) * sizeof(int);
    proxyChan.putWithSignal(offset, (pipelineSize - 1) * chunkSize / pipelineSize * sizeof(int));
  }
  localReduceScatterSm(buff, rank, nRanksPerNode, startChunkIndex, 0, chunkSize, chunkSize, nBlocksForReduceScatter);
  if (isComm) {
    proxyChan.wait();
  }
  deviceSyncer.sync(gridDim.x);
  // reduce to related rank, can not overlap since localReduceScatter also calculate the sum
  size_t offset = (rank * chunkSize + chunkSize / pipelineSize) * sizeof(int);
  int* dst = (int*)((char*)buff + offset);
  int* src = (int*)((char*)scratch + offset);
  vectorSum(dst, src, 2 * chunkSize / pipelineSize);
  if (isComm) {
    proxyChan.flush();
  }
}

// This kernel is the most performant when the number of blocks is a multiple of (nRanksPerNode - 1).
__device__ void localAllGatherSm(int rank, int nRanksPerNode, int startRankChunkIndex, uint64_t offsetInRankChunk,
                                 uint64_t rankChunkSize, uint64_t size, size_t nBlocks) {
  if (nRanksPerNode == 1) return;
  if (blockIdx.x >= nBlocks) return;
  const size_t nPeer = nRanksPerNode - 1;
  const size_t peerIdx = blockIdx.x % nPeer;
  const size_t nBlockForThisPeer = nBlocks / nPeer + (nBlocks % nPeer > peerIdx ? 1 : 0);
  const size_t peerLocalBlockIdx = blockIdx.x / nPeer;
  const size_t rankLocalIndex = rank % nRanksPerNode;
  const int remoteRankLocalIndex = (peerIdx < rankLocalIndex ? peerIdx : peerIdx + 1);

  // Split the data into chunks for aligned data access. Ignore the remainder here and let the last block handle it.
  constexpr size_t chunkBytes = 128;  // heuristic value
  const size_t nChunk = size / chunkBytes;
  const size_t nMinChunkPerBlock = nChunk / nBlockForThisPeer;
  const size_t nRemainderChunk = nChunk % nBlockForThisPeer;

  // Distribute chunks to blocks
  size_t nChunkForThisBlock;
  size_t offsetForThisBlock;
  if (peerLocalBlockIdx < nRemainderChunk) {
    nChunkForThisBlock = nMinChunkPerBlock + 1;
    offsetForThisBlock = (nMinChunkPerBlock + 1) * peerLocalBlockIdx;
  } else {
    nChunkForThisBlock = nMinChunkPerBlock;
    offsetForThisBlock =
        (nMinChunkPerBlock + 1) * nRemainderChunk + (peerLocalBlockIdx - nRemainderChunk) * nMinChunkPerBlock;
  }
  offsetForThisBlock *= chunkBytes;

  // Calculate the size of the data for this block
  size_t sizeForThisBlock = nChunkForThisBlock * chunkBytes;
  const size_t lastChunkSize = size - nChunk * chunkBytes;
  if (lastChunkSize > 0 && peerLocalBlockIdx == nBlockForThisPeer - 1) {
    sizeForThisBlock += lastChunkSize;
  }
  if (threadIdx.x == 0 && peerLocalBlockIdx == 0) {
    constSmInPlaceChans[peerIdx].signal();
    constSmInPlaceChans[peerIdx].wait();
  }
  allGatherDeviceSyncer.sync(nBlocks);
  size_t offset = rankChunkSize * (startRankChunkIndex + remoteRankLocalIndex) + offsetInRankChunk;
  constSmInPlaceChans[peerIdx].get(offset + offsetForThisBlock, sizeForThisBlock, threadIdx.x, blockDim.x);
}

__device__ void localRingAllGatherSm(int rank, int nRanksPerNode, uint64_t size, size_t nBlocks) {
  if (nRanksPerNode == 1) return;
  if (blockIdx.x >= nBlocks) return;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int nPeer = nRanksPerNode - 1;

  if (tid < nPeer) {
    constSmInPlaceChans[tid].signal();
  }
  int waitStart = nBlocks * blockDim.x - nPeer;
  if (tid >= waitStart && tid < (int)(nBlocks * blockDim.x)) {
    constSmInPlaceChans[tid - waitStart].wait();
  }
  allGatherDeviceSyncer.sync(nBlocks);
  for (int i = 0; i < nPeer; ++i) {
    int peerIdx = (i + rank) % nPeer;
    const int remoteRankLocalIndex = (peerIdx < rank ? peerIdx : peerIdx + 1);
    size_t offset = size * remoteRankLocalIndex;
    constSmInPlaceChans[peerIdx].get(offset, size, tid, blockDim.x * nBlocks);
  }
}

__device__ void localRingAllGatherSm2(size_t rank, size_t nRanksPerNode, size_t size, size_t nBlocks) {
  if (nRanksPerNode == 1) return;
  if (blockIdx.x >= nBlocks) return;

  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t nPeer = nRanksPerNode - 1;

  if (tid < nPeer) {
    constSmInPlaceChans[tid].signal();
  }
  size_t waitStart = nBlocks * blockDim.x - nPeer;
  if (tid >= waitStart && tid < nBlocks * blockDim.x) {
    constSmInPlaceChans[tid - waitStart].wait();
  }
  allGatherDeviceSyncer.sync(nBlocks);
  const size_t unitSize = 16 * blockDim.x * nBlocks;
  size_t base = 0;
  for (; base + unitSize < size; base += unitSize) {
    for (size_t i = 0; i < nPeer; ++i) {
      size_t peerIdx = (i + rank) % nPeer;
      const size_t remoteRankLocalIndex = (peerIdx < rank ? peerIdx : peerIdx + 1);
      size_t offset = size * remoteRankLocalIndex + base;
      constSmInPlaceChans[peerIdx].get(offset, unitSize, tid, blockDim.x * nBlocks);
    }
  }
  for (size_t i = 0; i < nPeer; ++i) {
    size_t peerIdx = (i + rank) % nPeer;
    const size_t remoteRankLocalIndex = (peerIdx < rank ? peerIdx : peerIdx + 1);
    size_t offset = size * remoteRankLocalIndex + base;
    constSmInPlaceChans[peerIdx].get(offset, size - base, tid, blockDim.x * nBlocks);
  }
}

// This is an allgather4 equivalent
__device__ void allGatherSm(int rank, int worldSize, int nRanksPerNode, size_t nelemsPerGPU) {
  // this allgather is a pipelined and hierarchical one and only works for two nodes
  // it is implemented as follows:
  // Step 1: each node does a local allgather and concurrently,
  // local GPU i exchange (piplineSize-1)/pipelineSize portion of their data with
  // its cross-node neighbor (local GPU i on the other node) via IB
  // Step 2: each node does a local allgather again with the data just received from its
  // cross-node neighbor in step 1, and concurrently, exchange the rest of the data with
  // its cross-node neighbor
  // Step 3: each node does a local allgather for the last time with the rest of the data

  int pipelineSize = 3;
  int peerRank = (rank + nRanksPerNode) % worldSize;
  int peerNodeId = peerRank / nRanksPerNode;
  int peer = (peerRank < rank) ? peerRank : peerRank - 1;
  DeviceHandle<mscclpp::SimpleProxyChannel>& proxyChan = constDevSndRoundChans[peer];
  const size_t nBlocksForLocalAllGather = gridDim.x / (nRanksPerNode - 1) * (nRanksPerNode - 1);
  const size_t rankChunkSize = nelemsPerGPU * sizeof(int);
  const int startRankIndexInLocalNode = (rank / nRanksPerNode) * nRanksPerNode;
  const int startRankIndexInPeerNode = (peerRank / nRanksPerNode) * nRanksPerNode;

  if (peerNodeId == rank / nRanksPerNode) {
    localAllGatherSm(rank, nRanksPerNode, 0, 0, rankChunkSize, rankChunkSize, gridDim.x);
    return;
  }

  constexpr size_t alignment = 128;
  size_t step1Bytes = (nelemsPerGPU * (pipelineSize - 1)) / pipelineSize * sizeof(int);
  step1Bytes = step1Bytes / alignment * alignment;
  const size_t step2Bytes = nelemsPerGPU * sizeof(int) - step1Bytes;

  // Step 1
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    proxyChan.putWithSignal(rank * nelemsPerGPU * sizeof(int), step1Bytes);
  }
  localAllGatherSm(rank, nRanksPerNode, startRankIndexInLocalNode, 0, rankChunkSize, rankChunkSize,
                   nBlocksForLocalAllGather);
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    proxyChan.wait();
    proxyChan.flush();
  }
  deviceSyncer.sync(gridDim.x);
  // Step 2
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    proxyChan.putWithSignal(rank * nelemsPerGPU * sizeof(int) + step1Bytes, step2Bytes);
  }
  localAllGatherSm(rank, nRanksPerNode, startRankIndexInPeerNode, 0, rankChunkSize, step1Bytes,
                   nBlocksForLocalAllGather);
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    proxyChan.wait();
    proxyChan.flush();
  }
  deviceSyncer.sync(gridDim.x);
  // Step 3
  localAllGatherSm(rank, nRanksPerNode, startRankIndexInPeerNode, step1Bytes, rankChunkSize, step2Bytes,
                   nBlocksForLocalAllGather);
}

__global__ void __launch_bounds__(1024)
    allreduce0(int* buff, int* scratch, int rank, int worldSize, size_t nelems, size_t scratchDataCount) {
  int peerId = blockIdx.x / BLOCKS_PER_PEER;
  int isComm = (threadIdx.x == 0) && (blockIdx.x % BLOCKS_PER_PEER == 0);
  int remoteRank = (peerId < rank) ? peerId : peerId + 1;

  // 1st communication phase: send data to the scratch buffer of the peer associated with this block
  DeviceHandle<mscclpp::SimpleProxyChannel>& devFstRoundChan = constDevFstRoundChans[peerId];
  Chunk toPeerChunk = getChunk(nelems, worldSize, remoteRank);
  // Now we need to figure out the offset of this chunk in the scratch buffer of the destination.
  // The destination will have allocated a scratch buffer of size numPeers() * toPeerChunk.size and
  // inside that each of the destination's peers send to the nth chunk, where n is the index of the
  // source peer from the destination's perspective.
  size_t dstOffset = (rank < remoteRank ? rank : rank - 1) * toPeerChunk.size;
  if (isComm) {
    // Write data to the peer
    devFstRoundChan.putWithSignalAndFlush(dstOffset * sizeof(int), toPeerChunk.offset * sizeof(int),
                                          toPeerChunk.size * sizeof(int));
    // Wait for data from the peer
    devFstRoundChan.wait();
  }

  deviceSyncer.sync(gridDim.x);

  // Local reduction: every block reduces a slice of each chunk in the scratch buffer into the user buffer
  DeviceHandle<mscclpp::SimpleProxyChannel>& devSndRoundChan = constDevSndRoundChans[peerId];
  Chunk rankChunk = getChunk(nelems, worldSize, rank);
  int* chunk = buff + rankChunk.offset;
  int numPeers = gridDim.x / BLOCKS_PER_PEER;
  int numBlocks = gridDim.x;
  Chunk blockUserChunk = getChunk(rankChunk.size, numBlocks, blockIdx.x);
  size_t scratchDataCountPerPeer = scratchDataCount / numPeers;
  Chunk blockScratchChunk = getChunk(scratchDataCountPerPeer, numBlocks, blockIdx.x);
  for (int peerIdx = 0; peerIdx < numPeers; ++peerIdx) {
    int* scratchChunk = scratch + peerIdx * scratchDataCountPerPeer;
    vectorSumSingleBlock(chunk + blockUserChunk.offset, scratchChunk + blockScratchChunk.offset,
                         blockScratchChunk.size);
  }

  deviceSyncer.sync(gridDim.x);

  // 2nd communication phase: send the now reduced data between the user buffers
  Chunk collectionChunk = getChunk(nelems, worldSize, rank);
  if (isComm) {
    // Write data to the peer
    devSndRoundChan.putWithSignalAndFlush(collectionChunk.offset * sizeof(int), collectionChunk.offset * sizeof(int),
                                          collectionChunk.size * sizeof(int));
    // Wait for data from the peer
    devSndRoundChan.wait();
  }
}

__global__ void __launch_bounds__(1024) allreduce1(int* buff, int* scratch, int rank, int worldSize, size_t nelems) {
  int isComm = (threadIdx.x == 0) && (blockIdx.x == 0);
  int remoteSendRank = (rank + 1) % worldSize;
  int remoteRecvRank = (rank + worldSize - 1) % worldSize;
  int peerSendId = (remoteSendRank < rank) ? remoteSendRank : remoteSendRank - 1;
  int peerRecvId = (remoteRecvRank < rank) ? remoteRecvRank : remoteRecvRank - 1;

  DeviceHandle<mscclpp::SimpleProxyChannel>& devFstSendChan = constDevFstRoundChans[peerSendId];
  DeviceHandle<mscclpp::SimpleProxyChannel>& devFstRecvChan = constDevFstRoundChans[peerRecvId];
  DeviceHandle<mscclpp::SimpleProxyChannel>& devSndSendChan = constDevSndRoundChans[peerSendId];
  DeviceHandle<mscclpp::SimpleProxyChannel>& devSndRecvChan = constDevSndRoundChans[peerRecvId];

  // Step 1
  size_t chunkIndex = (rank + worldSize - 1) % worldSize;
  size_t chunkNelem = nelems / worldSize;
  size_t offset = chunkIndex * chunkNelem * sizeof(int);
  if (isComm) {
    if (chunkNelem > 1) {
      devFstSendChan.putWithSignal(offset, chunkNelem / 2 * sizeof(int));
    }
  }

  // Step 2 ~ Step n-1
  for (int step = 2; step < worldSize; ++step) {
    if (isComm) {
      if (chunkNelem > 1) {
        devFstRecvChan.wait();
        devFstSendChan.flush();
      }
      devFstSendChan.putWithSignal(offset + chunkNelem / 2 * sizeof(int), (chunkNelem - chunkNelem / 2) * sizeof(int));
    }
    deviceSyncer.sync(gridDim.x);

    // Reduce
    chunkIndex = (rank + worldSize - step) % worldSize;
    offset = chunkIndex * chunkNelem * sizeof(int);
    int* dst = (int*)((char*)buff + offset);
    int* src = (int*)((char*)scratch + offset);
    vectorSum(dst, src, chunkNelem / 2);

    if (isComm) {
      devFstRecvChan.wait();
      devFstSendChan.flush();
      if (chunkNelem > 1) {
        devFstSendChan.putWithSignal(offset, chunkNelem / 2 * sizeof(int));
      }
    }
    deviceSyncer.sync(gridDim.x);

    dst += chunkNelem / 2;
    src += chunkNelem / 2;
    vectorSum(dst, src, chunkNelem - chunkNelem / 2);
  }

  // Step n
  if (isComm) {
    if (chunkNelem > 1) {
      devFstRecvChan.wait();
      devFstSendChan.flush();
    }
    devFstSendChan.putWithSignal(offset + chunkNelem / 2 * sizeof(int), (chunkNelem - chunkNelem / 2) * sizeof(int));
  }
  deviceSyncer.sync(gridDim.x);

  offset = rank * chunkNelem * sizeof(int);
  int* dst = (int*)((char*)buff + offset);
  int* src = (int*)((char*)scratch + offset);
  vectorSum(dst, src, chunkNelem / 2);

  if (isComm) {
    devFstRecvChan.wait();
    devFstSendChan.flush();
    if (chunkNelem > 1) {
      devSndSendChan.putWithSignal(offset, chunkNelem / 2 * sizeof(int));
    }
  }
  deviceSyncer.sync(gridDim.x);

  dst += chunkNelem / 2;
  src += chunkNelem / 2;
  vectorSum(dst, src, chunkNelem - chunkNelem / 2);

  if (isComm) {
    if (chunkNelem > 1) {
      devSndRecvChan.wait();
      devSndSendChan.flush();
    }
    devSndSendChan.putWithSignalAndFlush(offset + chunkNelem / 2 * sizeof(int),
                                         (chunkNelem - chunkNelem / 2) * sizeof(int));
  }

  // Step n+1 ~ Step 2n-2
  for (int i = 1; i < worldSize - 1; ++i) {
    if (isComm) {
      devSndRecvChan.wait();
    }
    deviceSyncer.sync(gridDim.x);

    // Copy
    chunkIndex = (rank + worldSize - i) % worldSize;
    if (isComm) {
      devSndSendChan.putWithSignalAndFlush(chunkIndex * chunkNelem * sizeof(int), chunkNelem * sizeof(int));
    }
  }

  // Final receive
  if (isComm) {
    devSndRecvChan.wait();
  }
}

__global__ void __launch_bounds__(1024)
    allreduce2(int* buff, void* scratch, void* putPktBuf, void* getPktBuf, void* result, int rank, int nRanksPerNode,
               int worldSize, size_t nelems) {
  int numPeersPerNode = nRanksPerNode - 1;
  size_t nPkts = nelems / 2;  // 2 elems per packet, assume nelems is even
  size_t pktBytes = nPkts * sizeof(mscclpp::LLPacket);

  // Channel to a local peer
  int smChanIdx = blockIdx.x / BLOCKS_PER_PEER;
  DeviceHandle<mscclpp::SmChannel> smChan = constSmOutOfPlaceChans[smChanIdx];

  // Channel to a remote peer that has the same local rank as me
  int localRank = rank % nRanksPerNode;
  DeviceHandle<mscclpp::SimpleProxyChannel> proxyChan = constDevFstRoundChans[localRank];

  // Flag for packets. Initially 1
  uint32_t flag = (uint32_t)globalFlag;

  int2* src = (int2*)buff;
  int2* res = (int2*)result;
  // double buffering
  size_t scratchOffset = (flag & 1) ? 0 : nPkts * max(numPeersPerNode, 1) * sizeof(mscclpp::LLPacket);
  mscclpp::LLPacket* scratchPtr = (mscclpp::LLPacket*)((char*)scratch + scratchOffset);
  size_t pktBufOffset = (flag & 1) ? 0 : nPkts * sizeof(mscclpp::LLPacket);
  mscclpp::LLPacket* getPktPtr = (mscclpp::LLPacket*)((char*)getPktBuf + pktBufOffset);
  mscclpp::LLPacket* putPktPtr = (mscclpp::LLPacket*)((char*)putPktBuf + pktBufOffset);

  // Phase 1: Local AllReduce. Read from buff, write to putPktBuf (for single node) or to result (for 2 nodes)
  if (numPeersPerNode == 0) {
    // One rank per node: write data to putPktBuf directly
    for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nPkts; idx += blockDim.x * gridDim.x) {
      putPktPtr[idx].write(src[idx].x, src[idx].y, flag);
    }
  } else {
    // Offset of the input data (buff) to read from
    size_t srcOffset =
        ((blockIdx.x % BLOCKS_PER_PEER) * nelems * sizeof(int) / BLOCKS_PER_PEER);  // offset for this block
    // Offset of the peer's scratch buffer (scratch) to write on
    size_t dstOffset = (scratchOffset) +                                                   // double buffering
                       ((smChanIdx < localRank ? localRank - 1 : localRank) * pktBytes) +  // offset for this rank
                       (srcOffset * 2);  // offset for this block: twice of srcOffset because 2 elems per packet
    // Write data to the peer's scratch
    smChan.putPackets(dstOffset, srcOffset, nelems / BLOCKS_PER_PEER * sizeof(int), threadIdx.x, blockDim.x, flag);
    // Read data from my scratch, reduce data with my buff, and write the result to my putPktBuf or to result
    const bool isSingleNode = (worldSize == nRanksPerNode);
    for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nPkts; idx += blockDim.x * gridDim.x) {
      int x = 0;
      int y = 0;
      for (int peerIdx = 0; peerIdx < numPeersPerNode / 2; ++peerIdx) {
        mscclpp::LLPacket* pkt0 = scratchPtr + 2 * peerIdx * nPkts;
        mscclpp::LLPacket* pkt1 = scratchPtr + (2 * peerIdx + 1) * nPkts;
        uint2 data0 = pkt0[idx].read(flag);
        uint2 data1 = pkt1[idx].read(flag);
        x += (int)data0.x;
        y += (int)data0.y;
        x += (int)data1.x;
        y += (int)data1.y;
      }
      if (numPeersPerNode & 1) {
        mscclpp::LLPacket* pkt = scratchPtr + (numPeersPerNode - 1) * nPkts;
        uint2 data = pkt[idx].read(flag);
        x += (int)data.x;
        y += (int)data.y;
      }
      if (isSingleNode) {
        res[idx].x = src[idx].x + x;
        res[idx].y = src[idx].y + y;
      } else {
        putPktPtr[idx].write(src[idx].x + x, src[idx].y + y, flag);
      }
    }
  }

  // If this is single node AllReduce, we are done.
  if (worldSize != nRanksPerNode) {
    // Phase 2: Inter-node AllReduce. Supports only 2 nodes. Read from putPktBuf, write to result

    // Wait for all threads to finish writing to putPktBuf in Phase 1
    deviceSyncer.sync(gridDim.x);

    // Phase 2 may need less blocks than Phase 1.
    constexpr int nBlocksPhase2 = 1;
    if (blockIdx.x >= nBlocksPhase2) return;

    // Write my putPktBuf to the remote peer's getPktBuf
    if (threadIdx.x == 0 && blockIdx.x == 0) {
      proxyChan.put(pktBufOffset, pktBytes);
      if ((flag & 63) == 0) {
        proxyChan.flush();
      }
    }

    // Read data from my getPktBuf, reduce data with my putPktBuf, and write the result to result
    for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nPkts; idx += blockDim.x * nBlocksPhase2) {
      uint2 data0 = putPktPtr[idx].read(flag);
      uint2 data1 = getPktPtr[idx].read(flag);
      res[idx].x = (int)data0.x + (int)data1.x;
      res[idx].y = (int)data0.y + (int)data1.y;
    }
  }

  if (threadIdx.x == 0 && blockIdx.x == 0) {
    globalFlag += 1;
  }
}

__global__ void __launch_bounds__(1024)
    allreduce3(int* buff, int* scratch, int rank, int nRanksPerNode, int worldSize, size_t nelems) {
  reduceScatter(buff, scratch, rank, nRanksPerNode, worldSize, nelems);
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    allGather(rank, worldSize, nRanksPerNode, nelems / worldSize);
  }
}

__global__ void __launch_bounds__(1024)
    allreduce4(int* buff, int* scratch, int rank, int nRanksPerNode, int worldSize, size_t nelems) {
  reduceScatterSm(buff, scratch, rank, nRanksPerNode, worldSize, nelems);
  deviceSyncer.sync(gridDim.x);
  allGatherSm(rank, worldSize, nRanksPerNode, nelems / worldSize);
}

__global__ void __launch_bounds__(1024)
    allreduce5(int* buff, int rank, int nRanksPerNode, int worldSize, size_t nelems) {
#if defined(__HIP_PLATFORM_AMD__)
  localReduceScatterSm3(buff, rank, nRanksPerNode, nelems / worldSize, nelems / worldSize, gridDim.x);
  deviceSyncer.sync(gridDim.x);
  localRingAllGatherSm2(rank, nRanksPerNode, nelems / worldSize * sizeof(int), gridDim.x);
#else
  localReduceScatterSm2(buff, rank, nRanksPerNode, nelems / worldSize, nelems / worldSize, gridDim.x);
  deviceSyncer.sync(gridDim.x);
  localRingAllGatherSm(rank, nRanksPerNode, nelems / worldSize * sizeof(int), gridDim.x);
#endif
}

__global__ void __launch_bounds__(1024)
    allreduce6(int* buff, int* scratch, void* resultBuff, int rank, int nRanksPerNode, int worldSize, size_t nelems) {
  // This version of allreduce only works for single nodes
  const int nPeers = nRanksPerNode - 1;
  const size_t nPkts = nelems / 2;
  const int nelemsPerRank = nelems / worldSize;
  const int nPktsPerRank = nelemsPerRank / 2;
  // flag for packets. Initially 1
  const uint32_t flag = (uint32_t)globalFlag;
  // thread block & channel info
  const int nBlocksPerPeer = gridDim.x / nPeers;
  const int localBlockIdx = blockIdx.x % nBlocksPerPeer;
  const int peerIdx = blockIdx.x / nBlocksPerPeer;
  const int remoteRank = peerIdx < rank ? peerIdx : peerIdx + 1;
  const int tid = threadIdx.x + localBlockIdx * blockDim.x;
  // double buffering
  size_t scratchBaseOffset = (flag & 1) ? 0 : nPkts * sizeof(mscclpp::LLPacket);
  void* scratchBuff = (void*)((char*)scratch + scratchBaseOffset);
  size_t scratchOffset = scratchBaseOffset + rank * nPktsPerRank * sizeof(mscclpp::LLPacket);
  size_t scratchResultOffset =
      (flag & 1) ? 2 * nPkts * sizeof(mscclpp::LLPacket) : 3 * nPkts * sizeof(mscclpp::LLPacket);
  size_t srcOffset = remoteRank * nelemsPerRank * sizeof(int);
  uint2* src = (uint2*)((char*)buff + rank * nelemsPerRank * sizeof(int));
  uint2* dst = (uint2*)((char*)resultBuff + rank * nelemsPerRank * sizeof(int));

  // step 1: write to scratch buffer
  constSmOutOfPlaceChans[peerIdx].putPackets(scratchOffset, srcOffset, nelemsPerRank * sizeof(int), tid,
                                             blockDim.x * nBlocksPerPeer, flag);
  // step 2: get data from scratch buffer, reduce data and write result to remote scratch buffer
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nPktsPerRank; idx += blockDim.x * gridDim.x) {
    uint2 data = make_uint2(0, 0);
    for (int index = 0; index < nPeers; index++) {
      const int remoteRank = index < rank ? index : index + 1;
      mscclpp::LLPacket* dstPkt = (mscclpp::LLPacket*)scratchBuff + remoteRank * nPktsPerRank;
      uint2 val = dstPkt[idx].read(flag);
      data.x += val.x;
      data.y += val.y;
    }
    data.x += src[idx].x;
    data.y += src[idx].y;
    dst[idx] = data;

    mscclpp::LLPacket packet;
    packet.data1 = data.x;
    packet.flag1 = flag;
    packet.data2 = data.y;
    packet.flag2 = flag;
    size_t offset = scratchResultOffset / sizeof(mscclpp::LLPacket) + (idx + rank * nPktsPerRank);
    for (int index = 0; index < nPeers; index++) {
      constSmOutOfPlaceChans[index].write(offset, packet);
    }
  }
  // step 3: get data result from scratch buffer
  mscclpp::LLPacket* dstPkt = (mscclpp::LLPacket*)((char*)scratch + scratchResultOffset);
  const int dstOffset = remoteRank * nPktsPerRank;
  uint2* result = (uint2*)((char*)resultBuff + remoteRank * nelemsPerRank * sizeof(int));
  for (int idx = threadIdx.x + localBlockIdx * blockDim.x; idx < nPktsPerRank; idx += blockDim.x * nBlocksPerPeer) {
    uint2 data = dstPkt[idx + dstOffset].read(flag);
    result[idx].x = data.x;
    result[idx].y = data.y;
  }
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    globalFlag += 1;
  }
}

__global__ void __launch_bounds__(1024)
    allreduce7(int* buff, int* scratch, void* resultBuff, int rank, int nRanksPerNode, int worldSize, size_t nelems) {
  // This version of allreduce only works for single nodes
  const int nPeers = nRanksPerNode - 1;
  const size_t nPkts = nelems;
  const int nelemsPerRank = nelems / worldSize;
  const int nPktsPerRank = nelemsPerRank;
  // flag for packets. Initially 1
  const uint32_t flag = (uint32_t)globalFlag;
  // thread block & channel info
  const int nBlocksPerPeer = gridDim.x / nPeers;
  const int localBlockIdx = blockIdx.x % nBlocksPerPeer;
  const int peerIdx = blockIdx.x / nBlocksPerPeer;
  const int remoteRank = peerIdx < rank ? peerIdx : peerIdx + 1;
  const int tid = threadIdx.x + localBlockIdx * blockDim.x;
  // double buffering
  size_t scratchBaseOffset = (flag & 1) ? 0 : nPkts * sizeof(mscclpp::LL8Packet);
  void* scratchBuff = (void*)((char*)scratch + scratchBaseOffset);
  size_t scratchOffset = scratchBaseOffset + rank * nPktsPerRank * sizeof(mscclpp::LL8Packet);
  size_t scratchResultOffset =
      (flag & 1) ? 2 * nPkts * sizeof(mscclpp::LL8Packet) : 3 * nPkts * sizeof(mscclpp::LL8Packet);
  size_t srcOffset = remoteRank * nelemsPerRank * sizeof(int);
  uint32_t* src = (uint32_t*)((char*)buff + rank * nelemsPerRank * sizeof(int));
  uint32_t* dst = (uint32_t*)((char*)resultBuff + rank * nelemsPerRank * sizeof(int));

  // step 1: write to scratch buffer
  constSmOutOfPlaceChans[peerIdx].putPackets<mscclpp::LL8Packet>(scratchOffset, srcOffset, nelemsPerRank * sizeof(int),
                                                                 tid, blockDim.x * nBlocksPerPeer, flag);
  // step 2: get data from scratch buffer, reduce data and write result to remote scratch buffer
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nPktsPerRank; idx += blockDim.x * gridDim.x) {
    uint32_t data = 0;
    for (int index = 0; index < nPeers; index++) {
      const int remoteRank = index < rank ? index : index + 1;
      mscclpp::LL8Packet* dstPkt = (mscclpp::LL8Packet*)scratchBuff + remoteRank * nPktsPerRank;
      uint32_t val = dstPkt[idx].read(flag);
      data += val;
    }
    data += src[idx];
    dst[idx] = data;

    mscclpp::LL8Packet packet;
    packet.data = data;
    packet.flag = flag;
    size_t offset = scratchResultOffset / sizeof(mscclpp::LL8Packet) + (idx + rank * nPktsPerRank);
    for (int index = 0; index < nPeers; index++) {
      constSmOutOfPlaceChans[index].write(offset, packet);
    }
  }
  // step 3: get data result from scratch buffer
  mscclpp::LL8Packet* dstPkt = (mscclpp::LL8Packet*)((char*)scratch + scratchResultOffset);
  const int dstOffset = remoteRank * nPktsPerRank;
  uint32_t* result = (uint32_t*)((char*)resultBuff + remoteRank * nelemsPerRank * sizeof(int));
  for (int idx = threadIdx.x + localBlockIdx * blockDim.x; idx < nPktsPerRank; idx += blockDim.x * nBlocksPerPeer) {
    uint32_t data = dstPkt[idx + dstOffset].read(flag);
    result[idx] = data;
  }
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    globalFlag += 1;
  }
}

class AllReduceTestColl : public BaseTestColl {
 public:
  AllReduceTestColl() = default;
  ~AllReduceTestColl() = default;

  void runColl(const TestArgs& args, hipStream_t stream) override;
  void initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) override;
  void getBw(const double deltaSec, double& algBw /*OUT*/, double& busBw /*OUT*/) override;
  void setupCollTest(size_t size) override;
  std::vector<KernelRestriction> getKernelRestrictions() override;
};

void AllReduceTestColl::runColl(const TestArgs& args, hipStream_t stream) {
  const int worldSize = args.totalRanks;
  const int rank = args.rank;
  const int kernelNum = args.kernelNum;
  const int nPeers = worldSize - 1;
  const Chunk chunk = getChunk(paramCount_, worldSize, rank);
  const size_t scratchDataCount = chunk.size * nPeers;

  int nBlocks;
  int nThreadsPerBlock;
  void* tmpBuff;
  if (kernelNum == 0) {
    nBlocks = nPeers * BLOCKS_PER_PEER;
    tmpBuff = scratchBuff;
    nThreadsPerBlock = 1024;
  } else if (kernelNum == 1 || kernelNum == 3) {
    nBlocks = 24;
    tmpBuff = scratchBuff;
    nThreadsPerBlock = 1024;
  } else if (kernelNum == 4) {
    nBlocks = 45;
    tmpBuff = scratchBuff;
    nThreadsPerBlock = 512;
  } else if (kernelNum == 5) {
    nBlocks = 24;
    tmpBuff = scratchBuff;
    nThreadsPerBlock = 1024;
  } else if (kernelNum == 6) {
    nBlocks = 21;
    tmpBuff = scratchPacketBuff;
    nThreadsPerBlock = 512;
  } else if (kernelNum == 7) {
    nBlocks = 28;
    tmpBuff = scratchPacketBuff;
    nThreadsPerBlock = 1024;
  } else {
    nBlocks = std::max(args.nRanksPerNode - 1, 1) * BLOCKS_PER_PEER;
    tmpBuff = scratchPacketBuff;
    nThreadsPerBlock = 1024;
  }
  if (kernelNum == 0)
    allreduce0<<<nBlocks, nThreadsPerBlock, 0, stream>>>((int*)inputBuff, (int*)tmpBuff, rank, worldSize, paramCount_,
                                                         scratchDataCount);
  else if (kernelNum == 1)
    allreduce1<<<nBlocks, nThreadsPerBlock, 0, stream>>>((int*)inputBuff, (int*)tmpBuff, rank, worldSize, paramCount_);
  else if (kernelNum == 2)
    allreduce2<<<nBlocks, nThreadsPerBlock, 0, stream>>>((int*)inputBuff, tmpBuff, putPacketBuff, getPacketBuff,
                                                         resultBuff, rank, args.nRanksPerNode, worldSize, paramCount_);
  else if (kernelNum == 3)
    allreduce3<<<nBlocks, nThreadsPerBlock, 0, stream>>>((int*)inputBuff, (int*)tmpBuff, rank, args.nRanksPerNode,
                                                         worldSize, paramCount_);
  else if (kernelNum == 4)
    allreduce4<<<nBlocks, nThreadsPerBlock, 0, stream>>>((int*)inputBuff, (int*)tmpBuff, rank, args.nRanksPerNode,
                                                         worldSize, paramCount_);
  else if (kernelNum == 5)
    allreduce5<<<nBlocks, nThreadsPerBlock, 0, stream>>>((int*)inputBuff, rank, args.nRanksPerNode, worldSize,
                                                         paramCount_);
  else if (kernelNum == 6)
    allreduce6<<<nBlocks, nThreadsPerBlock, 0, stream>>>((int*)inputBuff, (int*)tmpBuff, resultBuff, rank,
                                                         args.nRanksPerNode, worldSize, paramCount_);
  else if (kernelNum == 7)
    allreduce7<<<nBlocks, nThreadsPerBlock, 0, stream>>>((int*)inputBuff, (int*)tmpBuff, resultBuff, rank,
                                                         args.nRanksPerNode, worldSize, paramCount_);
}

void AllReduceTestColl::initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) {
  if (sendBuff.size() != 1) std::runtime_error("unexpected error");
  const int rank = args.rank;
  const int worldSize = args.totalRanks;
  std::vector<int> dataHost(std::max(sendCount_, recvCount_), rank);
  CUDATHROW(hipMemcpy(sendBuff[0], dataHost.data(), sendCount_ * typeSize_, hipMemcpyHostToDevice));

  for (size_t i = 0; i < recvCount_; i++) {
    dataHost[i] = worldSize * (worldSize - 1) / 2;
  }
  std::memcpy(expectedBuff, dataHost.data(), recvCount_ * typeSize_);
}

void AllReduceTestColl::getBw(const double deltaSec, double& algBw /*OUT*/, double& busBw /*OUT*/) {
  double baseBw = (double)(paramCount_ * typeSize_) / 1.0E9 / deltaSec;
  algBw = baseBw;
  double factor = (2 * (double)(worldSize_ - 1)) / ((double)worldSize_);
  busBw = baseBw * factor;
}

void AllReduceTestColl::setupCollTest(size_t size) {
  size_t count = size / typeSize_;
  sendCount_ = count;
  recvCount_ = count;
  paramCount_ = count;
  expectedCount_ = count;

  mscclpp::DeviceSyncer syncer = {};
  uint64_t initFlag = 1;
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(deviceSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(allGatherDeviceSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(reduceScatterDeviceSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(ibDeviceSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(globalFlag), &initFlag, sizeof(uint64_t)));
}

std::vector<KernelRestriction> AllReduceTestColl::getKernelRestrictions() {
  return {// {kernelNum, kernelName, compatibleWithMultiNodes, countDivisorForMultiNodes, alignedBytes}
          {0, "allreduce0", true, 1, 4 * worldSize_},
          {1, "allreduce1", true, 1, 4 * worldSize_},
          {2, "allreduce2", true, 1, 4 * worldSize_},
          {3, "allreduce3", true, 3, 4 * worldSize_},
          {
              4,
              "allreduce4",
              true,
              3,
              16 * worldSize_ /*use ulong2 to transfer data*/,
          },
          {5, "allreduce5", false, 1, 4 * worldSize_},
          {6, "allreduce6", false, 1, 4 * worldSize_},
          {7, "allreduce7", false, 1, 4 * worldSize_}};
}

class AllReduceTestEngine : public BaseTestEngine {
 public:
  AllReduceTestEngine(const TestArgs& args);
  ~AllReduceTestEngine() = default;

  void allocateBuffer() override;
  void setupConnections() override;

  bool isUsePacket() const;
  bool isInPlace() const;

  std::vector<void*> getSendBuff() override;
  void* getRecvBuff() override;
  void* getScratchBuff() override;

 private:
  void* getExpectedBuff() override;

  std::shared_ptr<int> inputBuff_;
  std::shared_ptr<int> scratchBuff_;
  std::shared_ptr<int> resultBuff_;
  std::shared_ptr<mscclpp::LLPacket> scratchPacketBuff_;
  std::shared_ptr<mscclpp::LLPacket> putPacketBuff_;
  std::shared_ptr<mscclpp::LLPacket> getPacketBuff_;
  std::shared_ptr<int[]> expectedBuff_;
  std::vector<mscclpp::SmChannel> smOutOfPlaceChannels_;
  std::vector<mscclpp::SmChannel> smInPlaceChannels_;
  std::vector<mscclpp::SmChannel> smOutOfPlaceGetChannels_;
};

AllReduceTestEngine::AllReduceTestEngine(const TestArgs& args) : BaseTestEngine(args, "allreduce") {
  inPlace_ = isInPlace();
}

bool AllReduceTestEngine::isUsePacket() const {
  return (args_.kernelNum == 2 || args_.kernelNum == 6 || args_.kernelNum == 7);
}

bool AllReduceTestEngine::isInPlace() const {
  return (args_.kernelNum != 2 && args_.kernelNum != 6 && args_.kernelNum != 7);
}

void AllReduceTestEngine::allocateBuffer() {
  inputBuff_ = mscclpp::allocExtSharedCuda<int>(args_.maxBytes / sizeof(int));
  resultBuff_ = mscclpp::allocExtSharedCuda<int>(args_.maxBytes / sizeof(int));
  inputBuff = inputBuff_.get();
  resultBuff = resultBuff_.get();

  if (args_.kernelNum == 0 || args_.kernelNum == 1 || args_.kernelNum == 3 || args_.kernelNum == 4) {
    scratchBuff_ = mscclpp::allocExtSharedCuda<int>(args_.maxBytes / sizeof(int));
    scratchBuff = scratchBuff_.get();
  } else if (args_.kernelNum == 2) {
    const size_t nPacket = (args_.maxBytes + sizeof(uint64_t) - 1) / sizeof(uint64_t);
    // 2x for double-buffering
    const size_t scratchBuffNelem = nPacket * std::max(args_.nRanksPerNode - 1, 1) * 2;
    scratchPacketBuff_ = mscclpp::allocExtSharedCuda<mscclpp::LLPacket>(scratchBuffNelem);
    scratchPacketBuff = scratchPacketBuff_.get();
    const size_t packetBuffNelem = nPacket * 2;
    putPacketBuff_ = mscclpp::allocExtSharedCuda<mscclpp::LLPacket>(packetBuffNelem);
    getPacketBuff_ = mscclpp::allocExtSharedCuda<mscclpp::LLPacket>(packetBuffNelem);
    putPacketBuff = putPacketBuff_.get();
    getPacketBuff = getPacketBuff_.get();
  } else if (args_.kernelNum == 6 || args_.kernelNum == 7) {
    const size_t nPacket = (args_.maxBytes + sizeof(uint64_t) - 1) / sizeof(uint64_t);
    // 2x for double-buffering, scratchBuff used to store original data and reduced results
    const size_t scratchBuffNelem = nPacket * 2 /*original data & reduced result */ * 2 /* double buffering*/;
    scratchPacketBuff_ = mscclpp::allocExtSharedCuda<mscclpp::LLPacket>(scratchBuffNelem);
    scratchPacketBuff = scratchPacketBuff_.get();
  }

  expectedBuff_ = std::shared_ptr<int[]>(new int[args_.maxBytes / sizeof(int)]);
}

void AllReduceTestEngine::setupConnections() {
  auto getChannelDeviceHandle = [](const std::vector<mscclpp::SmChannel>& in,
                                   std::vector<DeviceHandle<mscclpp::SmChannel>>& out) {
    return std::transform(in.begin(), in.end(), out.begin(),
                          [](const mscclpp::SmChannel& smChannel) { return mscclpp::deviceHandle(smChannel); });
  };
  if (isUsePacket()) {
    std::vector<DeviceHandle<mscclpp::SimpleProxyChannel>> proxyChannels;

    const size_t nPacket = (args_.maxBytes + sizeof(uint64_t) - 1) / sizeof(uint64_t);
    if (args_.kernelNum == 6 || args_.kernelNum == 7) {
      const size_t scratchPacketBuffBytes = nPacket * 2 * 2 * sizeof(mscclpp::LLPacket);
      setupMeshConnections(smOutOfPlaceChannels_, inputBuff_.get(), args_.maxBytes, scratchPacketBuff_.get(),
                           scratchPacketBuffBytes);
      std::vector<DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles(smOutOfPlaceChannels_.size());
      getChannelDeviceHandle(smOutOfPlaceChannels_, smChannelDeviceHandles);
      CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constSmOutOfPlaceChans), smChannelDeviceHandles.data(),
                                   sizeof(DeviceHandle<mscclpp::SmChannel>) * smChannelDeviceHandles.size()));
    }
    if (args_.kernelNum == 2) {
      const size_t scratchPacketBuffBytes =
          nPacket * std::max(args_.nRanksPerNode - 1, 1) * 2 * sizeof(mscclpp::LLPacket);
      const size_t packetBuffBytes = nPacket * 2 * sizeof(mscclpp::LLPacket);
      setupMeshConnections(smOutOfPlaceChannels_, proxyChannels, inputBuff_.get(), args_.maxBytes, putPacketBuff_.get(),
                           packetBuffBytes, getPacketBuff_.get(), packetBuffBytes, scratchPacketBuff_.get(),
                           scratchPacketBuffBytes);

      if (smOutOfPlaceChannels_.size() > sizeof(constSmOutOfPlaceChans) / sizeof(DeviceHandle<mscclpp::SmChannel>)) {
        std::runtime_error("unexpected error");
      }
      if (proxyChannels.size() > sizeof(constDevFstRoundChans) / sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>)) {
        std::runtime_error("unexpected error");
      }

      std::vector<DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles(smOutOfPlaceChannels_.size());
      getChannelDeviceHandle(smOutOfPlaceChannels_, smChannelDeviceHandles);
      CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constSmOutOfPlaceChans), smChannelDeviceHandles.data(),
                                   sizeof(DeviceHandle<mscclpp::SmChannel>) * smChannelDeviceHandles.size()));
      CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constDevFstRoundChans), proxyChannels.data(),
                                   sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>) * proxyChannels.size()));
    }
  } else {
    std::vector<DeviceHandle<mscclpp::SimpleProxyChannel>> fstRoundChannels;
    std::vector<DeviceHandle<mscclpp::SimpleProxyChannel>> sndRoundChannels;

    // Send data from local inputBuff to remote scratchBuff (out-of-place)
    setupMeshConnections(fstRoundChannels, inputBuff_.get(), args_.maxBytes, scratchBuff_.get(), args_.maxBytes);
    if (fstRoundChannels.size() > sizeof(constDevFstRoundChans) / sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>)) {
      std::runtime_error("unexpected error");
    }
    CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constDevFstRoundChans), fstRoundChannels.data(),
                                 sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>) * fstRoundChannels.size()));

    // Send data from local inputBuff to remote inputBuff (in-place)
    setupMeshConnections(sndRoundChannels, inputBuff_.get(), args_.maxBytes);
    if (sndRoundChannels.size() > sizeof(constDevSndRoundChans) / sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>)) {
      std::runtime_error("unexpected error");
    }
    CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constDevSndRoundChans), sndRoundChannels.data(),
                                 sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>) * sndRoundChannels.size()));

    setupMeshConnections(smOutOfPlaceChannels_, inputBuff_.get(), args_.maxBytes, scratchBuff_.get(), args_.maxBytes);
    if (smOutOfPlaceChannels_.size() > sizeof(constSmOutOfPlaceChans) / sizeof(DeviceHandle<mscclpp::SmChannel>)) {
      std::runtime_error("unexpected error");
    }
    std::vector<DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles(smOutOfPlaceChannels_.size());
    getChannelDeviceHandle(smOutOfPlaceChannels_, smChannelDeviceHandles);
    CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constSmOutOfPlaceChans), smChannelDeviceHandles.data(),
                                 sizeof(DeviceHandle<mscclpp::SmChannel>) * smChannelDeviceHandles.size()));

    setupMeshConnections(smInPlaceChannels_, inputBuff_.get(), args_.maxBytes);
    if (smInPlaceChannels_.size() > sizeof(constSmInPlaceChans) / sizeof(DeviceHandle<mscclpp::SmChannel>)) {
      std::runtime_error("unexpected error");
    }
    smChannelDeviceHandles.resize(smInPlaceChannels_.size());
    getChannelDeviceHandle(smInPlaceChannels_, smChannelDeviceHandles);
    CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constSmInPlaceChans), smChannelDeviceHandles.data(),
                                 sizeof(DeviceHandle<mscclpp::SmChannel>) * smChannelDeviceHandles.size()));

    setupMeshConnections(smOutOfPlaceGetChannels_, inputBuff_.get(), args_.maxBytes, scratchBuff_.get(), args_.maxBytes,
                         ChannelSemantic::GET);
    if (smOutOfPlaceGetChannels_.size() >
        sizeof(constSmOutOfPlaceGetChans) / sizeof(DeviceHandle<mscclpp::SmChannel>)) {
      std::runtime_error("unexpected error");
    }
    smChannelDeviceHandles.resize(smOutOfPlaceGetChannels_.size());
    getChannelDeviceHandle(smOutOfPlaceGetChannels_, smChannelDeviceHandles);
    CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constSmOutOfPlaceGetChans), smChannelDeviceHandles.data(),
                                 sizeof(DeviceHandle<mscclpp::SmChannel>) * smChannelDeviceHandles.size()));
  }
}

std::vector<void*> AllReduceTestEngine::getSendBuff() { return {inputBuff_.get()}; }

void* AllReduceTestEngine::getExpectedBuff() { return expectedBuff_.get(); }

void* AllReduceTestEngine::getRecvBuff() { return isInPlace() ? inputBuff_.get() : resultBuff_.get(); }

void* AllReduceTestEngine::getScratchBuff() { return scratchBuff_.get(); }

std::shared_ptr<BaseTestEngine> getTestEngine(const TestArgs& args) {
  return std::make_shared<AllReduceTestEngine>(args);
}

std::shared_ptr<BaseTestColl> getTestColl() { return std::make_shared<AllReduceTestColl>(); }
