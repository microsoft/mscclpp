#include "hip/hip_runtime.h"
#include <vector>

#include "common.hpp"

#define ALIGN 4

class AllReduceTestColl : public BaseTestColl {
 public:
  AllReduceTestColl() = default;
  ~AllReduceTestColl() = default;

  void runColl(const TestArgs& args, hipStream_t stream) override;
  void initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) override;
  void getBw(const double deltaSec, double& algBw /*OUT*/, double& busBw /*OUT*/) override;
  void setupCollTest(size_t size) override;
};

void AllReduceTestColl::runColl(const TestArgs& args, hipStream_t stream) {
}

void AllReduceTestColl::initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) {
  assert(sendBuff.size() == 1);
  const int rank = args.rank;
  const int worldSize = args.totalRanks;
  std::vector<int> dataHost(std::max(sendCount_, recvCount_), rank);
  CUDATHROW(hipMemcpy(sendBuff[0], dataHost.data(), sendCount_ * typeSize_, hipMemcpyHostToDevice));

  for (size_t i = 0; i < recvCount_; i++) {
    dataHost[i] = worldSize * (worldSize - 1) / 2;
  }
  std::memcpy(expectedBuff, dataHost.data(), recvCount_ * typeSize_);
}

void AllReduceTestColl::getBw(const double deltaSec, double& algBw /*OUT*/, double& busBw /*OUT*/)
{
  double baseBw = (double)(paramCount_ * typeSize_) / 1.0E9 / deltaSec;
  algBw = baseBw;
  double factor = (2 * (double)(worldSize_ - 1)) / ((double)worldSize_);
  busBw = baseBw * factor;
}

void AllReduceTestColl::setupCollTest(size_t size) {
  size_t count = size / typeSize_;
  size_t base = (count / ALIGN) * ALIGN;
  sendCount_ = base;
  recvCount_ = base;
  paramCount_ = base;
  recvCount_ = base;
}

class AllReduceEngine : public BaseTestEngine {
 public:
  AllReduceEngine() = default;
  ~AllReduceEngine() = default;

  void allocateBuffer() override;
  void setupConnections() override;

 private:
  std::vector<void*> getSendBuff() override;
  void* getExpectedBuff() override;
  void* getRecvBuff() override;

  std::shared_ptr<int> sendBuff_;
  std::shared_ptr<int> scratchBuff_;
  std::shared_ptr<int[]> expectedBuff_;
};

void AllReduceEngine::allocateBuffer() {
  sendBuff_ = mscclpp::makeSharedCuda<int>(args_.maxBytes / sizeof(int));
  scratchBuff_ = mscclpp::makeSharedCuda<int>(args_.maxBytes / sizeof(int));
  expectedBuff_ = std::shared_ptr<int[]>(new int[args_.maxBytes / sizeof(int)]);
}

void AllReduceEngine::setupConnections()
{
  const int rank = args_.rank;
  const int worldSize = args_.totalRanks;
  const int sendToRank = (args_.rank + 1) % worldSize;
  const int recvFromRank = (args_.rank - 1 + worldSize) % worldSize;

  std::vector<mscclpp::channel::SimpleDeviceChannel> fstRoundChannels;
  std::vector<mscclpp::channel::SimpleDeviceChannel> sndRoundChannels;
}
