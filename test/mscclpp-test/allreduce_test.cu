#include "hip/hip_runtime.h"
#include <cassert>
#include <mscclpp/concurrency.hpp>
#include <vector>

#include "common.hpp"

#define ALIGN 4
#define BLOCKS_PER_PEER 15

__constant__ mscclpp::channel::SimpleDeviceChannel constDevFstRoundChans[16];
__constant__ mscclpp::channel::SimpleDeviceChannel constDevSndRoundChans[16];

struct Chunk {
  size_t offset;
  size_t size;
};

__host__ __device__ Chunk getChunk(size_t dataCount, size_t numChunks, size_t chunkIdx) {
  size_t remainder = dataCount % numChunks;
  size_t smallChunkSize = dataCount / numChunks;
  size_t largeChunkSize = smallChunkSize + 1;
  size_t numRemainedLargeChunks = chunkIdx < remainder ? remainder - chunkIdx : 0;
  size_t offset = (remainder - numRemainedLargeChunks) * largeChunkSize +
                  (chunkIdx > remainder ? chunkIdx - remainder : 0) * smallChunkSize;
  return Chunk{offset, chunkIdx < remainder ? largeChunkSize : smallChunkSize};
}

__device__ void reduceSum(int* dst, int* src, size_t size) {
  for (int i = threadIdx.x; i < size; i += blockDim.x) {
    dst[i] += src[i];
  }
}

__device__ mscclpp::DeviceSyncer deviceSyncer;

__device__ void allreduce0(int rank, int worldSize, size_t nelems, size_t scratchDataCount) {
  int peerId = blockIdx.x / BLOCKS_PER_PEER;
  int isComm = (threadIdx.x == 0) && (blockIdx.x % BLOCKS_PER_PEER == 0);
  int remoteRank = (peerId < rank) ? peerId : peerId + 1;

  // 1st communication phase: send data to the scratch buffer of the peer associated with this block
  mscclpp::channel::SimpleDeviceChannel& devFstRoundChan = constDevFstRoundChans[peerId];
  Chunk toPeerChunk = getChunk(nelems, worldSize, remoteRank);
  // Now we need to figure out the offset of this chunk in the scratch buffer of the destination.
  // The destination will have allocated a scratch buffer of size numPeers() * toPeerChunk.size and
  // inside that each of the destination's peers send to the nth chunk, where n is the index of the
  // source peer from the destination's perspective.
  size_t dstOffset = (rank < remoteRank ? rank : rank - 1) * toPeerChunk.size;
  if (isComm) {
    // Write data to the peer
    devFstRoundChan.putWithSignalAndFlush(dstOffset * sizeof(int), toPeerChunk.offset * sizeof(int),
                                          toPeerChunk.size * sizeof(int));
    // Wait for data from the peer
    devFstRoundChan.wait();
  }

  deviceSyncer.sync(gridDim.x);

  // Local reduction: every block reduces a slice of each chunk in the scratch buffer into the user buffer
  mscclpp::channel::SimpleDeviceChannel& devSndRoundChan = constDevSndRoundChans[peerId];
  Chunk rankChunk = getChunk(nelems, worldSize, rank);
  int* chunk = (int*)devSndRoundChan.srcPtr_ + rankChunk.offset;
  int numPeers = gridDim.x / BLOCKS_PER_PEER;
  int numBlocks = gridDim.x;
  Chunk blockUserChunk = getChunk(rankChunk.size, numBlocks, blockIdx.x);
  size_t scratchDataCountPerPeer = scratchDataCount / numPeers;
  Chunk blockScratchChunk = getChunk(scratchDataCountPerPeer, numBlocks, blockIdx.x);
  for (int peerIdx = 0; peerIdx < numPeers; ++peerIdx) {
    int* scratchChunk = (int*)devFstRoundChan.tmpPtr_ + peerIdx * scratchDataCountPerPeer;
    reduceSum(chunk + blockUserChunk.offset, scratchChunk + blockScratchChunk.offset, blockScratchChunk.size);
  }

  deviceSyncer.sync(gridDim.x);

  // 2nd communication phase: send the now reduced data between the user buffers
  Chunk collectionChunk = getChunk(nelems, worldSize, rank);
  if (isComm) {
    // Write data to the peer
    devSndRoundChan.putWithSignalAndFlush(collectionChunk.offset * sizeof(int), collectionChunk.offset * sizeof(int),
                                          collectionChunk.size * sizeof(int));
    // Wait for data from the peer
    devSndRoundChan.wait();
  }
}

__forceinline__ __device__ void vectorSum(int* dst, int* src, size_t nElem) {
  size_t nInt4 = nElem / 4;
  size_t nLastInts = nElem % 4;
  int4* dst4 = (int4*)dst;
  int4* src4 = (int4*)src;
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < nInt4; i += blockDim.x * gridDim.x) {
    dst4[i].w += src4[i].w;
    dst4[i].x += src4[i].x;
    dst4[i].y += src4[i].y;
    dst4[i].z += src4[i].z;
  }
  if (nLastInts > 0) {
    int* dstLast = dst + nInt4 * 4;
    int* srcLast = src + nInt4 * 4;
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < nLastInts; i += blockDim.x * gridDim.x) {
      dstLast[i] += srcLast[i];
    }
  }
}

__device__ void allreduce1(int rank, int worldSize, size_t nelems, size_t scratchDataCount) {
  int isComm = (threadIdx.x == 0) && (blockIdx.x == 0);
  int remoteSendRank = (rank + 1) % worldSize;
  int remoteRecvRank = (rank + worldSize - 1) % worldSize;
  int peerSendId = (remoteSendRank < rank) ? remoteSendRank : remoteSendRank - 1;
  int peerRecvId = (remoteRecvRank < rank) ? remoteRecvRank : remoteRecvRank - 1;

  mscclpp::channel::SimpleDeviceChannel& devFstSendChan = constDevFstRoundChans[peerSendId];
  mscclpp::channel::SimpleDeviceChannel& devFstRecvChan = constDevFstRoundChans[peerRecvId];
  mscclpp::channel::SimpleDeviceChannel& devSndSendChan = constDevSndRoundChans[peerSendId];
  mscclpp::channel::SimpleDeviceChannel& devSndRecvChan = constDevSndRoundChans[peerRecvId];

  // Step 1
  size_t chunkIndex = (rank + worldSize - 1) % worldSize;
  size_t chunkNelem = nelems / worldSize;
  size_t offset = chunkIndex * chunkNelem * sizeof(int);
  if (isComm) {
    if (chunkNelem > 1) {
      devFstSendChan.putWithSignal(offset, chunkNelem / 2 * sizeof(int));
    }
  }

  // Step 2 ~ Step n-1
  for (int step = 2; step < worldSize; ++step) {
    if (isComm) {
      if (chunkNelem > 1) {
        devFstRecvChan.wait();
        devFstSendChan.flush();
      }
      devFstSendChan.putWithSignal(offset + chunkNelem / 2 * sizeof(int), (chunkNelem - chunkNelem / 2) * sizeof(int));
    }
    deviceSyncer.sync(gridDim.x);

    // Reduce
    chunkIndex = (rank + worldSize - step) % worldSize;
    offset = chunkIndex * chunkNelem * sizeof(int);
    int* dst = (int*)((char*)devFstSendChan.srcPtr_ + offset);
    int* src = (int*)((char*)devFstRecvChan.tmpPtr_ + offset);
    vectorSum(dst, src, chunkNelem / 2);

    if (isComm) {
      devFstRecvChan.wait();
      devFstSendChan.flush();
      if (chunkNelem > 1) {
        devFstSendChan.putWithSignal(offset, chunkNelem / 2 * sizeof(int));
      }
    }
    deviceSyncer.sync(gridDim.x);

    dst += chunkNelem / 2;
    src += chunkNelem / 2;
    vectorSum(dst, src, chunkNelem - chunkNelem / 2);
  }

  // Step n
  if (isComm) {
    if (chunkNelem > 1) {
      devFstRecvChan.wait();
      devFstSendChan.flush();
    }
    devFstSendChan.putWithSignal(offset + chunkNelem / 2 * sizeof(int), (chunkNelem - chunkNelem / 2) * sizeof(int));
  }
  deviceSyncer.sync(gridDim.x);

  offset = rank * chunkNelem * sizeof(int);
  int* dst = (int*)((char*)devFstSendChan.srcPtr_ + offset);
  int* src = (int*)((char*)devFstRecvChan.tmpPtr_ + offset);
  vectorSum(dst, src, chunkNelem / 2);

  if (isComm) {
    devFstRecvChan.wait();
    devFstSendChan.flush();
    if (chunkNelem > 1) {
      devSndSendChan.putWithSignal(offset, chunkNelem / 2 * sizeof(int));
    }
  }
  deviceSyncer.sync(gridDim.x);

  dst += chunkNelem / 2;
  src += chunkNelem / 2;
  vectorSum(dst, src, chunkNelem - chunkNelem / 2);

  if (isComm) {
    if (chunkNelem > 1) {
      devSndSendChan.flush();
    }
    devSndSendChan.putWithSignalAndFlush(offset + chunkNelem / 2 * sizeof(int),
                                         (chunkNelem - chunkNelem / 2) * sizeof(int));
  }

  // Step n+1 ~ Step 2n-2
  for (int i = 1; i < worldSize - 1; ++i) {
    if (isComm) {
      devSndRecvChan.wait();
    }
    deviceSyncer.sync(gridDim.x);

    // Copy
    chunkIndex = (rank + worldSize - i) % worldSize;
    if (isComm) {
      devSndSendChan.putWithSignalAndFlush(chunkIndex * chunkNelem * sizeof(int), chunkNelem * sizeof(int));
    }
  }

  // Final receive
  if (isComm) {
    devSndRecvChan.wait();
  }
}

__global__ void kernel(int rank, int worldSize, size_t nelems, size_t scratchDataCount, int kernel) {
  if (kernel == 0)
    allreduce0(rank, worldSize, nelems, scratchDataCount);
  else if (kernel == 1)
    allreduce1(rank, worldSize, nelems, scratchDataCount);
}

class AllReduceTestColl : public BaseTestColl {
 public:
  AllReduceTestColl() = default;
  ~AllReduceTestColl() = default;

  void runColl(const TestArgs& args, hipStream_t stream) override;
  void initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) override;
  void getBw(const double deltaSec, double& algBw /*OUT*/, double& busBw /*OUT*/) override;
  void setupCollTest(size_t size) override;
};

void AllReduceTestColl::runColl(const TestArgs& args, hipStream_t stream) {
  const int worldSize = args.totalRanks;
  const int rank = args.rank;
  const int kernelNum = args.kernelNum;
  const int nPeers = worldSize - 1;
  const Chunk chunk = getChunk(paramCount_, worldSize, rank);
  const size_t scratchDataCount = chunk.size * nPeers;
  const int nBlocks = (kernelNum == 0) ? nPeers * BLOCKS_PER_PEER : 24;
  kernel<<<nBlocks, 1024, 0, stream>>>(rank, worldSize, paramCount_, scratchDataCount, kernelNum);
}

void AllReduceTestColl::initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) {
  assert(sendBuff.size() == 1);
  const int rank = args.rank;
  const int worldSize = args.totalRanks;
  std::vector<int> dataHost(std::max(sendCount_, recvCount_), rank);
  CUDATHROW(hipMemcpy(sendBuff[0], dataHost.data(), sendCount_ * typeSize_, hipMemcpyHostToDevice));

  for (size_t i = 0; i < recvCount_; i++) {
    dataHost[i] = worldSize * (worldSize - 1) / 2;
  }
  std::memcpy(expectedBuff, dataHost.data(), recvCount_ * typeSize_);
}

void AllReduceTestColl::getBw(const double deltaSec, double& algBw /*OUT*/, double& busBw /*OUT*/) {
  double baseBw = (double)(paramCount_ * typeSize_) / 1.0E9 / deltaSec;
  algBw = baseBw;
  double factor = (2 * (double)(worldSize_ - 1)) / ((double)worldSize_);
  busBw = baseBw * factor;
}

void AllReduceTestColl::setupCollTest(size_t size) {
  size_t count = size / typeSize_;
  size_t base = (count / ALIGN) * ALIGN;
  sendCount_ = base;
  recvCount_ = base;
  paramCount_ = base;
  recvCount_ = base;

  mscclpp::DeviceSyncer syncer = {};
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(deviceSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));
}

class AllReduceTestEngine : public BaseTestEngine {
 public:
  AllReduceTestEngine() = default;
  ~AllReduceTestEngine() = default;

  void allocateBuffer() override;
  void setupConnections() override;

 private:
  std::vector<void*> getSendBuff() override;
  void* getExpectedBuff() override;
  void* getRecvBuff() override;

  std::shared_ptr<int> sendBuff_;
  std::shared_ptr<int> scratchBuff_;
  std::shared_ptr<int[]> expectedBuff_;
};

void AllReduceTestEngine::allocateBuffer() {
  sendBuff_ = mscclpp::allocSharedCuda<int>(args_.maxBytes / sizeof(int));
  scratchBuff_ = mscclpp::allocSharedCuda<int>(args_.maxBytes / sizeof(int));
  expectedBuff_ = std::shared_ptr<int[]>(new int[args_.maxBytes / sizeof(int)]);
}

void AllReduceTestEngine::setupConnections() {
  std::vector<mscclpp::channel::SimpleDeviceChannel> fstRoundChannels;
  std::vector<mscclpp::channel::SimpleDeviceChannel> sndRoundChannels;

  // Send data from local sendBuff to remote scratchBuff (out-of-place)
  setupMeshConnections(fstRoundChannels, sendBuff_.get(), args_.maxBytes, scratchBuff_.get(), args_.maxBytes);
  assert(fstRoundChannels.size() < sizeof(constDevFstRoundChans) / sizeof(mscclpp::channel::SimpleDeviceChannel));
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constDevFstRoundChans), fstRoundChannels.data(),
                               sizeof(mscclpp::channel::SimpleDeviceChannel) * fstRoundChannels.size()));

  // Send data from local sendBuff to remote sendBuff (in-place)
  setupMeshConnections(sndRoundChannels, sendBuff_.get(), args_.maxBytes);
  assert(sndRoundChannels.size() < sizeof(constDevSndRoundChans) / sizeof(mscclpp::channel::SimpleDeviceChannel));
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constDevSndRoundChans), sndRoundChannels.data(),
                               sizeof(mscclpp::channel::SimpleDeviceChannel) * sndRoundChannels.size()));
}

std::vector<void*> AllReduceTestEngine::getSendBuff() { return {sendBuff_.get()}; }

void* AllReduceTestEngine::getExpectedBuff() { return expectedBuff_.get(); }

void* AllReduceTestEngine::getRecvBuff() {
  // in-place operation reuse the send buffer
  return sendBuff_.get();
}

std::shared_ptr<BaseTestEngine> getTestEngine() { return std::make_shared<AllReduceTestEngine>(); }
std::shared_ptr<BaseTestColl> getTestColl() { return std::make_shared<AllReduceTestColl>(); }
