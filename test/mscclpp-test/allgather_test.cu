#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cassert>
#include <string>

#include "common.hpp"

#define ALIGN 4

namespace {
auto isUsingHostOffload = [](int kernelNum) { return kernelNum == 3; };
constexpr uint64_t MAGIC = 0xdeadbeef;
}  // namespace

__constant__ mscclpp::channel::SimpleDeviceChannel constDevChans[16];
__constant__ mscclpp::channel::DeviceChannel constRawDevChan[16];

__device__ void allgather0(mscclpp::channel::SimpleDeviceChannel devChan, int rank, int worldSize, int remoteRank,
                           size_t nelemsPerGPU) {
  // this allgather is really simple and implemented as an alltoall

  // this thread's role is a sender role
  // put your data asynchronously
  if (threadIdx.x % 32 == 0) devChan.putWithSignal(rank * nelemsPerGPU * sizeof(int), nelemsPerGPU * sizeof(int));
  // make sure everyone is put their data before some thread randomly blocks everyone else in signal
  __syncthreads();
  // push with flag and sync to make sure the data is received
  if (threadIdx.x % 32 == 0) devChan.flush();

  // this thread's role is a receiver role. wait on the semaphore to make sure the data is ready
  if (threadIdx.x % 32 == 0) devChan.wait();
}

__device__ void localAllGather(mscclpp::channel::SimpleDeviceChannel devChan, int rank, int worldSize,
                               int nranksPerNode, int remoteRank, uint64_t offset, uint64_t size) {
  // this allgather algorithm works as follows:
  // Step 1: GPU rank i sends data to GPU rank (i+1) % nranksPerNode
  // and waits for data from GPU rank (i-1) % nranksPerNode
  // Step 2: GPU rank i sends data to GPU rank (i+2) % nranksPerNode
  // ...
  // This order is much better for DMA engine for NVLinks
  for (int i = 1; i < nranksPerNode; i++) {
    if ((remoteRank % nranksPerNode) == ((rank + i) % nranksPerNode)) {
      // put your data to GPU (rank+i) % nranksPerNode and signal in one call
      if ((threadIdx.x % 32) == 0) devChan.putWithSignalAndFlush(offset, size);
    }
    // wait for the data from GPU (rank-i) % nranksPerNode to arrive
    if ((remoteRank % nranksPerNode) == ((rank - i + nranksPerNode) % nranksPerNode)) {
      if ((threadIdx.x % 32) == 0) devChan.wait();
    }
    asm volatile("bar.sync %0, %1;" ::"r"(11), "r"((nranksPerNode - 1) * 32) : "memory");
  }
}

__device__ void allgather1(mscclpp::channel::SimpleDeviceChannel devChan, int rank, int worldSize, int nranksPerNode,
                           int remoteRank, size_t nelemsPerGPU) {
  localAllGather(devChan, rank, worldSize, nranksPerNode, remoteRank, rank * nelemsPerGPU * sizeof(int),
                 nelemsPerGPU * sizeof(int));
}

__device__ void allgather2(mscclpp::channel::SimpleDeviceChannel devChan, int rank, int worldSize, int nranksPerNode,
                           int remoteRank, size_t nelemsPerGPU) {
  // this allgather is a pipelined and hierarchical one and only works for two nodes
  // it is implemented as follows:
  // Step 1: each node does a local allgather and concurrently,
  // local GPU i exchange (piplineSize-1)/pipelineSize portion of their data with
  // its cross-node neighbor (local GPU i on the other node) via IB
  // Step 2: each node does a local allgather again with the data just received from its
  // cross-node neighbor in step 1, and concurrently, exchange the rest of the data with
  // its cross-node neighbor
  // Step 3: each node does a local allgather for the last time with the rest of the data

  int pipelineSize = 3;

  // Step 1
  // local allgather
  if (remoteRank / nranksPerNode == rank / nranksPerNode) {
    localAllGather(devChan, rank, worldSize, nranksPerNode, remoteRank, rank * nelemsPerGPU * sizeof(int),
                   nelemsPerGPU * sizeof(int));
  }
  // cross-node exchange
  if (remoteRank % nranksPerNode == rank % nranksPerNode) {
    // opposite side
    if ((threadIdx.x % 32) == 0)
      devChan.putWithSignalAndFlush(rank * nelemsPerGPU * sizeof(int),
                                    (nelemsPerGPU * (pipelineSize - 1)) / pipelineSize * sizeof(int));
    if ((threadIdx.x % 32) == 0) devChan.wait();
  }

  __syncthreads();

  // Step 2
  // local allgather
  int otherNghr = (rank + nranksPerNode) % worldSize;
  if (remoteRank / nranksPerNode == rank / nranksPerNode) {
    localAllGather(devChan, rank, worldSize, nranksPerNode, remoteRank, otherNghr * nelemsPerGPU * sizeof(int),
                   (nelemsPerGPU * (pipelineSize - 1)) / pipelineSize * sizeof(int));
  }

  // cross-node exchange
  if (remoteRank % nranksPerNode == rank % nranksPerNode) {
    // opposite side
    if ((threadIdx.x % 32) == 0)
      devChan.putWithSignalAndFlush(
          (rank * nelemsPerGPU + (nelemsPerGPU * (pipelineSize - 1)) / pipelineSize) * sizeof(int),
          nelemsPerGPU / pipelineSize * sizeof(int));
    if ((threadIdx.x % 32) == 0) devChan.wait();
  }

  __syncthreads();

  // Step 3
  // local allgather
  if (remoteRank / nranksPerNode == rank / nranksPerNode) {
    localAllGather(devChan, rank, worldSize, nranksPerNode, remoteRank,
                   (otherNghr * nelemsPerGPU + (nelemsPerGPU * (pipelineSize - 1)) / pipelineSize) * sizeof(int),
                   nelemsPerGPU / pipelineSize * sizeof(int));
  }
}

__device__ void allgather3(mscclpp::channel::DeviceChannel devChan, int rank, int worldSize) {
  int tid = threadIdx.x;
  __syncthreads();
  if (tid == 0) {
    mscclpp::ProxyTrigger trigger;
    trigger.fst = MAGIC;
    // offload all the work to the proxy
    uint64_t currentFifoHead = devChan.fifo_.push(trigger);
    // wait for the work to be done in cpu side
    devChan.fifo_.sync(currentFifoHead);
  }
  if (tid % 32 == 0) {
    devChan.wait();
  }
}

__global__ void kernel(int rank, int worldSize, int nranksPerNode, size_t nelemsPerGPU, int kernel) {
  // find the mapping between remoteRank and devConns
  int warpId = threadIdx.x / 32;
  int remoteRank = (warpId < rank) ? warpId : warpId + 1;
  // Each warp is responsible for one of the remote ranks
  mscclpp::channel::SimpleDeviceChannel devChan = constDevChans[warpId];

  if (kernel == 0)
    allgather0(devChan, rank, worldSize, remoteRank, nelemsPerGPU);
  else if (kernel == 1)
    allgather1(devChan, rank, worldSize, nranksPerNode, remoteRank, nelemsPerGPU);
  else if (kernel == 2)
    allgather2(devChan, rank, worldSize, nranksPerNode, remoteRank, nelemsPerGPU);
  else if (kernel == 3) {
    mscclpp::channel::DeviceChannel devChan = constRawDevChan[warpId];
    allgather3(devChan, rank, worldSize);
  }
}

class AllGatherChannelService : public mscclpp::channel::BaseChannelService {
 public:
  AllGatherChannelService(mscclpp::Communicator& communicator, int worldSize, int rank, int cudaDevice);
  void startProxy() override { proxy_.start(); }
  void stopProxy() override { proxy_.stop(); }
  void setSendBytes(size_t sendBytes) { this->sendBytes_ = sendBytes; }
  void addRemoteMemory(mscclpp::RegisteredMemory memory) { remoteMemories_.push_back(memory); }
  void setLocalMemory(mscclpp::RegisteredMemory memory) { localMemory_ = memory; }
  mscclpp::channel::ChannelId addChannel(std::shared_ptr<mscclpp::Connection> connection) {
    channels_.push_back(mscclpp::channel::Channel(communicator_, connection));
    return channels_.size() - 1;
  }
  std::vector<mscclpp::channel::DeviceChannel> deviceChannels() {
    std::vector<mscclpp::channel::DeviceChannel> result;
    for (auto& channel : channels_) {
      result.push_back(mscclpp::channel::DeviceChannel(0, channel.epoch().deviceHandle(), proxy_.fifo().deviceFifo()));
    }
    return result;
  }

 private:
  int worldSize_;
  int rank_;
  int cudaDevice_;
  size_t sendBytes_;

  mscclpp::Proxy proxy_;
  mscclpp::Communicator& communicator_;
  std::vector<mscclpp::channel::Channel> channels_;
  std::vector<mscclpp::RegisteredMemory> remoteMemories_;
  mscclpp::RegisteredMemory localMemory_;

  mscclpp::ProxyHandlerResult handleTrigger(mscclpp::ProxyTrigger triggerRaw);
};

AllGatherChannelService::AllGatherChannelService(mscclpp::Communicator& communicator, int worldSize, int rank,
                                                 int cudaDevice)
    : communicator_(communicator),
      worldSize_(worldSize),
      sendBytes_(0),
      rank_(rank),
      cudaDevice_(cudaDevice),
      proxy_([&](mscclpp::ProxyTrigger triggerRaw) { return handleTrigger(triggerRaw); },
             [&]() {
               int deviceNumaNode = getDeviceNumaNode(cudaDevice_);
               numaBind(deviceNumaNode);
             }) {}

mscclpp::ProxyHandlerResult AllGatherChannelService::handleTrigger(mscclpp::ProxyTrigger triggerRaw) {
  size_t offset = rank_ * sendBytes_;
  if (triggerRaw.fst != MAGIC) {
    // this is not a valid trigger
    throw std::runtime_error("Invalid trigger");
  }
  for (int r = 0; r < worldSize_; r++) {
    if (r == rank_) {
      continue;
    }
    int index = (r < rank_) ? r : r - 1;
    auto& conn = channels_[index].connection();
    conn.write(remoteMemories_[index], offset, localMemory_, offset, sendBytes_);
    channels_[index].epoch().signal();
  }
  bool flushIpc = false;
  for (auto& chan : channels_) {
    auto& conn = chan.connection();
    if (conn.transport() == mscclpp::Transport::CudaIpc && !flushIpc) {
      // since all the cudaIpc channels are using the same cuda stream, we only need to flush one of them
      conn.flush();
      flushIpc = true;
    }
    if (mscclpp::AllIBTransports.has(conn.transport())) {
      conn.flush();
    }
  }
  return mscclpp::ProxyHandlerResult::FlushFifoTailAndContinue;
}

class AllGatherTestColl : public BaseTestColl {
 public:
  AllGatherTestColl() = default;
  ~AllGatherTestColl() override = default;

  void runColl(const TestArgs& args, hipStream_t stream) override;
  void initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) override;
  void getBw(const double deltaSec, double& algBw /*OUT*/, double& busBw /*OUT*/) override;
  void setupCollTest(size_t size) override;
};

void AllGatherTestColl::runColl(const TestArgs& args, hipStream_t stream) {
  const int worldSize = args.totalRanks;
  const int rank = args.rank;
  const int nRanksPerNode = args.nRanksPerNode;
  const int kernelNum = args.kernelNum;
  kernel<<<1, 32 * (worldSize - 1), 0, stream>>>(rank, worldSize, nRanksPerNode, paramCount_, kernelNum);
}

void AllGatherTestColl::initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) {
  assert(sendBuff.size() == 1);
  int rank = args.rank;
  std::vector<int> dataHost(std::max(sendCount_, recvCount_), 0);
  for (size_t i = 0; i < recvCount_; i++) {
    int val = i + 1;
    if (i / sendCount_ == (size_t)rank) {
      dataHost[i] = val;
    } else {
      dataHost[i] = 0;
    }
  }
  CUDATHROW(hipMemcpy(sendBuff[0], dataHost.data(), recvCount_ * typeSize_, hipMemcpyHostToDevice));

  for (size_t i = 0; i < recvCount_; i++) {
    dataHost[i] = static_cast<int>(i) + 1;
  }
  std::memcpy(expectedBuff, dataHost.data(), recvCount_ * typeSize_);
}

void AllGatherTestColl::getBw(const double deltaSec, double& algBw, double& busBw) {
  double baseBw = (double)(paramCount_ * typeSize_ * worldSize_) / 1.0E9 / deltaSec;

  algBw = baseBw;
  double factor = ((double)(worldSize_ - 1)) / ((double)worldSize_);
  busBw = baseBw * factor;
}

void AllGatherTestColl::setupCollTest(size_t size) {
  size_t count = size / typeSize_;
  size_t base = (count / (ALIGN * worldSize_)) * ALIGN;
  sendCount_ = base;
  recvCount_ = base * worldSize_;
  paramCount_ = base;
  expectedCount_ = recvCount_;
  if (isUsingHostOffload(kernelNum_)) {
    auto service = std::dynamic_pointer_cast<AllGatherChannelService>(chanService_);
    service->setSendBytes(sendCount_ * typeSize_);
  }
}

class AllGatherTestEngine : public BaseTestEngine {
 public:
  AllGatherTestEngine(const TestArgs& args);
  ~AllGatherTestEngine() override = default;

  void allocateBuffer() override;
  void setupConnections() override;

  std::vector<void*> getSendBuff() override;
  void* getRecvBuff() override;
  void* getScratchBuff() override;
  std::shared_ptr<mscclpp::channel::BaseChannelService> createChannelService() override;

 private:
  void* getExpectedBuff() override;

  std::shared_ptr<int> sendBuff_;
  std::shared_ptr<int[]> expectedBuff_;
};

AllGatherTestEngine::AllGatherTestEngine(const TestArgs& args) : BaseTestEngine(args) {}

void AllGatherTestEngine::allocateBuffer() {
  sendBuff_ = mscclpp::allocSharedCuda<int>(args_.maxBytes / sizeof(int));
  expectedBuff_ = std::shared_ptr<int[]>(new int[args_.maxBytes / sizeof(int)]);
}

void AllGatherTestEngine::setupConnections() {
  std::vector<mscclpp::channel::SimpleDeviceChannel> devChannels;
  if (!isUsingHostOffload(args_.kernelNum)) {
    setupMeshConnections(devChannels, sendBuff_.get(), args_.maxBytes);
    assert(devChannels.size() < sizeof(constDevChans) / sizeof(mscclpp::channel::SimpleDeviceChannel));
    CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constDevChans), devChannels.data(),
                                 sizeof(mscclpp::channel::SimpleDeviceChannel) * devChannels.size()));
  } else {
    auto service = std::dynamic_pointer_cast<AllGatherChannelService>(chanService_);
    setupMeshConnections(devChannels, sendBuff_.get(), args_.maxBytes, nullptr, 0,
                         [&](std::vector<std::shared_ptr<mscclpp::Connection>> conns,
                             std::vector<mscclpp::NonblockingFuture<mscclpp::RegisteredMemory>>& remoteMemories,
                             const mscclpp::RegisteredMemory& localMemory) {
                           std::vector<mscclpp::channel::ChannelId> channelIds;
                           for (int i = 0; i < conns.size(); ++i) {
                             service->addChannel(conns[i]);
                             service->addRemoteMemory(remoteMemories[i].get());
                           }
                           service->setLocalMemory(localMemory);
                           comm_->setup();
                         });
    auto devChannels = service->deviceChannels();
    assert(devChannels.size() < sizeof(constRawDevChan) / sizeof(mscclpp::channel::DeviceChannel));
    CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constRawDevChan), devChannels.data(),
                                 sizeof(mscclpp::channel::DeviceChannel) * devChannels.size()));
  }
}

std::shared_ptr<mscclpp::channel::BaseChannelService> AllGatherTestEngine::createChannelService() {
  if (isUsingHostOffload(args_.kernelNum)) {
    return std::make_shared<AllGatherChannelService>(*comm_, args_.totalRanks, args_.rank, args_.gpuNum);
  } else {
    return std::make_shared<mscclpp::channel::DeviceChannelService>(*comm_);
  }
}

std::vector<void*> AllGatherTestEngine::getSendBuff() { return {sendBuff_.get()}; }

void* AllGatherTestEngine::getExpectedBuff() { return expectedBuff_.get(); }

void* AllGatherTestEngine::getRecvBuff() {
  // in-place operation reuse the send buffer
  return sendBuff_.get();
}

void* AllGatherTestEngine::getScratchBuff() { return nullptr; }

std::shared_ptr<BaseTestEngine> getTestEngine(const TestArgs& args) {
  return std::make_shared<AllGatherTestEngine>(args);
}

std::shared_ptr<BaseTestColl> getTestColl() { return std::make_shared<AllGatherTestColl>(); }
