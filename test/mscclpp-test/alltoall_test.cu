#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <cstdint>
#include <mscclpp/concurrency_device.hpp>

#include "common.hpp"

template <class T>
using DeviceHandle = mscclpp::DeviceHandle<T>;
__constant__ DeviceHandle<mscclpp::SimpleProxyChannel> constProxyChans[16];
__device__ mscclpp::DeviceSyncer deviceSyncer;
void* localRecvBuff;
void* localSendBuff;

__device__ void localAlltoall(int rank, int nRanksPerNode, size_t nElements) {
  int remoteRank = ((int)blockIdx.x < rank) ? blockIdx.x : blockIdx.x + 1;
  for (int i = 1; i < nRanksPerNode; i++) {
    DeviceHandle<mscclpp::SimpleProxyChannel> proxyChan = constProxyChans[blockIdx.x];
    if (threadIdx.x == 0 && remoteRank % nRanksPerNode == (rank + i) % nRanksPerNode) {
      proxyChan.putWithSignalAndFlush(rank * nElements * sizeof(int), remoteRank * nElements * sizeof(int),
                                      nElements * sizeof(int));
    }
    // wait for the data from GPU (rank-i) % nranksPerNode to arrive
    if (threadIdx.x == 0 && remoteRank % nRanksPerNode == (rank - i + nRanksPerNode) % nRanksPerNode) {
      proxyChan.wait();
    }
    deviceSyncer.sync(nRanksPerNode - 1);
  }
}

__global__ void __launch_bounds__(1024) alltoall0(int rank, size_t nElements) {
  int remoteRank = ((int)blockIdx.x < rank) ? blockIdx.x : blockIdx.x + 1;
  DeviceHandle<mscclpp::SimpleProxyChannel> proxyChan = constProxyChans[blockIdx.x];
  if (threadIdx.x == 0) {
    proxyChan.putWithSignal(rank * nElements * sizeof(int), remoteRank * nElements * sizeof(int),
                            nElements * sizeof(int));
  }

  deviceSyncer.sync(gridDim.x);
  if (threadIdx.x == 0) {
    proxyChan.flush();
    proxyChan.wait();
  }
}

__global__ void __launch_bounds__(1024) alltoall1(int rank, int nRanksPerNode, size_t nElements) {
  localAlltoall(rank, nRanksPerNode, nElements);
}

class AllToAllTestColl : public BaseTestColl {
 public:
  AllToAllTestColl() = default;
  ~AllToAllTestColl() override = default;

  void runColl(const TestArgs& args, hipStream_t stream) override;
  void initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) override;
  void getBw(const double deltaSec, double& algBw /*OUT*/, double& busBw /*OUT*/) override;
  void setupCollTest(size_t size) override;
  std::vector<KernelRestriction> getKernelRestrictions() override;
};

void AllToAllTestColl::runColl(const TestArgs& args, hipStream_t stream) {
  const int worldSize = args.totalRanks;
  const int rank = args.rank;
  const int kernelNum = args.kernelNum;
  const int nRanksPerNode = args.nRanksPerNode;
  CUDATHROW(hipMemcpyAsync((int*)localRecvBuff + paramCount_ * rank, (int*)localSendBuff + paramCount_ * rank,
                            paramCount_ * sizeof(int), hipMemcpyDeviceToDevice, stream));
  if (kernelNum == 0) {
    alltoall0<<<worldSize - 1, 32, 0, stream>>>(rank, paramCount_);
  } else if (kernelNum == 1) {
    alltoall1<<<worldSize - 1, 32, 0, stream>>>(rank, nRanksPerNode, paramCount_);
  }
}

void AllToAllTestColl::initData(const TestArgs& args, std::vector<void*> sendBuff, void* expectedBuff) {
  if (sendBuff.size() != 1) std::runtime_error("unexpected error");
  const int rank = args.rank;
  std::vector<int> dataHost(recvCount_, 0);
  // For rank 0, the data is 0, 1, 2 ... recvCount_ - 1, for rank 1, the data is recvCount_, recvCount_ + 1, ...
  for (size_t i = 0; i < recvCount_; i++) {
    dataHost[i] = rank * recvCount_ + i;
  }
  CUDATHROW(hipMemcpy(sendBuff[0], dataHost.data(), sendCount_ * typeSize_, hipMemcpyHostToDevice));
  for (size_t i = 0; i < recvCount_ / paramCount_; i++) {
    for (size_t j = 0; j < paramCount_; j++) {
      dataHost[i * paramCount_ + j] = i * recvCount_ + rank * paramCount_ + j;
    }
  }
  std::memcpy(expectedBuff, dataHost.data(), recvCount_ * typeSize_);
}

void AllToAllTestColl::getBw(const double deltaSec, double& algBw, double& busBw) {
  double baseBw = (double)(paramCount_ * typeSize_ * worldSize_) / 1.0E9 / deltaSec;
  algBw = baseBw;
  double factor = ((double)(worldSize_ - 1)) / ((double)worldSize_);
  busBw = baseBw * factor;
}

void AllToAllTestColl::setupCollTest(size_t size) {
  size_t count = size / typeSize_;
  size_t base = count;
  sendCount_ = base;
  recvCount_ = base;
  paramCount_ = base / worldSize_;
  expectedCount_ = base;

  mscclpp::DeviceSyncer syncer = {};
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(deviceSyncer), &syncer, sizeof(mscclpp::DeviceSyncer)));
}

std::vector<KernelRestriction> AllToAllTestColl::getKernelRestrictions() {
  return {// {kernelNum, kernelName, compatibleWithMultiNodes, countDivisorForMultiNodes}
          {0, "alltoall0", true, 1, 4 * worldSize_},
          {1, "alltoall1", false, 1, 4 * worldSize_}};
}

class AllToAllTestEngine : public BaseTestEngine {
 public:
  AllToAllTestEngine(const TestArgs& args);
  ~AllToAllTestEngine() override = default;

  void allocateBuffer() override;
  void setupConnections() override;

  std::vector<void*> getSendBuff() override;
  void* getRecvBuff() override;
  void* getScratchBuff() override;

 private:
  void* getExpectedBuff() override;

  std::shared_ptr<int> sendBuff_;
  std::shared_ptr<int> recvBuff_;
  std::shared_ptr<int[]> expectedBuff_;
};

AllToAllTestEngine::AllToAllTestEngine(const TestArgs& args) : BaseTestEngine(args, "alltoall") { inPlace_ = false; }

void AllToAllTestEngine::allocateBuffer() {
  sendBuff_ = mscclpp::allocExtSharedCuda<int>(args_.maxBytes / sizeof(int));
  recvBuff_ = mscclpp::allocExtSharedCuda<int>(args_.maxBytes / sizeof(int));
  expectedBuff_ = std::shared_ptr<int[]>(new int[args_.maxBytes / sizeof(int)]);

  localSendBuff = sendBuff_.get();
  localRecvBuff = recvBuff_.get();
}

void AllToAllTestEngine::setupConnections() {
  std::vector<DeviceHandle<mscclpp::SimpleProxyChannel>> proxyChannels;
  setupMeshConnections(proxyChannels, sendBuff_.get(), args_.maxBytes, recvBuff_.get(), args_.maxBytes);

  if (proxyChannels.size() > sizeof(constProxyChans) / sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>)) {
    std::runtime_error("unexpected error");
  }
  CUDATHROW(hipMemcpyToSymbol(HIP_SYMBOL(constProxyChans), proxyChannels.data(),
                               sizeof(DeviceHandle<mscclpp::SimpleProxyChannel>) * proxyChannels.size()));
}

std::vector<void*> AllToAllTestEngine::getSendBuff() { return {sendBuff_.get()}; }
void* AllToAllTestEngine::getExpectedBuff() { return expectedBuff_.get(); }
void* AllToAllTestEngine::getRecvBuff() { return recvBuff_.get(); }
void* AllToAllTestEngine::getScratchBuff() { return nullptr; }

std::shared_ptr<BaseTestEngine> getTestEngine(const TestArgs& args) {
  return std::make_shared<AllToAllTestEngine>(args);
}
std::shared_ptr<BaseTestColl> getTestColl() { return std::make_shared<AllToAllTestColl>(); }
