#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <getopt.h>
#include <libgen.h>

#include <chrono>
#include <cstdint>
#include <cstdio>
#include <iostream>
#include <mscclpp/utils.hpp>
#include <string>
#include <type_traits>

#include "common.hpp"

int is_main_proc = 0;

mscclpp::Transport IBs[] = {mscclpp::Transport::IB0, mscclpp::Transport::IB1, mscclpp::Transport::IB2,
                            mscclpp::Transport::IB3, mscclpp::Transport::IB4, mscclpp::Transport::IB5,
                            mscclpp::Transport::IB6, mscclpp::Transport::IB7};

namespace {

// Command line parameter defaults
size_t minBytes = 32 * 1024 * 1024;
size_t maxBytes = 32 * 1024 * 1024;
size_t stepBytes = 1 * 1024 * 1024;
size_t stepFactor = 1;
int datacheck = 1;
int warmup_iters = 10;
int iters = 20;
// Report average iteration time: (0=RANK0,1=AVG,2=MIN,3=MAX)
int average = 1;
int kernel_num = 0;
int cudaGraphLaunches = 15;

double parseSize(const char* value) {
  long long int units;
  double size;
  char size_lit;
  int count = sscanf(value, "%lf %1s", &size, &size_lit);

  switch (count) {
    case 2:
      switch (size_lit) {
        case 'G':
        case 'g':
          units = 1024 * 1024 * 1024;
          break;
        case 'M':
        case 'm':
          units = 1024 * 1024;
          break;
        case 'K':
        case 'k':
          units = 1024;
          break;
        default:
          return -1.0;
      };
      break;
    case 1:
      units = 1;
      break;
    default:
      return -1.0;
  }

  return size * units;
}

double allreduceTime(int worldSize, double value, int average) {
  double accumulator = value;

  if (average != 0) {
    MPI_Op op;
    if (average == 1) {
      op = MPI_SUM;
    } else if (average == 2) {
      op = MPI_MIN;
    } else if (average == 3) {
      op = MPI_MAX;
    } else if (average == 4) {
      op = MPI_SUM;
    }
    MPI_Allreduce(MPI_IN_PLACE, (void*)&accumulator, 1, MPI_DOUBLE, op, MPI_COMM_WORLD);
  }

  if (average == 1) accumulator /= worldSize;
  return accumulator;
}
}  // namespace

BaseTestEngine::BaseTestEngine(const TestArgs& args) : args_(args), inPlace_(true), error_(0) {
  this->coll_ = getTestColl();
  CUDATHROW(hipStreamCreateWithFlags(&this->stream_, hipStreamNonBlocking));
}

BaseTestEngine::~BaseTestEngine() { hipStreamDestroy(stream_); }

void BaseTestColl::setupCollTest(const TestArgs& args, size_t size) {
  this->worldSize_ = args.totalRanks;
  this->typeSize_ = sizeof(int);
  this->setupCollTest(size);
}

double BaseTestEngine::benchTime() {
  // Performance Benchmark
  hipGraph_t graph;
  hipGraphExec_t graphExec;
  CUDATHROW(hipStreamBeginCapture(stream_, hipStreamCaptureModeGlobal));
  mscclpp::Timer timer;
  for (int iter = 0; iter < iters; iter++) {
    coll_->runColl(args_, stream_);
  }
  CUDATHROW(hipStreamEndCapture(stream_, &graph));
  CUDATHROW(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  this->barrier();
  timer.reset();
  for (int l = 0; l < cudaGraphLaunches; ++l) {
    CUDATHROW(hipGraphLaunch(graphExec, stream_));
  }
  CUDATHROW(hipStreamSynchronize(stream_));
  double deltaSec = timer.elapsed() * 1.e-6;
  deltaSec = deltaSec / (iters) / (cudaGraphLaunches);
  // all-reduce to get the average time
  allreduceTime(args_.totalRanks, deltaSec, average);
  CUDATHROW(hipGraphExecDestroy(graphExec));
  CUDATHROW(hipGraphDestroy(graph));
  return deltaSec;
}

void BaseTestEngine::barrier() { this->comm_->bootstrapper()->barrier(); }

void BaseTestEngine::runTest() {
  // warm-up for large size
  this->coll_->setupCollTest(args_, args_.maxBytes);
  this->barrier();
  for (int iter = 0; iter < warmup_iters; iter++) {
    this->coll_->runColl(args_, stream_);
  }
  CUDATHROW(hipDeviceSynchronize());

  // warm-up for small size
  this->coll_->setupCollTest(args_, args_.minBytes);
  this->barrier();
  for (int iter = 0; iter < warmup_iters; iter++) {
    this->coll_->runColl(args_, stream_);
  }
  CUDATHROW(hipDeviceSynchronize());

  PRINT("#\n");
  PRINT("# %10s  %12s           in-place                       out-of-place          \n", "", "");
  PRINT("# %10s  %12s  %7s  %6s  %6s  %6s  %7s  %6s  %6s  %6s\n", "size", "count", "time", "algbw", "busbw", "#wrong",
        "time", "algbw", "busbw", "#wrong");
  PRINT("# %10s  %12s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "(B)", "(elements)", "(us)", "(GB/s)", "(GB/s)", "",
        "(us)", "(GB/s)", "(GB/s)", "");

  // Benchmark
  for (size_t size = args_.minBytes; size <= args_.maxBytes;
       size = ((args_.stepFactor > 1) ? size * args_.stepFactor : size + args_.stepBytes)) {
    coll_->setupCollTest(args_, size);
    this->coll_->initData(this->args_, this->getSendBuff(), this->getExpectedBuff());
    PRINT("%12li  %12li", max(coll_->getSendBytes(), coll_->getExpectedBytes()), coll_->getParamBytes() / sizeof(int));
    double deltaSec = benchTime();

    size_t nErrors = 0;
    if (args_.reportErrors) {
      this->coll_->setupCollTest(args_, size);
      this->coll_->initData(this->args_, this->getSendBuff(), this->getExpectedBuff());
      this->barrier();
      this->coll_->runColl(args_, stream_);
      CUDATHROW(hipDeviceSynchronize());

      nErrors = this->checkData();
      if (nErrors > 0) {
        this->error_++;
      }
      MPI_Allreduce(MPI_IN_PLACE, &nErrors, 1, MPI_LONG, MPI_SUM, MPI_COMM_WORLD);
    }

    double timeUsec = deltaSec * 1e6;
    char timeStr[100];
    if (timeUsec >= 10000.0) {
      sprintf(timeStr, "%7.0f", timeUsec);
    } else if (timeUsec >= 100.0) {
      sprintf(timeStr, "%7.1f", timeUsec);
    } else {
      sprintf(timeStr, "%7.2f", timeUsec);
    }
    double algBw, busBw;
    this->coll_->getBw(deltaSec, algBw, busBw);
    if (!this->inPlace_) {
      PRINT("                                 ");
    }
    if (args_.reportErrors) {
      PRINT("  %7s  %6.2f  %6.2f  %5g", timeStr, algBw, busBw, (double)nErrors);
    } else {
      PRINT("  %7s  %6.2f  %6.2f  %5s", timeStr, algBw, busBw, "N/A");
    }
    PRINT("\n");
  }
  PRINT("\n");
}

void BaseTestEngine::bootstrap() {
  auto bootstrap = std::make_shared<mscclpp::Bootstrap>(args_.rank, args_.totalRanks);
  mscclpp::UniqueId id;
  if (bootstrap->getRank() == 0) id = bootstrap->createUniqueId();
  MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
  bootstrap->initialize(id);
  comm_ = std::make_shared<mscclpp::Communicator>(bootstrap);
  chanService_ = std::make_shared<mscclpp::channel::DeviceChannelService>(*comm_);
}

void BaseTestEngine::setupTest() {
  this->setupConnections();
  this->chanService_->startProxy();
}

size_t BaseTestEngine::checkData() {
  size_t nErrors = 0;
  void* recvBuff = this->getRecvBuff();
  void* expectedBuff = this->getExpectedBuff();

  size_t recvBytes = this->coll_->getRecvBytes();
  std::vector<int> recvData(recvBytes / sizeof(int), 0);
  CUDATHROW(hipMemcpy(recvData.data(), recvBuff, recvBytes, hipMemcpyDeviceToHost));
  for (size_t i = 0; i < recvData.size(); i++) {
    if (recvData[i] != ((int*)expectedBuff)[i]) {
      nErrors++;
    }
  }
  return nErrors;
}

// Create mesh connections between all ranks. If recvBuff is nullptr, assume in-place.
void BaseTestEngine::setupMeshConnections(std::vector<mscclpp::channel::SimpleDeviceChannel>& devChannels,
                                          void* sendBuff, size_t sendBuffBytes, void* recvBuff, size_t recvBuffBytes) {
  const int worldSize = args_.totalRanks;
  const int rank = args_.rank;
  const int nRanksPerNode = args_.nRanksPerNode;
  const int thisNode = rank / nRanksPerNode;
  const mscclpp::Transport ibTransport = IBs[args_.gpuNum];
  const bool isOutPlace = (recvBuff != nullptr);

  std::vector<mscclpp::channel::ChannelId> channelIds;
  std::vector<mscclpp::RegisteredMemory> localMemories;
  std::vector<mscclpp::RegisteredMemory> localTmpMemories;
  std::vector<mscclpp::NonblockingFuture<mscclpp::RegisteredMemory>> remoteMemories;

  auto rankToNode = [&](int rank) { return rank / nRanksPerNode; };
  for (int r = 0; r < worldSize; r++) {
    if (r == rank) {
      continue;
    }
    mscclpp::Transport transport;
    if (rankToNode(r) == thisNode) {
      transport = mscclpp::Transport::CudaIpc;
    } else {
      transport = ibTransport;
    }
    // Connect with all other ranks
    channelIds.push_back(chanService_->addChannel(comm_->connectOnSetup(r, 0, transport)));
    auto sendMemory = comm_->registerMemory(sendBuff, sendBuffBytes, mscclpp::Transport::CudaIpc | ibTransport);
    localMemories.push_back(sendMemory);
    if (isOutPlace) {
      auto recvMemory = comm_->registerMemory(recvBuff, recvBuffBytes, mscclpp::Transport::CudaIpc | ibTransport);
      comm_->sendMemoryOnSetup(recvMemory, r, 0);
      localTmpMemories.push_back(recvMemory);
    } else {
      comm_->sendMemoryOnSetup(sendMemory, r, 0);
    }
    remoteMemories.push_back(comm_->recvMemoryOnSetup(r, 0));
  }
  comm_->setup();

  for (size_t i = 0; i < channelIds.size(); ++i) {
    devChannels.push_back(mscclpp::channel::SimpleDeviceChannel(
        chanService_->deviceChannel(channelIds[i]), chanService_->addMemory(remoteMemories[i].get()),
        chanService_->addMemory(localMemories[i]), remoteMemories[i].get().data(), localMemories[i].data(),
        (isOutPlace ? localTmpMemories[i].data() : nullptr)));
  }
}

void run(int argc, char* argv[]);
int main(int argc, char* argv[]) {
  // Make sure everyline is flushed so that we see the progress of the test
  setlinebuf(stdout);

  // Parse args
  double parsed;
  int longindex;
  static option longopts[] = {{"minbytes", required_argument, 0, 'b'},
                              {"maxbytes", required_argument, 0, 'e'},
                              {"stepbytes", required_argument, 0, 'i'},
                              {"stepfactor", required_argument, 0, 'f'},
                              {"iters", required_argument, 0, 'n'},
                              {"warmup_iters", required_argument, 0, 'w'},
                              {"check", required_argument, 0, 'c'},
                              {"cudagraph", required_argument, 0, 'G'},
                              {"average", required_argument, 0, 'a'},
                              {"kernel_num", required_argument, 0, 'k'},
                              {"help", no_argument, 0, 'h'},
                              {}};

  while (1) {
    int c;
    c = getopt_long(argc, argv, "b:e:i:f:n:w:c:G:a:k:h:", longopts, &longindex);

    if (c == -1) break;

    switch (c) {
      case 'b':
        parsed = parseSize(optarg);
        if (parsed < 0) {
          fprintf(stderr, "invalid size specified for 'minbytes'\n");
          return -1;
        }
        minBytes = (size_t)parsed;
        break;
      case 'e':
        parsed = parseSize(optarg);
        if (parsed < 0) {
          fprintf(stderr, "invalid size specified for 'maxbytes'\n");
          return -1;
        }
        maxBytes = (size_t)parsed;
        break;
      case 'i':
        stepBytes = strtol(optarg, NULL, 0);
        break;
      case 'f':
        stepFactor = strtol(optarg, NULL, 0);
        break;
      case 'n':
        iters = (int)strtol(optarg, NULL, 0);
        break;
      case 'w':
        warmup_iters = (int)strtol(optarg, NULL, 0);
        break;
      case 'c':
        datacheck = (int)strtol(optarg, NULL, 0);
        break;
      case 'G':
        cudaGraphLaunches = strtol(optarg, NULL, 0);
        if (cudaGraphLaunches <= 0) {
          fprintf(stderr, "invalid number for 'cudaGraphLaunches'\n");
          return -1;
        }
        break;
      case 'a':
        average = (int)strtol(optarg, NULL, 0);
        break;
      case 'k':
        kernel_num = (int)strtol(optarg, NULL, 0);
        break;
      case 'h':
      default:
        if (c != 'h') printf("invalid option '%c'\n", c);
        printf(
            "USAGE: %s \n\t"
            "[-b,--minbytes <min size in bytes>] \n\t"
            "[-e,--maxbytes <max size in bytes>] \n\t"
            "[-i,--stepbytes <increment size>] \n\t"
            "[-f,--stepfactor <increment factor>] \n\t"
            "[-n,--iters <iteration count>] \n\t"
            "[-w,--warmup_iters <warmup iteration count>] \n\t"
            "[-c,--check <0/1>] \n\t"
            "[-T,--timeout <time in seconds>] \n\t"
            "[-G,--cudagraph <num graph launches>] \n\t"
            "[-C,--report_cputime <0/1>] \n\t"
            "[-a,--average <0/1/2/3> report average iteration time <0=RANK0/1=AVG/2=MIN/3=MAX>] \n\t"
            "[-k,--kernel_num <kernel number of commnication primitive>] \n\t"
            "[-h,--help]\n",
            basename(argv[0]));
        return 0;
    }
  }
  if (minBytes > maxBytes) {
    std::cerr << "invalid sizes for 'minbytes' and 'maxbytes': " << minBytes << " > " << maxBytes << std::endl;
    return -1;
  }
  run(argc, argv);
  return 0;
}

void run(int argc, char* argv[]) {
  int totalRanks = 1, rank = 0;
  int nRanksPerNode = 0, localRank = 0;
  std::string hostname = mscclpp::getHostName(1024, '.');

  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &totalRanks);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm shmcomm;
  MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0, MPI_INFO_NULL, &shmcomm);
  MPI_Comm_size(shmcomm, &nRanksPerNode);
  MPI_Comm_free(&shmcomm);
  localRank = rank % nRanksPerNode;
  is_main_proc = (rank == 0) ? 1 : 0;

  PRINT(
      "# minBytes %ld maxBytes %ld step: %ld(%s) warmup iters: %d iters: %d validation: %d graph: %d, "
      "kernel num: %d\n",
      minBytes, maxBytes, (stepFactor > 1) ? stepFactor : stepBytes, (stepFactor > 1) ? "factor" : "bytes",
      warmup_iters, iters, datacheck, cudaGraphLaunches, kernel_num);
  PRINT("#\n");
  PRINT("# Using devices\n");

  constexpr int MAX_LINE = 2048;
  char line[MAX_LINE];
  int len = 0;
  size_t maxMem = ~0;

  int cudaDev = localRank;
  hipDeviceProp_t prop;
  char busIdChar[] = "00000000:00:00.0";
  CUDATHROW(hipGetDeviceProperties(&prop, cudaDev));
  CUDATHROW(hipDeviceGetPCIBusId(busIdChar, sizeof(busIdChar), cudaDev));
  len += snprintf(line + len, MAX_LINE - len, "#  Rank %2d Pid %6d on %10s device %2d [%s] %s\n", rank, getpid(),
                  hostname.c_str(), cudaDev, busIdChar, prop.name);
  maxMem = std::min(maxMem, prop.totalGlobalMem);

  std::shared_ptr<char[]> lines(new char[totalRanks * MAX_LINE]);
  // Gather all output in rank order to root (0)
  MPI_Gather(line, MAX_LINE, MPI_BYTE, lines.get(), MAX_LINE, MPI_BYTE, 0, MPI_COMM_WORLD);
  if (rank == 0) {
    for (int r = 0; r < totalRanks; r++) PRINT("%s", &lines[MAX_LINE * r]);
  }
  MPI_Allreduce(MPI_IN_PLACE, &maxMem, 1, MPI_LONG, MPI_MIN, MPI_COMM_WORLD);

  // We need sendbuff, recvbuff, expected (when datacheck enabled), plus 1G for the rest.
  size_t memMaxBytes = (maxMem - (1 << 30)) / (datacheck ? 3 : 2);
  if (maxBytes > memMaxBytes) {
    maxBytes = memMaxBytes;
    PRINT("#\n# Reducing maxBytes to %ld due to memory limitation\n", maxBytes);
  }

  CUDATHROW(hipSetDevice(cudaDev));
  TestArgs args = {minBytes, maxBytes,  stepBytes,     stepFactor, totalRanks, rank,
                   cudaDev,  localRank, nRanksPerNode, kernel_num, datacheck};
  PRINT("#\n");
  PRINT("# Initializing MSCCL++\n");
  auto testEngine = getTestEngine(args);
  testEngine->bootstrap();
  testEngine->allocateBuffer();
  PRINT("# Setting up the connection in MSCCL++\n");
  testEngine->setupTest();
  testEngine->barrier();
  testEngine->runTest();

  fflush(stdout);

  int error = testEngine->getTestErrors();
  MPI_Allreduce(MPI_IN_PLACE, &error, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);

  PRINT("# Out of bounds values : %d %s\n", error, error ? "FAILED" : "OK");
  PRINT("#\n");

  MPI_Finalize();
}
