#include "hip/hip_runtime.h"
#include "mscclpp.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <unistd.h>

#include "common.h"

#define RANKS_PER_NODE 8
#define USE_DMA_FOR_P2P 1
#define TEST_CONN_TYPE 0 // 0: P2P(for local)+IB(for remote), 1: IB-Only

#define MSCCLPPCHECK(call)                                                                                             \
  do {                                                                                                                 \
    mscclppResult_t res = call;                                                                                        \
    if (res != mscclppSuccess && res != mscclppInProgress) {                                                           \
      /* Print the back trace*/                                                                                        \
      printf("Failure at %s:%d -> %d\n", __FILE__, __LINE__, res);                                                     \
      return res;                                                                                                      \
    }                                                                                                                  \
  } while (0);

// Check CUDA RT calls
#define CUDACHECK(cmd)                                                                                                 \
  do {                                                                                                                 \
    hipError_t err = cmd;                                                                                             \
    if (err != hipSuccess) {                                                                                          \
      printf("%s:%d Cuda failure '%s'\n", __FILE__, __LINE__, hipGetErrorString(err));                                \
      exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (false)

// Measure current time in second.
static double getTime(void)
{
  struct timespec tspec;
  if (clock_gettime(CLOCK_MONOTONIC, &tspec) == -1) {
    printf("clock_gettime failed\n");
    exit(EXIT_FAILURE);
  }
  return (tspec.tv_nsec / 1.0e9) + tspec.tv_sec;
}

__constant__ mscclppDevConn_t constDevConns[16];

__global__ void kernel(int rank, int world_size)
{
  if (threadIdx.x % 32 != 0)
    return;

  int warpId = threadIdx.x / 32;
  int remoteRank = (warpId < rank) ? warpId : warpId + 1;
  mscclppDevConn_t devConn = constDevConns[remoteRank];
  volatile int* data = (volatile int*)devConn.localBuff;
  volatile uint64_t* localFlag = devConn.localFlag;
#if (USE_DMA_FOR_P2P == 0)
  volatile uint64_t* remoteSignalEpochId = devConn.remoteSignalEpochId;
#endif
  volatile uint64_t* proxyFlag = devConn.proxyFlag;

  uint64_t baseFlag = *localFlag;

  if (threadIdx.x == 0) {
    // Set my data and flag
    *(data + rank) = rank + 1;
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    // Do we need a sys fence?
    // __threadfence_system();
    *localFlag = baseFlag + 1;
  }

  // get a thread-local trigger and a request for waiting on it
  // mscclppTrigger_t trig;
  // mscclppRequest_t req = devConn.fifo.getTrigger(&trig);

  // Each warp receives data from different ranks
#if (USE_DMA_FOR_P2P == 1)

  // Trigger sending data, flag and synchronize after
  auto req = devConn.fifo.putWithSignal(rank * sizeof(int), sizeof(int));

  // Wait on the request to make sure it is safe to reuse buffer and flag
  devConn.fifo.sync(req);

  // Wait for receiving data from remote rank
  while (*proxyFlag == baseFlag) {
  }

#else // USE_DMA_FOR_P2P == 0

  if (devConn.remoteBuff == NULL) { // IB
    // Wait until the proxy have sent my data and flag
    devConn.waitTrigger(trig);

    // Trigger sending data and flag
    devConn.setTrigger(trig, mscclppFlag | mscclppData, rank * sizeof(int), sizeof(int));

    // Wait for receiving data from remote rank
    while (*proxyFlag == baseFlag) {
    }
  } else { // P2P
    // Directly read data
    volatile int* remoteData = (volatile int*)devConn.remoteBuff;

    // Wait until the remote data is set
    while (*remoteSignalEpochId == baseFlag) {
    }

    // Read remote data
    data[remoteRank] = remoteData[remoteRank];
  }

#endif
}

int rankToLocalRank(int rank)
{
  return rank % RANKS_PER_NODE;
}

int rankToNode(int rank)
{
  return rank / RANKS_PER_NODE;
}

int cudaNumToIbNum(int cudaNum)
{
  int ibNum;
  if (cudaNum == 0) {
    ibNum = 0;
  } else if (cudaNum == 1) {
    ibNum = 4;
  } else if (cudaNum == 2) {
    ibNum = 1;
  } else if (cudaNum == 3) {
    ibNum = 5;
  } else if (cudaNum == 4) {
    ibNum = 2;
  } else if (cudaNum == 5) {
    ibNum = 6;
  } else if (cudaNum == 6) {
    ibNum = 3;
  } else if (cudaNum == 7) {
    ibNum = 7;
  } else {
    printf("Invalid cudaNum: %d\n", cudaNum);
    exit(EXIT_FAILURE);
  }
  return ibNum;
}

int main(int argc, const char* argv[])
{
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  MPI_Init(NULL, NULL);
#endif
  const char* ip_port;
  int rank, world_size;
  parse_arguments(argc, argv, &ip_port, &rank, &world_size);
  int localRank = rankToLocalRank(rank);
  int thisNode = rankToNode(rank);
  int cudaNum = localRank;
  int ibNum = cudaNumToIbNum(cudaNum);

  CUDACHECK(hipSetDevice(cudaNum));
  std::string ibDevStr = "mlx5_ib" + std::to_string(ibNum);

  mscclppComm_t comm;
  MSCCLPPCHECK(mscclppCommInitRank(&comm, world_size, rank, ip_port));

  int* data_d;
  uint64_t* flag_d;
  size_t data_size = sizeof(int) * world_size;
  CUDACHECK(hipMalloc(&data_d, data_size));
  CUDACHECK(hipMalloc(&flag_d, sizeof(uint64_t)));
  CUDACHECK(hipMemset(data_d, 0, data_size));
  CUDACHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));

  for (int r = 0; r < world_size; ++r) {
    if (r == rank)
      continue;
    mscclppTransport_t transportType = mscclppTransportIB;
    const char* ibDev = ibDevStr.c_str();
#if (TEST_CONN_TYPE == 0) // P2P+IB
    if (rankToNode(r) == thisNode) {
      transportType = mscclppTransportP2P;
      ibDev = NULL;
    }
#endif
    // Connect with all other ranks
    MSCCLPPCHECK(mscclppConnect(comm, r, 0, data_d, data_size, flag_d, transportType, ibDev));
  }

  MSCCLPPCHECK(mscclppConnectionSetup(comm));

  MSCCLPPCHECK(mscclppProxyLaunch(comm));

  mscclppDevConn_t* devConns;
  int nCons;
  MSCCLPPCHECK(mscclppGetAllDeviceConnections(comm, &devConns, &nCons));

  CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(constDevConns), devConns, sizeof(mscclppDevConn_t) * world_size));

  hipStream_t stream;
  CUDACHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  kernel<<<1, 32 * (world_size - 1), 0, stream>>>(rank, world_size);
  CUDACHECK(hipDeviceSynchronize());

  // Read results from GPU
  int* buf = (int*)calloc(world_size, sizeof(int));
  if (buf == nullptr) {
    printf("calloc failed\n");
    return -1;
  }
  CUDACHECK(hipMemcpy(buf, data_d, sizeof(int) * world_size, hipMemcpyDeviceToHost));

  bool failed = false;
  for (int i = 0; i < world_size; ++i) {
    if (buf[i] != i + 1) {
      printf("rank: %d, wrong data: %d, expected %d\n", rank, buf[i], i + 1);
      failed = true;
    }
  }
  if (failed) {
    return -1;
  }

  // Perf test
  hipEvent_t ev_start;
  hipEvent_t ev_end;
  CUDACHECK(hipEventCreate(&ev_start));
  CUDACHECK(hipEventCreate(&ev_end));

  // warm up
  // int warmupiter = 10;
  //  for (int i = 0; i < warmupiter; ++i) {
  //    kernel<<<1, 32 * (world_size - 1), 0, stream>>>(rank, world_size);
  //  }

  // cudaGraph Capture
  hipGraph_t graph;
  hipGraphExec_t instance;
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  int cudagraphiter = 100;
  for (int i = 0; i < cudagraphiter; ++i) {
    kernel<<<1, 32 * (world_size - 1), 0, stream>>>(rank, world_size);
  }
  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

  int cudagraphwarmup = 10;
  for (int i = 0; i < cudagraphwarmup; ++i) {
    hipGraphLaunch(instance, stream);
  }
  CUDACHECK(hipStreamSynchronize(stream));

  // measure runtime
  //  CUDACHECK(hipEventRecord(ev_start, stream));
  double t0 = getTime();
  int cudagraphlaunch = 10;
  for (int i = 0; i < cudagraphlaunch; ++i) {
    // kernel<<<1, 32 * (world_size - 1), 0, stream>>>(rank, world_size);
    hipGraphLaunch(instance, stream);
  }
  //  CUDACHECK(hipEventRecord(ev_end, stream));
  CUDACHECK(hipStreamSynchronize(stream));

  double t1 = getTime();
  float ms = (t1 - t0) * 1000.0;
  //  CUDACHECK(hipEventElapsedTime(&ms, ev_start, ev_end));
  printf("rank: %d, time: %f us/iter\n", rank, ms * 1000. / (float)cudagraphlaunch / (float)cudagraphiter);

  MSCCLPPCHECK(mscclppProxyStop(comm));

  MSCCLPPCHECK(mscclppCommDestroy(comm));

#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  if (argc == 2) {
    MPI_Finalize();
  }
#endif
  printf("Succeeded! %d\n", rank);
  return 0;
}
